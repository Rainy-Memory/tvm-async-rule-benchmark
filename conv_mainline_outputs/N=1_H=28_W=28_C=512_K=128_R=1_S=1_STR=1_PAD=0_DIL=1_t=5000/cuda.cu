
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[8];
  __shared__ float PadInput_shared[512];
  __shared__ float weight_shared[1024];
  for (int h_3_init = 0; h_3_init < 4; ++h_3_init) {
    for (int co_3_init = 0; co_3_init < 2; ++co_3_init) {
      conv2d_nhwc_local[((h_3_init * 2) + co_3_init)] = 0.000000e+00f;
    }
  }
  for (int rc_0 = 0; rc_0 < 16; ++rc_0) {
    __syncthreads();
    for (int ax0_ax1_ax2_ax3_fused_0 = 0; ax0_ax1_ax2_ax3_fused_0 < 4; ++ax0_ax1_ax2_ax3_fused_0) {
      *(float2*)(PadInput_shared + ((ax0_ax1_ax2_ax3_fused_0 * 128) + (((int)threadIdx.x) * 2))) = *(float2*)(inputs + (((((((((int)blockIdx.x) / 28) * 57344) + (ax0_ax1_ax2_ax3_fused_0 * 14336)) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)));
    }
    for (int ax0_ax1_ax2_ax3_fused_0_1 = 0; ax0_ax1_ax2_ax3_fused_0_1 < 4; ++ax0_ax1_ax2_ax3_fused_0_1) {
      *(float4*)(weight_shared + ((ax0_ax1_ax2_ax3_fused_0_1 * 256) + (((int)threadIdx.x) * 4))) = *(float4*)(weight + (((((rc_0 * 4096) + (ax0_ax1_ax2_ax3_fused_0_1 * 1024)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)));
    }
    __syncthreads();
    for (int h_3 = 0; h_3 < 4; ++h_3) {
      for (int co_3 = 0; co_3 < 2; ++co_3) {
        for (int rc_2 = 0; rc_2 < 32; ++rc_2) {
          conv2d_nhwc_local[((h_3 * 2) + co_3)] = (conv2d_nhwc_local[((h_3 * 2) + co_3)] + (PadInput_shared[(((h_3 * 128) + ((((int)threadIdx.x) >> 4) * 32)) + rc_2)] * weight_shared[(((rc_2 * 32) + ((((int)threadIdx.x) & 15) * 2)) + co_3)]));
        }
      }
    }
  }
  for (int ax1 = 0; ax1 < 4; ++ax1) {
    for (int ax3 = 0; ax3 < 2; ++ax3) {
      conv2d_nhwc[((((((((((int)blockIdx.x) / 28) * 14336) + (ax1 * 3584)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)threadIdx.x) >> 4) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ax3)] = conv2d_nhwc_local[((ax1 * 2) + ax3)];
    }
  }
}


