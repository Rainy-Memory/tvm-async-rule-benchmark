
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[64];
  __shared__ float A_shared[768];
  __shared__ float B_shared[3072];
  for (int j_4_init = 0; j_4_init < 2; ++j_4_init) {
    for (int vthread_x_s = 0; vthread_x_s < 32; ++vthread_x_s) {
      Y_local[((vthread_x_s * 2) + j_4_init)] = 0.000000e+00f;
    }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((int)blockIdx.x) / 24) * 24576) + ((((int)threadIdx.x) >> 1) * 768)) + ((((int)threadIdx.x) & 1) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((int)threadIdx.x) >> 5) * 3072) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 5) * 3072) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 6144))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 5) * 3072) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 12288))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 5) * 3072) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 18432))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 24576) + ((((int)threadIdx.x) >> 1) * 768)) + ((((int)threadIdx.x) & 1) * 4)) + 8))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 5) * 3072) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 24576))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 5) * 3072) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 30720))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 5) * 3072) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 36864))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 5) * 3072) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 43008))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int k_0_fused = 0; k_0_fused < 94; ++k_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((k_0_fused + 2) % 3) * 256) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 24) * 24576) + ((((int)threadIdx.x) >> 1) * 768)) + (k_0_fused * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 16))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((k_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 24576) + ((((int)threadIdx.x) >> 5) * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 49152))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 24576) + ((((int)threadIdx.x) >> 5) * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 55296))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 24576) + ((((int)threadIdx.x) >> 5) * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 61440))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4)) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 24576) + ((((int)threadIdx.x) >> 5) * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 67584))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    for (int k_1 = 0; k_1 < 2; ++k_1) {
      for (int k_2 = 0; k_2 < 4; ++k_2) {
        for (int j_4 = 0; j_4 < 2; ++j_4) {
          for (int vthread_x_s_1 = 0; vthread_x_s_1 < 32; ++vthread_x_s_1) {
            Y_local[((vthread_x_s_1 * 2) + j_4)] = (Y_local[((vthread_x_s_1 * 2) + j_4)] + (A_shared[((((((k_0_fused % 3) * 256) + ((vthread_x_s_1 >> 2) * 32)) + ((((int)threadIdx.x) >> 4) * 8)) + (k_1 * 4)) + k_2)] * B_shared[(((((((k_0_fused % 3) * 1024) + (k_1 * 512)) + (k_2 * 128)) + ((vthread_x_s_1 & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + j_4)]));
          }
        }
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  for (int k_1_1 = 0; k_1_1 < 2; ++k_1_1) {
    for (int k_2_1 = 0; k_2_1 < 4; ++k_2_1) {
      for (int j_4_1 = 0; j_4_1 < 2; ++j_4_1) {
        for (int vthread_x_s_2 = 0; vthread_x_s_2 < 32; ++vthread_x_s_2) {
          Y_local[((vthread_x_s_2 * 2) + j_4_1)] = (Y_local[((vthread_x_s_2 * 2) + j_4_1)] + (A_shared[((((((vthread_x_s_2 >> 2) * 32) + ((((int)threadIdx.x) >> 4) * 8)) + (k_1_1 * 4)) + k_2_1) + 256)] * B_shared[((((((k_1_1 * 512) + (k_2_1 * 128)) + ((vthread_x_s_2 & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + j_4_1) + 1024)]));
        }
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int k_1_2 = 0; k_1_2 < 2; ++k_1_2) {
    for (int k_2_2 = 0; k_2_2 < 4; ++k_2_2) {
      for (int j_4_2 = 0; j_4_2 < 2; ++j_4_2) {
        for (int vthread_x_s_3 = 0; vthread_x_s_3 < 32; ++vthread_x_s_3) {
          Y_local[((vthread_x_s_3 * 2) + j_4_2)] = (Y_local[((vthread_x_s_3 * 2) + j_4_2)] + (A_shared[((((((vthread_x_s_3 >> 2) * 32) + ((((int)threadIdx.x) >> 4) * 8)) + (k_1_2 * 4)) + k_2_2) + 512)] * B_shared[((((((k_1_2 * 512) + (k_2_2 * 128)) + ((vthread_x_s_3 & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + j_4_2) + 2048)]));
        }
      }
    }
  }
  for (int ax1 = 0; ax1 < 2; ++ax1) {
    for (int vthread_x_s_4 = 0; vthread_x_s_4 < 32; ++vthread_x_s_4) {
      Y[((((((((((int)blockIdx.x) / 24) * 98304) + ((vthread_x_s_4 >> 2) * 12288)) + ((((int)threadIdx.x) >> 4) * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((vthread_x_s_4 & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ax1)] = Y_local[((vthread_x_s_4 * 2) + ax1)];
    }
  }
}


