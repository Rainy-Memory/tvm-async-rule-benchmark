
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[128];
  __shared__ float A_shared[1536];
  __shared__ float B_shared[3072];
  for (int i_4_init = 0; i_4_init < 16; ++i_4_init) {
    for (int j_4_init = 0; j_4_init < 4; ++j_4_init) {
      Y_local[((i_4_init * 4) + j_4_init)] = 0.000000e+00f;
      Y_local[(((i_4_init * 4) + j_4_init) + 64)] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 2; ++ax0_ax1_fused_0) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((ax0_ax1_fused_0 * 256) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 4) * 131072) + (ax0_ax1_fused_0 * 65536)) + ((((int)threadIdx.x) >> 1) * 2048)) + ((((int)threadIdx.x) & 1) * 4)))), "n"(16)
    );
  }
  }
  for (int ax0_ax1_fused_0_1 = 0; ax0_ax1_fused_0_1 < 4; ++ax0_ax1_fused_0_1) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((ax0_ax1_fused_0_1 * 256) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((ax0_ax1_fused_0_1 * 4096) + ((((int)threadIdx.x) >> 5) * 2048)) + ((((int)blockIdx.x) & 15) * 128)) + ((((int)threadIdx.x) & 31) * 4)))), "n"(16)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int ax0_ax1_fused_0_2 = 0; ax0_ax1_fused_0_2 < 2; ++ax0_ax1_fused_0_2) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((ax0_ax1_fused_0_2 * 256) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) >> 4) * 131072) + (ax0_ax1_fused_0_2 * 65536)) + ((((int)threadIdx.x) >> 1) * 2048)) + ((((int)threadIdx.x) & 1) * 4)) + 8))), "n"(16)
    );
  }
  }
  for (int ax0_ax1_fused_0_3 = 0; ax0_ax1_fused_0_3 < 4; ++ax0_ax1_fused_0_3) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((ax0_ax1_fused_0_3 * 256) + (((int)threadIdx.x) * 4)) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((ax0_ax1_fused_0_3 * 4096) + ((((int)threadIdx.x) >> 5) * 2048)) + ((((int)blockIdx.x) & 15) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 16384))), "n"(16)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int k_0_fused = 0; k_0_fused < 254; ++k_0_fused) {
    __syncthreads();
    for (int ax0_ax1_fused_0_4 = 0; ax0_ax1_fused_0_4 < 2; ++ax0_ax1_fused_0_4) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 2) % 3) * 512) + (ax0_ax1_fused_0_4 * 256)) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((((int)blockIdx.x) >> 4) * 131072) + (ax0_ax1_fused_0_4 * 65536)) + ((((int)threadIdx.x) >> 1) * 2048)) + (k_0_fused * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 16))), "n"(16)
    );
  }
    }
    for (int ax0_ax1_fused_0_5 = 0; ax0_ax1_fused_0_5 < 4; ++ax0_ax1_fused_0_5) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 1024) + (ax0_ax1_fused_0_5 * 256)) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((k_0_fused * 16384) + (ax0_ax1_fused_0_5 * 4096)) + ((((int)threadIdx.x) >> 5) * 2048)) + ((((int)blockIdx.x) & 15) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 32768))), "n"(16)
    );
  }
    }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    for (int k_1 = 0; k_1 < 4; ++k_1) {
      for (int k_2 = 0; k_2 < 2; ++k_2) {
        for (int i_4 = 0; i_4 < 16; ++i_4) {
          for (int j_4 = 0; j_4 < 4; ++j_4) {
            Y_local[((i_4 * 4) + j_4)] = (Y_local[((i_4 * 4) + j_4)] + (A_shared[((((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 4) * 128)) + (i_4 * 8)) + (k_1 * 2)) + k_2)] * B_shared[((((((k_0_fused % 3) * 1024) + (k_1 * 256)) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + j_4)]));
            Y_local[(((i_4 * 4) + j_4) + 64)] = (Y_local[(((i_4 * 4) + j_4) + 64)] + (A_shared[((((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 4) * 128)) + (i_4 * 8)) + (k_1 * 2)) + k_2)] * B_shared[(((((((k_0_fused % 3) * 1024) + (k_1 * 256)) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + j_4) + 64)]));
          }
        }
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  for (int k_1_1 = 0; k_1_1 < 4; ++k_1_1) {
    for (int k_2_1 = 0; k_2_1 < 2; ++k_2_1) {
      for (int i_4_1 = 0; i_4_1 < 16; ++i_4_1) {
        for (int j_4_1 = 0; j_4_1 < 4; ++j_4_1) {
          Y_local[((i_4_1 * 4) + j_4_1)] = (Y_local[((i_4_1 * 4) + j_4_1)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 128) + (i_4_1 * 8)) + (k_1_1 * 2)) + k_2_1) + 1024)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + j_4_1) + 2048)]));
          Y_local[(((i_4_1 * 4) + j_4_1) + 64)] = (Y_local[(((i_4_1 * 4) + j_4_1) + 64)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 128) + (i_4_1 * 8)) + (k_1_1 * 2)) + k_2_1) + 1024)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + j_4_1) + 2112)]));
        }
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int k_1_2 = 0; k_1_2 < 4; ++k_1_2) {
    for (int k_2_2 = 0; k_2_2 < 2; ++k_2_2) {
      for (int i_4_2 = 0; i_4_2 < 16; ++i_4_2) {
        for (int j_4_2 = 0; j_4_2 < 4; ++j_4_2) {
          Y_local[((i_4_2 * 4) + j_4_2)] = (Y_local[((i_4_2 * 4) + j_4_2)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 128) + (i_4_2 * 8)) + (k_1_2 * 2)) + k_2_2)] * B_shared[((((k_1_2 * 256) + (k_2_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + j_4_2)]));
          Y_local[(((i_4_2 * 4) + j_4_2) + 64)] = (Y_local[(((i_4_2 * 4) + j_4_2) + 64)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 128) + (i_4_2 * 8)) + (k_1_2 * 2)) + k_2_2)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + j_4_2) + 64)]));
        }
      }
    }
  }
  for (int ax0 = 0; ax0 < 16; ++ax0) {
    for (int ax1 = 0; ax1 < 4; ++ax1) {
      Y[(((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 32768)) + (ax0 * 2048)) + ((((int)blockIdx.x) & 15) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + ax1)] = Y_local[((ax0 * 4) + ax1)];
      Y[((((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 32768)) + (ax0 * 2048)) + ((((int)blockIdx.x) & 15) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + ax1) + 64)] = Y_local[(((ax0 * 4) + ax1) + 64)];
    }
  }
}


