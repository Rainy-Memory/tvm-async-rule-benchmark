
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[96];
  __shared__ float A_shared[6144];
  __shared__ float B_shared[4096];
  for (int j_3_init = 0; j_3_init < 2; ++j_3_init) {
    Y_local[j_3_init] = 0.000000e+00f;
    Y_local[(j_3_init + 6)] = 0.000000e+00f;
    Y_local[(j_3_init + 12)] = 0.000000e+00f;
    Y_local[(j_3_init + 18)] = 0.000000e+00f;
    Y_local[(j_3_init + 24)] = 0.000000e+00f;
    Y_local[(j_3_init + 30)] = 0.000000e+00f;
    Y_local[(j_3_init + 36)] = 0.000000e+00f;
    Y_local[(j_3_init + 42)] = 0.000000e+00f;
    Y_local[(j_3_init + 48)] = 0.000000e+00f;
    Y_local[(j_3_init + 54)] = 0.000000e+00f;
    Y_local[(j_3_init + 60)] = 0.000000e+00f;
    Y_local[(j_3_init + 66)] = 0.000000e+00f;
    Y_local[(j_3_init + 72)] = 0.000000e+00f;
    Y_local[(j_3_init + 78)] = 0.000000e+00f;
    Y_local[(j_3_init + 84)] = 0.000000e+00f;
    Y_local[(j_3_init + 90)] = 0.000000e+00f;
    Y_local[(j_3_init + 2)] = 0.000000e+00f;
    Y_local[(j_3_init + 8)] = 0.000000e+00f;
    Y_local[(j_3_init + 14)] = 0.000000e+00f;
    Y_local[(j_3_init + 20)] = 0.000000e+00f;
    Y_local[(j_3_init + 26)] = 0.000000e+00f;
    Y_local[(j_3_init + 32)] = 0.000000e+00f;
    Y_local[(j_3_init + 38)] = 0.000000e+00f;
    Y_local[(j_3_init + 44)] = 0.000000e+00f;
    Y_local[(j_3_init + 50)] = 0.000000e+00f;
    Y_local[(j_3_init + 56)] = 0.000000e+00f;
    Y_local[(j_3_init + 62)] = 0.000000e+00f;
    Y_local[(j_3_init + 68)] = 0.000000e+00f;
    Y_local[(j_3_init + 74)] = 0.000000e+00f;
    Y_local[(j_3_init + 80)] = 0.000000e+00f;
    Y_local[(j_3_init + 86)] = 0.000000e+00f;
    Y_local[(j_3_init + 92)] = 0.000000e+00f;
    Y_local[(j_3_init + 4)] = 0.000000e+00f;
    Y_local[(j_3_init + 10)] = 0.000000e+00f;
    Y_local[(j_3_init + 16)] = 0.000000e+00f;
    Y_local[(j_3_init + 22)] = 0.000000e+00f;
    Y_local[(j_3_init + 28)] = 0.000000e+00f;
    Y_local[(j_3_init + 34)] = 0.000000e+00f;
    Y_local[(j_3_init + 40)] = 0.000000e+00f;
    Y_local[(j_3_init + 46)] = 0.000000e+00f;
    Y_local[(j_3_init + 52)] = 0.000000e+00f;
    Y_local[(j_3_init + 58)] = 0.000000e+00f;
    Y_local[(j_3_init + 64)] = 0.000000e+00f;
    Y_local[(j_3_init + 70)] = 0.000000e+00f;
    Y_local[(j_3_init + 76)] = 0.000000e+00f;
    Y_local[(j_3_init + 82)] = 0.000000e+00f;
    Y_local[(j_3_init + 88)] = 0.000000e+00f;
    Y_local[(j_3_init + 94)] = 0.000000e+00f;
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 24576))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 49152))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 73728))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 98304))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 122880))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((int)threadIdx.x) >> 4) * 1536) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1536) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 6144))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1536) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 12288))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1536) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 18432))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 16))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 24592))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 49168))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 2304)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 73744))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 2560)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 98320))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 2816)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 122896))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1536) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 24576))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1536) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 30720))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1536) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 36864))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1536) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 43008))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 3072)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 32))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 3328)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 24608))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 3584)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 49184))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 3840)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 73760))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 4096)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 98336))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 4352)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + ((((int)threadIdx.x) & 3) * 4)) + 122912))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1536) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 49152))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 2304)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1536) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 55296))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 2560)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1536) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 61440))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 2816)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1536) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 67584))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int k_0_fused = 0; k_0_fused < 93; ++k_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((k_0_fused + 3) & 3) * 1536) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + (k_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 48))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 3) & 3) * 1536) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + (k_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 24624))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 3) & 3) * 1536) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + (k_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 49200))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 3) & 3) * 1536) + (((int)threadIdx.x) * 4)) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + (k_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 73776))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 3) & 3) * 1536) + (((int)threadIdx.x) * 4)) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + (k_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 98352))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 3) & 3) * 1536) + (((int)threadIdx.x) * 4)) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 2) * 1536)) + (k_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 122928))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((k_0_fused + 3) & 3) * 1024) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 24576) + ((((int)threadIdx.x) >> 4) * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 73728))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 3) & 3) * 1024) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 24576) + ((((int)threadIdx.x) >> 4) * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 79872))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 3) & 3) * 1024) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 24576) + ((((int)threadIdx.x) >> 4) * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 86016))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 3) & 3) * 1024) + (((int)threadIdx.x) * 4)) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 24576) + ((((int)threadIdx.x) >> 4) * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 92160))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

    __syncthreads();
    for (int k_1 = 0; k_1 < 4; ++k_1) {
      for (int j_3 = 0; j_3 < 2; ++j_3) {
        for (int k_2 = 0; k_2 < 4; ++k_2) {
          Y_local[j_3] = (Y_local[j_3] + (A_shared[(((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2)] * B_shared[((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3)]));
          Y_local[(j_3 + 6)] = (Y_local[(j_3 + 6)] + (A_shared[(((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 16)]));
          Y_local[(j_3 + 12)] = (Y_local[(j_3 + 12)] + (A_shared[(((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 32)]));
          Y_local[(j_3 + 18)] = (Y_local[(j_3 + 18)] + (A_shared[(((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 48)]));
          Y_local[(j_3 + 24)] = (Y_local[(j_3 + 24)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 384)] * B_shared[((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3)]));
          Y_local[(j_3 + 30)] = (Y_local[(j_3 + 30)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 384)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 16)]));
          Y_local[(j_3 + 36)] = (Y_local[(j_3 + 36)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 384)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 32)]));
          Y_local[(j_3 + 42)] = (Y_local[(j_3 + 42)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 384)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 48)]));
          Y_local[(j_3 + 48)] = (Y_local[(j_3 + 48)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 768)] * B_shared[((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3)]));
          Y_local[(j_3 + 54)] = (Y_local[(j_3 + 54)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 768)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 16)]));
          Y_local[(j_3 + 60)] = (Y_local[(j_3 + 60)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 768)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 32)]));
          Y_local[(j_3 + 66)] = (Y_local[(j_3 + 66)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 768)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 48)]));
          Y_local[(j_3 + 72)] = (Y_local[(j_3 + 72)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 1152)] * B_shared[((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3)]));
          Y_local[(j_3 + 78)] = (Y_local[(j_3 + 78)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 1152)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 16)]));
          Y_local[(j_3 + 84)] = (Y_local[(j_3 + 84)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 1152)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 32)]));
          Y_local[(j_3 + 90)] = (Y_local[(j_3 + 90)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 1152)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 48)]));
          Y_local[(j_3 + 2)] = (Y_local[(j_3 + 2)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 16)] * B_shared[((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3)]));
          Y_local[(j_3 + 8)] = (Y_local[(j_3 + 8)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 16)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 16)]));
          Y_local[(j_3 + 14)] = (Y_local[(j_3 + 14)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 16)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 32)]));
          Y_local[(j_3 + 20)] = (Y_local[(j_3 + 20)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 16)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 48)]));
          Y_local[(j_3 + 26)] = (Y_local[(j_3 + 26)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 400)] * B_shared[((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3)]));
          Y_local[(j_3 + 32)] = (Y_local[(j_3 + 32)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 400)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 16)]));
          Y_local[(j_3 + 38)] = (Y_local[(j_3 + 38)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 400)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 32)]));
          Y_local[(j_3 + 44)] = (Y_local[(j_3 + 44)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 400)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 48)]));
          Y_local[(j_3 + 50)] = (Y_local[(j_3 + 50)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 784)] * B_shared[((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3)]));
          Y_local[(j_3 + 56)] = (Y_local[(j_3 + 56)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 784)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 16)]));
          Y_local[(j_3 + 62)] = (Y_local[(j_3 + 62)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 784)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 32)]));
          Y_local[(j_3 + 68)] = (Y_local[(j_3 + 68)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 784)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 48)]));
          Y_local[(j_3 + 74)] = (Y_local[(j_3 + 74)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 1168)] * B_shared[((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3)]));
          Y_local[(j_3 + 80)] = (Y_local[(j_3 + 80)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 1168)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 16)]));
          Y_local[(j_3 + 86)] = (Y_local[(j_3 + 86)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 1168)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 32)]));
          Y_local[(j_3 + 92)] = (Y_local[(j_3 + 92)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 1168)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 48)]));
          Y_local[(j_3 + 4)] = (Y_local[(j_3 + 4)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 32)] * B_shared[((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3)]));
          Y_local[(j_3 + 10)] = (Y_local[(j_3 + 10)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 32)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 16)]));
          Y_local[(j_3 + 16)] = (Y_local[(j_3 + 16)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 32)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 32)]));
          Y_local[(j_3 + 22)] = (Y_local[(j_3 + 22)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 32)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 48)]));
          Y_local[(j_3 + 28)] = (Y_local[(j_3 + 28)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 416)] * B_shared[((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3)]));
          Y_local[(j_3 + 34)] = (Y_local[(j_3 + 34)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 416)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 16)]));
          Y_local[(j_3 + 40)] = (Y_local[(j_3 + 40)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 416)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 32)]));
          Y_local[(j_3 + 46)] = (Y_local[(j_3 + 46)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 416)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 48)]));
          Y_local[(j_3 + 52)] = (Y_local[(j_3 + 52)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 800)] * B_shared[((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3)]));
          Y_local[(j_3 + 58)] = (Y_local[(j_3 + 58)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 800)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 16)]));
          Y_local[(j_3 + 64)] = (Y_local[(j_3 + 64)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 800)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 32)]));
          Y_local[(j_3 + 70)] = (Y_local[(j_3 + 70)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 800)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 48)]));
          Y_local[(j_3 + 76)] = (Y_local[(j_3 + 76)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 1184)] * B_shared[((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3)]));
          Y_local[(j_3 + 82)] = (Y_local[(j_3 + 82)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 1184)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 16)]));
          Y_local[(j_3 + 88)] = (Y_local[(j_3 + 88)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 1184)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 32)]));
          Y_local[(j_3 + 94)] = (Y_local[(j_3 + 94)] + (A_shared[((((((k_0_fused & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 48)) + (k_1 * 4)) + k_2) + 1184)] * B_shared[(((((((k_0_fused & 3) * 1024) + (k_1 * 256)) + (k_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 48)]));
        }
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  for (int k_1_1 = 0; k_1_1 < 4; ++k_1_1) {
    for (int j_3_1 = 0; j_3_1 < 2; ++j_3_1) {
      for (int k_2_1 = 0; k_2_1 < 4; ++k_2_1) {
        Y_local[j_3_1] = (Y_local[j_3_1] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1536)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1024)]));
        Y_local[(j_3_1 + 6)] = (Y_local[(j_3_1 + 6)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1536)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1040)]));
        Y_local[(j_3_1 + 12)] = (Y_local[(j_3_1 + 12)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1536)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1056)]));
        Y_local[(j_3_1 + 18)] = (Y_local[(j_3_1 + 18)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1536)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1072)]));
        Y_local[(j_3_1 + 24)] = (Y_local[(j_3_1 + 24)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1920)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1024)]));
        Y_local[(j_3_1 + 30)] = (Y_local[(j_3_1 + 30)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1920)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1040)]));
        Y_local[(j_3_1 + 36)] = (Y_local[(j_3_1 + 36)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1920)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1056)]));
        Y_local[(j_3_1 + 42)] = (Y_local[(j_3_1 + 42)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1920)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1072)]));
        Y_local[(j_3_1 + 48)] = (Y_local[(j_3_1 + 48)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2304)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1024)]));
        Y_local[(j_3_1 + 54)] = (Y_local[(j_3_1 + 54)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2304)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1040)]));
        Y_local[(j_3_1 + 60)] = (Y_local[(j_3_1 + 60)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2304)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1056)]));
        Y_local[(j_3_1 + 66)] = (Y_local[(j_3_1 + 66)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2304)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1072)]));
        Y_local[(j_3_1 + 72)] = (Y_local[(j_3_1 + 72)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2688)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1024)]));
        Y_local[(j_3_1 + 78)] = (Y_local[(j_3_1 + 78)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2688)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1040)]));
        Y_local[(j_3_1 + 84)] = (Y_local[(j_3_1 + 84)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2688)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1056)]));
        Y_local[(j_3_1 + 90)] = (Y_local[(j_3_1 + 90)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2688)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1072)]));
        Y_local[(j_3_1 + 2)] = (Y_local[(j_3_1 + 2)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1552)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1024)]));
        Y_local[(j_3_1 + 8)] = (Y_local[(j_3_1 + 8)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1552)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1040)]));
        Y_local[(j_3_1 + 14)] = (Y_local[(j_3_1 + 14)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1552)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1056)]));
        Y_local[(j_3_1 + 20)] = (Y_local[(j_3_1 + 20)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1552)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1072)]));
        Y_local[(j_3_1 + 26)] = (Y_local[(j_3_1 + 26)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1936)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1024)]));
        Y_local[(j_3_1 + 32)] = (Y_local[(j_3_1 + 32)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1936)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1040)]));
        Y_local[(j_3_1 + 38)] = (Y_local[(j_3_1 + 38)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1936)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1056)]));
        Y_local[(j_3_1 + 44)] = (Y_local[(j_3_1 + 44)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1936)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1072)]));
        Y_local[(j_3_1 + 50)] = (Y_local[(j_3_1 + 50)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2320)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1024)]));
        Y_local[(j_3_1 + 56)] = (Y_local[(j_3_1 + 56)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2320)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1040)]));
        Y_local[(j_3_1 + 62)] = (Y_local[(j_3_1 + 62)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2320)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1056)]));
        Y_local[(j_3_1 + 68)] = (Y_local[(j_3_1 + 68)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2320)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1072)]));
        Y_local[(j_3_1 + 74)] = (Y_local[(j_3_1 + 74)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2704)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1024)]));
        Y_local[(j_3_1 + 80)] = (Y_local[(j_3_1 + 80)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2704)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1040)]));
        Y_local[(j_3_1 + 86)] = (Y_local[(j_3_1 + 86)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2704)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1056)]));
        Y_local[(j_3_1 + 92)] = (Y_local[(j_3_1 + 92)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2704)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1072)]));
        Y_local[(j_3_1 + 4)] = (Y_local[(j_3_1 + 4)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1568)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1024)]));
        Y_local[(j_3_1 + 10)] = (Y_local[(j_3_1 + 10)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1568)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1040)]));
        Y_local[(j_3_1 + 16)] = (Y_local[(j_3_1 + 16)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1568)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1056)]));
        Y_local[(j_3_1 + 22)] = (Y_local[(j_3_1 + 22)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1568)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1072)]));
        Y_local[(j_3_1 + 28)] = (Y_local[(j_3_1 + 28)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1952)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1024)]));
        Y_local[(j_3_1 + 34)] = (Y_local[(j_3_1 + 34)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1952)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1040)]));
        Y_local[(j_3_1 + 40)] = (Y_local[(j_3_1 + 40)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1952)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1056)]));
        Y_local[(j_3_1 + 46)] = (Y_local[(j_3_1 + 46)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 1952)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1072)]));
        Y_local[(j_3_1 + 52)] = (Y_local[(j_3_1 + 52)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2336)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1024)]));
        Y_local[(j_3_1 + 58)] = (Y_local[(j_3_1 + 58)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2336)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1040)]));
        Y_local[(j_3_1 + 64)] = (Y_local[(j_3_1 + 64)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2336)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1056)]));
        Y_local[(j_3_1 + 70)] = (Y_local[(j_3_1 + 70)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2336)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1072)]));
        Y_local[(j_3_1 + 76)] = (Y_local[(j_3_1 + 76)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2720)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1024)]));
        Y_local[(j_3_1 + 82)] = (Y_local[(j_3_1 + 82)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2720)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1040)]));
        Y_local[(j_3_1 + 88)] = (Y_local[(j_3_1 + 88)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2720)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1056)]));
        Y_local[(j_3_1 + 94)] = (Y_local[(j_3_1 + 94)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_1 * 4)) + k_2_1) + 2720)] * B_shared[(((((k_1_1 * 256) + (k_2_1 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_1) + 1072)]));
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  for (int k_1_2 = 0; k_1_2 < 4; ++k_1_2) {
    for (int j_3_2 = 0; j_3_2 < 2; ++j_3_2) {
      for (int k_2_2 = 0; k_2_2 < 4; ++k_2_2) {
        Y_local[j_3_2] = (Y_local[j_3_2] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3072)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2048)]));
        Y_local[(j_3_2 + 6)] = (Y_local[(j_3_2 + 6)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3072)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2064)]));
        Y_local[(j_3_2 + 12)] = (Y_local[(j_3_2 + 12)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3072)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2080)]));
        Y_local[(j_3_2 + 18)] = (Y_local[(j_3_2 + 18)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3072)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2096)]));
        Y_local[(j_3_2 + 24)] = (Y_local[(j_3_2 + 24)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3456)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2048)]));
        Y_local[(j_3_2 + 30)] = (Y_local[(j_3_2 + 30)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3456)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2064)]));
        Y_local[(j_3_2 + 36)] = (Y_local[(j_3_2 + 36)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3456)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2080)]));
        Y_local[(j_3_2 + 42)] = (Y_local[(j_3_2 + 42)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3456)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2096)]));
        Y_local[(j_3_2 + 48)] = (Y_local[(j_3_2 + 48)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3840)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2048)]));
        Y_local[(j_3_2 + 54)] = (Y_local[(j_3_2 + 54)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3840)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2064)]));
        Y_local[(j_3_2 + 60)] = (Y_local[(j_3_2 + 60)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3840)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2080)]));
        Y_local[(j_3_2 + 66)] = (Y_local[(j_3_2 + 66)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3840)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2096)]));
        Y_local[(j_3_2 + 72)] = (Y_local[(j_3_2 + 72)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 4224)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2048)]));
        Y_local[(j_3_2 + 78)] = (Y_local[(j_3_2 + 78)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 4224)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2064)]));
        Y_local[(j_3_2 + 84)] = (Y_local[(j_3_2 + 84)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 4224)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2080)]));
        Y_local[(j_3_2 + 90)] = (Y_local[(j_3_2 + 90)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 4224)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2096)]));
        Y_local[(j_3_2 + 2)] = (Y_local[(j_3_2 + 2)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3088)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2048)]));
        Y_local[(j_3_2 + 8)] = (Y_local[(j_3_2 + 8)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3088)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2064)]));
        Y_local[(j_3_2 + 14)] = (Y_local[(j_3_2 + 14)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3088)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2080)]));
        Y_local[(j_3_2 + 20)] = (Y_local[(j_3_2 + 20)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3088)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2096)]));
        Y_local[(j_3_2 + 26)] = (Y_local[(j_3_2 + 26)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3472)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2048)]));
        Y_local[(j_3_2 + 32)] = (Y_local[(j_3_2 + 32)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3472)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2064)]));
        Y_local[(j_3_2 + 38)] = (Y_local[(j_3_2 + 38)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3472)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2080)]));
        Y_local[(j_3_2 + 44)] = (Y_local[(j_3_2 + 44)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3472)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2096)]));
        Y_local[(j_3_2 + 50)] = (Y_local[(j_3_2 + 50)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3856)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2048)]));
        Y_local[(j_3_2 + 56)] = (Y_local[(j_3_2 + 56)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3856)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2064)]));
        Y_local[(j_3_2 + 62)] = (Y_local[(j_3_2 + 62)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3856)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2080)]));
        Y_local[(j_3_2 + 68)] = (Y_local[(j_3_2 + 68)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3856)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2096)]));
        Y_local[(j_3_2 + 74)] = (Y_local[(j_3_2 + 74)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 4240)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2048)]));
        Y_local[(j_3_2 + 80)] = (Y_local[(j_3_2 + 80)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 4240)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2064)]));
        Y_local[(j_3_2 + 86)] = (Y_local[(j_3_2 + 86)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 4240)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2080)]));
        Y_local[(j_3_2 + 92)] = (Y_local[(j_3_2 + 92)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 4240)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2096)]));
        Y_local[(j_3_2 + 4)] = (Y_local[(j_3_2 + 4)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3104)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2048)]));
        Y_local[(j_3_2 + 10)] = (Y_local[(j_3_2 + 10)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3104)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2064)]));
        Y_local[(j_3_2 + 16)] = (Y_local[(j_3_2 + 16)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3104)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2080)]));
        Y_local[(j_3_2 + 22)] = (Y_local[(j_3_2 + 22)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3104)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2096)]));
        Y_local[(j_3_2 + 28)] = (Y_local[(j_3_2 + 28)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3488)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2048)]));
        Y_local[(j_3_2 + 34)] = (Y_local[(j_3_2 + 34)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3488)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2064)]));
        Y_local[(j_3_2 + 40)] = (Y_local[(j_3_2 + 40)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3488)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2080)]));
        Y_local[(j_3_2 + 46)] = (Y_local[(j_3_2 + 46)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3488)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2096)]));
        Y_local[(j_3_2 + 52)] = (Y_local[(j_3_2 + 52)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3872)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2048)]));
        Y_local[(j_3_2 + 58)] = (Y_local[(j_3_2 + 58)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3872)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2064)]));
        Y_local[(j_3_2 + 64)] = (Y_local[(j_3_2 + 64)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3872)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2080)]));
        Y_local[(j_3_2 + 70)] = (Y_local[(j_3_2 + 70)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 3872)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2096)]));
        Y_local[(j_3_2 + 76)] = (Y_local[(j_3_2 + 76)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 4256)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2048)]));
        Y_local[(j_3_2 + 82)] = (Y_local[(j_3_2 + 82)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 4256)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2064)]));
        Y_local[(j_3_2 + 88)] = (Y_local[(j_3_2 + 88)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 4256)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2080)]));
        Y_local[(j_3_2 + 94)] = (Y_local[(j_3_2 + 94)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_2 * 4)) + k_2_2) + 4256)] * B_shared[(((((k_1_2 * 256) + (k_2_2 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_2) + 2096)]));
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int k_1_3 = 0; k_1_3 < 4; ++k_1_3) {
    for (int j_3_3 = 0; j_3_3 < 2; ++j_3_3) {
      for (int k_2_3 = 0; k_2_3 < 4; ++k_2_3) {
        Y_local[j_3_3] = (Y_local[j_3_3] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4608)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3072)]));
        Y_local[(j_3_3 + 6)] = (Y_local[(j_3_3 + 6)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4608)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3088)]));
        Y_local[(j_3_3 + 12)] = (Y_local[(j_3_3 + 12)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4608)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3104)]));
        Y_local[(j_3_3 + 18)] = (Y_local[(j_3_3 + 18)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4608)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3120)]));
        Y_local[(j_3_3 + 24)] = (Y_local[(j_3_3 + 24)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4992)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3072)]));
        Y_local[(j_3_3 + 30)] = (Y_local[(j_3_3 + 30)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4992)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3088)]));
        Y_local[(j_3_3 + 36)] = (Y_local[(j_3_3 + 36)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4992)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3104)]));
        Y_local[(j_3_3 + 42)] = (Y_local[(j_3_3 + 42)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4992)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3120)]));
        Y_local[(j_3_3 + 48)] = (Y_local[(j_3_3 + 48)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5376)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3072)]));
        Y_local[(j_3_3 + 54)] = (Y_local[(j_3_3 + 54)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5376)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3088)]));
        Y_local[(j_3_3 + 60)] = (Y_local[(j_3_3 + 60)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5376)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3104)]));
        Y_local[(j_3_3 + 66)] = (Y_local[(j_3_3 + 66)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5376)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3120)]));
        Y_local[(j_3_3 + 72)] = (Y_local[(j_3_3 + 72)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5760)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3072)]));
        Y_local[(j_3_3 + 78)] = (Y_local[(j_3_3 + 78)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5760)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3088)]));
        Y_local[(j_3_3 + 84)] = (Y_local[(j_3_3 + 84)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5760)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3104)]));
        Y_local[(j_3_3 + 90)] = (Y_local[(j_3_3 + 90)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5760)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3120)]));
        Y_local[(j_3_3 + 2)] = (Y_local[(j_3_3 + 2)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4624)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3072)]));
        Y_local[(j_3_3 + 8)] = (Y_local[(j_3_3 + 8)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4624)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3088)]));
        Y_local[(j_3_3 + 14)] = (Y_local[(j_3_3 + 14)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4624)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3104)]));
        Y_local[(j_3_3 + 20)] = (Y_local[(j_3_3 + 20)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4624)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3120)]));
        Y_local[(j_3_3 + 26)] = (Y_local[(j_3_3 + 26)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5008)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3072)]));
        Y_local[(j_3_3 + 32)] = (Y_local[(j_3_3 + 32)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5008)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3088)]));
        Y_local[(j_3_3 + 38)] = (Y_local[(j_3_3 + 38)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5008)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3104)]));
        Y_local[(j_3_3 + 44)] = (Y_local[(j_3_3 + 44)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5008)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3120)]));
        Y_local[(j_3_3 + 50)] = (Y_local[(j_3_3 + 50)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5392)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3072)]));
        Y_local[(j_3_3 + 56)] = (Y_local[(j_3_3 + 56)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5392)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3088)]));
        Y_local[(j_3_3 + 62)] = (Y_local[(j_3_3 + 62)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5392)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3104)]));
        Y_local[(j_3_3 + 68)] = (Y_local[(j_3_3 + 68)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5392)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3120)]));
        Y_local[(j_3_3 + 74)] = (Y_local[(j_3_3 + 74)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5776)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3072)]));
        Y_local[(j_3_3 + 80)] = (Y_local[(j_3_3 + 80)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5776)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3088)]));
        Y_local[(j_3_3 + 86)] = (Y_local[(j_3_3 + 86)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5776)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3104)]));
        Y_local[(j_3_3 + 92)] = (Y_local[(j_3_3 + 92)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5776)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3120)]));
        Y_local[(j_3_3 + 4)] = (Y_local[(j_3_3 + 4)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4640)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3072)]));
        Y_local[(j_3_3 + 10)] = (Y_local[(j_3_3 + 10)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4640)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3088)]));
        Y_local[(j_3_3 + 16)] = (Y_local[(j_3_3 + 16)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4640)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3104)]));
        Y_local[(j_3_3 + 22)] = (Y_local[(j_3_3 + 22)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 4640)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3120)]));
        Y_local[(j_3_3 + 28)] = (Y_local[(j_3_3 + 28)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5024)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3072)]));
        Y_local[(j_3_3 + 34)] = (Y_local[(j_3_3 + 34)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5024)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3088)]));
        Y_local[(j_3_3 + 40)] = (Y_local[(j_3_3 + 40)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5024)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3104)]));
        Y_local[(j_3_3 + 46)] = (Y_local[(j_3_3 + 46)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5024)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3120)]));
        Y_local[(j_3_3 + 52)] = (Y_local[(j_3_3 + 52)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5408)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3072)]));
        Y_local[(j_3_3 + 58)] = (Y_local[(j_3_3 + 58)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5408)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3088)]));
        Y_local[(j_3_3 + 64)] = (Y_local[(j_3_3 + 64)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5408)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3104)]));
        Y_local[(j_3_3 + 70)] = (Y_local[(j_3_3 + 70)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5408)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3120)]));
        Y_local[(j_3_3 + 76)] = (Y_local[(j_3_3 + 76)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5792)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3072)]));
        Y_local[(j_3_3 + 82)] = (Y_local[(j_3_3 + 82)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5792)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3088)]));
        Y_local[(j_3_3 + 88)] = (Y_local[(j_3_3 + 88)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5792)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3104)]));
        Y_local[(j_3_3 + 94)] = (Y_local[(j_3_3 + 94)] + (A_shared[(((((((int)threadIdx.x) >> 3) * 48) + (k_1_3 * 4)) + k_2_3) + 5792)] * B_shared[(((((k_1_3 * 256) + (k_2_3 * 64)) + ((((int)threadIdx.x) & 7) * 2)) + j_3_3) + 3120)]));
      }
    }
  }
  for (int ax0 = 0; ax0 < 3; ++ax0) {
    Y[((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2))] = Y_local[(ax0 * 2)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 16)] = Y_local[((ax0 * 2) + 6)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 32)] = Y_local[((ax0 * 2) + 12)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 48)] = Y_local[((ax0 * 2) + 18)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 36864)] = Y_local[((ax0 * 2) + 24)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 36880)] = Y_local[((ax0 * 2) + 30)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 36896)] = Y_local[((ax0 * 2) + 36)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 36912)] = Y_local[((ax0 * 2) + 42)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 73728)] = Y_local[((ax0 * 2) + 48)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 73744)] = Y_local[((ax0 * 2) + 54)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 73760)] = Y_local[((ax0 * 2) + 60)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 73776)] = Y_local[((ax0 * 2) + 66)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 110592)] = Y_local[((ax0 * 2) + 72)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 110608)] = Y_local[((ax0 * 2) + 78)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 110624)] = Y_local[((ax0 * 2) + 84)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 110640)] = Y_local[((ax0 * 2) + 90)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 1)] = Y_local[((ax0 * 2) + 1)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 17)] = Y_local[((ax0 * 2) + 7)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 33)] = Y_local[((ax0 * 2) + 13)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 49)] = Y_local[((ax0 * 2) + 19)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 36865)] = Y_local[((ax0 * 2) + 25)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 36881)] = Y_local[((ax0 * 2) + 31)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 36897)] = Y_local[((ax0 * 2) + 37)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 36913)] = Y_local[((ax0 * 2) + 43)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 73729)] = Y_local[((ax0 * 2) + 49)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 73745)] = Y_local[((ax0 * 2) + 55)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 73761)] = Y_local[((ax0 * 2) + 61)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 73777)] = Y_local[((ax0 * 2) + 67)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 110593)] = Y_local[((ax0 * 2) + 73)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 110609)] = Y_local[((ax0 * 2) + 79)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 110625)] = Y_local[((ax0 * 2) + 85)];
    Y[(((((((((int)blockIdx.x) / 24) * 147456) + ((((int)threadIdx.x) >> 3) * 4608)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 24) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 110641)] = Y_local[((ax0 * 2) + 91)];
  }
}


