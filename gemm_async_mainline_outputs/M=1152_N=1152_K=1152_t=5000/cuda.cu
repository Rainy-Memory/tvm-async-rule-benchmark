
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[54];
  __shared__ float A_shared[864];
  __shared__ float B_shared[2304];
  Y_local[0] = 0.000000e+00f;
  Y_local[2] = 0.000000e+00f;
  Y_local[4] = 0.000000e+00f;
  Y_local[6] = 0.000000e+00f;
  Y_local[8] = 0.000000e+00f;
  Y_local[10] = 0.000000e+00f;
  Y_local[12] = 0.000000e+00f;
  Y_local[14] = 0.000000e+00f;
  Y_local[16] = 0.000000e+00f;
  Y_local[18] = 0.000000e+00f;
  Y_local[20] = 0.000000e+00f;
  Y_local[22] = 0.000000e+00f;
  Y_local[24] = 0.000000e+00f;
  Y_local[26] = 0.000000e+00f;
  Y_local[28] = 0.000000e+00f;
  Y_local[30] = 0.000000e+00f;
  Y_local[32] = 0.000000e+00f;
  Y_local[34] = 0.000000e+00f;
  Y_local[36] = 0.000000e+00f;
  Y_local[38] = 0.000000e+00f;
  Y_local[40] = 0.000000e+00f;
  Y_local[42] = 0.000000e+00f;
  Y_local[44] = 0.000000e+00f;
  Y_local[46] = 0.000000e+00f;
  Y_local[48] = 0.000000e+00f;
  Y_local[50] = 0.000000e+00f;
  Y_local[52] = 0.000000e+00f;
  Y_local[1] = 0.000000e+00f;
  Y_local[3] = 0.000000e+00f;
  Y_local[5] = 0.000000e+00f;
  Y_local[7] = 0.000000e+00f;
  Y_local[9] = 0.000000e+00f;
  Y_local[11] = 0.000000e+00f;
  Y_local[13] = 0.000000e+00f;
  Y_local[15] = 0.000000e+00f;
  Y_local[17] = 0.000000e+00f;
  Y_local[19] = 0.000000e+00f;
  Y_local[21] = 0.000000e+00f;
  Y_local[23] = 0.000000e+00f;
  Y_local[25] = 0.000000e+00f;
  Y_local[27] = 0.000000e+00f;
  Y_local[29] = 0.000000e+00f;
  Y_local[31] = 0.000000e+00f;
  Y_local[33] = 0.000000e+00f;
  Y_local[35] = 0.000000e+00f;
  Y_local[37] = 0.000000e+00f;
  Y_local[39] = 0.000000e+00f;
  Y_local[41] = 0.000000e+00f;
  Y_local[43] = 0.000000e+00f;
  Y_local[45] = 0.000000e+00f;
  Y_local[47] = 0.000000e+00f;
  Y_local[49] = 0.000000e+00f;
  Y_local[51] = 0.000000e+00f;
  Y_local[53] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((int)threadIdx.x)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (((int)threadIdx.x) & 7)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((int)threadIdx.x) + 64)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (((int)threadIdx.x) & 7)) + 9216))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((int)threadIdx.x) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (((int)threadIdx.x) & 7)) + 18432))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((int)threadIdx.x) + 192)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (((int)threadIdx.x) & 7)) + 27648))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((int)threadIdx.x) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (((int)threadIdx.x) & 7)) + 36864))), "n"(4)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((int)threadIdx.x) / 24) * 1152) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) % 24) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((((int)threadIdx.x) * 4) + 256) / 96) * 1152) + ((((int)blockIdx.x) % 12) * 96)) + (((((int)threadIdx.x) * 4) + 64) % 96)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((((int)threadIdx.x) * 4) + 512) / 96) * 1152) + ((((int)blockIdx.x) % 12) * 96)) + (((((int)threadIdx.x) * 4) + 32) % 96)))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((int)threadIdx.x) + 288)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (((int)threadIdx.x) & 7)) + 8))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((int)threadIdx.x) + 352)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (((int)threadIdx.x) & 7)) + 9224))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((int)threadIdx.x) + 416)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (((int)threadIdx.x) & 7)) + 18440))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((int)threadIdx.x) + 480)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (((int)threadIdx.x) & 7)) + 27656))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((int)threadIdx.x) + 544)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (((int)threadIdx.x) & 7)) + 36872))), "n"(4)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) / 24) * 1152) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) % 24) * 4)) + 9216))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((((int)threadIdx.x) * 4) + 256) / 96) * 1152) + ((((int)blockIdx.x) % 12) * 96)) + (((((int)threadIdx.x) * 4) + 64) % 96)) + 9216))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((((int)threadIdx.x) * 4) + 512) / 96) * 1152) + ((((int)blockIdx.x) % 12) * 96)) + (((((int)threadIdx.x) * 4) + 32) % 96)) + 9216))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int k_0_fused = 0; k_0_fused < 142; ++k_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((k_0_fused + 2) % 3) * 288) + ((int)threadIdx.x))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (k_0_fused * 8)) + (((int)threadIdx.x) & 7)) + 16))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 2) % 3) * 288) + ((int)threadIdx.x)) + 64)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (k_0_fused * 8)) + (((int)threadIdx.x) & 7)) + 9232))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 2) % 3) * 288) + ((int)threadIdx.x)) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (k_0_fused * 8)) + (((int)threadIdx.x) & 7)) + 18448))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 2) % 3) * 288) + ((int)threadIdx.x)) + 192)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (k_0_fused * 8)) + (((int)threadIdx.x) & 7)) + 27664))), "n"(4)
    );
  }
    if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 2) % 3) * 288) + ((int)threadIdx.x)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 3) * 1152)) + (k_0_fused * 8)) + (((int)threadIdx.x) & 7)) + 36880))), "n"(4)
    );
  }
    }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((k_0_fused + 2) % 3) * 768) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 9216) + ((((int)threadIdx.x) / 24) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) % 24) * 4)) + 18432))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 768) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 9216) + ((((((int)threadIdx.x) * 4) + 256) / 96) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + (((((int)threadIdx.x) * 4) + 64) % 96)) + 18432))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 768) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 9216) + ((((((int)threadIdx.x) * 4) + 512) / 96) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + (((((int)threadIdx.x) * 4) + 32) % 96)) + 18432))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    Y_local[0] = (Y_local[0] + (A_shared[(((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8))] * B_shared[(((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2))]));
    Y_local[2] = (Y_local[2] + (A_shared[(((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8))] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    Y_local[4] = (Y_local[4] + (A_shared[(((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8))] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 32)] * B_shared[(((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2))]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 32)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 32)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 64)] * B_shared[(((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2))]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 64)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    Y_local[16] = (Y_local[16] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 64)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    Y_local[18] = (Y_local[18] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 96)] * B_shared[(((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2))]));
    Y_local[20] = (Y_local[20] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 96)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    Y_local[22] = (Y_local[22] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 96)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    Y_local[24] = (Y_local[24] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 128)] * B_shared[(((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2))]));
    Y_local[26] = (Y_local[26] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 128)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    Y_local[28] = (Y_local[28] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 128)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    Y_local[30] = (Y_local[30] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 160)] * B_shared[(((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2))]));
    Y_local[32] = (Y_local[32] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 160)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    Y_local[34] = (Y_local[34] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 160)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    Y_local[36] = (Y_local[36] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 192)] * B_shared[(((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2))]));
    Y_local[38] = (Y_local[38] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 192)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    Y_local[40] = (Y_local[40] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 192)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    Y_local[42] = (Y_local[42] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 224)] * B_shared[(((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2))]));
    Y_local[44] = (Y_local[44] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 224)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    Y_local[46] = (Y_local[46] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 224)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    Y_local[48] = (Y_local[48] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 256)] * B_shared[(((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2))]));
    Y_local[50] = (Y_local[50] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 256)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    Y_local[52] = (Y_local[52] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 256)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    Y_local[1] = (Y_local[1] + (A_shared[(((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8))] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    Y_local[3] = (Y_local[3] + (A_shared[(((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8))] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    Y_local[5] = (Y_local[5] + (A_shared[(((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8))] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 32)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 32)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 32)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 64)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 64)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    Y_local[17] = (Y_local[17] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 64)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    Y_local[19] = (Y_local[19] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 96)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    Y_local[21] = (Y_local[21] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 96)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    Y_local[23] = (Y_local[23] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 96)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    Y_local[25] = (Y_local[25] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 128)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    Y_local[27] = (Y_local[27] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 128)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    Y_local[29] = (Y_local[29] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 128)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    Y_local[31] = (Y_local[31] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 160)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    Y_local[33] = (Y_local[33] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 160)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    Y_local[35] = (Y_local[35] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 160)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    Y_local[37] = (Y_local[37] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 192)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    Y_local[39] = (Y_local[39] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 192)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    Y_local[41] = (Y_local[41] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 192)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    Y_local[43] = (Y_local[43] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 224)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    Y_local[45] = (Y_local[45] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 224)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    Y_local[47] = (Y_local[47] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 224)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    Y_local[49] = (Y_local[49] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 256)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    Y_local[51] = (Y_local[51] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 256)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    Y_local[53] = (Y_local[53] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 256)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 1)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 1)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 1)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 33)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 33)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 33)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 65)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 65)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    Y_local[16] = (Y_local[16] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 65)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    Y_local[18] = (Y_local[18] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 97)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    Y_local[20] = (Y_local[20] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 97)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    Y_local[22] = (Y_local[22] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 97)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    Y_local[24] = (Y_local[24] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 129)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    Y_local[26] = (Y_local[26] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 129)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    Y_local[28] = (Y_local[28] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 129)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    Y_local[30] = (Y_local[30] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 161)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    Y_local[32] = (Y_local[32] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 161)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    Y_local[34] = (Y_local[34] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 161)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    Y_local[36] = (Y_local[36] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 193)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    Y_local[38] = (Y_local[38] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 193)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    Y_local[40] = (Y_local[40] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 193)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    Y_local[42] = (Y_local[42] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 225)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    Y_local[44] = (Y_local[44] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 225)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    Y_local[46] = (Y_local[46] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 225)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    Y_local[48] = (Y_local[48] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 257)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    Y_local[50] = (Y_local[50] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 257)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    Y_local[52] = (Y_local[52] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 257)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 1)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 1)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 1)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 33)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 33)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 33)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 65)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 65)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    Y_local[17] = (Y_local[17] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 65)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    Y_local[19] = (Y_local[19] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 97)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    Y_local[21] = (Y_local[21] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 97)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    Y_local[23] = (Y_local[23] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 97)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    Y_local[25] = (Y_local[25] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 129)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    Y_local[27] = (Y_local[27] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 129)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    Y_local[29] = (Y_local[29] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 129)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    Y_local[31] = (Y_local[31] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 161)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    Y_local[33] = (Y_local[33] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 161)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    Y_local[35] = (Y_local[35] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 161)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    Y_local[37] = (Y_local[37] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 193)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    Y_local[39] = (Y_local[39] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 193)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    Y_local[41] = (Y_local[41] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 193)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    Y_local[43] = (Y_local[43] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 225)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    Y_local[45] = (Y_local[45] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 225)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    Y_local[47] = (Y_local[47] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 225)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    Y_local[49] = (Y_local[49] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 257)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    Y_local[51] = (Y_local[51] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 257)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    Y_local[53] = (Y_local[53] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 257)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 2)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 2)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 2)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 34)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 34)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 34)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 66)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 66)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    Y_local[16] = (Y_local[16] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 66)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    Y_local[18] = (Y_local[18] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 98)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    Y_local[20] = (Y_local[20] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 98)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    Y_local[22] = (Y_local[22] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 98)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    Y_local[24] = (Y_local[24] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 130)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    Y_local[26] = (Y_local[26] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 130)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    Y_local[28] = (Y_local[28] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 130)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    Y_local[30] = (Y_local[30] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 162)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    Y_local[32] = (Y_local[32] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 162)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    Y_local[34] = (Y_local[34] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 162)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    Y_local[36] = (Y_local[36] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 194)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    Y_local[38] = (Y_local[38] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 194)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    Y_local[40] = (Y_local[40] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 194)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    Y_local[42] = (Y_local[42] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 226)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    Y_local[44] = (Y_local[44] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 226)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    Y_local[46] = (Y_local[46] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 226)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    Y_local[48] = (Y_local[48] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 258)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    Y_local[50] = (Y_local[50] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 258)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    Y_local[52] = (Y_local[52] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 258)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 2)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 2)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 2)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 34)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 34)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 34)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 66)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 66)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    Y_local[17] = (Y_local[17] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 66)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    Y_local[19] = (Y_local[19] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 98)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    Y_local[21] = (Y_local[21] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 98)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    Y_local[23] = (Y_local[23] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 98)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    Y_local[25] = (Y_local[25] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 130)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    Y_local[27] = (Y_local[27] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 130)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    Y_local[29] = (Y_local[29] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 130)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    Y_local[31] = (Y_local[31] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 162)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    Y_local[33] = (Y_local[33] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 162)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    Y_local[35] = (Y_local[35] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 162)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    Y_local[37] = (Y_local[37] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 194)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    Y_local[39] = (Y_local[39] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 194)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    Y_local[41] = (Y_local[41] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 194)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    Y_local[43] = (Y_local[43] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 226)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    Y_local[45] = (Y_local[45] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 226)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    Y_local[47] = (Y_local[47] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 226)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    Y_local[49] = (Y_local[49] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 258)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    Y_local[51] = (Y_local[51] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 258)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    Y_local[53] = (Y_local[53] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 258)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 3)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 3)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 3)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 35)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 35)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 35)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 67)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 67)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    Y_local[16] = (Y_local[16] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 67)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    Y_local[18] = (Y_local[18] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 99)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    Y_local[20] = (Y_local[20] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 99)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    Y_local[22] = (Y_local[22] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 99)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    Y_local[24] = (Y_local[24] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 131)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    Y_local[26] = (Y_local[26] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 131)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    Y_local[28] = (Y_local[28] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 131)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    Y_local[30] = (Y_local[30] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 163)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    Y_local[32] = (Y_local[32] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 163)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    Y_local[34] = (Y_local[34] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 163)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    Y_local[36] = (Y_local[36] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 195)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    Y_local[38] = (Y_local[38] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 195)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    Y_local[40] = (Y_local[40] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 195)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    Y_local[42] = (Y_local[42] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 227)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    Y_local[44] = (Y_local[44] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 227)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    Y_local[46] = (Y_local[46] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 227)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    Y_local[48] = (Y_local[48] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 259)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    Y_local[50] = (Y_local[50] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 259)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    Y_local[52] = (Y_local[52] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 259)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 3)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 3)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 3)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 35)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 35)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 35)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 67)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 67)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    Y_local[17] = (Y_local[17] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 67)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    Y_local[19] = (Y_local[19] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 99)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    Y_local[21] = (Y_local[21] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 99)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    Y_local[23] = (Y_local[23] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 99)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    Y_local[25] = (Y_local[25] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 131)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    Y_local[27] = (Y_local[27] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 131)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    Y_local[29] = (Y_local[29] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 131)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    Y_local[31] = (Y_local[31] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 163)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    Y_local[33] = (Y_local[33] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 163)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    Y_local[35] = (Y_local[35] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 163)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    Y_local[37] = (Y_local[37] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 195)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    Y_local[39] = (Y_local[39] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 195)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    Y_local[41] = (Y_local[41] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 195)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    Y_local[43] = (Y_local[43] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 227)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    Y_local[45] = (Y_local[45] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 227)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    Y_local[47] = (Y_local[47] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 227)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    Y_local[49] = (Y_local[49] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 259)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    Y_local[51] = (Y_local[51] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 259)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    Y_local[53] = (Y_local[53] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 259)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 4)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 4)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 4)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 36)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 36)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 36)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 68)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 68)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    Y_local[16] = (Y_local[16] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 68)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    Y_local[18] = (Y_local[18] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 100)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    Y_local[20] = (Y_local[20] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 100)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    Y_local[22] = (Y_local[22] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 100)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    Y_local[24] = (Y_local[24] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 132)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    Y_local[26] = (Y_local[26] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 132)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    Y_local[28] = (Y_local[28] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 132)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    Y_local[30] = (Y_local[30] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 164)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    Y_local[32] = (Y_local[32] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 164)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    Y_local[34] = (Y_local[34] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 164)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    Y_local[36] = (Y_local[36] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 196)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    Y_local[38] = (Y_local[38] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 196)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    Y_local[40] = (Y_local[40] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 196)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    Y_local[42] = (Y_local[42] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 228)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    Y_local[44] = (Y_local[44] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 228)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    Y_local[46] = (Y_local[46] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 228)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    Y_local[48] = (Y_local[48] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 260)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    Y_local[50] = (Y_local[50] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 260)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    Y_local[52] = (Y_local[52] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 260)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 4)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 4)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 4)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 36)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 36)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 36)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 68)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 68)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    Y_local[17] = (Y_local[17] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 68)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    Y_local[19] = (Y_local[19] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 100)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    Y_local[21] = (Y_local[21] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 100)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    Y_local[23] = (Y_local[23] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 100)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    Y_local[25] = (Y_local[25] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 132)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    Y_local[27] = (Y_local[27] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 132)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    Y_local[29] = (Y_local[29] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 132)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    Y_local[31] = (Y_local[31] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 164)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    Y_local[33] = (Y_local[33] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 164)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    Y_local[35] = (Y_local[35] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 164)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    Y_local[37] = (Y_local[37] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 196)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    Y_local[39] = (Y_local[39] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 196)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    Y_local[41] = (Y_local[41] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 196)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    Y_local[43] = (Y_local[43] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 228)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    Y_local[45] = (Y_local[45] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 228)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    Y_local[47] = (Y_local[47] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 228)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    Y_local[49] = (Y_local[49] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 260)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    Y_local[51] = (Y_local[51] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 260)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    Y_local[53] = (Y_local[53] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 260)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 5)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 5)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 5)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 37)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 37)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 37)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 69)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 69)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    Y_local[16] = (Y_local[16] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 69)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    Y_local[18] = (Y_local[18] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 101)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    Y_local[20] = (Y_local[20] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 101)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    Y_local[22] = (Y_local[22] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 101)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    Y_local[24] = (Y_local[24] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 133)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    Y_local[26] = (Y_local[26] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 133)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    Y_local[28] = (Y_local[28] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 133)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    Y_local[30] = (Y_local[30] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 165)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    Y_local[32] = (Y_local[32] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 165)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    Y_local[34] = (Y_local[34] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 165)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    Y_local[36] = (Y_local[36] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 197)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    Y_local[38] = (Y_local[38] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 197)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    Y_local[40] = (Y_local[40] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 197)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    Y_local[42] = (Y_local[42] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 229)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    Y_local[44] = (Y_local[44] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 229)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    Y_local[46] = (Y_local[46] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 229)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    Y_local[48] = (Y_local[48] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 261)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    Y_local[50] = (Y_local[50] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 261)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    Y_local[52] = (Y_local[52] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 261)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 5)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 5)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 5)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 37)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 37)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 37)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 69)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 69)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    Y_local[17] = (Y_local[17] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 69)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    Y_local[19] = (Y_local[19] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 101)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    Y_local[21] = (Y_local[21] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 101)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    Y_local[23] = (Y_local[23] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 101)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    Y_local[25] = (Y_local[25] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 133)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    Y_local[27] = (Y_local[27] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 133)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    Y_local[29] = (Y_local[29] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 133)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    Y_local[31] = (Y_local[31] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 165)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    Y_local[33] = (Y_local[33] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 165)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    Y_local[35] = (Y_local[35] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 165)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    Y_local[37] = (Y_local[37] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 197)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    Y_local[39] = (Y_local[39] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 197)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    Y_local[41] = (Y_local[41] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 197)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    Y_local[43] = (Y_local[43] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 229)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    Y_local[45] = (Y_local[45] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 229)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    Y_local[47] = (Y_local[47] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 229)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    Y_local[49] = (Y_local[49] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 261)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    Y_local[51] = (Y_local[51] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 261)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    Y_local[53] = (Y_local[53] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 261)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 6)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 6)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 6)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 38)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 38)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 38)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 70)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 70)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    Y_local[16] = (Y_local[16] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 70)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    Y_local[18] = (Y_local[18] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 102)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    Y_local[20] = (Y_local[20] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 102)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    Y_local[22] = (Y_local[22] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 102)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    Y_local[24] = (Y_local[24] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 134)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    Y_local[26] = (Y_local[26] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 134)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    Y_local[28] = (Y_local[28] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 134)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    Y_local[30] = (Y_local[30] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 166)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    Y_local[32] = (Y_local[32] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 166)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    Y_local[34] = (Y_local[34] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 166)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    Y_local[36] = (Y_local[36] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 198)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    Y_local[38] = (Y_local[38] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 198)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    Y_local[40] = (Y_local[40] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 198)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    Y_local[42] = (Y_local[42] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 230)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    Y_local[44] = (Y_local[44] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 230)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    Y_local[46] = (Y_local[46] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 230)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    Y_local[48] = (Y_local[48] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 262)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    Y_local[50] = (Y_local[50] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 262)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    Y_local[52] = (Y_local[52] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 262)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 6)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 6)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 6)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 38)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 38)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 38)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 70)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 70)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    Y_local[17] = (Y_local[17] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 70)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    Y_local[19] = (Y_local[19] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 102)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    Y_local[21] = (Y_local[21] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 102)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    Y_local[23] = (Y_local[23] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 102)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    Y_local[25] = (Y_local[25] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 134)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    Y_local[27] = (Y_local[27] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 134)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    Y_local[29] = (Y_local[29] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 134)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    Y_local[31] = (Y_local[31] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 166)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    Y_local[33] = (Y_local[33] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 166)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    Y_local[35] = (Y_local[35] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 166)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    Y_local[37] = (Y_local[37] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 198)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    Y_local[39] = (Y_local[39] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 198)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    Y_local[41] = (Y_local[41] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 198)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    Y_local[43] = (Y_local[43] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 230)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    Y_local[45] = (Y_local[45] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 230)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    Y_local[47] = (Y_local[47] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 230)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    Y_local[49] = (Y_local[49] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 262)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    Y_local[51] = (Y_local[51] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 262)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    Y_local[53] = (Y_local[53] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 262)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 7)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 7)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 7)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 39)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 39)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 39)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 71)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 71)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    Y_local[16] = (Y_local[16] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 71)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    Y_local[18] = (Y_local[18] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 103)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    Y_local[20] = (Y_local[20] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 103)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    Y_local[22] = (Y_local[22] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 103)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    Y_local[24] = (Y_local[24] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 135)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    Y_local[26] = (Y_local[26] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 135)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    Y_local[28] = (Y_local[28] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 135)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    Y_local[30] = (Y_local[30] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 167)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    Y_local[32] = (Y_local[32] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 167)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    Y_local[34] = (Y_local[34] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 167)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    Y_local[36] = (Y_local[36] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 199)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    Y_local[38] = (Y_local[38] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 199)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    Y_local[40] = (Y_local[40] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 199)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    Y_local[42] = (Y_local[42] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 231)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    Y_local[44] = (Y_local[44] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 231)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    Y_local[46] = (Y_local[46] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 231)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    Y_local[48] = (Y_local[48] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 263)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    Y_local[50] = (Y_local[50] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 263)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    Y_local[52] = (Y_local[52] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 263)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 7)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 7)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 7)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 39)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 39)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 39)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 71)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 71)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    Y_local[17] = (Y_local[17] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 71)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    Y_local[19] = (Y_local[19] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 103)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    Y_local[21] = (Y_local[21] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 103)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    Y_local[23] = (Y_local[23] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 103)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    Y_local[25] = (Y_local[25] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 135)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    Y_local[27] = (Y_local[27] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 135)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    Y_local[29] = (Y_local[29] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 135)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    Y_local[31] = (Y_local[31] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 167)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    Y_local[33] = (Y_local[33] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 167)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    Y_local[35] = (Y_local[35] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 167)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    Y_local[37] = (Y_local[37] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 199)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    Y_local[39] = (Y_local[39] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 199)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    Y_local[41] = (Y_local[41] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 199)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    Y_local[43] = (Y_local[43] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 231)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    Y_local[45] = (Y_local[45] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 231)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    Y_local[47] = (Y_local[47] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 231)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    Y_local[49] = (Y_local[49] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 263)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    Y_local[51] = (Y_local[51] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 263)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    Y_local[53] = (Y_local[53] + (A_shared[((((k_0_fused % 3) * 288) + ((((int)threadIdx.x) >> 4) * 8)) + 263)] * B_shared[((((k_0_fused % 3) * 768) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 288)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 288)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 288)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 320)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 320)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 320)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 352)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 352)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 352)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 384)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 384)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 384)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 416)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 416)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 416)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 448)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 448)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 448)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 480)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 480)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 480)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 512)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 512)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 512)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 544)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 544)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 544)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 288)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 288)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 288)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 320)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 320)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 320)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 352)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 352)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 352)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 384)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 384)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 384)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 416)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 416)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 416)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 448)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 448)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 448)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 480)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 480)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 480)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 512)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 512)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 512)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 544)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 544)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 544)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 289)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 289)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 289)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 321)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 321)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 321)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 353)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 353)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 353)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 385)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 385)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 385)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 417)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 417)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 417)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 449)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 449)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 449)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 481)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 481)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 481)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 513)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 513)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 513)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 545)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 545)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 545)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 289)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 289)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 289)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 321)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 321)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 321)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 353)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 353)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 353)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 385)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 385)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 385)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 417)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 417)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 417)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 449)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 449)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 449)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 481)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 481)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 481)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 513)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 513)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 513)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 545)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 545)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 545)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 290)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 290)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 290)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 322)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 322)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 322)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 354)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 354)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 354)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 386)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 386)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 386)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 418)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 418)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 418)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 450)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 450)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 450)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 482)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 482)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 482)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 514)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 514)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 514)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 546)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 546)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 546)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 290)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 290)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 290)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 322)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 322)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 322)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 354)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 354)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 354)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 386)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 386)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 386)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 418)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 418)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 418)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 450)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 450)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 450)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 482)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 482)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 482)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 514)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 514)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 514)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 546)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 546)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 546)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 291)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 291)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 291)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 323)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 323)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 323)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 355)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 355)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 355)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 387)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 387)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 387)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 419)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 419)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 419)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 451)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 451)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 451)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 483)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 483)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 483)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 515)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 515)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 515)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 547)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 547)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 547)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 291)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 291)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 291)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 323)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 323)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 323)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 355)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 355)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 355)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 387)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 387)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 387)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 419)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 419)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 419)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 451)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 451)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 451)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 483)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 483)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 483)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 515)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 515)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 515)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 547)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 547)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 547)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 292)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 292)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 292)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 324)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 324)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 324)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 356)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 356)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 356)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 388)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 388)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 388)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 420)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 420)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 420)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 452)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 452)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 452)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 484)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 484)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 484)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 516)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 516)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 516)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 548)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 548)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 548)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 292)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 292)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 292)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 324)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 324)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 324)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 356)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 356)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 356)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 388)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 388)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 388)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 420)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 420)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 420)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 452)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 452)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 452)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 484)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 484)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 484)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 516)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 516)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 516)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 548)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 548)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 548)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 293)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 293)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 293)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 325)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 325)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 325)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 357)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 357)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 357)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 389)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 389)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 389)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 421)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 421)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 421)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 453)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 453)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 453)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 485)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 485)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 485)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 517)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 517)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 517)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 549)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 549)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 549)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 293)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 293)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 293)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 325)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 325)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 325)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 357)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 357)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 357)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 389)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 389)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 389)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 421)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 421)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 421)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 453)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 453)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 453)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 485)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 485)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 485)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 517)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 517)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 517)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 549)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 549)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 549)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 294)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 294)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 294)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 326)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 326)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 326)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 358)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 358)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 358)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 390)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 390)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 390)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 422)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 422)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 422)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 454)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 454)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 454)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 486)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 486)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 486)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 518)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 518)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 518)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 550)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 550)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 550)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 294)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 294)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 294)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 326)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 326)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 326)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 358)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 358)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 358)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 390)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 390)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 390)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 422)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 422)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 422)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 454)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 454)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 454)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 486)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 486)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 486)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 518)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 518)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 518)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 550)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 550)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 550)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 295)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 295)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 295)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 327)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 327)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 327)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 359)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 359)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 359)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 391)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 391)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 391)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 423)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 423)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 423)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 455)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 455)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 455)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 487)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 487)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 487)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 519)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 519)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 519)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 551)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 551)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 551)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 295)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 295)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 295)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 327)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 327)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 327)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 359)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 359)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 359)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 391)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 391)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 391)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 423)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 423)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 423)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 455)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 455)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 455)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 487)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 487)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 487)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 519)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 519)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 519)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 551)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 551)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 551)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 576)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 576)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 576)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 608)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 608)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 608)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 640)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 640)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 640)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 672)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 672)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 672)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 704)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 704)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 704)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 736)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 736)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 736)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 768)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 768)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 768)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 800)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 800)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 800)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 832)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 832)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 832)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 576)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 576)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 576)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 608)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 608)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 608)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 640)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 640)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 640)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 672)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 672)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 672)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 704)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 704)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 704)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 736)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 736)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 736)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 768)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 768)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 768)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 800)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 800)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 800)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 832)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 832)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 832)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 577)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 577)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 577)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 609)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 609)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 609)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 641)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 641)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 641)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 673)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 673)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 673)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 705)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 705)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 705)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 737)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 737)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 737)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 769)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 769)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 769)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 801)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 801)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 801)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 833)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 833)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 833)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 577)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 577)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 577)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 609)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 609)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 609)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 641)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 641)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 641)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 673)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 673)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 673)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 705)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 705)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 705)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 737)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 737)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 737)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 769)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 769)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 769)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 801)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 801)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 801)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 833)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 833)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 833)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 578)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 578)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 578)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 610)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 610)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 610)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 642)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 642)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 642)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 674)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 674)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 674)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 706)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 706)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 706)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 738)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 738)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 738)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 770)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 770)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 770)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 802)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 802)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 802)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 834)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 834)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 834)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 578)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 578)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 578)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 610)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 610)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 610)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 642)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 642)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 642)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 674)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 674)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 674)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 706)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 706)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 706)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 738)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 738)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 738)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 770)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 770)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 770)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 802)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 802)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 802)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 834)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 834)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 834)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 579)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 579)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 579)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 611)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 611)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 611)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 643)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 643)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 643)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 675)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 675)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 675)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 707)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 707)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 707)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 739)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 739)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 739)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 771)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 771)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 771)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 803)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 803)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 803)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 835)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 835)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 835)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 579)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 579)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 579)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 611)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 611)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 611)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 643)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 643)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 643)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 675)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 675)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 675)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 707)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 707)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 707)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 739)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 739)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 739)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 771)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 771)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 771)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 803)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 803)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 803)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 835)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 835)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 835)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 580)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 580)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 580)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 612)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 612)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 612)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 644)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 644)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 644)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 676)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 676)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 676)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 708)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 708)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 708)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 740)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 740)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 740)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 772)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 772)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 772)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 804)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 804)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 804)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 836)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 836)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 836)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 580)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 580)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 580)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 612)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 612)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 612)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 644)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 644)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 644)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 676)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 676)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 676)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 708)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 708)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 708)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 740)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 740)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 740)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 772)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 772)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 772)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 804)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 804)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 804)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 836)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 836)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 836)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 581)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 581)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 581)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 613)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 613)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 613)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 645)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 645)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 645)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 677)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 677)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 677)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 709)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 709)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 709)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 741)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 741)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 741)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 773)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 773)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 773)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 805)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 805)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 805)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 837)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 837)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 837)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 581)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 581)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 581)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 613)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 613)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 613)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 645)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 645)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 645)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 677)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 677)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 677)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 709)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 709)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 709)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 741)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 741)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 741)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 773)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 773)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 773)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 805)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 805)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 805)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 837)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 837)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 837)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 582)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 582)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 582)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 614)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 614)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 614)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 646)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 646)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 646)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 678)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 678)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 678)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 710)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 710)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 710)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 742)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 742)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 742)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 774)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 774)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 774)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 806)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 806)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 806)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 838)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 838)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 838)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 582)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 582)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 582)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 614)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 614)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 614)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 646)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 646)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 646)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 678)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 678)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 678)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 710)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 710)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 710)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 742)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 742)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 742)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 774)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 774)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 774)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 806)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 806)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 806)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 838)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 838)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 838)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 583)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 583)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 583)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 615)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 615)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 615)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 647)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 647)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  Y_local[16] = (Y_local[16] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 647)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  Y_local[18] = (Y_local[18] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 679)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  Y_local[20] = (Y_local[20] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 679)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  Y_local[22] = (Y_local[22] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 679)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  Y_local[24] = (Y_local[24] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 711)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  Y_local[26] = (Y_local[26] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 711)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  Y_local[28] = (Y_local[28] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 711)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  Y_local[30] = (Y_local[30] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 743)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  Y_local[32] = (Y_local[32] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 743)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  Y_local[34] = (Y_local[34] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 743)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  Y_local[36] = (Y_local[36] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 775)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  Y_local[38] = (Y_local[38] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 775)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  Y_local[40] = (Y_local[40] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 775)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  Y_local[42] = (Y_local[42] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 807)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  Y_local[44] = (Y_local[44] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 807)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  Y_local[46] = (Y_local[46] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 807)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  Y_local[48] = (Y_local[48] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 839)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  Y_local[50] = (Y_local[50] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 839)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  Y_local[52] = (Y_local[52] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 839)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 583)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 583)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 583)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 615)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 615)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 615)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 647)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 647)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  Y_local[17] = (Y_local[17] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 647)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  Y_local[19] = (Y_local[19] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 679)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  Y_local[21] = (Y_local[21] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 679)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  Y_local[23] = (Y_local[23] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 679)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  Y_local[25] = (Y_local[25] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 711)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  Y_local[27] = (Y_local[27] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 711)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  Y_local[29] = (Y_local[29] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 711)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  Y_local[31] = (Y_local[31] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 743)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  Y_local[33] = (Y_local[33] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 743)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  Y_local[35] = (Y_local[35] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 743)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  Y_local[37] = (Y_local[37] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 775)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  Y_local[39] = (Y_local[39] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 775)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  Y_local[41] = (Y_local[41] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 775)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  Y_local[43] = (Y_local[43] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 807)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  Y_local[45] = (Y_local[45] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 807)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  Y_local[47] = (Y_local[47] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 807)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  Y_local[49] = (Y_local[49] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 839)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  Y_local[51] = (Y_local[51] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 839)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  Y_local[53] = (Y_local[53] + (A_shared[(((((int)threadIdx.x) >> 4) * 8) + 839)] * B_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  Y[(((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2))] = Y_local[0];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 32)] = Y_local[2];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 64)] = Y_local[4];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 4608)] = Y_local[6];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 4640)] = Y_local[8];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 4672)] = Y_local[10];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 9216)] = Y_local[12];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 9248)] = Y_local[14];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 9280)] = Y_local[16];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 13824)] = Y_local[18];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 13856)] = Y_local[20];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 13888)] = Y_local[22];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 18432)] = Y_local[24];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 18464)] = Y_local[26];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 18496)] = Y_local[28];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 23040)] = Y_local[30];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 23072)] = Y_local[32];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 23104)] = Y_local[34];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 27648)] = Y_local[36];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 27680)] = Y_local[38];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 27712)] = Y_local[40];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 32256)] = Y_local[42];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 32288)] = Y_local[44];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 32320)] = Y_local[46];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 36864)] = Y_local[48];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 36896)] = Y_local[50];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 36928)] = Y_local[52];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 1)] = Y_local[1];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 33)] = Y_local[3];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 65)] = Y_local[5];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 4609)] = Y_local[7];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 4641)] = Y_local[9];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 4673)] = Y_local[11];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 9217)] = Y_local[13];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 9249)] = Y_local[15];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 9281)] = Y_local[17];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 13825)] = Y_local[19];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 13857)] = Y_local[21];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 13889)] = Y_local[23];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 18433)] = Y_local[25];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 18465)] = Y_local[27];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 18497)] = Y_local[29];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 23041)] = Y_local[31];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 23073)] = Y_local[33];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 23105)] = Y_local[35];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 27649)] = Y_local[37];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 27681)] = Y_local[39];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 27713)] = Y_local[41];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 32257)] = Y_local[43];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 32289)] = Y_local[45];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 32321)] = Y_local[47];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 36865)] = Y_local[49];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 36897)] = Y_local[51];
  Y[((((((((int)blockIdx.x) / 12) * 41472) + ((((int)threadIdx.x) >> 4) * 1152)) + ((((int)blockIdx.x) % 12) * 96)) + ((((int)threadIdx.x) & 15) * 2)) + 36929)] = Y_local[53];
}


