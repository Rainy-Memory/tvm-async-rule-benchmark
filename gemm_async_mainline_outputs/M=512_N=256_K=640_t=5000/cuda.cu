
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[16];
  __shared__ float A_shared[1536];
  __shared__ float B_shared[6144];
  Y_local[0] = 0.000000e+00f;
  Y_local[8] = 0.000000e+00f;
  Y_local[1] = 0.000000e+00f;
  Y_local[9] = 0.000000e+00f;
  Y_local[2] = 0.000000e+00f;
  Y_local[10] = 0.000000e+00f;
  Y_local[3] = 0.000000e+00f;
  Y_local[11] = 0.000000e+00f;
  Y_local[4] = 0.000000e+00f;
  Y_local[12] = 0.000000e+00f;
  Y_local[5] = 0.000000e+00f;
  Y_local[13] = 0.000000e+00f;
  Y_local[6] = 0.000000e+00f;
  Y_local[14] = 0.000000e+00f;
  Y_local[7] = 0.000000e+00f;
  Y_local[15] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((int)blockIdx.x) >> 2) * 10240) + ((((int)threadIdx.x) >> 3) * 640)) + ((((int)threadIdx.x) & 7) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 2) * 10240) + ((((int)threadIdx.x) >> 3) * 640)) + ((((int)threadIdx.x) & 7) * 4)) + 5120))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 1024))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 2048))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 3072))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 4096))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 5120))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 6144))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 7168))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 2) * 10240) + ((((int)threadIdx.x) >> 3) * 640)) + ((((int)threadIdx.x) & 7) * 4)) + 32))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 2) * 10240) + ((((int)threadIdx.x) >> 3) * 640)) + ((((int)threadIdx.x) & 7) * 4)) + 5152))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 8192))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 2304)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 9216))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 2560)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 10240))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 2816)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 11264))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 3072)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 12288))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 3328)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 13312))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 3584)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 14336))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 3840)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 15360))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int k_0_fused = 0; k_0_fused < 18; ++k_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((k_0_fused + 2) % 3) * 512) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) >> 2) * 10240) + ((((int)threadIdx.x) >> 3) * 640)) + (k_0_fused * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 64))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 2) % 3) * 512) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) >> 2) * 10240) + ((((int)threadIdx.x) >> 3) * 640)) + (k_0_fused * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 5184))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((k_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 16384))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 17408))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 18432))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 19456))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 20480))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 21504))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 22528))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 8192) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 23552))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    Y_local[0] = (Y_local[0] + (A_shared[(((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128))] * B_shared[(((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 256)] * B_shared[(((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    Y_local[1] = (Y_local[1] + (A_shared[(((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128))] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 256)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 32)] * B_shared[(((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 288)] * B_shared[(((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 32)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 288)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 1)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 257)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 1)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 257)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 33)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 289)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 33)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 289)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 2)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 258)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 2)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 258)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 34)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 290)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 34)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 290)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 3)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 259)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 3)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 259)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 35)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 291)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 35)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 291)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 4)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 260)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 4)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 260)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 36)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 292)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 36)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 292)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 5)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 261)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 5)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 261)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 37)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 293)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 37)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 293)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 6)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 262)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 6)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 262)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 38)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 294)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 38)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 294)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 7)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 263)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 7)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 263)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 39)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 295)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 39)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 295)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 8)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 264)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 8)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 264)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 40)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 296)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 40)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 296)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 9)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 265)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 9)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 265)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 41)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 297)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 41)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 297)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 10)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 266)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 10)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 266)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 42)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 298)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 42)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 298)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 11)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 267)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 11)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 267)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 43)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 299)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 43)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 299)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 12)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 268)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 12)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 268)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 44)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 300)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 44)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 300)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 13)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 269)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 13)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 269)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 45)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 301)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 45)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 301)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 14)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 270)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 14)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 270)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 46)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 302)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 46)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 302)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 15)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 271)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 15)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 271)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 47)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 303)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 47)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 303)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 16)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 272)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 16)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 272)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 48)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 304)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 48)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 304)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 17)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 273)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 17)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 273)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 49)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 305)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 49)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 305)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 18)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 274)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 18)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 274)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 50)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 306)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 50)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 306)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 19)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 275)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 19)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 275)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 51)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 307)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 51)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 307)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 20)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 276)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 20)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 276)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 52)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 308)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 52)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 308)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 21)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 277)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 21)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 277)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 53)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 309)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 53)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 309)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 22)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 278)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 22)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 278)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 54)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 310)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 54)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 310)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 23)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 279)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 23)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 279)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 55)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 311)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 55)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 311)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 24)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 280)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 24)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 280)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 56)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 312)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 56)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 312)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 25)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 281)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 25)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 281)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 57)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 313)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 57)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 313)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 26)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 282)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 26)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 282)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 58)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 314)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 58)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 314)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 27)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 283)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 27)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 283)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 59)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 315)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 59)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 315)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 28)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 284)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 28)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 284)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 60)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 316)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 60)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 316)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 29)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 285)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 29)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 285)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 61)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 317)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 61)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 317)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 30)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 286)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 30)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 286)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 62)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 318)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 62)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 318)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    Y_local[0] = (Y_local[0] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 31)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    Y_local[8] = (Y_local[8] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 287)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    Y_local[1] = (Y_local[1] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 31)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
    Y_local[9] = (Y_local[9] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 287)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
    Y_local[2] = (Y_local[2] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 63)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    Y_local[10] = (Y_local[10] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 319)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    Y_local[3] = (Y_local[3] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 63)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
    Y_local[11] = (Y_local[11] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 319)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 64)] * B_shared[(((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 320)] * B_shared[(((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 64)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 320)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 96)] * B_shared[(((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 352)] * B_shared[(((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 96)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 352)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 65)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 321)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 65)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 321)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 97)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 353)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 97)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 353)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 66)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 322)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 66)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 322)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 98)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 354)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 98)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 354)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 67)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 323)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 67)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 323)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 99)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 355)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 99)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 355)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 68)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 324)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 68)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 324)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 100)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 356)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 100)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 356)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 69)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 325)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 69)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 325)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 101)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 357)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 101)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 357)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 70)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 326)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 70)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 326)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 102)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 358)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 102)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 358)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 71)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 327)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 71)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 327)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 103)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 359)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 103)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 359)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 72)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 328)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 72)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 328)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 104)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 360)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 104)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 360)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 73)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 329)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 73)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 329)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 105)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 361)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 105)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 361)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 74)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 330)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 74)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 330)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 106)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 362)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 106)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 362)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 75)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 331)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 75)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 331)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 107)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 363)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 107)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 363)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 76)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 332)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 76)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 332)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 108)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 364)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 108)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 364)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 77)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 333)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 77)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 333)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 109)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 365)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 109)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 365)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 78)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 334)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 78)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 334)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 110)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 366)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 110)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 366)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 79)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 335)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 79)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 335)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 111)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 367)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 111)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 367)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 80)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 336)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 80)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 336)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 112)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 368)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 112)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 368)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 81)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 337)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 81)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 337)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 113)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 369)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 113)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 369)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 82)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 338)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 82)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 338)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 114)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 370)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 114)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 370)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 83)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 339)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 83)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 339)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 115)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 371)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 115)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 371)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 84)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 340)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 84)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 340)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 116)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 372)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 116)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 372)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 85)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 341)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 85)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 341)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 117)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 373)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 117)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 373)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 86)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 342)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 86)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 342)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 118)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 374)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 118)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 374)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 87)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 343)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 87)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 343)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 119)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 375)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 119)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 375)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 88)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 344)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 88)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 344)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 120)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 376)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 120)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 376)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 89)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 345)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 89)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 345)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 121)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 377)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 121)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 377)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 90)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 346)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 90)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 346)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 122)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 378)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 122)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 378)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 91)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 347)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 91)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 347)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 123)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 379)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 123)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 379)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 92)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 348)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 92)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 348)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 124)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 380)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 124)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 380)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 93)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 349)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 93)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 349)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 125)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 381)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 125)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 381)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 94)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 350)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 94)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 350)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 126)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 382)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 126)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 382)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    Y_local[4] = (Y_local[4] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 95)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    Y_local[12] = (Y_local[12] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 351)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    Y_local[5] = (Y_local[5] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 95)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
    Y_local[13] = (Y_local[13] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 351)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
    Y_local[6] = (Y_local[6] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 127)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    Y_local[14] = (Y_local[14] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 383)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    Y_local[7] = (Y_local[7] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 127)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
    Y_local[15] = (Y_local[15] + (A_shared[((((k_0_fused % 3) * 512) + ((((int)threadIdx.x) >> 5) * 128)) + 383)] * B_shared[((((k_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  Y_local[0] = (Y_local[0] + (A_shared[((((int)threadIdx.x) >> 5) * 128)] * B_shared[((((int)threadIdx.x) & 31) * 2)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 256)] * B_shared[((((int)threadIdx.x) & 31) * 2)]));
  Y_local[1] = (Y_local[1] + (A_shared[((((int)threadIdx.x) >> 5) * 128)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 256)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 32)] * B_shared[((((int)threadIdx.x) & 31) * 2)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 288)] * B_shared[((((int)threadIdx.x) & 31) * 2)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 32)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 288)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 1)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 257)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 1)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 257)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 33)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 289)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 33)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 289)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 2)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 258)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 2)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 258)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 34)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 290)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 34)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 290)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 3)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 259)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 3)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 259)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 35)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 291)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 35)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 291)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 4)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 260)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 4)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 260)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 36)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 292)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 36)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 292)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 5)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 261)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 5)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 261)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 37)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 293)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 37)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 293)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 6)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 262)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 6)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 262)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 38)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 294)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 38)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 294)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 7)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 263)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 7)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 263)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 39)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 295)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 39)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 295)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 8)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 264)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 8)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 264)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 40)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 296)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 40)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 296)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 9)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 265)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 9)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 265)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 41)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 297)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 41)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 297)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 10)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 266)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 10)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 266)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 42)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 298)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 42)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 298)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 11)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 267)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 11)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 267)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 43)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 299)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 43)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 299)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 12)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 268)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 12)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 268)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 44)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 300)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 44)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 300)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 13)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 269)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 13)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 269)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 45)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 301)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 45)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 301)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 14)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 270)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 14)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 270)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 46)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 302)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 46)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 302)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 15)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 271)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 15)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 271)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 47)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 303)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 47)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 303)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 16)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 272)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 16)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 272)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 48)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 304)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 48)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 304)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 17)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 273)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 17)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 273)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 49)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 305)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 49)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 305)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 18)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 274)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 18)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 274)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 50)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 306)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 50)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 306)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 19)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 275)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 19)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 275)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 51)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 307)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 51)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 307)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 20)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 276)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 20)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 276)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 52)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 308)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 52)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 308)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 21)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 277)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 21)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 277)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 53)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 309)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 53)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 309)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 22)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 278)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 22)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 278)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 54)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 310)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 54)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 310)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 23)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 279)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 23)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 279)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 55)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 311)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 55)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 311)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 24)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 280)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 24)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 280)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 56)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 312)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 56)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 312)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 25)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 281)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 25)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 281)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 57)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 313)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 57)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 313)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 26)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 282)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 26)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 282)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 58)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 314)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 58)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 314)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 27)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 283)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 27)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 283)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 59)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 315)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 59)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 315)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 28)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 284)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 28)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 284)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 60)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 316)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 60)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 316)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 29)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 285)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 29)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 285)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 61)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 317)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 61)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 317)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 30)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 286)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 30)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 286)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 62)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 318)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 62)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 318)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 31)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 287)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 31)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 287)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 63)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 319)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 63)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 319)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 64)] * B_shared[((((int)threadIdx.x) & 31) * 2)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 320)] * B_shared[((((int)threadIdx.x) & 31) * 2)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 64)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 320)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 96)] * B_shared[((((int)threadIdx.x) & 31) * 2)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 352)] * B_shared[((((int)threadIdx.x) & 31) * 2)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 96)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 352)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 65)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 321)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 65)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 321)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 97)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 353)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 97)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 353)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 66)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 322)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 66)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 322)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 98)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 354)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 98)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 354)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 67)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 323)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 67)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 323)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 99)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 355)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 99)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 355)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 68)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 324)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 68)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 324)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 100)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 356)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 100)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 356)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 69)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 325)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 69)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 325)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 101)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 357)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 101)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 357)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 70)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 326)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 70)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 326)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 102)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 358)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 102)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 358)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 71)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 327)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 71)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 327)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 103)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 359)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 103)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 359)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 72)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 328)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 72)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 328)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 104)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 360)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 104)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 360)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 73)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 329)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 73)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 329)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 105)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 361)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 105)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 361)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 74)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 330)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 74)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 330)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 106)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 362)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 106)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 362)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 75)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 331)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 75)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 331)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 107)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 363)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 107)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 363)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 76)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 332)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 76)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 332)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 108)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 364)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 108)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 364)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 77)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 333)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 77)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 333)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 109)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 365)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 109)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 365)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 78)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 334)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 78)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 334)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 110)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 366)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 110)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 366)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 79)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 335)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 79)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 335)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 111)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 367)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 111)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 367)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 80)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 336)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 80)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 336)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 112)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 368)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 112)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 368)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 81)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 337)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 81)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 337)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 113)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 369)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 113)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 369)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 82)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 338)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 82)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 338)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 114)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 370)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 114)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 370)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 83)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 339)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 83)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 339)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 115)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 371)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 115)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 371)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 84)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 340)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 84)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 340)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 116)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 372)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 116)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 372)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 85)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 341)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 85)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 341)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 117)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 373)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 117)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 373)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 86)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 342)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 86)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 342)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 118)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 374)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 118)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 374)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 87)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 343)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 87)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 343)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 119)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 375)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 119)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 375)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 88)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 344)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 88)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 344)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 120)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 376)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 120)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 376)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 89)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 345)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 89)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 345)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 121)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 377)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 121)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 377)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 90)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 346)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 90)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 346)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 122)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 378)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 122)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 378)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 91)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 347)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 91)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 347)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 123)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 379)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 123)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 379)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 92)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 348)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 92)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 348)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 124)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 380)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 124)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 380)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 93)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 349)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 93)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 349)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 125)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 381)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 125)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 381)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 94)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 350)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 94)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 350)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 126)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 382)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 126)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 382)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 95)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 351)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 95)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 351)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 127)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 383)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 127)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 383)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 512)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2048)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 768)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2048)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 512)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2049)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 768)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2049)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 544)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2048)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 800)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2048)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 544)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2049)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 800)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2049)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 513)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2112)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 769)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2112)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 513)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2113)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 769)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2113)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 545)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2112)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 801)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2112)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 545)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2113)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 801)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2113)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 514)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2176)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 770)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2176)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 514)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2177)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 770)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2177)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 546)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2176)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 802)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2176)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 546)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2177)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 802)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2177)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 515)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2240)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 771)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2240)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 515)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2241)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 771)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2241)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 547)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2240)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 803)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2240)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 547)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2241)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 803)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2241)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 516)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2304)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 772)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2304)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 516)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2305)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 772)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2305)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 548)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2304)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 804)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2304)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 548)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2305)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 804)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2305)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 517)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2368)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 773)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2368)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 517)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2369)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 773)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2369)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 549)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2368)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 805)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2368)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 549)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2369)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 805)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2369)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 518)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2432)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 774)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2432)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 518)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2433)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 774)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2433)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 550)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2432)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 806)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2432)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 550)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2433)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 806)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2433)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 519)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2496)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 775)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2496)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 519)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2497)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 775)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2497)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 551)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2496)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 807)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2496)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 551)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2497)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 807)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2497)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 520)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2560)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 776)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2560)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 520)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2561)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 776)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2561)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 552)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2560)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 808)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2560)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 552)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2561)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 808)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2561)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 521)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2624)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 777)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2624)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 521)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2625)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 777)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2625)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 553)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2624)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 809)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2624)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 553)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2625)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 809)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2625)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 522)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2688)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 778)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2688)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 522)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2689)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 778)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2689)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 554)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2688)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 810)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2688)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 554)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2689)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 810)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2689)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 523)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2752)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 779)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2752)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 523)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2753)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 779)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2753)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 555)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2752)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 811)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2752)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 555)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2753)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 811)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2753)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 524)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2816)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 780)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2816)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 524)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2817)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 780)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2817)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 556)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2816)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 812)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2816)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 556)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2817)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 812)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2817)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 525)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2880)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 781)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2880)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 525)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2881)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 781)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2881)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 557)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2880)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 813)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2880)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 557)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2881)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 813)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2881)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 526)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2944)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 782)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2944)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 526)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2945)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 782)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2945)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 558)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2944)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 814)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2944)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 558)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2945)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 814)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2945)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 527)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3008)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 783)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3008)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 527)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3009)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 783)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3009)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 559)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3008)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 815)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3008)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 559)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3009)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 815)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3009)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 528)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3072)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 784)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3072)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 528)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3073)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 784)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3073)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 560)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3072)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 816)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3072)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 560)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3073)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 816)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3073)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 529)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3136)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 785)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3136)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 529)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3137)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 785)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3137)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 561)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3136)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 817)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3136)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 561)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3137)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 817)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3137)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 530)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3200)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 786)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3200)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 530)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3201)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 786)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3201)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 562)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3200)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 818)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3200)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 562)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3201)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 818)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3201)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 531)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3264)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 787)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3264)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 531)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3265)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 787)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3265)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 563)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3264)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 819)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3264)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 563)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3265)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 819)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3265)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 532)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3328)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 788)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3328)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 532)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3329)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 788)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3329)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 564)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3328)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 820)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3328)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 564)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3329)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 820)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3329)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 533)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3392)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 789)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3392)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 533)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3393)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 789)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3393)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 565)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3392)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 821)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3392)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 565)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3393)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 821)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3393)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 534)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3456)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 790)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3456)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 534)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3457)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 790)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3457)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 566)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3456)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 822)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3456)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 566)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3457)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 822)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3457)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 535)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3520)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 791)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3520)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 535)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3521)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 791)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3521)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 567)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3520)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 823)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3520)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 567)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3521)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 823)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3521)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 536)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3584)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 792)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3584)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 536)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3585)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 792)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3585)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 568)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3584)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 824)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3584)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 568)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3585)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 824)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3585)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 537)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3648)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 793)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3648)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 537)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3649)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 793)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3649)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 569)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3648)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 825)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3648)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 569)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3649)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 825)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3649)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 538)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3712)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 794)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3712)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 538)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3713)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 794)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3713)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 570)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3712)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 826)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3712)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 570)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3713)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 826)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3713)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 539)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3776)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 795)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3776)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 539)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3777)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 795)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3777)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 571)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3776)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 827)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3776)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 571)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3777)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 827)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3777)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 540)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3840)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 796)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3840)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 540)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3841)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 796)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3841)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 572)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3840)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 828)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3840)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 572)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3841)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 828)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3841)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 541)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3904)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 797)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3904)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 541)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3905)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 797)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3905)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 573)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3904)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 829)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3904)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 573)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3905)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 829)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3905)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 542)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3968)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 798)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3968)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 542)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3969)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 798)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3969)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 574)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3968)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 830)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3968)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 574)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3969)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 830)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3969)]));
  Y_local[0] = (Y_local[0] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 543)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4032)]));
  Y_local[8] = (Y_local[8] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 799)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4032)]));
  Y_local[1] = (Y_local[1] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 543)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4033)]));
  Y_local[9] = (Y_local[9] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 799)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4033)]));
  Y_local[2] = (Y_local[2] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 575)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4032)]));
  Y_local[10] = (Y_local[10] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 831)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4032)]));
  Y_local[3] = (Y_local[3] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 575)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4033)]));
  Y_local[11] = (Y_local[11] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 831)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4033)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 576)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2048)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 832)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2048)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 576)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2049)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 832)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2049)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 608)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2048)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 864)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2048)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 608)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2049)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 864)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2049)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 577)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2112)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 833)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2112)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 577)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2113)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 833)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2113)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 609)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2112)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 865)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2112)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 609)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2113)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 865)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2113)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 578)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2176)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 834)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2176)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 578)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2177)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 834)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2177)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 610)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2176)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 866)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2176)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 610)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2177)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 866)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2177)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 579)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2240)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 835)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2240)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 579)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2241)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 835)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2241)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 611)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2240)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 867)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2240)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 611)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2241)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 867)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2241)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 580)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2304)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 836)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2304)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 580)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2305)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 836)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2305)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 612)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2304)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 868)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2304)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 612)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2305)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 868)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2305)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 581)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2368)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 837)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2368)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 581)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2369)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 837)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2369)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 613)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2368)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 869)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2368)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 613)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2369)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 869)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2369)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 582)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2432)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 838)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2432)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 582)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2433)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 838)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2433)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 614)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2432)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 870)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2432)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 614)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2433)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 870)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2433)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 583)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2496)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 839)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2496)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 583)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2497)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 839)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2497)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 615)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2496)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 871)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2496)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 615)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2497)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 871)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2497)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 584)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2560)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 840)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2560)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 584)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2561)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 840)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2561)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 616)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2560)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 872)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2560)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 616)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2561)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 872)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2561)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 585)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2624)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 841)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2624)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 585)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2625)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 841)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2625)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 617)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2624)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 873)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2624)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 617)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2625)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 873)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2625)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 586)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2688)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 842)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2688)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 586)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2689)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 842)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2689)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 618)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2688)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 874)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2688)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 618)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2689)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 874)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2689)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 587)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2752)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 843)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2752)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 587)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2753)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 843)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2753)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 619)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2752)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 875)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2752)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 619)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2753)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 875)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2753)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 588)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2816)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 844)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2816)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 588)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2817)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 844)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2817)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 620)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2816)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 876)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2816)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 620)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2817)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 876)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2817)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 589)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2880)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 845)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2880)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 589)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2881)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 845)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2881)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 621)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2880)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 877)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2880)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 621)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2881)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 877)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2881)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 590)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2944)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 846)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2944)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 590)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2945)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 846)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2945)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 622)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2944)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 878)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2944)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 622)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2945)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 878)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 2945)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 591)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3008)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 847)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3008)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 591)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3009)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 847)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3009)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 623)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3008)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 879)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3008)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 623)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3009)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 879)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3009)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 592)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3072)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 848)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3072)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 592)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3073)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 848)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3073)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 624)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3072)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 880)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3072)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 624)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3073)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 880)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3073)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 593)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3136)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 849)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3136)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 593)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3137)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 849)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3137)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 625)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3136)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 881)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3136)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 625)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3137)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 881)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3137)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 594)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3200)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 850)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3200)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 594)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3201)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 850)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3201)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 626)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3200)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 882)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3200)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 626)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3201)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 882)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3201)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 595)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3264)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 851)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3264)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 595)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3265)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 851)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3265)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 627)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3264)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 883)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3264)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 627)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3265)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 883)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3265)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 596)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3328)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 852)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3328)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 596)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3329)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 852)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3329)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 628)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3328)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 884)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3328)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 628)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3329)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 884)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3329)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 597)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3392)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 853)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3392)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 597)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3393)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 853)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3393)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 629)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3392)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 885)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3392)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 629)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3393)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 885)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3393)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 598)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3456)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 854)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3456)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 598)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3457)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 854)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3457)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 630)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3456)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 886)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3456)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 630)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3457)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 886)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3457)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 599)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3520)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 855)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3520)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 599)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3521)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 855)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3521)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 631)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3520)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 887)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3520)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 631)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3521)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 887)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3521)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 600)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3584)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 856)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3584)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 600)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3585)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 856)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3585)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 632)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3584)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 888)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3584)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 632)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3585)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 888)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3585)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 601)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3648)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 857)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3648)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 601)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3649)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 857)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3649)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 633)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3648)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 889)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3648)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 633)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3649)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 889)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3649)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 602)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3712)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 858)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3712)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 602)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3713)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 858)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3713)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 634)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3712)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 890)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3712)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 634)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3713)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 890)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3713)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 603)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3776)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 859)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3776)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 603)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3777)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 859)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3777)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 635)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3776)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 891)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3776)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 635)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3777)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 891)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3777)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 604)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3840)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 860)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3840)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 604)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3841)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 860)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3841)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 636)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3840)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 892)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3840)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 636)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3841)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 892)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3841)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 605)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3904)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 861)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3904)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 605)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3905)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 861)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3905)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 637)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3904)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 893)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3904)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 637)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3905)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 893)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3905)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 606)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3968)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 862)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3968)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 606)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3969)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 862)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3969)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 638)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3968)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 894)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3968)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 638)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3969)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 894)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 3969)]));
  Y_local[4] = (Y_local[4] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 607)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4032)]));
  Y_local[12] = (Y_local[12] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 863)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4032)]));
  Y_local[5] = (Y_local[5] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 607)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4033)]));
  Y_local[13] = (Y_local[13] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 863)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4033)]));
  Y_local[6] = (Y_local[6] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 639)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4032)]));
  Y_local[14] = (Y_local[14] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 895)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4032)]));
  Y_local[7] = (Y_local[7] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 639)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4033)]));
  Y_local[15] = (Y_local[15] + (A_shared[(((((int)threadIdx.x) >> 5) * 128) + 895)] * B_shared[(((((int)threadIdx.x) & 31) * 2) + 4033)]));
  Y[(((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2))] = Y_local[0];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 2048)] = Y_local[8];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 1)] = Y_local[1];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 2049)] = Y_local[9];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 256)] = Y_local[2];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 2304)] = Y_local[10];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 257)] = Y_local[3];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 2305)] = Y_local[11];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 512)] = Y_local[4];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 2560)] = Y_local[12];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 513)] = Y_local[5];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 2561)] = Y_local[13];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 768)] = Y_local[6];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 2816)] = Y_local[14];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 769)] = Y_local[7];
  Y[((((((((int)blockIdx.x) >> 2) * 4096) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 2817)] = Y_local[15];
}


