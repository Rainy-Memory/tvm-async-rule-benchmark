
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[64];
  __shared__ float A_shared[3072];
  __shared__ float B_shared[3072];
  for (int j_4_init = 0; j_4_init < 2; ++j_4_init) {
    for (int vthread_x_s = 0; vthread_x_s < 32; ++vthread_x_s) {
      Y_local[((vthread_x_s * 2) + j_4_init)] = 0.000000e+00f;
    }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + ((((int)threadIdx.x) & 3) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + ((((int)threadIdx.x) & 3) * 4)) + 16384))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + ((((int)threadIdx.x) & 3) * 4)) + 32768))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + ((((int)threadIdx.x) & 3) * 4)) + 49152))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((int)threadIdx.x) >> 4) * 1024) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1024) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 4096))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1024) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 8192))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1024) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 12288))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + ((((int)threadIdx.x) & 3) * 4)) + 16))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + ((((int)threadIdx.x) & 3) * 4)) + 16400))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + ((((int)threadIdx.x) & 3) * 4)) + 32784))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((int)threadIdx.x) * 4) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + ((((int)threadIdx.x) & 3) * 4)) + 49168))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1024) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 16384))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1024) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 20480))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1024) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 24576))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((int)threadIdx.x) * 4) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 1024) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 28672))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int k_0_fused = 0; k_0_fused < 62; ++k_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + ((((k_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 32))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 16416))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 32800))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(A_shared + (((((k_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4)) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_0_fused * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 49184))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + ((((k_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 16384) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 32768))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 16384) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 36864))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 16384) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 40960))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(B_shared + (((((k_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4)) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0_fused * 16384) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 45056))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    for (int k_1 = 0; k_1 < 2; ++k_1) {
      for (int k_2 = 0; k_2 < 8; ++k_2) {
        for (int j_4 = 0; j_4 < 2; ++j_4) {
          for (int vthread_x_s_1 = 0; vthread_x_s_1 < 32; ++vthread_x_s_1) {
            Y_local[((vthread_x_s_1 * 2) + j_4)] = (Y_local[((vthread_x_s_1 * 2) + j_4)] + (A_shared[((((((k_0_fused % 3) * 1024) + ((vthread_x_s_1 >> 2) * 128)) + ((((int)threadIdx.x) >> 3) * 16)) + (k_1 * 8)) + k_2)] * B_shared[(((((((k_0_fused % 3) * 1024) + (k_1 * 512)) + (k_2 * 64)) + ((vthread_x_s_1 & 3) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + j_4)]));
          }
        }
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  for (int k_1_1 = 0; k_1_1 < 2; ++k_1_1) {
    for (int k_2_1 = 0; k_2_1 < 8; ++k_2_1) {
      for (int j_4_1 = 0; j_4_1 < 2; ++j_4_1) {
        for (int vthread_x_s_2 = 0; vthread_x_s_2 < 32; ++vthread_x_s_2) {
          Y_local[((vthread_x_s_2 * 2) + j_4_1)] = (Y_local[((vthread_x_s_2 * 2) + j_4_1)] + (A_shared[((((((vthread_x_s_2 >> 2) * 128) + ((((int)threadIdx.x) >> 3) * 16)) + (k_1_1 * 8)) + k_2_1) + 2048)] * B_shared[((((((k_1_1 * 512) + (k_2_1 * 64)) + ((vthread_x_s_2 & 3) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + j_4_1) + 2048)]));
        }
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int k_1_2 = 0; k_1_2 < 2; ++k_1_2) {
    for (int k_2_2 = 0; k_2_2 < 8; ++k_2_2) {
      for (int j_4_2 = 0; j_4_2 < 2; ++j_4_2) {
        for (int vthread_x_s_3 = 0; vthread_x_s_3 < 32; ++vthread_x_s_3) {
          Y_local[((vthread_x_s_3 * 2) + j_4_2)] = (Y_local[((vthread_x_s_3 * 2) + j_4_2)] + (A_shared[(((((vthread_x_s_3 >> 2) * 128) + ((((int)threadIdx.x) >> 3) * 16)) + (k_1_2 * 8)) + k_2_2)] * B_shared[(((((k_1_2 * 512) + (k_2_2 * 64)) + ((vthread_x_s_3 & 3) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + j_4_2)]));
        }
      }
    }
  }
  for (int ax1 = 0; ax1 < 2; ++ax1) {
    for (int vthread_x_s_4 = 0; vthread_x_s_4 < 32; ++vthread_x_s_4) {
      Y[((((((((((int)blockIdx.x) >> 4) * 65536) + ((vthread_x_s_4 >> 2) * 8192)) + ((((int)threadIdx.x) >> 3) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((vthread_x_s_4 & 3) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax1)] = Y_local[((vthread_x_s_4 * 2) + ax1)];
    }
  }
}


