
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[14];
  __shared__ float PadInput_shared[1296];
  __shared__ float weight_shared[4608];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  conv2d_nhwc_local[8] = 0.000000e+00f;
  conv2d_nhwc_local[10] = 0.000000e+00f;
  conv2d_nhwc_local[12] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[7] = 0.000000e+00f;
  conv2d_nhwc_local[9] = 0.000000e+00f;
  conv2d_nhwc_local[11] = 0.000000e+00f;
  conv2d_nhwc_local[13] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((int)threadIdx.x)))
    );
    int src_bytes = (((1 <= (((((int)blockIdx.x) >> 4) * 7) + (((int)threadIdx.x) / 36))) && (1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + ((((int)threadIdx.x) % 36) >> 2)))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + ((((int)threadIdx.x) % 36) >> 2)) < 29)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) / 36) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + (((((int)threadIdx.x) % 36) >> 2) * 128)) + (((int)threadIdx.x) & 3)) - 3712))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((((int)threadIdx.x) + 112) / 36) * 36) + ((((((int)threadIdx.x) >> 2) + 1) % 9) * 4)) + (((int)threadIdx.x) & 3))))
    );
    int src_bytes = ((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 1) % 9))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 1) % 9)) < 29)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 25088) + (((((int)threadIdx.x) + 112) / 36) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((((int)threadIdx.x) >> 2) + 1) % 9) * 128)) + (((int)threadIdx.x) & 3)) - 3712))), "n"(4), "r"(src_bytes)
    );
  }
  if (((int)threadIdx.x) < 100) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((((int)threadIdx.x) + 224) / 36) * 36) + ((((((int)threadIdx.x) >> 2) + 2) % 9) * 4)) + (((int)threadIdx.x) & 3))))
    );
    int src_bytes = ((((((((int)blockIdx.x) >> 4) * 7) + ((((int)threadIdx.x) + 224) / 36)) < 29) && (1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 2) % 9)))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 2) % 9)) < 29)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 25088) + (((((int)threadIdx.x) + 224) / 36) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((((int)threadIdx.x) >> 2) + 2) % 9) * 128)) + (((int)threadIdx.x) & 3)) - 3712))), "n"(4), "r"(src_bytes)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((int)threadIdx.x)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 5) * 128) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 112)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 112) >> 7) * 16384) + (((((((int)threadIdx.x) >> 4) + 7) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 224) >> 7) * 16384) + ((((((int)threadIdx.x) >> 5) + 3) & 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 336)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 336) >> 7) * 16384) + (((((((int)threadIdx.x) >> 4) + 5) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 448) >> 7) * 16384) + ((((((int)threadIdx.x) >> 5) + 2) & 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 560)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 560) >> 7) * 16384) + (((((((int)threadIdx.x) >> 4) + 3) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 672) >> 7) * 16384) + ((((((int)threadIdx.x) >> 5) + 1) & 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 784)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 784) >> 7) * 16384) + ((((((int)threadIdx.x) >> 4) + 1) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 128) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 114688))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1008)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1008) >> 7) * 16384) + (((((((int)threadIdx.x) >> 4) + 7) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1120)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1120) >> 7) * 16384) + ((((int)blockIdx.x) & 3) * 32)) + ((int)threadIdx.x)) + 384))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 324)))
    );
    int src_bytes = (((1 <= (((((int)blockIdx.x) >> 4) * 7) + (((int)threadIdx.x) / 36))) && (1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + ((((int)threadIdx.x) % 36) >> 2)))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + ((((int)threadIdx.x) % 36) >> 2)) < 29)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) / 36) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + (((((int)threadIdx.x) % 36) >> 2) * 128)) + (((int)threadIdx.x) & 3)) - 3708))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((((int)threadIdx.x) + 112) / 36) * 36) + ((((((int)threadIdx.x) >> 2) + 1) % 9) * 4)) + (((int)threadIdx.x) & 3)) + 324)))
    );
    int src_bytes = ((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 1) % 9))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 1) % 9)) < 29)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 25088) + (((((int)threadIdx.x) + 112) / 36) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((((int)threadIdx.x) >> 2) + 1) % 9) * 128)) + (((int)threadIdx.x) & 3)) - 3708))), "n"(4), "r"(src_bytes)
    );
  }
  if (((int)threadIdx.x) < 100) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((((int)threadIdx.x) + 224) / 36) * 36) + ((((((int)threadIdx.x) >> 2) + 2) % 9) * 4)) + (((int)threadIdx.x) & 3)) + 324)))
    );
    int src_bytes = ((((((((int)blockIdx.x) >> 4) * 7) + ((((int)threadIdx.x) + 224) / 36)) < 29) && (1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 2) % 9)))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 2) % 9)) < 29)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 25088) + (((((int)threadIdx.x) + 224) / 36) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((((int)threadIdx.x) >> 2) + 2) % 9) * 128)) + (((int)threadIdx.x) & 3)) - 3708))), "n"(4), "r"(src_bytes)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1152)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 128) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 512))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1264)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 112) >> 7) * 16384) + (((((((int)threadIdx.x) >> 4) + 7) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 512))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1376)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 224) >> 7) * 16384) + ((((((int)threadIdx.x) >> 5) + 3) & 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 512))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1488)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 336) >> 7) * 16384) + (((((((int)threadIdx.x) >> 4) + 5) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 512))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1600)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 448) >> 7) * 16384) + ((((((int)threadIdx.x) >> 5) + 2) & 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 512))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1712)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 560) >> 7) * 16384) + (((((((int)threadIdx.x) >> 4) + 3) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 512))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1824)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 672) >> 7) * 16384) + ((((((int)threadIdx.x) >> 5) + 1) & 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 512))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1936)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 784) >> 7) * 16384) + ((((((int)threadIdx.x) >> 4) + 1) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 512))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 128) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 115200))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2160)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 1008) >> 7) * 16384) + (((((((int)threadIdx.x) >> 4) + 7) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 512))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2272)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1120) >> 7) * 16384) + ((((int)blockIdx.x) & 3) * 32)) + ((int)threadIdx.x)) + 896))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 648)))
    );
    int src_bytes = (((1 <= (((((int)blockIdx.x) >> 4) * 7) + (((int)threadIdx.x) / 36))) && (1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + ((((int)threadIdx.x) % 36) >> 2)))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + ((((int)threadIdx.x) % 36) >> 2)) < 29)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) / 36) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + (((((int)threadIdx.x) % 36) >> 2) * 128)) + (((int)threadIdx.x) & 3)) - 3704))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((((int)threadIdx.x) + 112) / 36) * 36) + ((((((int)threadIdx.x) >> 2) + 1) % 9) * 4)) + (((int)threadIdx.x) & 3)) + 648)))
    );
    int src_bytes = ((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 1) % 9))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 1) % 9)) < 29)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 25088) + (((((int)threadIdx.x) + 112) / 36) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((((int)threadIdx.x) >> 2) + 1) % 9) * 128)) + (((int)threadIdx.x) & 3)) - 3704))), "n"(4), "r"(src_bytes)
    );
  }
  if (((int)threadIdx.x) < 100) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((((int)threadIdx.x) + 224) / 36) * 36) + ((((((int)threadIdx.x) >> 2) + 2) % 9) * 4)) + (((int)threadIdx.x) & 3)) + 648)))
    );
    int src_bytes = ((((((((int)blockIdx.x) >> 4) * 7) + ((((int)threadIdx.x) + 224) / 36)) < 29) && (1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 2) % 9)))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 2) % 9)) < 29)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 25088) + (((((int)threadIdx.x) + 224) / 36) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((((int)threadIdx.x) >> 2) + 2) % 9) * 128)) + (((int)threadIdx.x) & 3)) - 3704))), "n"(4), "r"(src_bytes)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2304)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 128) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 1024))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2416)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 112) >> 7) * 16384) + (((((((int)threadIdx.x) >> 4) + 7) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 1024))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2528)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 224) >> 7) * 16384) + ((((((int)threadIdx.x) >> 5) + 3) & 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 1024))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2640)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 336) >> 7) * 16384) + (((((((int)threadIdx.x) >> 4) + 5) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 1024))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2752)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 448) >> 7) * 16384) + ((((((int)threadIdx.x) >> 5) + 2) & 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 1024))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2864)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 560) >> 7) * 16384) + (((((((int)threadIdx.x) >> 4) + 3) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 1024))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2976)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 672) >> 7) * 16384) + ((((((int)threadIdx.x) >> 5) + 1) & 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 1024))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3088)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 784) >> 7) * 16384) + ((((((int)threadIdx.x) >> 4) + 1) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 1024))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3200)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 128) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 115712))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3312)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 1008) >> 7) * 16384) + (((((((int)threadIdx.x) >> 4) + 7) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 1024))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3424)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1120) >> 7) * 16384) + ((((int)blockIdx.x) & 3) * 32)) + ((int)threadIdx.x)) + 1408))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int rh_0_rw_0_rc_0_fused = 0; rh_0_rw_0_rc_0_fused < 29; ++rh_0_rw_0_rc_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 324) + ((int)threadIdx.x))))
    );
    int src_bytes = (((1 <= (((((int)blockIdx.x) >> 4) * 7) + (((int)threadIdx.x) / 36))) && (1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + ((((int)threadIdx.x) % 36) >> 2)))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + ((((int)threadIdx.x) % 36) >> 2)) < 29)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) / 36) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + (((((int)threadIdx.x) % 36) >> 2) * 128)) + (rh_0_rw_0_rc_0_fused * 4)) + (((int)threadIdx.x) & 3)) - 3700))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((rh_0_rw_0_rc_0_fused + 3) & 3) * 324) + (((((int)threadIdx.x) + 112) / 36) * 36)) + ((((((int)threadIdx.x) >> 2) + 1) % 9) * 4)) + (((int)threadIdx.x) & 3))))
    );
    int src_bytes = ((1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 1) % 9))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 1) % 9)) < 29)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)blockIdx.x) >> 4) * 25088) + (((((int)threadIdx.x) + 112) / 36) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((((int)threadIdx.x) >> 2) + 1) % 9) * 128)) + (rh_0_rw_0_rc_0_fused * 4)) + (((int)threadIdx.x) & 3)) - 3700))), "n"(4), "r"(src_bytes)
    );
  }
    if (((int)threadIdx.x) < 100) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((rh_0_rw_0_rc_0_fused + 3) & 3) * 324) + (((((int)threadIdx.x) + 224) / 36) * 36)) + ((((((int)threadIdx.x) >> 2) + 2) % 9) * 4)) + (((int)threadIdx.x) & 3))))
    );
    int src_bytes = ((((((((int)blockIdx.x) >> 4) * 7) + ((((int)threadIdx.x) + 224) / 36)) < 29) && (1 <= ((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 2) % 9)))) && (((((((int)blockIdx.x) & 15) >> 2) * 7) + (((((int)threadIdx.x) >> 2) + 2) % 9)) < 29)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)blockIdx.x) >> 4) * 25088) + (((((int)threadIdx.x) + 224) / 36) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((((int)threadIdx.x) >> 2) + 2) % 9) * 128)) + (rh_0_rw_0_rc_0_fused * 4)) + (((int)threadIdx.x) & 3)) - 3700))), "n"(4), "r"(src_bytes)
    );
  }
    }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1152) + ((int)threadIdx.x))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 512) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 1536))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1152) + ((int)threadIdx.x)) + 112)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((((int)threadIdx.x) + 112) >> 7) * 16384) + (rh_0_rw_0_rc_0_fused * 512)) + (((((((int)threadIdx.x) >> 4) + 7) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 1536))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1152) + ((int)threadIdx.x)) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((((int)threadIdx.x) + 224) >> 7) * 16384) + (rh_0_rw_0_rc_0_fused * 512)) + ((((((int)threadIdx.x) >> 5) + 3) & 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 1536))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1152) + ((int)threadIdx.x)) + 336)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((((int)threadIdx.x) + 336) >> 7) * 16384) + (rh_0_rw_0_rc_0_fused * 512)) + (((((((int)threadIdx.x) >> 4) + 5) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 1536))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1152) + ((int)threadIdx.x)) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((((int)threadIdx.x) + 448) >> 7) * 16384) + (rh_0_rw_0_rc_0_fused * 512)) + ((((((int)threadIdx.x) >> 5) + 2) & 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 1536))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1152) + ((int)threadIdx.x)) + 560)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((((int)threadIdx.x) + 560) >> 7) * 16384) + (rh_0_rw_0_rc_0_fused * 512)) + (((((((int)threadIdx.x) >> 4) + 3) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 1536))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1152) + ((int)threadIdx.x)) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((((int)threadIdx.x) + 672) >> 7) * 16384) + (rh_0_rw_0_rc_0_fused * 512)) + ((((((int)threadIdx.x) >> 5) + 1) & 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 1536))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1152) + ((int)threadIdx.x)) + 784)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((((int)threadIdx.x) + 784) >> 7) * 16384) + (rh_0_rw_0_rc_0_fused * 512)) + ((((((int)threadIdx.x) >> 4) + 1) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 1536))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1152) + ((int)threadIdx.x)) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 512) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 116224))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1152) + ((int)threadIdx.x)) + 1008)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((((int)threadIdx.x) + 1008) >> 7) * 16384) + (rh_0_rw_0_rc_0_fused * 512)) + (((((((int)threadIdx.x) >> 4) + 7) & 7) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 1536))), "n"(4)
    );
  }
    if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1152) + ((int)threadIdx.x)) + 1120)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((((int)threadIdx.x) + 1120) >> 7) * 16384) + (rh_0_rw_0_rc_0_fused * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((int)threadIdx.x)) + 1920))), "n"(4)
    );
  }
    }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

    __syncthreads();
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36))] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 4)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 8)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 12)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 16)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 20)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 24)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 32)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36))] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 32)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1024)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1024)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1024)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1024)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1024)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1024)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1024)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1025)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1025)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1025)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1025)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1025)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1025)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1025)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1056)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1056)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1056)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1056)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1056)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1056)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1056)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1057)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1057)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1057)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1057)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1057)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1057)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1057)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1088)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1088)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1088)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1088)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1088)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1088)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1088)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1089)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1089)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1089)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1089)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1089)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1089)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1089)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1120)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1120)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1120)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1120)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1120)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1120)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1120)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1121)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1121)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1121)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1121)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1121)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1121)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 324) + ((((int)threadIdx.x) >> 4) * 36)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1152) + ((((int)threadIdx.x) & 15) * 2)) + 1121)]));
  }
__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 324)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 328)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 336)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 340)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 344)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 348)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 328)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 336)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 340)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 344)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 348)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 352)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 336)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 340)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 344)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 348)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 352)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 356)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 324)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 328)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 336)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 340)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 344)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 348)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 328)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 336)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 340)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 344)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 348)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 352)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 336)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 340)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 344)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 348)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 352)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 356)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 325)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 329)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 337)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 341)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 345)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 349)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 329)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 337)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 341)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 345)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 349)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 353)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 337)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 341)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 345)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 349)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 353)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 357)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 325)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 329)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 337)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 341)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 345)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 349)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 329)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 337)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 341)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 345)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 349)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 353)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 337)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 341)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 345)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 349)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 353)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 357)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 326)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 330)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 338)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 342)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 346)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 350)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 330)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 338)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 342)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 346)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 350)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 354)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 338)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 342)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 346)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 350)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 354)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 358)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 326)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 330)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 338)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 342)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 346)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 350)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 330)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 338)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 342)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 346)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 350)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 354)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 338)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 342)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 346)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 350)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 354)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 358)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 327)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 331)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 339)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 343)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 347)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 351)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 331)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 339)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 343)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 347)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 351)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 355)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 339)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 343)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 347)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 351)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 355)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 359)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 327)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 331)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 339)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 343)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 347)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 351)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 331)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 339)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 343)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 347)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 351)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 355)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 339)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 343)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 347)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 351)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 355)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 359)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 360)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 364)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 368)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 372)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 376)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 380)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 384)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 364)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 368)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 372)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 376)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 380)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 384)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 388)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 368)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 372)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 376)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 380)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 384)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 388)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 392)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 360)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 364)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 368)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 372)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 376)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 380)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 384)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 364)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 368)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 372)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 376)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 380)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 384)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 388)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 368)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 372)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 376)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 380)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 384)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 388)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 392)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 361)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 365)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 369)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 373)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 377)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 381)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 385)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 365)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 369)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 373)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 377)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 381)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 385)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 389)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 369)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 373)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 377)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 381)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 385)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 389)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 393)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 361)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 365)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 369)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 373)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 377)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 381)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 385)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 365)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 369)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 373)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 377)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 381)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 385)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 389)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 369)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 373)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 377)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 381)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 385)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 389)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 393)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 362)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 366)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 370)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 374)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 378)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 382)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 386)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 366)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 370)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 374)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 378)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 382)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 386)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 390)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 370)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 374)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 378)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 382)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 386)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 390)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 394)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 362)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 366)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 370)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 374)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 378)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 382)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 386)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 366)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 370)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 374)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 378)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 382)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 386)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 390)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 370)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 374)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 378)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 382)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 386)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 390)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 394)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 363)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 367)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 371)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 375)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 379)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 383)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 387)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 367)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 371)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 375)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 379)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 383)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 387)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 391)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 371)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 375)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 379)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 383)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 387)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 391)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 395)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 363)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 367)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 371)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 375)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 379)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 383)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 387)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 367)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 371)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 375)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 379)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 383)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 387)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 391)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 371)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 375)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 379)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 383)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 387)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 391)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 395)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 396)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 400)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 404)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 408)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 412)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 416)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 420)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 400)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 404)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 408)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 412)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 416)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 420)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 424)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 404)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 408)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 412)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 416)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 420)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 424)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 428)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 396)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 400)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 404)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 408)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 412)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 416)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 420)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 400)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 404)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 408)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 412)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 416)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 420)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 424)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 404)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 408)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 412)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 416)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 420)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 424)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 428)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 397)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 401)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 405)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 409)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 413)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 417)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 421)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 401)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 405)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 409)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 413)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 417)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 421)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 425)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 405)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 409)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 413)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 417)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 421)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 425)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 429)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 397)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 401)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 405)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 409)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 413)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 417)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 421)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 401)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 405)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 409)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 413)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 417)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 421)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 425)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 405)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 409)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 413)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 417)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 421)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 425)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 429)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 398)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 402)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 406)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 410)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 414)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 418)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 422)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 402)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 406)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 410)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 414)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 418)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 422)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 426)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 406)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 410)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 414)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 418)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 422)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 426)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 430)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 398)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 402)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 406)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 410)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 414)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 418)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 422)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 402)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 406)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 410)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 414)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 418)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 422)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 426)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 406)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 410)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 414)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 418)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 422)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 426)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 430)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 399)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 403)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 407)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 411)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 415)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 419)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 423)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 403)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 407)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 411)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 415)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 419)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 423)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 427)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 407)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 411)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 415)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 419)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 423)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 427)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 431)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 399)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 403)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 407)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 411)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 415)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 419)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 423)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 403)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 407)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 411)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 415)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 419)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 423)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 427)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 407)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 411)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 415)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 419)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 423)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 427)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 431)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 648)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 652)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 656)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 664)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 668)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 672)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 652)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 656)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 664)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 668)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 672)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 676)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 656)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 664)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 668)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 672)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 676)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 680)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 648)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 652)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 656)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 664)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 668)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 672)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 652)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 656)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 664)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 668)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 672)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 676)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 656)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 664)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 668)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 672)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 676)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 680)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 649)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 653)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 665)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 669)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 673)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 653)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 665)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 669)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 673)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 677)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 665)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 669)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 673)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 677)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 681)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 649)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 653)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 665)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 669)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 673)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 653)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 665)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 669)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 673)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 677)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 665)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 669)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 673)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 677)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 681)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 650)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 654)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 666)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 670)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 674)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 654)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 666)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 670)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 674)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 678)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 666)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 670)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 674)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 678)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 682)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 650)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 654)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 666)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 670)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 674)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 654)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 666)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 670)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 674)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 678)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 666)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 670)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 674)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 678)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 682)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 651)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 655)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 667)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 671)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 675)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 655)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 667)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 671)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 675)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 679)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 667)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 671)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 675)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 679)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 683)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 651)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 655)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 667)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 671)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 675)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 655)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 667)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 671)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 675)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 679)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 667)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 671)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 675)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 679)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 683)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 684)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 688)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 692)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 696)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 700)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 704)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 708)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 688)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 692)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 696)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 700)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 704)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 708)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 712)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 692)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 696)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 700)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 704)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 708)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 712)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 716)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 684)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 688)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 692)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 696)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 700)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 704)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 708)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 688)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 692)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 696)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 700)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 704)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 708)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 712)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 692)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 696)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 700)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 704)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 708)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 712)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 716)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 685)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 689)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 693)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 697)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 701)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 705)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 709)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 689)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 693)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 697)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 701)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 705)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 709)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 713)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 693)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 697)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 701)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 705)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 709)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 713)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 717)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 685)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 689)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 693)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 697)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 701)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 705)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 709)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 689)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 693)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 697)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 701)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 705)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 709)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 713)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 693)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 697)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 701)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 705)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 709)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 713)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 717)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 686)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 690)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 694)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 698)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 702)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 706)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 710)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 690)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 694)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 698)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 702)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 706)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 710)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 714)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 694)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 698)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 702)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 706)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 710)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 714)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 718)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 686)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 690)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 694)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 698)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 702)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 706)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 710)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 690)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 694)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 698)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 702)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 706)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 710)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 714)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 694)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 698)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 702)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 706)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 710)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 714)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 718)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 687)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 691)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 695)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 699)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 703)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 707)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 711)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 691)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 695)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 699)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 703)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 707)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 711)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 715)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 695)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 699)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 703)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 707)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 711)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 715)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 719)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 687)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 691)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 695)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 699)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 703)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 707)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 711)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 691)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 695)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 699)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 703)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 707)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 711)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 715)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 695)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 699)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 703)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 707)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 711)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 715)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 719)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 720)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 724)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 724)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 748)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 748)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 752)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 720)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 724)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 724)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 748)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 748)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 752)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 721)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 725)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 725)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 749)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 749)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 753)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 721)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 725)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 725)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 749)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 749)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 753)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 722)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 726)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 726)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 750)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 750)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 754)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 722)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 726)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 726)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 750)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 750)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 754)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 723)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 727)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 727)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 751)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 751)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 755)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 723)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 727)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 727)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 751)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 751)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 755)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 972)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 976)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 980)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 984)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 988)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 992)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 996)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 976)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 980)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 984)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 988)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 992)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 996)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1000)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 980)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 984)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 988)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 992)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 996)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1000)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1004)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 972)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 976)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 980)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 984)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 988)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 992)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 996)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 976)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 980)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 984)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 988)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 992)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 996)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1000)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 980)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 984)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 988)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 992)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 996)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1000)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1004)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 973)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 977)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 981)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 985)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 989)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 993)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 997)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 977)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 981)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 985)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 989)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 993)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 997)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1001)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 981)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 985)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 989)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 993)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 997)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1001)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1005)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 973)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 977)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 981)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 985)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 989)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 993)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 997)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 977)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 981)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 985)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 989)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 993)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 997)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1001)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 981)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 985)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 989)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 993)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 997)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1001)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1005)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 974)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 978)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 982)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 986)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 990)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 994)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 998)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 978)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 982)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 986)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 990)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 994)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 998)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1002)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 982)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 986)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 990)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 994)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 998)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1002)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1006)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 974)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 978)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 982)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 986)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 990)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 994)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 998)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 978)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 982)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 986)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 990)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 994)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 998)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1002)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 982)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 986)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 990)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 994)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 998)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1002)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1006)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 975)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 979)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 983)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 987)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 991)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 995)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 999)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 979)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 983)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 987)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 991)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 995)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 999)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1003)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 983)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 987)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 991)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 995)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 999)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1003)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1007)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 975)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 979)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 983)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 987)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 991)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 995)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 999)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 979)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 983)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 987)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 991)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 995)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 999)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1003)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 983)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 987)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 991)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 995)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 999)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1003)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1007)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1008)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1012)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1016)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1020)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1024)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1028)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1032)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1012)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1016)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1020)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1024)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1028)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1032)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1036)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1016)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4096)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1020)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4096)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1024)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4096)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1028)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4096)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1032)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4096)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1036)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4096)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1040)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4096)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1008)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1012)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1016)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1020)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1024)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1028)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1032)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1012)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1016)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1020)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1024)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1028)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1032)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1036)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1016)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4097)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1020)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4097)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1024)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4097)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1028)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4097)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1032)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4097)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1036)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4097)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1040)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4097)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1009)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1013)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1017)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1021)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1025)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1029)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1033)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1013)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1017)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1021)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1025)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1029)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1033)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1037)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1017)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4128)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1021)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4128)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1025)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4128)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1029)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4128)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1033)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4128)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1037)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4128)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1041)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4128)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1009)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1013)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1017)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1021)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1025)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1029)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1033)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1013)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1017)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1021)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1025)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1029)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1033)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1037)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1017)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4129)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1021)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4129)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1025)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4129)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1029)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4129)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1033)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4129)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1037)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4129)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1041)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4129)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1010)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1014)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1018)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1022)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1026)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1030)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1034)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1014)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1018)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1022)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1026)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1030)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1034)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1038)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1018)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4160)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1022)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4160)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1026)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4160)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1030)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4160)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1034)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4160)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1038)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4160)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1042)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4160)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1010)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1014)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1018)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1022)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1026)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1030)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1034)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1014)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1018)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1022)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1026)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1030)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1034)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1038)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1018)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4161)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1022)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4161)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1026)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4161)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1030)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4161)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1034)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4161)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1038)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4161)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1042)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4161)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1011)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1015)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1019)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1023)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1027)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1031)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1035)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1015)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1019)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1023)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1027)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1031)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1035)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1039)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1019)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4192)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1023)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4192)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1027)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4192)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1031)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4192)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1035)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4192)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1039)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4192)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1043)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4192)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1011)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1015)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1019)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1023)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1027)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1031)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1035)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1015)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1019)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1023)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1027)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1031)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1035)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1039)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1019)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4193)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1023)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4193)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1027)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4193)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1031)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4193)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1035)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4193)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1039)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4193)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1043)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4193)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1044)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4224)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1048)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4224)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4224)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4224)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4224)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4224)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4224)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1048)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4352)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4352)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4352)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4352)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4352)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4352)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1072)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4352)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4480)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4480)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4480)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4480)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4480)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1072)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4480)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1076)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4480)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1044)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4225)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1048)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4225)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4225)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4225)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4225)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4225)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4225)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1048)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4353)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4353)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4353)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4353)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4353)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4353)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1072)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4353)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4481)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4481)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4481)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4481)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4481)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1072)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4481)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1076)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4481)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1045)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4256)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1049)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4256)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4256)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4256)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4256)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4256)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4256)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1049)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4384)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4384)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4384)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4384)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4384)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4384)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1073)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4384)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4512)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4512)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4512)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4512)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4512)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1073)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4512)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1077)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4512)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1045)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4257)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1049)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4257)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4257)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4257)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4257)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4257)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4257)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1049)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4385)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4385)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4385)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4385)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4385)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4385)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1073)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4385)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4513)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4513)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4513)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4513)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4513)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1073)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4513)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1077)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4513)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1046)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4288)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1050)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4288)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4288)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4288)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4288)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4288)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4288)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1050)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4416)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4416)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4416)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4416)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4416)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4416)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1074)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4416)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4544)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4544)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4544)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4544)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4544)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1074)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4544)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1078)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4544)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1046)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4289)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1050)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4289)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4289)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4289)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4289)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4289)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4289)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1050)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4417)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4417)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4417)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4417)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4417)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4417)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1074)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4417)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4545)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4545)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4545)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4545)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4545)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1074)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4545)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1078)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4545)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1047)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4320)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1051)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4320)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4320)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4320)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4320)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4320)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4320)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1051)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4448)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4448)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4448)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4448)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4448)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4448)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1075)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4448)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4576)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4576)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4576)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4576)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4576)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1075)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4576)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1079)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4576)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1047)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4321)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1051)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4321)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4321)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4321)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4321)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4321)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4321)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1051)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4449)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4449)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4449)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4449)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4449)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4449)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1075)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4449)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4577)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4577)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4577)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4577)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4577)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1075)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4577)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 36) + 1079)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4577)]));
  conv2d_nhwc[((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2))] = conv2d_nhwc_local[0];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 1)] = conv2d_nhwc_local[1];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 128)] = conv2d_nhwc_local[2];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 129)] = conv2d_nhwc_local[3];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 256)] = conv2d_nhwc_local[4];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 257)] = conv2d_nhwc_local[5];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 384)] = conv2d_nhwc_local[6];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 385)] = conv2d_nhwc_local[7];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 512)] = conv2d_nhwc_local[8];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 513)] = conv2d_nhwc_local[9];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 640)] = conv2d_nhwc_local[10];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 641)] = conv2d_nhwc_local[11];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 768)] = conv2d_nhwc_local[12];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 25088) + ((((int)threadIdx.x) >> 4) * 3584)) + (((((int)blockIdx.x) & 15) >> 2) * 896)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 769)] = conv2d_nhwc_local[13];
}


