
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[28];
  __shared__ float PadInput_shared[1344];
  __shared__ float weight_shared[1536];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[8] = 0.000000e+00f;
  conv2d_nhwc_local[9] = 0.000000e+00f;
  conv2d_nhwc_local[12] = 0.000000e+00f;
  conv2d_nhwc_local[13] = 0.000000e+00f;
  conv2d_nhwc_local[16] = 0.000000e+00f;
  conv2d_nhwc_local[17] = 0.000000e+00f;
  conv2d_nhwc_local[20] = 0.000000e+00f;
  conv2d_nhwc_local[21] = 0.000000e+00f;
  conv2d_nhwc_local[24] = 0.000000e+00f;
  conv2d_nhwc_local[25] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  conv2d_nhwc_local[7] = 0.000000e+00f;
  conv2d_nhwc_local[10] = 0.000000e+00f;
  conv2d_nhwc_local[11] = 0.000000e+00f;
  conv2d_nhwc_local[14] = 0.000000e+00f;
  conv2d_nhwc_local[15] = 0.000000e+00f;
  conv2d_nhwc_local[18] = 0.000000e+00f;
  conv2d_nhwc_local[19] = 0.000000e+00f;
  conv2d_nhwc_local[22] = 0.000000e+00f;
  conv2d_nhwc_local[23] = 0.000000e+00f;
  conv2d_nhwc_local[26] = 0.000000e+00f;
  conv2d_nhwc_local[27] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) * 2)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 64)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + 3584))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + 7168))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 192)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + 10752))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + 14336))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 320)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + 17920))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + 21504))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2048))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 4096))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 6144))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + 16))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + 3600))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 576)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + 7184))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + 10768))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 704)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + 14352))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + 17936))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 832)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + 21520))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 8192))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 10240))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 12288))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14336))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int rh_0_rw_0_rc_0_fused = 0; rh_0_rw_0_rc_0_fused < 6; ++rh_0_rw_0_rc_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((rh_0_rw_0_rc_0_fused + 2) % 3) * 448) + (((int)threadIdx.x) * 2))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + (rh_0_rw_0_rc_0_fused * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 32))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 448) + (((int)threadIdx.x) * 2)) + 64)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + (rh_0_rw_0_rc_0_fused * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 3616))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 448) + (((int)threadIdx.x) * 2)) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + (rh_0_rw_0_rc_0_fused * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 7200))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 448) + (((int)threadIdx.x) * 2)) + 192)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + (rh_0_rw_0_rc_0_fused * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 10784))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 448) + (((int)threadIdx.x) * 2)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + (rh_0_rw_0_rc_0_fused * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 14368))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 448) + (((int)threadIdx.x) * 2)) + 320)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + (rh_0_rw_0_rc_0_fused * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 17952))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 448) + (((int)threadIdx.x) * 2)) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 112) * 25088) + (((((int)blockIdx.x) % 112) >> 4) * 512)) + ((((int)threadIdx.x) >> 3) * 128)) + (rh_0_rw_0_rc_0_fused * 16)) + ((((int)threadIdx.x) & 7) * 2)) + 21536))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((rh_0_rw_0_rc_0_fused + 2) % 3) * 512) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 16384))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 512) + (((int)threadIdx.x) * 4)) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 18432))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 512) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 20480))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 512) + (((int)threadIdx.x) * 4)) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 22528))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16))] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16))] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 64)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 128)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 192)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 256)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 256)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 320)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 320)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 384)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4))]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 384)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 1)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 257)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 257)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 321)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 321)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 385)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 32)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 385)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 33)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 258)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 258)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 322)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 322)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 386)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 64)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 386)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 65)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 259)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 259)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 323)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 323)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 387)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 96)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 387)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 97)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 260)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 260)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 324)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 324)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 388)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 128)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 388)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 129)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 261)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 261)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 325)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 325)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 389)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 160)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 389)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 161)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 262)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 262)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 326)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 326)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 390)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 192)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 390)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 193)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 263)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 263)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 327)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 327)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 391)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 224)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 391)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 225)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 264)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 264)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 328)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 328)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 392)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 256)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 392)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 257)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 265)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 265)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 329)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 329)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 393)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 288)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 393)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 289)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 266)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 266)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 330)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 330)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 394)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 320)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 394)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 321)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 267)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 267)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 331)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 331)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 395)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 352)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 395)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 353)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 268)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 268)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 332)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 332)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 396)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 384)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 396)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 385)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 269)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 269)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 333)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 333)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 397)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 416)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 397)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 417)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 270)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 270)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 334)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 334)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 398)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 448)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 398)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 449)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 271)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 271)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 335)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 335)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 399)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 480)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 399)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 481)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16))] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16))] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 256)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 256)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 320)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 320)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 384)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 2)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 384)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 3)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 257)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 257)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 321)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 321)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 385)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 34)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 385)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 35)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 258)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 258)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 322)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 322)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 386)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 66)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 386)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 67)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 259)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 259)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 323)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 323)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 387)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 98)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 387)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 99)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 260)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 260)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 324)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 324)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 388)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 130)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 388)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 131)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 261)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 261)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 325)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 325)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 389)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 162)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 389)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 163)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 262)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 262)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 326)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 326)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 390)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 194)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 390)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 195)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 263)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 263)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 327)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 327)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 391)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 226)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 391)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 227)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 264)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 264)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 328)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 328)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 392)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 258)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 392)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 259)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 265)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 265)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 329)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 329)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 393)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 290)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 393)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 291)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 266)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 266)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 330)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 330)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 394)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 322)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 394)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 323)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 267)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 267)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 331)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 331)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 395)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 354)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 395)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 355)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 268)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 268)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 332)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 332)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 396)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 386)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 396)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 387)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 269)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 269)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 333)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 333)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 397)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 418)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 397)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 419)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 270)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 270)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 334)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 334)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 398)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 450)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 398)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 451)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 271)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 271)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 335)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 335)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 399)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 482)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 3) * 16)) + 399)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 512) + ((((int)threadIdx.x) & 7) * 4)) + 483)]));
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((int)threadIdx.x) >> 3) * 16)] * weight_shared[((((int)threadIdx.x) & 7) * 4)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((int)threadIdx.x) >> 3) * 16)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 64)] * weight_shared[((((int)threadIdx.x) & 7) * 4)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 64)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 128)] * weight_shared[((((int)threadIdx.x) & 7) * 4)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 128)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 192)] * weight_shared[((((int)threadIdx.x) & 7) * 4)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 192)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 256)] * weight_shared[((((int)threadIdx.x) & 7) * 4)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 256)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 320)] * weight_shared[((((int)threadIdx.x) & 7) * 4)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 320)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 384)] * weight_shared[((((int)threadIdx.x) & 7) * 4)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 384)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 1)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 32)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 33)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 65)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 32)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 65)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 33)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 129)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 32)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 129)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 33)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 193)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 32)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 193)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 33)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 257)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 32)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 257)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 33)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 321)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 32)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 321)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 33)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 385)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 32)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 385)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 33)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 2)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 64)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 2)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 65)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 66)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 64)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 66)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 65)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 130)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 64)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 130)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 65)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 194)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 64)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 194)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 65)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 258)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 64)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 258)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 65)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 322)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 64)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 322)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 65)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 386)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 64)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 386)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 65)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 3)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 96)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 3)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 97)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 67)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 96)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 67)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 97)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 131)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 96)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 131)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 97)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 195)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 96)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 195)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 97)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 259)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 96)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 259)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 97)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 323)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 96)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 323)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 97)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 387)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 96)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 387)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 97)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 4)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 128)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 4)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 129)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 68)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 128)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 68)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 129)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 132)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 128)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 132)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 129)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 196)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 128)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 196)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 129)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 260)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 128)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 260)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 129)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 324)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 128)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 324)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 129)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 388)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 128)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 388)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 129)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 5)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 160)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 5)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 161)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 69)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 160)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 69)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 161)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 133)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 160)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 133)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 161)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 197)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 160)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 197)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 161)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 261)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 160)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 261)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 161)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 325)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 160)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 325)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 161)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 389)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 160)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 389)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 161)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 6)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 192)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 6)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 193)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 70)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 192)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 70)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 193)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 134)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 192)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 134)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 193)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 198)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 192)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 198)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 193)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 262)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 192)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 262)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 193)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 326)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 192)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 326)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 193)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 390)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 192)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 390)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 193)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 7)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 224)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 7)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 225)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 71)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 224)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 71)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 225)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 135)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 224)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 135)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 225)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 199)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 224)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 199)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 225)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 263)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 224)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 263)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 225)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 327)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 224)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 327)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 225)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 391)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 224)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 391)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 225)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 8)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 256)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 8)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 257)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 72)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 256)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 72)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 257)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 136)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 256)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 136)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 257)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 200)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 256)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 200)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 257)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 264)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 256)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 264)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 257)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 328)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 256)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 328)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 257)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 392)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 256)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 392)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 257)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 9)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 288)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 9)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 289)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 73)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 288)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 73)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 289)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 137)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 288)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 137)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 289)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 201)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 288)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 201)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 289)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 265)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 288)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 265)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 289)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 329)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 288)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 329)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 289)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 393)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 288)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 393)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 289)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 10)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 320)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 10)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 321)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 74)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 320)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 74)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 321)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 138)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 320)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 138)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 321)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 202)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 320)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 202)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 321)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 266)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 320)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 266)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 321)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 330)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 320)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 330)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 321)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 394)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 320)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 394)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 321)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 11)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 352)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 11)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 353)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 75)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 352)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 75)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 353)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 139)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 352)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 139)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 353)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 203)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 352)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 203)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 353)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 267)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 352)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 267)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 353)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 331)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 352)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 331)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 353)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 395)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 352)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 395)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 353)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 12)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 384)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 12)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 385)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 76)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 384)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 76)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 385)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 140)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 384)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 140)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 385)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 204)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 384)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 204)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 385)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 268)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 384)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 268)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 385)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 332)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 384)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 332)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 385)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 396)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 384)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 396)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 385)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 13)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 416)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 13)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 417)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 77)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 416)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 77)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 417)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 141)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 416)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 141)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 417)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 205)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 416)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 205)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 417)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 269)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 416)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 269)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 417)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 333)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 416)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 333)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 417)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 397)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 416)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 397)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 417)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 14)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 448)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 14)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 449)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 78)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 448)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 78)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 449)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 142)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 448)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 142)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 449)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 206)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 448)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 206)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 449)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 270)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 448)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 270)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 449)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 334)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 448)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 334)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 449)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 398)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 448)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 398)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 449)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 15)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 480)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 15)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 481)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 79)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 480)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 79)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 481)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 143)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 480)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 143)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 481)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 207)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 480)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 207)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 481)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 271)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 480)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 271)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 481)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 335)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 480)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 335)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 481)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 399)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 480)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 399)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 481)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((int)threadIdx.x) >> 3) * 16)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((int)threadIdx.x) >> 3) * 16)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 3)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 64)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 64)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 3)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 128)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 128)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 3)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 192)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 192)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 3)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 256)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 256)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 3)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 320)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 320)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 3)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 384)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 2)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 384)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 3)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 34)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 1)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 35)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 65)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 34)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 65)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 35)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 129)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 34)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 129)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 35)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 193)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 34)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 193)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 35)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 257)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 34)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 257)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 35)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 321)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 34)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 321)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 35)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 385)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 34)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 385)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 35)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 2)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 66)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 2)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 67)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 66)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 66)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 66)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 67)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 130)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 66)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 130)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 67)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 194)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 66)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 194)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 67)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 258)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 66)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 258)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 67)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 322)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 66)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 322)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 67)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 386)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 66)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 386)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 67)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 3)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 98)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 3)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 99)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 67)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 98)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 67)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 99)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 131)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 98)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 131)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 99)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 195)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 98)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 195)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 99)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 259)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 98)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 259)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 99)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 323)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 98)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 323)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 99)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 387)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 98)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 387)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 99)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 4)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 130)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 4)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 131)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 68)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 130)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 68)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 131)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 132)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 130)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 132)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 131)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 196)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 130)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 196)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 131)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 260)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 130)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 260)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 131)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 324)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 130)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 324)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 131)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 388)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 130)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 388)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 131)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 5)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 162)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 5)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 163)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 69)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 162)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 69)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 163)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 133)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 162)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 133)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 163)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 197)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 162)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 197)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 163)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 261)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 162)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 261)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 163)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 325)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 162)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 325)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 163)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 389)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 162)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 389)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 163)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 6)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 194)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 6)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 195)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 70)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 194)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 70)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 195)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 134)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 194)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 134)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 195)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 198)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 194)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 198)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 195)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 262)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 194)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 262)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 195)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 326)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 194)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 326)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 195)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 390)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 194)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 390)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 195)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 7)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 226)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 7)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 227)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 71)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 226)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 71)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 227)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 135)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 226)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 135)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 227)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 199)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 226)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 199)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 227)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 263)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 226)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 263)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 227)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 327)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 226)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 327)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 227)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 391)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 226)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 391)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 227)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 8)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 258)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 8)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 259)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 72)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 258)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 72)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 259)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 136)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 258)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 136)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 259)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 200)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 258)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 200)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 259)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 264)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 258)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 264)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 259)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 328)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 258)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 328)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 259)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 392)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 258)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 392)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 259)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 9)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 290)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 9)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 291)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 73)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 290)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 73)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 291)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 137)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 290)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 137)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 291)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 201)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 290)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 201)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 291)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 265)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 290)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 265)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 291)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 329)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 290)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 329)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 291)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 393)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 290)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 393)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 291)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 10)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 322)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 10)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 323)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 74)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 322)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 74)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 323)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 138)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 322)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 138)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 323)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 202)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 322)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 202)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 323)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 266)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 322)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 266)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 323)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 330)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 322)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 330)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 323)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 394)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 322)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 394)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 323)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 11)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 354)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 11)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 355)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 75)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 354)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 75)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 355)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 139)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 354)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 139)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 355)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 203)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 354)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 203)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 355)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 267)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 354)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 267)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 355)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 331)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 354)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 331)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 355)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 395)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 354)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 395)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 355)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 12)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 386)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 12)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 387)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 76)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 386)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 76)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 387)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 140)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 386)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 140)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 387)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 204)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 386)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 204)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 387)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 268)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 386)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 268)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 387)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 332)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 386)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 332)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 387)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 396)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 386)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 396)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 387)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 13)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 418)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 13)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 419)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 77)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 418)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 77)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 419)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 141)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 418)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 141)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 419)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 205)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 418)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 205)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 419)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 269)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 418)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 269)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 419)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 333)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 418)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 333)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 419)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 397)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 418)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 397)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 419)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 14)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 450)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 14)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 451)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 78)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 450)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 78)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 451)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 142)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 450)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 142)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 451)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 206)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 450)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 206)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 451)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 270)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 450)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 270)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 451)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 334)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 450)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 334)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 451)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 398)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 450)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 398)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 451)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 15)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 482)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 15)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 483)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 79)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 482)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 79)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 483)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 143)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 482)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 143)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 483)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 207)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 482)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 207)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 483)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 271)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 482)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 271)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 483)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 335)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 482)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 335)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 483)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 399)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 482)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 399)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 483)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 448)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 448)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 512)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 512)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 576)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 576)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 640)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 640)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 704)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 704)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 768)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 768)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 832)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 512)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 832)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 513)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 449)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 449)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 513)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 513)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 577)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 577)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 641)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 641)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 705)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 705)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 769)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 769)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 833)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 544)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 833)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 545)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 450)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 450)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 514)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 514)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 578)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 578)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 642)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 642)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 706)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 706)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 770)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 770)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 834)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 576)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 834)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 577)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 451)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 451)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 515)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 515)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 579)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 579)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 643)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 643)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 707)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 707)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 771)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 771)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 835)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 608)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 835)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 609)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 452)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 452)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 516)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 516)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 580)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 580)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 644)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 644)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 708)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 708)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 772)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 772)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 836)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 640)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 836)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 641)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 453)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 453)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 517)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 517)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 581)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 581)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 645)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 645)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 709)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 709)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 773)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 773)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 837)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 672)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 837)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 673)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 454)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 454)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 518)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 518)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 582)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 582)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 646)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 646)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 710)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 710)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 774)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 774)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 838)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 704)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 838)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 705)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 455)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 455)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 519)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 519)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 583)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 583)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 647)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 647)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 711)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 711)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 775)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 775)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 839)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 736)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 839)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 737)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 456)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 456)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 520)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 520)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 584)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 584)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 648)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 648)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 712)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 712)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 776)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 776)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 840)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 768)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 840)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 769)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 457)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 457)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 521)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 521)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 585)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 585)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 649)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 649)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 713)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 713)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 777)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 777)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 841)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 800)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 841)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 801)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 458)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 458)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 522)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 522)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 586)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 586)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 650)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 650)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 714)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 714)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 778)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 778)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 842)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 832)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 842)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 833)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 459)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 459)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 523)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 523)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 587)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 587)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 651)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 651)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 715)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 715)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 779)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 779)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 843)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 864)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 843)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 865)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 460)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 460)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 524)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 524)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 588)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 588)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 652)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 652)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 716)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 716)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 780)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 780)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 844)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 896)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 844)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 897)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 461)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 461)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 525)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 525)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 589)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 589)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 653)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 653)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 717)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 717)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 781)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 781)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 845)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 928)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 845)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 929)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 462)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 462)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 526)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 526)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 590)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 590)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 654)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 654)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 718)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 718)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 782)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 782)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 846)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 960)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 846)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 961)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 463)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 463)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 527)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 527)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 591)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 591)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 655)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 655)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 719)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 719)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 783)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 783)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 847)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 992)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 847)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 993)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 448)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 448)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 512)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 512)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 576)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 576)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 640)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 640)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 704)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 704)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 768)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 768)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 832)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 514)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 832)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 515)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 449)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 449)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 513)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 513)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 577)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 577)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 641)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 641)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 705)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 705)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 769)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 769)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 833)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 546)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 833)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 547)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 450)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 450)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 514)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 514)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 578)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 578)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 642)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 642)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 706)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 706)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 770)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 770)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 834)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 578)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 834)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 579)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 451)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 451)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 515)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 515)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 579)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 579)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 643)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 643)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 707)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 707)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 771)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 771)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 835)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 610)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 835)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 611)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 452)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 452)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 516)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 516)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 580)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 580)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 644)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 644)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 708)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 708)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 772)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 772)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 836)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 642)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 836)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 643)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 453)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 453)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 517)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 517)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 581)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 581)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 645)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 645)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 709)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 709)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 773)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 773)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 837)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 674)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 837)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 675)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 454)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 454)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 518)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 518)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 582)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 582)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 646)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 646)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 710)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 710)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 774)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 774)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 838)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 706)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 838)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 707)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 455)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 455)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 519)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 519)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 583)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 583)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 647)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 647)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 711)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 711)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 775)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 775)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 839)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 738)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 839)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 739)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 456)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 456)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 520)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 520)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 584)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 584)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 648)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 648)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 712)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 712)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 776)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 776)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 840)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 770)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 840)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 771)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 457)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 457)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 521)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 521)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 585)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 585)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 649)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 649)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 713)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 713)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 777)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 777)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 841)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 802)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 841)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 803)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 458)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 458)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 522)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 522)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 586)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 586)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 650)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 650)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 714)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 714)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 778)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 778)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 842)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 834)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 842)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 835)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 459)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 459)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 523)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 523)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 587)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 587)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 651)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 651)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 715)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 715)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 779)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 779)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 843)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 866)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 843)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 867)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 460)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 460)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 524)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 524)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 588)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 588)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 652)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 652)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 716)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 716)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 780)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 780)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 844)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 898)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 844)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 899)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 461)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 461)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 525)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 525)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 589)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 589)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 653)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 653)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 717)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 717)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 781)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 781)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 845)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 930)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 845)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 931)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 462)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 462)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 526)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 526)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 590)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 590)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 654)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 654)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 718)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 718)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 782)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 782)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 846)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 962)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 846)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 963)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 463)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 463)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 527)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 527)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 591)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 591)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 655)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 655)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 719)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 719)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 783)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 783)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 847)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 994)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 3) * 16) + 847)] * weight_shared[(((((int)threadIdx.x) & 7) * 4) + 995)]));
  conv2d_nhwc[((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4))] = conv2d_nhwc_local[0];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1)] = conv2d_nhwc_local[1];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2)] = conv2d_nhwc_local[2];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3)] = conv2d_nhwc_local[3];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14336)] = conv2d_nhwc_local[4];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14337)] = conv2d_nhwc_local[5];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14338)] = conv2d_nhwc_local[6];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14339)] = conv2d_nhwc_local[7];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 28672)] = conv2d_nhwc_local[8];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 28673)] = conv2d_nhwc_local[9];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 28674)] = conv2d_nhwc_local[10];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 28675)] = conv2d_nhwc_local[11];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 43008)] = conv2d_nhwc_local[12];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 43009)] = conv2d_nhwc_local[13];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 43010)] = conv2d_nhwc_local[14];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 43011)] = conv2d_nhwc_local[15];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 57344)] = conv2d_nhwc_local[16];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 57345)] = conv2d_nhwc_local[17];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 57346)] = conv2d_nhwc_local[18];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 57347)] = conv2d_nhwc_local[19];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 71680)] = conv2d_nhwc_local[20];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 71681)] = conv2d_nhwc_local[21];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 71682)] = conv2d_nhwc_local[22];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 71683)] = conv2d_nhwc_local[23];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 86016)] = conv2d_nhwc_local[24];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 86017)] = conv2d_nhwc_local[25];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 86018)] = conv2d_nhwc_local[26];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 112) * 100352) + (((((int)blockIdx.x) % 112) >> 4) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 86019)] = conv2d_nhwc_local[27];
}


