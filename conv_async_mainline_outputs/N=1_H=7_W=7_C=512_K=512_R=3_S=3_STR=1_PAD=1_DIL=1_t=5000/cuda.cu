
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[7];
  __shared__ float PadInput_shared[864];
  __shared__ float weight_shared[9216];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((int)threadIdx.x)))
    );
    int src_bytes = (((24 <= ((int)threadIdx.x)) && (1 <= ((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3)))) && (((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)threadIdx.x) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + (((((int)threadIdx.x) % 24) >> 3) * 512)) + (((int)threadIdx.x) & 7)) - 4096))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((((int)threadIdx.x) + 32) / 24) * 24) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 8)) + (((int)threadIdx.x) & 7))))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) + 32) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 512)) + (((int)threadIdx.x) & 7)) - 4096))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((((int)threadIdx.x) + 64) / 24) * 24) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 8)) + (((int)threadIdx.x) & 7))))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) + 64) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 512)) + (((int)threadIdx.x) & 7)) - 4096))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 96)))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3))) && (((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)threadIdx.x) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + (((((int)threadIdx.x) % 24) >> 3) * 512)) + (((int)threadIdx.x) & 7)) + 10240))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((((int)threadIdx.x) + 128) / 24) * 24) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 8)) + (((int)threadIdx.x) & 7))))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) + 128) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 512)) + (((int)threadIdx.x) & 7)) - 4096))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((((int)threadIdx.x) + 160) / 24) * 24) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 8)) + (((int)threadIdx.x) & 7))))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) + 160) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 512)) + (((int)threadIdx.x) & 7)) - 4096))), "n"(4), "r"(src_bytes)
    );
  }
  if (((int)threadIdx.x) < 24) {
    PadInput_shared[(((int)threadIdx.x) + 192)] = 0.000000e+00f;
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2048))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 262144))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 264192))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 524288))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 526336))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 786432))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 788480))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1048576))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1152)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1050624))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1310720))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1408)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1312768))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1572864))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1664)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1574912))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1835008))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1920)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1837056))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2097152))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2176)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2099200))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 216)))
    );
    int src_bytes = (((24 <= ((int)threadIdx.x)) && (1 <= ((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3)))) && (((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)threadIdx.x) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + (((((int)threadIdx.x) % 24) >> 3) * 512)) + (((int)threadIdx.x) & 7)) - 4088))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((((int)threadIdx.x) + 32) / 24) * 24) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 8)) + (((int)threadIdx.x) & 7)) + 216)))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) + 32) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 512)) + (((int)threadIdx.x) & 7)) - 4088))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((((int)threadIdx.x) + 64) / 24) * 24) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 8)) + (((int)threadIdx.x) & 7)) + 216)))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) + 64) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 512)) + (((int)threadIdx.x) & 7)) - 4088))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 312)))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3))) && (((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)threadIdx.x) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + (((((int)threadIdx.x) % 24) >> 3) * 512)) + (((int)threadIdx.x) & 7)) + 10248))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((((int)threadIdx.x) + 128) / 24) * 24) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 8)) + (((int)threadIdx.x) & 7)) + 216)))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) + 128) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 512)) + (((int)threadIdx.x) & 7)) - 4088))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((((int)threadIdx.x) + 160) / 24) * 24) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 8)) + (((int)threadIdx.x) & 7)) + 216)))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) + 160) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 512)) + (((int)threadIdx.x) & 7)) - 4088))), "n"(4), "r"(src_bytes)
    );
  }
  if (((int)threadIdx.x) < 24) {
    PadInput_shared[(((int)threadIdx.x) + 408)] = 0.000000e+00f;
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2304)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 4096))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2432)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 6144))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2560)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 266240))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2688)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 268288))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2816)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 528384))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2944)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 530432))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3072)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 790528))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3200)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 792576))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3328)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1052672))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3456)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1054720))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3584)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1314816))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3712)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1316864))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3840)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1576960))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3968)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1579008))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4096)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1839104))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4224)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1841152))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4352)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2101248))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4480)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2103296))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 432)))
    );
    int src_bytes = (((24 <= ((int)threadIdx.x)) && (1 <= ((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3)))) && (((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)threadIdx.x) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + (((((int)threadIdx.x) % 24) >> 3) * 512)) + (((int)threadIdx.x) & 7)) - 4080))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((((int)threadIdx.x) + 32) / 24) * 24) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 8)) + (((int)threadIdx.x) & 7)) + 432)))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) + 32) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 512)) + (((int)threadIdx.x) & 7)) - 4080))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((((int)threadIdx.x) + 64) / 24) * 24) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 8)) + (((int)threadIdx.x) & 7)) + 432)))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) + 64) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 512)) + (((int)threadIdx.x) & 7)) - 4080))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 528)))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3))) && (((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)threadIdx.x) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + (((((int)threadIdx.x) % 24) >> 3) * 512)) + (((int)threadIdx.x) & 7)) + 10256))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((((int)threadIdx.x) + 128) / 24) * 24) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 8)) + (((int)threadIdx.x) & 7)) + 432)))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) + 128) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 512)) + (((int)threadIdx.x) & 7)) - 4080))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((((int)threadIdx.x) + 160) / 24) * 24) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 8)) + (((int)threadIdx.x) & 7)) + 432)))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) + 160) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 512)) + (((int)threadIdx.x) & 7)) - 4080))), "n"(4), "r"(src_bytes)
    );
  }
  if (((int)threadIdx.x) < 24) {
    PadInput_shared[(((int)threadIdx.x) + 624)] = 0.000000e+00f;
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4608)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 8192))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4736)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 10240))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4864)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 270336))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4992)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 272384))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5120)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 532480))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5248)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 534528))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5376)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 794624))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5504)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 796672))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5632)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1056768))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5760)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1058816))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5888)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1318912))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 6016)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1320960))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 6144)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1581056))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 6272)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1583104))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 6400)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1843200))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 6528)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1845248))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 6656)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2105344))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 6784)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2107392))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int rh_0_rw_0_rc_0_fused = 0; rh_0_rw_0_rc_0_fused < 61; ++rh_0_rw_0_rc_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 216) + ((int)threadIdx.x))))
    );
    int src_bytes = (((24 <= ((int)threadIdx.x)) && (1 <= ((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3)))) && (((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + (((((int)threadIdx.x) % 24) >> 3) * 512)) + (rh_0_rw_0_rc_0_fused * 8)) + (((int)threadIdx.x) & 7)) - 4072))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((rh_0_rw_0_rc_0_fused + 3) & 3) * 216) + (((((int)threadIdx.x) + 32) / 24) * 24)) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 8)) + (((int)threadIdx.x) & 7))))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)threadIdx.x) + 32) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 512)) + (rh_0_rw_0_rc_0_fused * 8)) + (((int)threadIdx.x) & 7)) - 4072))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((rh_0_rw_0_rc_0_fused + 3) & 3) * 216) + (((((int)threadIdx.x) + 64) / 24) * 24)) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 8)) + (((int)threadIdx.x) & 7))))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)threadIdx.x) + 64) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 512)) + (rh_0_rw_0_rc_0_fused * 8)) + (((int)threadIdx.x) & 7)) - 4072))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 216) + ((int)threadIdx.x)) + 96)))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3))) && (((((int)blockIdx.x) >> 4) + ((((int)threadIdx.x) % 24) >> 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)threadIdx.x) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + (((((int)threadIdx.x) % 24) >> 3) * 512)) + (rh_0_rw_0_rc_0_fused * 8)) + (((int)threadIdx.x) & 7)) + 10264))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((rh_0_rw_0_rc_0_fused + 3) & 3) * 216) + (((((int)threadIdx.x) + 128) / 24) * 24)) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 8)) + (((int)threadIdx.x) & 7))))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 1) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)threadIdx.x) + 128) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 512)) + (rh_0_rw_0_rc_0_fused * 8)) + (((int)threadIdx.x) & 7)) - 4072))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((((rh_0_rw_0_rc_0_fused + 3) & 3) * 216) + (((((int)threadIdx.x) + 160) / 24) * 24)) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 8)) + (((int)threadIdx.x) & 7))))
    );
    int src_bytes = ((1 <= ((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3))) && (((((int)blockIdx.x) >> 4) + (((((int)threadIdx.x) >> 3) + 2) % 3)) < 8)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)threadIdx.x) + 160) / 24) * 3584) + ((((int)blockIdx.x) >> 4) * 512)) + ((((((int)threadIdx.x) >> 3) + 2) % 3) * 512)) + (rh_0_rw_0_rc_0_fused * 8)) + (((int)threadIdx.x) & 7)) - 4072))), "n"(4), "r"(src_bytes)
    );
  }
    if (((int)threadIdx.x) < 24) {
      PadInput_shared[(((((rh_0_rw_0_rc_0_fused + 3) & 3) * 216) + ((int)threadIdx.x)) + 192)] = 0.000000e+00f;
    }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 12288))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14336))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 274432))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 276480))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 536576))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 538624))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 798720))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 800768))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1060864))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 1152)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1062912))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1323008))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 1408)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1325056))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1585152))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 1664)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1587200))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1847296))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 1920)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1849344))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2109440))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 2176)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2111488))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

    __syncthreads();
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((rh_0_rw_0_rc_0_fused & 3) * 216)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 24)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 48)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 72)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 120)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 96)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 120)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 144)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 120)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 144)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 168)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 144)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 168)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 121)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 121)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 121)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 169)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 169)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 122)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 122)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 122)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 170)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 170)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 123)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 123)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 123)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 171)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 171)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 124)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 124)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 124)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 172)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 172)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 125)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 125)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 125)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 173)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 173)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 126)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 126)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 126)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 174)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 174)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 127)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 127)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 127)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 175)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 175)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 32)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 32)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 176)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 176)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 177)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 177)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 178)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 178)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 179)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 179)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 180)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 180)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 181)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 181)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 182)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 182)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2016)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2016)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2016)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2016)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2016)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 183)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2016)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 183)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2016)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2048)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2048)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 112)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2048)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 112)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2048)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 112)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 160)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2048)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 160)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 184)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2048)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 160)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 184)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 208)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2048)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2080)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2080)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 113)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2080)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 113)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2080)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 113)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 161)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2080)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 161)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 185)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2080)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 161)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 185)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 209)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2080)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2112)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2112)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 114)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2112)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 114)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2112)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 114)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 162)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2112)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 162)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 186)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2112)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 162)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 186)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 210)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2112)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2144)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2144)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 115)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2144)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 115)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2144)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 115)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 163)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2144)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 163)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 187)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2144)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 163)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 187)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 211)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2144)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2176)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2176)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 116)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2176)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 116)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2176)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 116)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 164)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2176)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 164)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 188)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2176)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 164)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 188)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 212)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2176)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2208)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2208)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 117)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2208)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 117)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2208)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 117)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 165)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2208)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 165)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 189)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2208)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 165)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 189)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 213)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2208)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2240)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2240)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 118)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2240)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 118)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2240)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 118)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 166)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2240)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 166)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 190)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2240)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 166)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 190)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 214)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2240)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2272)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2272)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 119)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2272)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 119)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2272)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 119)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 167)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2272)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 167)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 191)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2272)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 167)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 191)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 216) + 215)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + ((int)threadIdx.x)) + 2272)]));
  }
__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[216] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[240] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[264] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[240] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[264] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[288] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[264] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[288] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[312] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[288] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[312] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[336] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[312] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[336] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[360] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[336] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[360] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[384] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[360] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[384] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[408] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[217] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[241] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[265] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[241] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[265] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[289] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[265] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[289] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[313] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[289] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[313] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[337] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[313] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[337] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[361] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[337] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[361] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[385] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[361] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[385] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[409] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[218] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[242] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[266] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[242] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[266] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[290] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[266] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[290] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[314] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[290] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[314] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[338] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[314] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[338] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[362] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[338] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[362] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[386] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[362] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[386] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[410] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[219] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[243] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[267] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[243] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[267] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[291] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[267] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[291] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[315] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[291] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[315] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[339] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[315] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[339] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[363] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[339] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[363] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[387] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[363] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[387] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[411] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[220] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[244] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[268] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[244] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[268] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[292] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[268] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[292] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[316] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[292] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[316] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[340] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[316] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[340] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[364] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[340] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[364] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[388] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[364] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[388] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[412] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[221] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[245] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[269] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[245] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[269] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[293] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[269] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[293] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[317] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[293] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[317] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[341] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[317] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[341] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[365] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[341] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[365] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[389] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[365] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[389] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[413] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[222] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[246] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[270] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[246] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[270] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[294] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[270] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[294] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[318] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[294] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[318] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[342] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[318] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[342] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[366] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[342] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[366] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[390] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[366] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[390] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[414] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[223] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[247] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[271] * weight_shared[(((int)threadIdx.x) + 4064)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[247] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[271] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[295] * weight_shared[(((int)threadIdx.x) + 4064)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[271] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[295] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[319] * weight_shared[(((int)threadIdx.x) + 4064)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[295] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[319] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[343] * weight_shared[(((int)threadIdx.x) + 4064)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[319] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[343] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[367] * weight_shared[(((int)threadIdx.x) + 4064)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[343] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[367] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[391] * weight_shared[(((int)threadIdx.x) + 4064)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[367] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[391] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[415] * weight_shared[(((int)threadIdx.x) + 4064)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[224] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[248] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[272] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[248] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[272] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[296] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[272] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[296] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[320] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[296] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[320] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[344] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[320] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[344] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[368] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[344] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[368] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[392] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[368] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[392] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[416] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[225] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[249] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[273] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[249] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[273] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[297] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[273] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[297] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[321] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[297] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[321] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[345] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[321] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[345] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[369] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[345] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[369] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[393] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[369] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[393] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[417] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[226] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[250] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[274] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[250] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[274] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[298] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[274] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[298] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[322] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[298] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[322] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[346] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[322] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[346] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[370] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[346] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[370] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[394] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[370] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[394] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[418] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[227] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[251] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[275] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[251] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[275] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[299] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[275] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[299] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[323] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[299] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[323] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[347] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[323] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[347] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[371] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[347] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[371] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[395] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[371] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[395] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[419] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[228] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[252] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[276] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[252] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[276] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[300] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[276] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[300] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[324] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[300] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[324] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[348] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[324] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[348] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[372] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[348] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[372] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[396] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[372] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[396] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[420] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[229] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[253] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[277] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[253] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[277] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[301] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[277] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[301] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[325] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[301] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[325] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[349] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[325] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[349] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[373] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[349] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[373] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[397] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[373] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[397] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[421] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[230] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[254] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[278] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[254] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[278] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[302] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[278] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[302] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[326] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[302] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[326] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[350] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[326] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[350] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[374] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[350] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[374] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[398] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[374] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[398] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[422] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[231] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[255] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[279] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[255] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[279] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[303] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[279] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[303] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[327] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[303] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[327] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[351] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[327] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[351] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[375] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[351] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[375] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[399] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[375] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[399] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[423] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[232] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[256] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[280] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[256] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[280] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[304] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[280] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[304] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[328] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[304] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[328] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[352] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[328] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[352] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[376] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[352] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[376] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[400] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[376] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[400] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[424] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[233] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[257] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[281] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[257] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[281] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[305] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[281] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[305] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[329] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[305] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[329] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[353] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[329] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[353] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[377] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[353] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[377] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[401] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[377] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[401] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[425] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[234] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[258] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[282] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[258] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[282] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[306] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[282] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[306] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[330] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[306] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[330] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[354] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[330] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[354] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[378] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[354] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[378] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[402] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[378] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[402] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[426] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[235] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[259] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[283] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[259] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[283] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[307] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[283] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[307] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[331] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[307] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[331] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[355] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[331] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[355] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[379] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[355] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[379] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[403] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[379] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[403] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[427] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[236] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[260] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[284] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[260] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[284] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[308] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[284] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[308] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[332] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[308] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[332] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[356] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[332] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[356] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[380] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[356] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[380] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[404] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[380] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[404] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[428] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[237] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[261] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[285] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[261] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[285] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[309] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[285] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[309] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[333] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[309] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[333] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[357] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[333] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[357] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[381] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[357] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[381] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[405] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[381] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[405] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[429] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[238] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[262] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[286] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[262] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[286] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[310] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[286] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[310] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[334] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[310] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[334] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[358] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[334] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[358] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[382] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[358] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[382] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[406] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[382] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[406] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[430] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[239] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[263] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[287] * weight_shared[(((int)threadIdx.x) + 4576)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[263] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[287] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[311] * weight_shared[(((int)threadIdx.x) + 4576)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[287] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[311] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[335] * weight_shared[(((int)threadIdx.x) + 4576)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[311] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[335] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[359] * weight_shared[(((int)threadIdx.x) + 4576)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[335] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[359] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[383] * weight_shared[(((int)threadIdx.x) + 4576)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[359] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[383] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[407] * weight_shared[(((int)threadIdx.x) + 4576)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[383] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[407] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[431] * weight_shared[(((int)threadIdx.x) + 4576)]));
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[432] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[456] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[480] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[456] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[480] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[504] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[480] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[504] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[528] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[504] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[528] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[552] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[528] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[552] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[576] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[552] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[576] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[600] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[576] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[600] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[624] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[433] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[457] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[481] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[457] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[481] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[505] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[481] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[505] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[529] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[505] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[529] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[553] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[529] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[553] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[577] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[553] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[577] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[601] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[577] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[601] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[625] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[434] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[458] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[482] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[458] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[482] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[506] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[482] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[506] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[530] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[506] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[530] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[554] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[530] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[554] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[578] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[554] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[578] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[602] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[578] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[602] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[626] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[435] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[459] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[483] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[459] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[483] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[507] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[483] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[507] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[531] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[507] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[531] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[555] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[531] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[555] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[579] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[555] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[579] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[603] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[579] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[603] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[627] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[436] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[460] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[484] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[460] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[484] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[508] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[484] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[508] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[532] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[508] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[532] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[556] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[532] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[556] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[580] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[556] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[580] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[604] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[580] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[604] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[628] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[437] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[461] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[485] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[461] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[485] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[509] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[485] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[509] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[533] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[509] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[533] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[557] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[533] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[557] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[581] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[557] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[581] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[605] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[581] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[605] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[629] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[438] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[462] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[486] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[462] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[486] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[510] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[486] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[510] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[534] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[510] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[534] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[558] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[534] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[558] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[582] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[558] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[582] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[606] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[582] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[606] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[630] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[439] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[463] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[487] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[463] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[487] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[511] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[487] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[511] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[535] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[511] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[535] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[559] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[535] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[559] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[583] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[559] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[583] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[607] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[583] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[607] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[631] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[440] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[464] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[488] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[464] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[488] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[512] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[488] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[512] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[536] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[512] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[536] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[560] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[536] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[560] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[584] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[560] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[584] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[608] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[584] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[608] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[632] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[441] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[465] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[489] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[465] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[489] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[513] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[489] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[513] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[537] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[513] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[537] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[561] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[537] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[561] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[585] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[561] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[585] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[609] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[585] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[609] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[633] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[442] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[466] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[490] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[466] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[490] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[514] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[490] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[514] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[538] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[514] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[538] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[562] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[538] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[562] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[586] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[562] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[586] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[610] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[586] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[610] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[634] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[443] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[467] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[491] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[467] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[491] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[515] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[491] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[515] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[539] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[515] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[539] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[563] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[539] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[563] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[587] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[563] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[587] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[611] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[587] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[611] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[635] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[444] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[468] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[492] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[468] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[492] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[516] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[492] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[516] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[540] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[516] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[540] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[564] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[540] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[564] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[588] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[564] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[588] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[612] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[588] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[612] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[636] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[445] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[469] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[493] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[469] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[493] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[517] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[493] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[517] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[541] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[517] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[541] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[565] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[541] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[565] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[589] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[565] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[589] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[613] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[589] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[613] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[637] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[446] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[470] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[494] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[470] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[494] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[518] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[494] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[518] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[542] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[518] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[542] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[566] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[542] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[566] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[590] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[566] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[590] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[614] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[590] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[614] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[638] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[447] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[471] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[495] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[471] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[495] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[519] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[495] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[519] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[543] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[519] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[543] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[567] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[543] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[567] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[591] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[567] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[591] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[615] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[591] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[615] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[639] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[448] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[472] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[496] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[472] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[496] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[520] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[496] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[520] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[544] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[520] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[544] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[568] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[544] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[568] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[592] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[568] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[592] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[616] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[592] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[616] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[640] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[449] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[473] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[497] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[473] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[497] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[521] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[497] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[521] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[545] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[521] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[545] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[569] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[545] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[569] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[593] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[569] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[593] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[617] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[593] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[617] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[641] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[450] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[474] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[498] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[474] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[498] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[522] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[498] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[522] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[546] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[522] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[546] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[570] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[546] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[570] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[594] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[570] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[594] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[618] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[594] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[618] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[642] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[451] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[475] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[499] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[475] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[499] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[523] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[499] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[523] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[547] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[523] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[547] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[571] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[547] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[571] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[595] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[571] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[595] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[619] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[595] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[619] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[643] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[452] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[476] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[500] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[476] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[500] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[524] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[500] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[524] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[548] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[524] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[548] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[572] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[548] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[572] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[596] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[572] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[596] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[620] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[596] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[620] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[644] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[453] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[477] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[501] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[477] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[501] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[525] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[501] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[525] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[549] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[525] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[549] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[573] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[549] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[573] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[597] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[573] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[597] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[621] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[597] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[621] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[645] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[454] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[478] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[502] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[478] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[502] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[526] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[502] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[526] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[550] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[526] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[550] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[574] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[550] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[574] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[598] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[574] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[598] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[622] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[598] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[622] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[646] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[455] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[479] * weight_shared[(((int)threadIdx.x) + 6112)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[503] * weight_shared[(((int)threadIdx.x) + 6880)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[479] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[503] * weight_shared[(((int)threadIdx.x) + 6112)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[527] * weight_shared[(((int)threadIdx.x) + 6880)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[503] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[527] * weight_shared[(((int)threadIdx.x) + 6112)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[551] * weight_shared[(((int)threadIdx.x) + 6880)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[527] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[551] * weight_shared[(((int)threadIdx.x) + 6112)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[575] * weight_shared[(((int)threadIdx.x) + 6880)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[551] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[575] * weight_shared[(((int)threadIdx.x) + 6112)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[599] * weight_shared[(((int)threadIdx.x) + 6880)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[575] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[599] * weight_shared[(((int)threadIdx.x) + 6112)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[623] * weight_shared[(((int)threadIdx.x) + 6880)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[599] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[623] * weight_shared[(((int)threadIdx.x) + 6112)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[647] * weight_shared[(((int)threadIdx.x) + 6880)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[648] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[672] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[696] * weight_shared[(((int)threadIdx.x) + 8448)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[672] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[696] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[720] * weight_shared[(((int)threadIdx.x) + 8448)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[696] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[720] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[744] * weight_shared[(((int)threadIdx.x) + 8448)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[720] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[744] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[768] * weight_shared[(((int)threadIdx.x) + 8448)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[744] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[768] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[792] * weight_shared[(((int)threadIdx.x) + 8448)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[768] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[792] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[816] * weight_shared[(((int)threadIdx.x) + 8448)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[792] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[816] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[840] * weight_shared[(((int)threadIdx.x) + 8448)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[649] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[673] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[697] * weight_shared[(((int)threadIdx.x) + 8480)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[673] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[697] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[721] * weight_shared[(((int)threadIdx.x) + 8480)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[697] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[721] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[745] * weight_shared[(((int)threadIdx.x) + 8480)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[721] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[745] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[769] * weight_shared[(((int)threadIdx.x) + 8480)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[745] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[769] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[793] * weight_shared[(((int)threadIdx.x) + 8480)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[769] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[793] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[817] * weight_shared[(((int)threadIdx.x) + 8480)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[793] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[817] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[841] * weight_shared[(((int)threadIdx.x) + 8480)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[650] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[674] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[698] * weight_shared[(((int)threadIdx.x) + 8512)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[674] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[698] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[722] * weight_shared[(((int)threadIdx.x) + 8512)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[698] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[722] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[746] * weight_shared[(((int)threadIdx.x) + 8512)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[722] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[746] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[770] * weight_shared[(((int)threadIdx.x) + 8512)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[746] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[770] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[794] * weight_shared[(((int)threadIdx.x) + 8512)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[770] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[794] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[818] * weight_shared[(((int)threadIdx.x) + 8512)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[794] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[818] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[842] * weight_shared[(((int)threadIdx.x) + 8512)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[651] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[675] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[699] * weight_shared[(((int)threadIdx.x) + 8544)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[675] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[699] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[723] * weight_shared[(((int)threadIdx.x) + 8544)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[699] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[723] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[747] * weight_shared[(((int)threadIdx.x) + 8544)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[723] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[747] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[771] * weight_shared[(((int)threadIdx.x) + 8544)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[747] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[771] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[795] * weight_shared[(((int)threadIdx.x) + 8544)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[771] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[795] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[819] * weight_shared[(((int)threadIdx.x) + 8544)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[795] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[819] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[843] * weight_shared[(((int)threadIdx.x) + 8544)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[652] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[676] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[700] * weight_shared[(((int)threadIdx.x) + 8576)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[676] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[700] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[724] * weight_shared[(((int)threadIdx.x) + 8576)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[700] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[724] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[748] * weight_shared[(((int)threadIdx.x) + 8576)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[724] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[748] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[772] * weight_shared[(((int)threadIdx.x) + 8576)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[748] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[772] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[796] * weight_shared[(((int)threadIdx.x) + 8576)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[772] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[796] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[820] * weight_shared[(((int)threadIdx.x) + 8576)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[796] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[820] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[844] * weight_shared[(((int)threadIdx.x) + 8576)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[653] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[677] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[701] * weight_shared[(((int)threadIdx.x) + 8608)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[677] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[701] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[725] * weight_shared[(((int)threadIdx.x) + 8608)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[701] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[725] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[749] * weight_shared[(((int)threadIdx.x) + 8608)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[725] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[749] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[773] * weight_shared[(((int)threadIdx.x) + 8608)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[749] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[773] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[797] * weight_shared[(((int)threadIdx.x) + 8608)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[773] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[797] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[821] * weight_shared[(((int)threadIdx.x) + 8608)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[797] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[821] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[845] * weight_shared[(((int)threadIdx.x) + 8608)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[654] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[678] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[702] * weight_shared[(((int)threadIdx.x) + 8640)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[678] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[702] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[726] * weight_shared[(((int)threadIdx.x) + 8640)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[702] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[726] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[750] * weight_shared[(((int)threadIdx.x) + 8640)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[726] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[750] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[774] * weight_shared[(((int)threadIdx.x) + 8640)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[750] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[774] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[798] * weight_shared[(((int)threadIdx.x) + 8640)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[774] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[798] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[822] * weight_shared[(((int)threadIdx.x) + 8640)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[798] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[822] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[846] * weight_shared[(((int)threadIdx.x) + 8640)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[655] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[679] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[703] * weight_shared[(((int)threadIdx.x) + 8672)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[679] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[703] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[727] * weight_shared[(((int)threadIdx.x) + 8672)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[703] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[727] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[751] * weight_shared[(((int)threadIdx.x) + 8672)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[727] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[751] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[775] * weight_shared[(((int)threadIdx.x) + 8672)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[751] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[775] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[799] * weight_shared[(((int)threadIdx.x) + 8672)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[775] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[799] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[823] * weight_shared[(((int)threadIdx.x) + 8672)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[799] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[823] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[847] * weight_shared[(((int)threadIdx.x) + 8672)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[656] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[680] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[704] * weight_shared[(((int)threadIdx.x) + 8704)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[680] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[704] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[728] * weight_shared[(((int)threadIdx.x) + 8704)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[704] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[728] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[752] * weight_shared[(((int)threadIdx.x) + 8704)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[728] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[752] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[776] * weight_shared[(((int)threadIdx.x) + 8704)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[752] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[776] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[800] * weight_shared[(((int)threadIdx.x) + 8704)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[776] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[800] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[824] * weight_shared[(((int)threadIdx.x) + 8704)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[800] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[824] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[848] * weight_shared[(((int)threadIdx.x) + 8704)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[657] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[681] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[705] * weight_shared[(((int)threadIdx.x) + 8736)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[681] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[705] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[729] * weight_shared[(((int)threadIdx.x) + 8736)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[705] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[729] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[753] * weight_shared[(((int)threadIdx.x) + 8736)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[729] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[753] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[777] * weight_shared[(((int)threadIdx.x) + 8736)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[753] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[777] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[801] * weight_shared[(((int)threadIdx.x) + 8736)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[777] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[801] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[825] * weight_shared[(((int)threadIdx.x) + 8736)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[801] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[825] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[849] * weight_shared[(((int)threadIdx.x) + 8736)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[658] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[682] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[706] * weight_shared[(((int)threadIdx.x) + 8768)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[682] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[706] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[730] * weight_shared[(((int)threadIdx.x) + 8768)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[706] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[730] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[754] * weight_shared[(((int)threadIdx.x) + 8768)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[730] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[754] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[778] * weight_shared[(((int)threadIdx.x) + 8768)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[754] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[778] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[802] * weight_shared[(((int)threadIdx.x) + 8768)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[778] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[802] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[826] * weight_shared[(((int)threadIdx.x) + 8768)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[802] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[826] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[850] * weight_shared[(((int)threadIdx.x) + 8768)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[659] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[683] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[707] * weight_shared[(((int)threadIdx.x) + 8800)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[683] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[707] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[731] * weight_shared[(((int)threadIdx.x) + 8800)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[707] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[731] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[755] * weight_shared[(((int)threadIdx.x) + 8800)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[731] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[755] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[779] * weight_shared[(((int)threadIdx.x) + 8800)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[755] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[779] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[803] * weight_shared[(((int)threadIdx.x) + 8800)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[779] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[803] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[827] * weight_shared[(((int)threadIdx.x) + 8800)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[803] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[827] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[851] * weight_shared[(((int)threadIdx.x) + 8800)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[660] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[684] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[708] * weight_shared[(((int)threadIdx.x) + 8832)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[684] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[708] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[732] * weight_shared[(((int)threadIdx.x) + 8832)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[708] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[732] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[756] * weight_shared[(((int)threadIdx.x) + 8832)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[732] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[756] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[780] * weight_shared[(((int)threadIdx.x) + 8832)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[756] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[780] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[804] * weight_shared[(((int)threadIdx.x) + 8832)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[780] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[804] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[828] * weight_shared[(((int)threadIdx.x) + 8832)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[804] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[828] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[852] * weight_shared[(((int)threadIdx.x) + 8832)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[661] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[685] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[709] * weight_shared[(((int)threadIdx.x) + 8864)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[685] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[709] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[733] * weight_shared[(((int)threadIdx.x) + 8864)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[709] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[733] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[757] * weight_shared[(((int)threadIdx.x) + 8864)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[733] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[757] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[781] * weight_shared[(((int)threadIdx.x) + 8864)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[757] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[781] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[805] * weight_shared[(((int)threadIdx.x) + 8864)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[781] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[805] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[829] * weight_shared[(((int)threadIdx.x) + 8864)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[805] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[829] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[853] * weight_shared[(((int)threadIdx.x) + 8864)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[662] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[686] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[710] * weight_shared[(((int)threadIdx.x) + 8896)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[686] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[710] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[734] * weight_shared[(((int)threadIdx.x) + 8896)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[710] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[734] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[758] * weight_shared[(((int)threadIdx.x) + 8896)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[734] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[758] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[782] * weight_shared[(((int)threadIdx.x) + 8896)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[758] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[782] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[806] * weight_shared[(((int)threadIdx.x) + 8896)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[782] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[806] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[830] * weight_shared[(((int)threadIdx.x) + 8896)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[806] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[830] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[854] * weight_shared[(((int)threadIdx.x) + 8896)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[663] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[687] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[711] * weight_shared[(((int)threadIdx.x) + 8928)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[687] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[711] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[735] * weight_shared[(((int)threadIdx.x) + 8928)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[711] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[735] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[759] * weight_shared[(((int)threadIdx.x) + 8928)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[735] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[759] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[783] * weight_shared[(((int)threadIdx.x) + 8928)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[759] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[783] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[807] * weight_shared[(((int)threadIdx.x) + 8928)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[783] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[807] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[831] * weight_shared[(((int)threadIdx.x) + 8928)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[807] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[831] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[855] * weight_shared[(((int)threadIdx.x) + 8928)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[664] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[688] * weight_shared[(((int)threadIdx.x) + 8192)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[712] * weight_shared[(((int)threadIdx.x) + 8960)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[688] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[712] * weight_shared[(((int)threadIdx.x) + 8192)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[736] * weight_shared[(((int)threadIdx.x) + 8960)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[712] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[736] * weight_shared[(((int)threadIdx.x) + 8192)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[760] * weight_shared[(((int)threadIdx.x) + 8960)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[736] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[760] * weight_shared[(((int)threadIdx.x) + 8192)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[784] * weight_shared[(((int)threadIdx.x) + 8960)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[760] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[784] * weight_shared[(((int)threadIdx.x) + 8192)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[808] * weight_shared[(((int)threadIdx.x) + 8960)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[784] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[808] * weight_shared[(((int)threadIdx.x) + 8192)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[832] * weight_shared[(((int)threadIdx.x) + 8960)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[808] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[832] * weight_shared[(((int)threadIdx.x) + 8192)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[856] * weight_shared[(((int)threadIdx.x) + 8960)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[665] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[689] * weight_shared[(((int)threadIdx.x) + 8224)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[713] * weight_shared[(((int)threadIdx.x) + 8992)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[689] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[713] * weight_shared[(((int)threadIdx.x) + 8224)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[737] * weight_shared[(((int)threadIdx.x) + 8992)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[713] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[737] * weight_shared[(((int)threadIdx.x) + 8224)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[761] * weight_shared[(((int)threadIdx.x) + 8992)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[737] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[761] * weight_shared[(((int)threadIdx.x) + 8224)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[785] * weight_shared[(((int)threadIdx.x) + 8992)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[761] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[785] * weight_shared[(((int)threadIdx.x) + 8224)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[809] * weight_shared[(((int)threadIdx.x) + 8992)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[785] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[809] * weight_shared[(((int)threadIdx.x) + 8224)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[833] * weight_shared[(((int)threadIdx.x) + 8992)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[809] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[833] * weight_shared[(((int)threadIdx.x) + 8224)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[857] * weight_shared[(((int)threadIdx.x) + 8992)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[666] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[690] * weight_shared[(((int)threadIdx.x) + 8256)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[714] * weight_shared[(((int)threadIdx.x) + 9024)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[690] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[714] * weight_shared[(((int)threadIdx.x) + 8256)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[738] * weight_shared[(((int)threadIdx.x) + 9024)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[714] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[738] * weight_shared[(((int)threadIdx.x) + 8256)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[762] * weight_shared[(((int)threadIdx.x) + 9024)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[738] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[762] * weight_shared[(((int)threadIdx.x) + 8256)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[786] * weight_shared[(((int)threadIdx.x) + 9024)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[762] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[786] * weight_shared[(((int)threadIdx.x) + 8256)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[810] * weight_shared[(((int)threadIdx.x) + 9024)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[786] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[810] * weight_shared[(((int)threadIdx.x) + 8256)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[834] * weight_shared[(((int)threadIdx.x) + 9024)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[810] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[834] * weight_shared[(((int)threadIdx.x) + 8256)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[858] * weight_shared[(((int)threadIdx.x) + 9024)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[667] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[691] * weight_shared[(((int)threadIdx.x) + 8288)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[715] * weight_shared[(((int)threadIdx.x) + 9056)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[691] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[715] * weight_shared[(((int)threadIdx.x) + 8288)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[739] * weight_shared[(((int)threadIdx.x) + 9056)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[715] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[739] * weight_shared[(((int)threadIdx.x) + 8288)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[763] * weight_shared[(((int)threadIdx.x) + 9056)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[739] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[763] * weight_shared[(((int)threadIdx.x) + 8288)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[787] * weight_shared[(((int)threadIdx.x) + 9056)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[763] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[787] * weight_shared[(((int)threadIdx.x) + 8288)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[811] * weight_shared[(((int)threadIdx.x) + 9056)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[787] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[811] * weight_shared[(((int)threadIdx.x) + 8288)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[835] * weight_shared[(((int)threadIdx.x) + 9056)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[811] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[835] * weight_shared[(((int)threadIdx.x) + 8288)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[859] * weight_shared[(((int)threadIdx.x) + 9056)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[668] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[692] * weight_shared[(((int)threadIdx.x) + 8320)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[716] * weight_shared[(((int)threadIdx.x) + 9088)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[692] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[716] * weight_shared[(((int)threadIdx.x) + 8320)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[740] * weight_shared[(((int)threadIdx.x) + 9088)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[716] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[740] * weight_shared[(((int)threadIdx.x) + 8320)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[764] * weight_shared[(((int)threadIdx.x) + 9088)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[740] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[764] * weight_shared[(((int)threadIdx.x) + 8320)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[788] * weight_shared[(((int)threadIdx.x) + 9088)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[764] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[788] * weight_shared[(((int)threadIdx.x) + 8320)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[812] * weight_shared[(((int)threadIdx.x) + 9088)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[788] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[812] * weight_shared[(((int)threadIdx.x) + 8320)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[836] * weight_shared[(((int)threadIdx.x) + 9088)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[812] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[836] * weight_shared[(((int)threadIdx.x) + 8320)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[860] * weight_shared[(((int)threadIdx.x) + 9088)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[669] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[693] * weight_shared[(((int)threadIdx.x) + 8352)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[717] * weight_shared[(((int)threadIdx.x) + 9120)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[693] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[717] * weight_shared[(((int)threadIdx.x) + 8352)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[741] * weight_shared[(((int)threadIdx.x) + 9120)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[717] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[741] * weight_shared[(((int)threadIdx.x) + 8352)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[765] * weight_shared[(((int)threadIdx.x) + 9120)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[741] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[765] * weight_shared[(((int)threadIdx.x) + 8352)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[789] * weight_shared[(((int)threadIdx.x) + 9120)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[765] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[789] * weight_shared[(((int)threadIdx.x) + 8352)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[813] * weight_shared[(((int)threadIdx.x) + 9120)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[789] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[813] * weight_shared[(((int)threadIdx.x) + 8352)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[837] * weight_shared[(((int)threadIdx.x) + 9120)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[813] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[837] * weight_shared[(((int)threadIdx.x) + 8352)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[861] * weight_shared[(((int)threadIdx.x) + 9120)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[670] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[694] * weight_shared[(((int)threadIdx.x) + 8384)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[718] * weight_shared[(((int)threadIdx.x) + 9152)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[694] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[718] * weight_shared[(((int)threadIdx.x) + 8384)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[742] * weight_shared[(((int)threadIdx.x) + 9152)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[718] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[742] * weight_shared[(((int)threadIdx.x) + 8384)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[766] * weight_shared[(((int)threadIdx.x) + 9152)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[742] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[766] * weight_shared[(((int)threadIdx.x) + 8384)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[790] * weight_shared[(((int)threadIdx.x) + 9152)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[766] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[790] * weight_shared[(((int)threadIdx.x) + 8384)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[814] * weight_shared[(((int)threadIdx.x) + 9152)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[790] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[814] * weight_shared[(((int)threadIdx.x) + 8384)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[838] * weight_shared[(((int)threadIdx.x) + 9152)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[814] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[838] * weight_shared[(((int)threadIdx.x) + 8384)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[862] * weight_shared[(((int)threadIdx.x) + 9152)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[671] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[695] * weight_shared[(((int)threadIdx.x) + 8416)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[719] * weight_shared[(((int)threadIdx.x) + 9184)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[695] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[719] * weight_shared[(((int)threadIdx.x) + 8416)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[743] * weight_shared[(((int)threadIdx.x) + 9184)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[719] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[743] * weight_shared[(((int)threadIdx.x) + 8416)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[767] * weight_shared[(((int)threadIdx.x) + 9184)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[743] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[767] * weight_shared[(((int)threadIdx.x) + 8416)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[791] * weight_shared[(((int)threadIdx.x) + 9184)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[767] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[791] * weight_shared[(((int)threadIdx.x) + 8416)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[815] * weight_shared[(((int)threadIdx.x) + 9184)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[791] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[815] * weight_shared[(((int)threadIdx.x) + 8416)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[839] * weight_shared[(((int)threadIdx.x) + 9184)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[815] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[839] * weight_shared[(((int)threadIdx.x) + 8416)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[863] * weight_shared[(((int)threadIdx.x) + 9184)]));
  conv2d_nhwc[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = conv2d_nhwc_local[0];
  conv2d_nhwc[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 3584)] = conv2d_nhwc_local[1];
  conv2d_nhwc[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 7168)] = conv2d_nhwc_local[2];
  conv2d_nhwc[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 10752)] = conv2d_nhwc_local[3];
  conv2d_nhwc[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 14336)] = conv2d_nhwc_local[4];
  conv2d_nhwc[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 17920)] = conv2d_nhwc_local[5];
  conv2d_nhwc[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 21504)] = conv2d_nhwc_local[6];
}


