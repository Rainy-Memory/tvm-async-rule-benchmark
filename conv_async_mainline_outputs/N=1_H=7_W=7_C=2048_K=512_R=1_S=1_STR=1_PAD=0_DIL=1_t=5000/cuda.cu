
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[7];
  __shared__ float PadInput_shared[1792];
  __shared__ float weight_shared[8192];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) * 2)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 64)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 2048))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 4096))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 192)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 6144))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 8192))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 320)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 10240))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 12288))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2048))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 4096))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 6144))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 8192))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 10240))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 12288))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14336))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 16384))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1152)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 18432))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 20480))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1408)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 22528))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 24576))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1664)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 26624))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 28672))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1920)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 30720))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 64))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 2112))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 576)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 4160))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 6208))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 704)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 8256))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 10304))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 832)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 12352))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 32768))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2176)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 34816))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2304)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 36864))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2432)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 38912))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2560)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 40960))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2688)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 43008))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2816)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 45056))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2944)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 47104))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3072)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 49152))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3200)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 51200))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3328)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 53248))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3456)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 55296))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3584)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 57344))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3712)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 59392))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3840)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 61440))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3968)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 63488))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 128))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 960)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 2176))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 4224))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1088)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 6272))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1152)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 8320))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1216)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 10368))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 14336) + (((int)threadIdx.x) * 2)) + 12416))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4096)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 65536))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4224)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 67584))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4352)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 69632))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4480)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 71680))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4608)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 73728))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4736)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 75776))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4864)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 77824))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4992)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 79872))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5120)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 81920))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5248)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 83968))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5376)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 86016))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5504)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 88064))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5632)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 90112))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5760)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 92160))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5888)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 94208))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 6016)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 96256))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int rh_0_rw_0_rc_0_fused = 0; rh_0_rw_0_rc_0_fused < 29; ++rh_0_rw_0_rc_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 448) + (((int)threadIdx.x) * 2))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 14336) + (rh_0_rw_0_rc_0_fused * 64)) + (((int)threadIdx.x) * 2)) + 192))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 448) + (((int)threadIdx.x) * 2)) + 64)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 14336) + (rh_0_rw_0_rc_0_fused * 64)) + (((int)threadIdx.x) * 2)) + 2240))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 448) + (((int)threadIdx.x) * 2)) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 14336) + (rh_0_rw_0_rc_0_fused * 64)) + (((int)threadIdx.x) * 2)) + 4288))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 448) + (((int)threadIdx.x) * 2)) + 192)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 14336) + (rh_0_rw_0_rc_0_fused * 64)) + (((int)threadIdx.x) * 2)) + 6336))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 448) + (((int)threadIdx.x) * 2)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 14336) + (rh_0_rw_0_rc_0_fused * 64)) + (((int)threadIdx.x) * 2)) + 8384))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 448) + (((int)threadIdx.x) * 2)) + 320)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 14336) + (rh_0_rw_0_rc_0_fused * 64)) + (((int)threadIdx.x) * 2)) + 10432))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 448) + (((int)threadIdx.x) * 2)) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 14336) + (rh_0_rw_0_rc_0_fused * 64)) + (((int)threadIdx.x) * 2)) + 12480))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 98304))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 100352))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 102400))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 104448))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 106496))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 108544))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 110592))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 112640))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 114688))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 1152)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 116736))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 118784))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 1408)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 120832))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 122880))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 1664)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 124928))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 126976))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + (((int)threadIdx.x) * 4)) + 1920)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 129024))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

    __syncthreads();
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((rh_0_rw_0_rc_0_fused & 3) * 448)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 64)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 128)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 192)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 256)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 257)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 320)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 321)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 384)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x))]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 385)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 32)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 258)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 259)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 322)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 323)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 386)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 64)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 387)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 96)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 260)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 261)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 324)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 325)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 388)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 128)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 389)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 160)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 262)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 263)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 326)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 327)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 390)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 192)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 391)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 224)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 264)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 265)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 328)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 329)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 392)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 256)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 393)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 288)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 266)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 267)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 330)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 331)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 394)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 320)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 395)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 352)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 268)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 269)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 332)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 333)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 396)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 384)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 397)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 416)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 270)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 271)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 334)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 335)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 398)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 448)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 399)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 480)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 144)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 208)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 209)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 272)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 273)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 336)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 337)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 400)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 512)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 401)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 544)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 210)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 211)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 274)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 275)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 338)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 339)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 402)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 576)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 403)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 608)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 212)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 213)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 276)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 277)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 340)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 341)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 404)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 640)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 405)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 672)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 214)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 215)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 278)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 279)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 342)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 343)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 406)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 704)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 407)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 736)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 216)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 217)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 280)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 281)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 344)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 345)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 408)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 768)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 409)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 800)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 218)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 219)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 282)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 283)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 346)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 347)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 410)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 832)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 411)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 864)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 220)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 221)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 284)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 285)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 348)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 349)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 412)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 896)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 413)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 928)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 222)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 223)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 286)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 287)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 350)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 351)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 414)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 960)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 415)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 992)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 32)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 160)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 161)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 224)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 225)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 288)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 289)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 352)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 353)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 416)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1024)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 417)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1056)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 162)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 163)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 226)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 227)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 290)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 291)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 354)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 355)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 418)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1088)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 419)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1120)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 164)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 165)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 228)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 229)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 292)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 293)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 356)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 357)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 420)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1152)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 421)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1184)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 166)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 167)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 230)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 231)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 294)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 295)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 358)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 359)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 422)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1216)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 423)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1248)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 168)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 169)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 232)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 233)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 296)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 297)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 360)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 361)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 424)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1280)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 425)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1312)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 170)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 171)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 234)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 235)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 298)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 299)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 362)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 363)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 426)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1344)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 427)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1376)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 172)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 173)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 236)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 237)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 300)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 301)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 364)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 365)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 428)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1408)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 429)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1440)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 174)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 175)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 238)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 239)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 302)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 303)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 366)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 367)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 430)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1472)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 431)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1504)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 112)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 113)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 176)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 177)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 240)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 241)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 304)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 305)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 368)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 369)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 432)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1536)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 433)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1568)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 114)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 115)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 178)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 179)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 242)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 243)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 306)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 307)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 370)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 371)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 434)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1600)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 435)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1632)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 116)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 117)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 180)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 181)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 244)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 245)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 308)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 309)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 372)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 373)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 436)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1664)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 437)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1696)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 118)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 119)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 182)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 183)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 246)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 247)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 310)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 311)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 374)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 375)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 438)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1728)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 439)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1760)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 120)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 121)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 184)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 185)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 248)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 249)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 312)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 313)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 376)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 377)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 440)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1792)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 441)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1824)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 122)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 123)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 186)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 187)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 250)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 251)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 314)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 315)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 378)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 379)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 442)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1856)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 443)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1888)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 124)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 125)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 188)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 189)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 252)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 253)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 316)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 317)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 380)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 381)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 444)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1920)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 445)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1952)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 2016)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 126)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 127)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 2016)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 190)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 191)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 2016)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 254)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 255)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 2016)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 318)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 319)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 2016)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 382)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 383)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 2016)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 446)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 1984)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 448) + 447)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((int)threadIdx.x)) + 2016)]));
  }
__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[448] * weight_shared[(((int)threadIdx.x) + 2048)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[449] * weight_shared[(((int)threadIdx.x) + 2080)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[512] * weight_shared[(((int)threadIdx.x) + 2048)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[513] * weight_shared[(((int)threadIdx.x) + 2080)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[576] * weight_shared[(((int)threadIdx.x) + 2048)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[577] * weight_shared[(((int)threadIdx.x) + 2080)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[640] * weight_shared[(((int)threadIdx.x) + 2048)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[641] * weight_shared[(((int)threadIdx.x) + 2080)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[704] * weight_shared[(((int)threadIdx.x) + 2048)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[705] * weight_shared[(((int)threadIdx.x) + 2080)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[768] * weight_shared[(((int)threadIdx.x) + 2048)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[769] * weight_shared[(((int)threadIdx.x) + 2080)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[832] * weight_shared[(((int)threadIdx.x) + 2048)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[833] * weight_shared[(((int)threadIdx.x) + 2080)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[450] * weight_shared[(((int)threadIdx.x) + 2112)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[451] * weight_shared[(((int)threadIdx.x) + 2144)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[514] * weight_shared[(((int)threadIdx.x) + 2112)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[515] * weight_shared[(((int)threadIdx.x) + 2144)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[578] * weight_shared[(((int)threadIdx.x) + 2112)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[579] * weight_shared[(((int)threadIdx.x) + 2144)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[642] * weight_shared[(((int)threadIdx.x) + 2112)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[643] * weight_shared[(((int)threadIdx.x) + 2144)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[706] * weight_shared[(((int)threadIdx.x) + 2112)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[707] * weight_shared[(((int)threadIdx.x) + 2144)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[770] * weight_shared[(((int)threadIdx.x) + 2112)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[771] * weight_shared[(((int)threadIdx.x) + 2144)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[834] * weight_shared[(((int)threadIdx.x) + 2112)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[835] * weight_shared[(((int)threadIdx.x) + 2144)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[452] * weight_shared[(((int)threadIdx.x) + 2176)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[453] * weight_shared[(((int)threadIdx.x) + 2208)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[516] * weight_shared[(((int)threadIdx.x) + 2176)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[517] * weight_shared[(((int)threadIdx.x) + 2208)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[580] * weight_shared[(((int)threadIdx.x) + 2176)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[581] * weight_shared[(((int)threadIdx.x) + 2208)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[644] * weight_shared[(((int)threadIdx.x) + 2176)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[645] * weight_shared[(((int)threadIdx.x) + 2208)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[708] * weight_shared[(((int)threadIdx.x) + 2176)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[709] * weight_shared[(((int)threadIdx.x) + 2208)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[772] * weight_shared[(((int)threadIdx.x) + 2176)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[773] * weight_shared[(((int)threadIdx.x) + 2208)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[836] * weight_shared[(((int)threadIdx.x) + 2176)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[837] * weight_shared[(((int)threadIdx.x) + 2208)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[454] * weight_shared[(((int)threadIdx.x) + 2240)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[455] * weight_shared[(((int)threadIdx.x) + 2272)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[518] * weight_shared[(((int)threadIdx.x) + 2240)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[519] * weight_shared[(((int)threadIdx.x) + 2272)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[582] * weight_shared[(((int)threadIdx.x) + 2240)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[583] * weight_shared[(((int)threadIdx.x) + 2272)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[646] * weight_shared[(((int)threadIdx.x) + 2240)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[647] * weight_shared[(((int)threadIdx.x) + 2272)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[710] * weight_shared[(((int)threadIdx.x) + 2240)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[711] * weight_shared[(((int)threadIdx.x) + 2272)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[774] * weight_shared[(((int)threadIdx.x) + 2240)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[775] * weight_shared[(((int)threadIdx.x) + 2272)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[838] * weight_shared[(((int)threadIdx.x) + 2240)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[839] * weight_shared[(((int)threadIdx.x) + 2272)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[456] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[457] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[520] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[521] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[584] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[585] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[648] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[649] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[712] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[713] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[776] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[777] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[840] * weight_shared[(((int)threadIdx.x) + 2304)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[841] * weight_shared[(((int)threadIdx.x) + 2336)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[458] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[459] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[522] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[523] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[586] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[587] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[650] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[651] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[714] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[715] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[778] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[779] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[842] * weight_shared[(((int)threadIdx.x) + 2368)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[843] * weight_shared[(((int)threadIdx.x) + 2400)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[460] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[461] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[524] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[525] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[588] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[589] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[652] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[653] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[716] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[717] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[780] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[781] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[844] * weight_shared[(((int)threadIdx.x) + 2432)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[845] * weight_shared[(((int)threadIdx.x) + 2464)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[462] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[463] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[526] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[527] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[590] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[591] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[654] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[655] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[718] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[719] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[782] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[783] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[846] * weight_shared[(((int)threadIdx.x) + 2496)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[847] * weight_shared[(((int)threadIdx.x) + 2528)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[464] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[465] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[528] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[529] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[592] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[593] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[656] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[657] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[720] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[721] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[784] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[785] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[848] * weight_shared[(((int)threadIdx.x) + 2560)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[849] * weight_shared[(((int)threadIdx.x) + 2592)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[466] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[467] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[530] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[531] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[594] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[595] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[658] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[659] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[722] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[723] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[786] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[787] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[850] * weight_shared[(((int)threadIdx.x) + 2624)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[851] * weight_shared[(((int)threadIdx.x) + 2656)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[468] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[469] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[532] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[533] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[596] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[597] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[660] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[661] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[724] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[725] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[788] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[789] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[852] * weight_shared[(((int)threadIdx.x) + 2688)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[853] * weight_shared[(((int)threadIdx.x) + 2720)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[470] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[471] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[534] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[535] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[598] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[599] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[662] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[663] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[726] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[727] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[790] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[791] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[854] * weight_shared[(((int)threadIdx.x) + 2752)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[855] * weight_shared[(((int)threadIdx.x) + 2784)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[472] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[473] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[536] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[537] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[600] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[601] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[664] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[665] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[728] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[729] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[792] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[793] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[856] * weight_shared[(((int)threadIdx.x) + 2816)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[857] * weight_shared[(((int)threadIdx.x) + 2848)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[474] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[475] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[538] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[539] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[602] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[603] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[666] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[667] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[730] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[731] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[794] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[795] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[858] * weight_shared[(((int)threadIdx.x) + 2880)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[859] * weight_shared[(((int)threadIdx.x) + 2912)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[476] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[477] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[540] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[541] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[604] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[605] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[668] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[669] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[732] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[733] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[796] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[797] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[860] * weight_shared[(((int)threadIdx.x) + 2944)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[861] * weight_shared[(((int)threadIdx.x) + 2976)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[478] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[479] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[542] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[543] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[606] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[607] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[670] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[671] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[734] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[735] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[798] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[799] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[862] * weight_shared[(((int)threadIdx.x) + 3008)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[863] * weight_shared[(((int)threadIdx.x) + 3040)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[480] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[481] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[544] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[545] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[608] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[609] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[672] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[673] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[736] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[737] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[800] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[801] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[864] * weight_shared[(((int)threadIdx.x) + 3072)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[865] * weight_shared[(((int)threadIdx.x) + 3104)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[482] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[483] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[546] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[547] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[610] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[611] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[674] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[675] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[738] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[739] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[802] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[803] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[866] * weight_shared[(((int)threadIdx.x) + 3136)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[867] * weight_shared[(((int)threadIdx.x) + 3168)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[484] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[485] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[548] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[549] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[612] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[613] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[676] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[677] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[740] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[741] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[804] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[805] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[868] * weight_shared[(((int)threadIdx.x) + 3200)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[869] * weight_shared[(((int)threadIdx.x) + 3232)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[486] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[487] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[550] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[551] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[614] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[615] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[678] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[679] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[742] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[743] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[806] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[807] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[870] * weight_shared[(((int)threadIdx.x) + 3264)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[871] * weight_shared[(((int)threadIdx.x) + 3296)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[488] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[489] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[552] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[553] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[616] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[617] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[680] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[681] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[744] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[745] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[808] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[809] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[872] * weight_shared[(((int)threadIdx.x) + 3328)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[873] * weight_shared[(((int)threadIdx.x) + 3360)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[490] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[491] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[554] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[555] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[618] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[619] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[682] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[683] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[746] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[747] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[810] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[811] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[874] * weight_shared[(((int)threadIdx.x) + 3392)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[875] * weight_shared[(((int)threadIdx.x) + 3424)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[492] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[493] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[556] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[557] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[620] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[621] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[684] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[685] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[748] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[749] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[812] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[813] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[876] * weight_shared[(((int)threadIdx.x) + 3456)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[877] * weight_shared[(((int)threadIdx.x) + 3488)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[494] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[495] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[558] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[559] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[622] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[623] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[686] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[687] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[750] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[751] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[814] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[815] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[878] * weight_shared[(((int)threadIdx.x) + 3520)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[879] * weight_shared[(((int)threadIdx.x) + 3552)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[496] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[497] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[560] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[561] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[624] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[625] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[688] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[689] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[752] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[753] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[816] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[817] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[880] * weight_shared[(((int)threadIdx.x) + 3584)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[881] * weight_shared[(((int)threadIdx.x) + 3616)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[498] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[499] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[562] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[563] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[626] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[627] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[690] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[691] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[754] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[755] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[818] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[819] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[882] * weight_shared[(((int)threadIdx.x) + 3648)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[883] * weight_shared[(((int)threadIdx.x) + 3680)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[500] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[501] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[564] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[565] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[628] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[629] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[692] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[693] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[756] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[757] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[820] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[821] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[884] * weight_shared[(((int)threadIdx.x) + 3712)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[885] * weight_shared[(((int)threadIdx.x) + 3744)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[502] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[503] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[566] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[567] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[630] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[631] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[694] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[695] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[758] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[759] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[822] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[823] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[886] * weight_shared[(((int)threadIdx.x) + 3776)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[887] * weight_shared[(((int)threadIdx.x) + 3808)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[504] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[505] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[568] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[569] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[632] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[633] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[696] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[697] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[760] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[761] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[824] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[825] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[888] * weight_shared[(((int)threadIdx.x) + 3840)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[889] * weight_shared[(((int)threadIdx.x) + 3872)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[506] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[507] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[570] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[571] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[634] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[635] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[698] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[699] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[762] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[763] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[826] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[827] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[890] * weight_shared[(((int)threadIdx.x) + 3904)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[891] * weight_shared[(((int)threadIdx.x) + 3936)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[508] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[509] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[572] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[573] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[636] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[637] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[700] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[701] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[764] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[765] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[828] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[829] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[892] * weight_shared[(((int)threadIdx.x) + 3968)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[893] * weight_shared[(((int)threadIdx.x) + 4000)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[510] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[511] * weight_shared[(((int)threadIdx.x) + 4064)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[574] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[575] * weight_shared[(((int)threadIdx.x) + 4064)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[638] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[639] * weight_shared[(((int)threadIdx.x) + 4064)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[702] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[703] * weight_shared[(((int)threadIdx.x) + 4064)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[766] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[767] * weight_shared[(((int)threadIdx.x) + 4064)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[830] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[831] * weight_shared[(((int)threadIdx.x) + 4064)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[894] * weight_shared[(((int)threadIdx.x) + 4032)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[895] * weight_shared[(((int)threadIdx.x) + 4064)]));
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[896] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[897] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[960] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[961] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1024] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1025] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1088] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1089] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1152] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1153] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1216] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1217] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1280] * weight_shared[(((int)threadIdx.x) + 4096)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1281] * weight_shared[(((int)threadIdx.x) + 4128)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[898] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[899] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[962] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[963] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1026] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1027] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1090] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1091] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1154] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1155] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1218] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1219] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1282] * weight_shared[(((int)threadIdx.x) + 4160)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1283] * weight_shared[(((int)threadIdx.x) + 4192)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[900] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[901] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[964] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[965] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1028] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1029] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1092] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1093] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1156] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1157] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1220] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1221] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1284] * weight_shared[(((int)threadIdx.x) + 4224)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1285] * weight_shared[(((int)threadIdx.x) + 4256)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[902] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[903] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[966] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[967] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1030] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1031] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1094] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1095] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1158] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1159] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1222] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1223] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1286] * weight_shared[(((int)threadIdx.x) + 4288)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1287] * weight_shared[(((int)threadIdx.x) + 4320)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[904] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[905] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[968] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[969] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1032] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1033] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1096] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1097] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1160] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1161] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1224] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1225] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1288] * weight_shared[(((int)threadIdx.x) + 4352)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1289] * weight_shared[(((int)threadIdx.x) + 4384)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[906] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[907] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[970] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[971] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1034] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1035] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1098] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1099] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1162] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1163] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1226] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1227] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1290] * weight_shared[(((int)threadIdx.x) + 4416)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1291] * weight_shared[(((int)threadIdx.x) + 4448)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[908] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[909] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[972] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[973] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1036] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1037] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1100] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1101] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1164] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1165] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1228] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1229] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1292] * weight_shared[(((int)threadIdx.x) + 4480)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1293] * weight_shared[(((int)threadIdx.x) + 4512)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[910] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[911] * weight_shared[(((int)threadIdx.x) + 4576)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[974] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[975] * weight_shared[(((int)threadIdx.x) + 4576)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1038] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1039] * weight_shared[(((int)threadIdx.x) + 4576)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1102] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1103] * weight_shared[(((int)threadIdx.x) + 4576)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1166] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1167] * weight_shared[(((int)threadIdx.x) + 4576)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1230] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1231] * weight_shared[(((int)threadIdx.x) + 4576)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1294] * weight_shared[(((int)threadIdx.x) + 4544)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1295] * weight_shared[(((int)threadIdx.x) + 4576)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[912] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[913] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[976] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[977] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1040] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1041] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1104] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1105] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1168] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1169] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1232] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1233] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1296] * weight_shared[(((int)threadIdx.x) + 4608)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1297] * weight_shared[(((int)threadIdx.x) + 4640)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[914] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[915] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[978] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[979] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1042] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1043] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1106] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1107] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1170] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1171] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1234] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1235] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1298] * weight_shared[(((int)threadIdx.x) + 4672)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1299] * weight_shared[(((int)threadIdx.x) + 4704)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[916] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[917] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[980] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[981] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1044] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1045] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1108] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1109] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1172] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1173] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1236] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1237] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1300] * weight_shared[(((int)threadIdx.x) + 4736)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1301] * weight_shared[(((int)threadIdx.x) + 4768)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[918] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[919] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[982] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[983] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1046] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1047] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1110] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1111] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1174] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1175] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1238] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1239] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1302] * weight_shared[(((int)threadIdx.x) + 4800)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1303] * weight_shared[(((int)threadIdx.x) + 4832)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[920] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[921] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[984] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[985] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1048] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1049] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1112] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1113] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1176] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1177] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1240] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1241] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1304] * weight_shared[(((int)threadIdx.x) + 4864)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1305] * weight_shared[(((int)threadIdx.x) + 4896)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[922] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[923] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[986] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[987] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1050] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1051] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1114] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1115] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1178] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1179] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1242] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1243] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1306] * weight_shared[(((int)threadIdx.x) + 4928)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1307] * weight_shared[(((int)threadIdx.x) + 4960)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[924] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[925] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[988] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[989] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1052] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1053] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1116] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1117] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1180] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1181] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1244] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1245] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1308] * weight_shared[(((int)threadIdx.x) + 4992)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1309] * weight_shared[(((int)threadIdx.x) + 5024)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[926] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[927] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[990] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[991] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1054] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1055] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1118] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1119] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1182] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1183] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1246] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1247] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1310] * weight_shared[(((int)threadIdx.x) + 5056)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1311] * weight_shared[(((int)threadIdx.x) + 5088)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[928] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[929] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[992] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[993] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1056] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1057] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1120] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1121] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1184] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1185] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1248] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1249] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1312] * weight_shared[(((int)threadIdx.x) + 5120)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1313] * weight_shared[(((int)threadIdx.x) + 5152)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[930] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[931] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[994] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[995] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1058] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1059] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1122] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1123] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1186] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1187] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1250] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1251] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1314] * weight_shared[(((int)threadIdx.x) + 5184)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1315] * weight_shared[(((int)threadIdx.x) + 5216)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[932] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[933] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[996] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[997] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1060] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1061] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1124] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1125] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1188] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1189] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1252] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1253] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1316] * weight_shared[(((int)threadIdx.x) + 5248)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1317] * weight_shared[(((int)threadIdx.x) + 5280)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[934] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[935] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[998] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[999] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1062] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1063] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1126] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1127] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1190] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1191] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1254] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1255] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1318] * weight_shared[(((int)threadIdx.x) + 5312)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1319] * weight_shared[(((int)threadIdx.x) + 5344)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[936] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[937] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1000] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1001] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1064] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1065] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1128] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1129] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1192] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1193] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1256] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1257] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1320] * weight_shared[(((int)threadIdx.x) + 5376)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1321] * weight_shared[(((int)threadIdx.x) + 5408)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[938] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[939] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1002] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1003] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1066] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1067] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1130] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1131] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1194] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1195] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1258] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1259] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1322] * weight_shared[(((int)threadIdx.x) + 5440)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1323] * weight_shared[(((int)threadIdx.x) + 5472)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[940] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[941] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1004] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1005] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1068] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1069] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1132] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1133] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1196] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1197] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1260] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1261] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1324] * weight_shared[(((int)threadIdx.x) + 5504)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1325] * weight_shared[(((int)threadIdx.x) + 5536)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[942] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[943] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1006] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1007] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1070] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1071] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1134] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1135] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1198] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1199] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1262] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1263] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1326] * weight_shared[(((int)threadIdx.x) + 5568)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1327] * weight_shared[(((int)threadIdx.x) + 5600)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[944] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[945] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1008] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1009] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1072] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1073] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1136] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1137] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1200] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1201] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1264] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1265] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1328] * weight_shared[(((int)threadIdx.x) + 5632)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1329] * weight_shared[(((int)threadIdx.x) + 5664)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[946] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[947] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1010] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1011] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1074] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1075] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1138] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1139] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1202] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1203] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1266] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1267] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1330] * weight_shared[(((int)threadIdx.x) + 5696)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1331] * weight_shared[(((int)threadIdx.x) + 5728)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[948] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[949] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1012] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1013] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1076] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1077] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1140] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1141] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1204] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1205] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1268] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1269] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1332] * weight_shared[(((int)threadIdx.x) + 5760)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1333] * weight_shared[(((int)threadIdx.x) + 5792)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[950] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[951] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1014] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1015] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1078] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1079] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1142] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1143] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1206] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1207] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1270] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1271] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1334] * weight_shared[(((int)threadIdx.x) + 5824)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1335] * weight_shared[(((int)threadIdx.x) + 5856)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[952] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[953] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1016] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1017] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1080] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1081] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1144] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1145] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1208] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1209] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1272] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1273] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1336] * weight_shared[(((int)threadIdx.x) + 5888)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1337] * weight_shared[(((int)threadIdx.x) + 5920)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[954] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[955] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1018] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1019] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1082] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1083] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1146] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1147] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1210] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1211] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1274] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1275] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1338] * weight_shared[(((int)threadIdx.x) + 5952)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1339] * weight_shared[(((int)threadIdx.x) + 5984)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[956] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[957] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1020] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1021] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1084] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1085] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1148] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1149] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1212] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1213] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1276] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1277] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1340] * weight_shared[(((int)threadIdx.x) + 6016)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1341] * weight_shared[(((int)threadIdx.x) + 6048)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[958] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[959] * weight_shared[(((int)threadIdx.x) + 6112)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1022] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1023] * weight_shared[(((int)threadIdx.x) + 6112)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1086] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1087] * weight_shared[(((int)threadIdx.x) + 6112)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1150] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1151] * weight_shared[(((int)threadIdx.x) + 6112)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1214] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1215] * weight_shared[(((int)threadIdx.x) + 6112)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1278] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1279] * weight_shared[(((int)threadIdx.x) + 6112)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1342] * weight_shared[(((int)threadIdx.x) + 6080)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1343] * weight_shared[(((int)threadIdx.x) + 6112)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1344] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1345] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1408] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1409] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1472] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1473] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1536] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1537] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1600] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1601] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1664] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1665] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1728] * weight_shared[(((int)threadIdx.x) + 6144)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1729] * weight_shared[(((int)threadIdx.x) + 6176)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1346] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1347] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1410] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1411] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1474] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1475] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1538] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1539] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1602] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1603] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1666] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1667] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1730] * weight_shared[(((int)threadIdx.x) + 6208)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1731] * weight_shared[(((int)threadIdx.x) + 6240)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1348] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1349] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1412] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1413] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1476] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1477] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1540] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1541] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1604] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1605] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1668] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1669] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1732] * weight_shared[(((int)threadIdx.x) + 6272)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1733] * weight_shared[(((int)threadIdx.x) + 6304)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1350] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1351] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1414] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1415] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1478] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1479] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1542] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1543] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1606] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1607] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1670] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1671] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1734] * weight_shared[(((int)threadIdx.x) + 6336)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1735] * weight_shared[(((int)threadIdx.x) + 6368)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1352] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1353] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1416] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1417] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1480] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1481] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1544] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1545] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1608] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1609] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1672] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1673] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1736] * weight_shared[(((int)threadIdx.x) + 6400)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1737] * weight_shared[(((int)threadIdx.x) + 6432)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1354] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1355] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1418] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1419] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1482] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1483] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1546] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1547] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1610] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1611] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1674] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1675] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1738] * weight_shared[(((int)threadIdx.x) + 6464)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1739] * weight_shared[(((int)threadIdx.x) + 6496)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1356] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1357] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1420] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1421] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1484] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1485] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1548] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1549] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1612] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1613] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1676] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1677] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1740] * weight_shared[(((int)threadIdx.x) + 6528)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1741] * weight_shared[(((int)threadIdx.x) + 6560)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1358] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1359] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1422] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1423] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1486] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1487] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1550] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1551] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1614] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1615] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1678] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1679] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1742] * weight_shared[(((int)threadIdx.x) + 6592)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1743] * weight_shared[(((int)threadIdx.x) + 6624)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1360] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1361] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1424] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1425] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1488] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1489] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1552] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1553] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1616] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1617] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1680] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1681] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1744] * weight_shared[(((int)threadIdx.x) + 6656)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1745] * weight_shared[(((int)threadIdx.x) + 6688)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1362] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1363] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1426] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1427] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1490] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1491] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1554] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1555] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1618] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1619] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1682] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1683] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1746] * weight_shared[(((int)threadIdx.x) + 6720)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1747] * weight_shared[(((int)threadIdx.x) + 6752)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1364] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1365] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1428] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1429] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1492] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1493] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1556] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1557] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1620] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1621] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1684] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1685] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1748] * weight_shared[(((int)threadIdx.x) + 6784)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1749] * weight_shared[(((int)threadIdx.x) + 6816)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1366] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1367] * weight_shared[(((int)threadIdx.x) + 6880)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1430] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1431] * weight_shared[(((int)threadIdx.x) + 6880)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1494] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1495] * weight_shared[(((int)threadIdx.x) + 6880)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1558] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1559] * weight_shared[(((int)threadIdx.x) + 6880)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1622] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1623] * weight_shared[(((int)threadIdx.x) + 6880)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1686] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1687] * weight_shared[(((int)threadIdx.x) + 6880)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1750] * weight_shared[(((int)threadIdx.x) + 6848)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1751] * weight_shared[(((int)threadIdx.x) + 6880)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1368] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1369] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1432] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1433] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1496] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1497] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1560] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1561] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1624] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1625] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1688] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1689] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1752] * weight_shared[(((int)threadIdx.x) + 6912)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1753] * weight_shared[(((int)threadIdx.x) + 6944)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1370] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1371] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1434] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1435] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1498] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1499] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1562] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1563] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1626] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1627] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1690] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1691] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1754] * weight_shared[(((int)threadIdx.x) + 6976)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1755] * weight_shared[(((int)threadIdx.x) + 7008)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1372] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1373] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1436] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1437] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1500] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1501] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1564] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1565] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1628] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1629] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1692] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1693] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1756] * weight_shared[(((int)threadIdx.x) + 7040)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1757] * weight_shared[(((int)threadIdx.x) + 7072)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1374] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1375] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1438] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1439] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1502] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1503] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1566] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1567] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1630] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1631] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1694] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1695] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1758] * weight_shared[(((int)threadIdx.x) + 7104)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1759] * weight_shared[(((int)threadIdx.x) + 7136)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1376] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1377] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1440] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1441] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1504] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1505] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1568] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1569] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1632] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1633] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1696] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1697] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1760] * weight_shared[(((int)threadIdx.x) + 7168)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1761] * weight_shared[(((int)threadIdx.x) + 7200)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1378] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1379] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1442] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1443] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1506] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1507] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1570] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1571] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1634] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1635] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1698] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1699] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1762] * weight_shared[(((int)threadIdx.x) + 7232)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1763] * weight_shared[(((int)threadIdx.x) + 7264)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1380] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1381] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1444] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1445] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1508] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1509] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1572] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1573] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1636] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1637] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1700] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1701] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1764] * weight_shared[(((int)threadIdx.x) + 7296)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1765] * weight_shared[(((int)threadIdx.x) + 7328)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1382] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1383] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1446] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1447] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1510] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1511] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1574] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1575] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1638] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1639] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1702] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1703] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1766] * weight_shared[(((int)threadIdx.x) + 7360)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1767] * weight_shared[(((int)threadIdx.x) + 7392)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1384] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1385] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1448] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1449] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1512] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1513] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1576] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1577] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1640] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1641] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1704] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1705] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1768] * weight_shared[(((int)threadIdx.x) + 7424)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1769] * weight_shared[(((int)threadIdx.x) + 7456)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1386] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1387] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1450] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1451] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1514] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1515] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1578] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1579] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1642] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1643] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1706] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1707] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1770] * weight_shared[(((int)threadIdx.x) + 7488)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1771] * weight_shared[(((int)threadIdx.x) + 7520)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1388] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1389] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1452] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1453] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1516] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1517] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1580] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1581] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1644] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1645] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1708] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1709] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1772] * weight_shared[(((int)threadIdx.x) + 7552)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1773] * weight_shared[(((int)threadIdx.x) + 7584)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1390] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1391] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1454] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1455] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1518] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1519] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1582] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1583] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1646] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1647] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1710] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1711] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1774] * weight_shared[(((int)threadIdx.x) + 7616)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1775] * weight_shared[(((int)threadIdx.x) + 7648)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1392] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1393] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1456] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1457] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1520] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1521] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1584] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1585] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1648] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1649] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1712] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1713] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1776] * weight_shared[(((int)threadIdx.x) + 7680)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1777] * weight_shared[(((int)threadIdx.x) + 7712)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1394] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1395] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1458] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1459] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1522] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1523] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1586] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1587] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1650] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1651] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1714] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1715] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1778] * weight_shared[(((int)threadIdx.x) + 7744)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1779] * weight_shared[(((int)threadIdx.x) + 7776)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1396] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1397] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1460] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1461] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1524] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1525] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1588] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1589] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1652] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1653] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1716] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1717] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1780] * weight_shared[(((int)threadIdx.x) + 7808)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1781] * weight_shared[(((int)threadIdx.x) + 7840)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1398] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1399] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1462] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1463] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1526] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1527] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1590] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1591] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1654] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1655] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1718] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1719] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1782] * weight_shared[(((int)threadIdx.x) + 7872)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1783] * weight_shared[(((int)threadIdx.x) + 7904)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1400] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1401] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1464] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1465] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1528] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1529] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1592] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1593] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1656] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1657] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1720] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1721] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1784] * weight_shared[(((int)threadIdx.x) + 7936)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1785] * weight_shared[(((int)threadIdx.x) + 7968)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1402] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1403] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1466] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1467] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1530] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1531] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1594] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1595] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1658] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1659] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1722] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1723] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1786] * weight_shared[(((int)threadIdx.x) + 8000)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1787] * weight_shared[(((int)threadIdx.x) + 8032)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1404] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1405] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1468] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1469] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1532] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1533] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1596] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1597] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1660] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1661] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1724] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1725] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1788] * weight_shared[(((int)threadIdx.x) + 8064)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1789] * weight_shared[(((int)threadIdx.x) + 8096)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1406] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[1407] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1470] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[1471] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1534] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[1535] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1598] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[1599] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1662] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[1663] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1726] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[1727] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1790] * weight_shared[(((int)threadIdx.x) + 8128)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[1791] * weight_shared[(((int)threadIdx.x) + 8160)]));
  conv2d_nhwc[((((((int)blockIdx.x) >> 4) * 3584) + ((((int)blockIdx.x) & 15) * 32)) + ((int)threadIdx.x))] = conv2d_nhwc_local[0];
  conv2d_nhwc[(((((((int)blockIdx.x) >> 4) * 3584) + ((((int)blockIdx.x) & 15) * 32)) + ((int)threadIdx.x)) + 512)] = conv2d_nhwc_local[1];
  conv2d_nhwc[(((((((int)blockIdx.x) >> 4) * 3584) + ((((int)blockIdx.x) & 15) * 32)) + ((int)threadIdx.x)) + 1024)] = conv2d_nhwc_local[2];
  conv2d_nhwc[(((((((int)blockIdx.x) >> 4) * 3584) + ((((int)blockIdx.x) & 15) * 32)) + ((int)threadIdx.x)) + 1536)] = conv2d_nhwc_local[3];
  conv2d_nhwc[(((((((int)blockIdx.x) >> 4) * 3584) + ((((int)blockIdx.x) & 15) * 32)) + ((int)threadIdx.x)) + 2048)] = conv2d_nhwc_local[4];
  conv2d_nhwc[(((((((int)blockIdx.x) >> 4) * 3584) + ((((int)blockIdx.x) & 15) * 32)) + ((int)threadIdx.x)) + 2560)] = conv2d_nhwc_local[5];
  conv2d_nhwc[(((((((int)blockIdx.x) >> 4) * 3584) + ((((int)blockIdx.x) & 15) * 32)) + ((int)threadIdx.x)) + 3072)] = conv2d_nhwc_local[6];
}


