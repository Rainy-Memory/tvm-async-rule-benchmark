
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[28];
  __shared__ float PadInput_shared[7168];
  __shared__ float weight_shared[2048];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  conv2d_nhwc_local[7] = 0.000000e+00f;
  conv2d_nhwc_local[8] = 0.000000e+00f;
  conv2d_nhwc_local[9] = 0.000000e+00f;
  conv2d_nhwc_local[10] = 0.000000e+00f;
  conv2d_nhwc_local[11] = 0.000000e+00f;
  conv2d_nhwc_local[12] = 0.000000e+00f;
  conv2d_nhwc_local[13] = 0.000000e+00f;
  conv2d_nhwc_local[14] = 0.000000e+00f;
  conv2d_nhwc_local[15] = 0.000000e+00f;
  conv2d_nhwc_local[16] = 0.000000e+00f;
  conv2d_nhwc_local[17] = 0.000000e+00f;
  conv2d_nhwc_local[18] = 0.000000e+00f;
  conv2d_nhwc_local[19] = 0.000000e+00f;
  conv2d_nhwc_local[20] = 0.000000e+00f;
  conv2d_nhwc_local[21] = 0.000000e+00f;
  conv2d_nhwc_local[22] = 0.000000e+00f;
  conv2d_nhwc_local[23] = 0.000000e+00f;
  conv2d_nhwc_local[24] = 0.000000e+00f;
  conv2d_nhwc_local[25] = 0.000000e+00f;
  conv2d_nhwc_local[26] = 0.000000e+00f;
  conv2d_nhwc_local[27] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) * 2)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 7168))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 14336))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 21504))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 28672))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 35840))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 43008))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 16))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 7184))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2304)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 14352))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2560)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 21520))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2816)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 28688))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 3072)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 35856))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 3328)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 43024))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1024))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 3584)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 32))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 3840)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 7200))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 4096)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 14368))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 4352)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 21536))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 4608)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 28704))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 4864)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 35872))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 5120)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 43040))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2048))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 5376)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 48))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 5632)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 7216))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 5888)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 14384))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 6144)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 21552))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 6400)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 28720))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 6656)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 35888))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 6912)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 43056))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 64) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3072))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32))] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32))] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 16)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 16)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 17)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 17)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 18)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 18)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 19)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 19)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 128)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 128)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 129)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 129)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 130)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 130)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 131)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 131)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 144)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 144)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 256)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 256)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 257)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 257)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 258)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 258)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 259)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 259)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 272)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 272)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 273)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 273)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 274)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 274)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 275)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 275)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 384)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 384)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 385)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 385)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 386)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 386)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 387)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 387)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 400)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 400)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 401)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 401)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 402)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 402)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 403)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 403)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 512)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 512)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 513)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 513)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 514)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 514)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 515)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 515)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 528)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 528)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 529)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 529)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 530)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 530)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 531)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 531)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 640)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 640)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 641)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 641)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 642)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 642)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 643)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 643)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 656)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 656)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 768)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 768)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 769)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 769)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 770)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 770)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 771)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 771)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 784)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 784)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 785)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 785)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 786)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 786)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 787)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 787)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 7)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 7)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 20)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 20)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 21)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 21)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 22)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 22)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 23)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 23)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 132)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 132)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 133)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 133)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 134)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 134)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 135)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 135)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 260)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 260)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 261)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 261)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 262)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 262)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 263)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 263)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 276)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 276)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 277)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 277)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 278)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 278)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 279)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 279)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 388)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 388)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 389)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 389)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 390)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 390)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 391)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 391)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 404)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 404)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 405)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 405)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 406)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 406)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 407)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 407)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 516)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 516)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 517)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 517)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 518)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 518)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 519)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 519)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 532)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 532)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 533)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 533)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 534)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 534)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 535)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 535)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 644)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 644)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 645)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 645)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 646)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 646)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 647)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 647)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 772)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 772)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 773)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 773)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 774)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 774)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 775)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 775)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 788)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 788)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 789)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 789)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 790)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 790)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 791)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 791)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 8)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 8)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 9)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 9)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 10)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 10)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 11)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 11)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 24)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 24)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 25)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 25)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 26)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 26)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 27)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 27)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 136)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 136)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 137)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 137)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 138)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 138)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 139)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 139)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 264)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 264)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 265)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 265)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 266)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 266)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 267)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 267)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 280)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 280)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 281)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 281)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 282)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 282)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 283)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 283)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 392)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 392)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 393)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 393)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 394)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 394)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 395)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 395)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 408)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 408)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 409)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 409)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 410)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 410)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 411)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 411)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 520)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 520)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 521)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 521)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 522)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 522)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 523)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 523)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 536)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 536)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 537)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 537)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 538)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 538)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 539)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 539)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 648)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 648)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 649)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 649)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 650)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 650)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 651)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 651)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 664)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 664)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 665)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 665)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 666)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 666)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 667)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 667)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 776)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 776)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 777)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 777)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 778)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 778)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 779)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 779)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 792)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 792)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 793)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 793)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 794)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 794)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 795)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 795)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 12)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 12)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 13)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 13)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 14)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 14)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 15)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 15)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 28)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 28)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 29)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 29)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 30)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 30)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 31)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 31)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 140)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 140)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 141)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 141)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 142)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 142)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 143)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 143)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 268)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 268)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 269)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 269)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 270)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 270)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 271)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 271)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 284)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 284)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 285)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 285)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 286)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 286)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 287)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 287)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 396)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 396)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 397)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 397)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 398)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 398)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 399)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 399)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 412)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 412)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 413)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 413)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 414)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 414)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 415)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 415)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 524)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 524)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 525)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 525)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 526)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 526)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 527)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 527)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 540)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 540)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 541)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 541)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 542)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 542)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 543)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 543)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 652)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 652)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 653)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 653)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 654)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 654)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 655)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 655)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 668)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 668)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 669)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 669)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 670)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 670)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 671)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 671)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 780)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 780)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 781)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 781)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 782)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 782)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 783)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 783)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 796)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 796)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 797)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 797)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 798)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 798)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 799)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 799)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1792)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1792)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1793)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1793)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1794)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1794)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1795)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1795)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1808)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1808)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1809)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1809)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1810)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1810)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1811)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1811)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1920)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1920)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1921)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1921)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1922)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1922)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1923)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1923)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1936)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1936)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1937)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1937)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1938)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1938)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1939)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1939)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2048)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2048)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2049)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2049)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2050)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2050)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2051)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2051)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2176)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2176)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2177)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2177)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2178)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2178)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2179)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2179)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2304)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2304)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2305)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2305)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2306)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2306)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2307)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2307)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2320)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2320)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2321)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2321)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2322)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2322)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2323)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2323)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2432)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2432)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2433)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2433)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2434)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2434)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2435)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2435)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2448)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2448)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2449)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2449)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2450)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2450)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2451)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2451)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2560)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2560)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2561)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2561)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2562)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2562)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2563)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2563)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2576)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2576)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2577)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2577)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2578)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2578)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2579)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2579)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1796)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1796)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1797)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1797)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1798)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1798)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1799)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1799)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1812)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1812)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1813)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1813)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1814)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1814)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1815)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1815)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1924)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1924)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1925)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1925)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1926)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1926)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1927)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1927)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1940)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1940)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1941)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1941)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1942)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1942)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1943)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1943)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2180)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2180)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2181)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2181)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2182)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2182)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2183)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2183)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2308)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2308)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2309)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2309)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2310)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2310)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2311)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2311)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2324)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2324)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2325)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2325)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2326)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2326)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2327)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2327)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2436)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2436)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2437)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2437)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2438)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2438)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2439)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2439)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2452)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2452)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2453)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2453)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2454)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2454)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2455)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2455)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2564)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2564)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2565)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2565)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2566)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2566)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2567)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2567)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2580)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2580)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2581)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2581)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2582)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2582)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2583)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2583)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1800)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1800)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1801)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1801)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1802)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1802)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1803)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1803)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1816)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1816)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1817)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1817)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1818)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1818)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1819)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1819)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1928)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1928)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1929)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1929)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1930)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1930)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1931)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1931)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1944)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1944)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1945)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1945)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1946)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1946)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1947)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1947)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2072)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2072)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2073)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2073)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2074)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2074)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2075)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2075)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2184)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2184)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2185)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2185)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2186)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2186)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2187)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2187)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2312)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2312)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2313)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2313)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2314)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2314)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2315)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2315)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2328)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2328)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2329)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2329)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2330)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2330)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2331)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2331)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2440)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2440)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2441)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2441)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2442)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2442)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2443)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2443)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2456)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2456)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2457)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2457)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2458)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2458)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2459)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2459)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2568)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2568)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2569)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2569)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2570)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2570)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2571)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2571)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2584)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2584)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2585)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2585)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2586)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2586)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2587)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2587)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1804)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1804)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1805)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1805)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1806)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1806)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1807)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1807)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1820)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1820)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1821)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1821)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1822)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1822)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1823)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1823)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1932)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1932)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1933)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1933)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1934)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1934)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1935)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1935)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1948)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1948)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1949)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1949)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1950)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1950)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1951)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 1951)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2076)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2076)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2077)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2077)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2078)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2078)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2079)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2079)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2188)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2188)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2189)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2189)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2190)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2190)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2191)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2191)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2316)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2316)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2317)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2317)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2318)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2318)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2319)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2319)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2444)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2444)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2445)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2445)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2446)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2446)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2447)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2447)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2460)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2460)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2461)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2461)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2462)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2462)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2463)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2463)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2572)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2572)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2573)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2573)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2574)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2574)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2575)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2575)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2588)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2588)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2589)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2589)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2590)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2590)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2591)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 2591)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3584)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3584)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3585)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3585)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3586)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3586)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3587)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3587)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3600)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3600)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3601)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3601)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3602)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3602)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3603)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3603)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3712)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3712)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3713)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3713)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3714)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3714)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3715)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3715)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3840)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3840)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3841)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3841)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3842)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3842)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3843)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3843)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3856)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3856)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3857)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3857)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3858)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3858)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3859)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3859)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3968)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3968)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3969)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3969)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3970)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3970)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3971)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3971)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3984)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3984)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3985)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3985)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3986)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3986)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3987)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3987)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4096)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4096)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4097)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4097)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4098)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4098)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4099)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4099)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4112)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4112)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4113)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4113)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4114)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4114)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4115)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4115)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4224)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4224)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4225)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4225)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4226)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4226)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4227)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4227)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4240)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4240)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4241)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4241)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4242)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4242)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4243)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4243)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4352)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4352)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4353)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4353)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4354)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4354)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4355)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4355)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4368)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4368)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4369)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4369)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4370)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4370)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4371)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4371)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3588)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3588)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3589)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3589)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3590)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3590)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3591)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3591)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3604)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3604)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3605)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3605)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3606)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3606)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3607)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3607)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3716)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3716)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3717)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3717)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3718)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3718)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3719)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3719)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3844)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3844)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3845)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3845)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3846)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3846)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3847)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3847)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3860)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3860)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3861)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3861)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3862)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3862)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3863)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3863)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3972)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3972)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3973)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3973)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3974)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3974)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3975)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3975)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3988)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3988)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3989)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3989)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3990)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3990)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3991)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3991)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4100)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4100)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4101)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4101)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4102)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4102)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4103)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4103)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4116)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4116)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4117)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4117)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4118)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4118)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4119)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4119)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4228)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4228)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4229)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4229)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4230)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4230)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4231)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4231)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4244)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4244)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4245)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4245)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4246)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4246)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4247)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4247)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4356)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4356)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4357)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4357)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4358)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4358)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4359)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4359)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4372)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4372)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4373)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4373)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4374)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4374)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4375)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4375)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3592)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3592)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3593)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3593)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3594)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3594)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3595)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3595)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3608)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3608)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3609)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3609)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3610)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3610)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3611)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3611)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3720)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3720)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3721)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3721)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3722)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3722)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3723)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3723)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3848)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3848)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3849)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3849)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3850)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3850)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3851)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3851)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3864)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3864)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3865)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3865)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3866)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3866)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3867)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3867)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3976)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3976)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3977)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3977)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3978)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3978)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3979)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3979)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3992)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3992)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3993)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3993)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3994)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3994)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3995)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3995)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4104)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4104)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4105)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4105)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4106)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4106)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4107)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4107)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4120)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4120)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4121)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4121)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4122)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4122)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4123)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4123)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4232)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4232)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4233)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4233)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4234)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4234)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4235)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4235)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4248)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4248)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4249)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4249)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4250)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4250)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4251)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4251)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4360)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4360)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4361)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4361)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4362)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4362)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4363)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4363)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4376)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4376)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4377)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4377)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4378)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4378)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4379)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4379)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3596)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3596)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3597)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3597)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3598)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3598)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3599)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3599)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3612)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3612)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3613)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3613)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3614)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3614)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3615)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3615)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3724)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3724)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3725)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3725)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3726)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3726)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3727)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3727)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3852)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3852)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3853)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3853)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3854)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3854)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3855)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3855)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3868)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3868)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3869)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3869)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3870)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3870)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3871)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3871)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3980)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3980)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3981)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3981)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3982)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3982)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3983)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3983)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3996)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3996)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3997)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3997)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3998)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3998)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3999)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 3999)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4108)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4108)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4109)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4109)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4110)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4110)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4111)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4111)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4124)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4124)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4125)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4125)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4126)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4126)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4127)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4127)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4236)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4236)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4237)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4237)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4238)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4238)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4239)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4239)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4252)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4252)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4253)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4253)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4254)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4254)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4255)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4255)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4364)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4364)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4365)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4365)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4366)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4366)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4367)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4367)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4380)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4380)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4381)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4381)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4382)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4382)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4383)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 4383)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5376)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5376)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5377)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5377)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5378)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5378)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5379)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5379)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5392)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5392)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5393)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5393)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5394)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5394)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5395)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5395)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5504)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5504)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5505)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5505)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5506)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5506)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5507)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5507)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5520)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5520)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5521)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5521)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5522)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5522)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5523)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5523)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5632)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5632)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5633)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5633)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5634)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5634)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5635)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5635)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5648)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5648)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5649)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5649)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5650)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5650)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5651)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5651)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5760)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5760)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5761)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5761)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5762)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5762)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5763)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5763)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5776)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5776)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5777)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5777)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5778)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5778)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5779)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5779)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5888)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5888)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5889)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5889)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5890)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5890)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5891)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5891)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5904)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5904)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5905)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5905)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5906)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5906)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5907)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5907)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6016)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6016)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6017)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6017)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6018)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6018)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6019)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6019)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6032)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6032)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6033)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6033)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6034)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6034)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6035)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6035)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6144)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6144)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6160)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6160)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6161)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6161)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6162)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6162)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6163)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6163)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5380)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5380)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5381)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5381)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5382)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5382)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5383)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5383)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5396)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5396)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5397)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5397)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5398)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5398)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5399)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5399)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5508)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5508)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5509)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5509)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5510)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5510)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5511)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5511)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5524)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5524)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5525)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5525)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5526)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5526)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5527)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5527)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5636)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5636)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5637)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5637)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5638)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5638)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5639)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5639)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5652)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5652)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5653)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5653)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5654)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5654)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5655)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5655)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5764)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5764)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5765)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5765)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5766)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5766)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5767)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5767)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5780)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5780)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5781)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5781)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5782)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5782)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5783)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5783)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5892)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5892)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5893)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5893)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5894)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5894)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5895)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5895)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5908)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5908)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5909)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5909)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5910)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5910)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5911)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5911)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6020)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6020)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6021)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6021)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6022)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6022)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6023)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6023)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6036)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6036)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6037)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6037)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6038)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6038)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6039)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6039)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6164)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6164)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6165)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6165)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6166)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6166)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6167)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6167)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5384)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5384)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5385)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5385)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5386)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5386)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5387)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5387)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5400)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5400)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5401)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5401)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5402)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5402)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5403)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5403)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5512)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5512)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5513)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5513)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5514)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5514)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5515)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5515)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5528)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5528)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5529)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5529)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5530)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5530)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5531)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5531)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5640)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5640)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5641)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5641)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5642)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5642)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5643)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5643)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5656)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5656)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5768)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5768)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5769)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5769)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5770)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5770)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5771)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5771)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5784)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5784)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5785)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5785)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5786)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5786)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5787)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5787)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5896)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5896)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5897)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5897)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5898)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5898)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5899)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5899)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5912)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5912)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5913)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5913)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5914)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5914)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5915)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5915)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6024)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6024)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6025)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6025)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6026)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6026)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6027)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6027)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6040)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6040)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6041)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6041)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6042)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6042)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6043)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6043)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6168)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6168)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6169)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6169)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6170)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6170)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6171)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6171)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5388)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5388)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5389)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5389)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5390)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5390)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5391)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5391)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5404)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5404)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5405)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5405)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5406)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5406)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5407)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5407)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5516)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5516)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5517)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5517)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5518)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5518)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5519)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5519)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5532)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5532)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5533)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5533)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5534)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5534)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5535)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5535)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5644)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5644)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5645)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5645)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5646)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5646)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5647)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5647)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5772)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5772)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5773)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5773)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5774)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5774)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5775)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5775)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5788)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5788)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5789)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5789)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5790)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5790)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5791)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5791)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5900)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5900)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5901)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5901)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5902)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5902)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5903)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5903)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5916)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5916)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5917)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5917)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5918)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5918)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5919)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 5919)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6028)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6028)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6029)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6029)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6030)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6030)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6031)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6031)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6044)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6044)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6045)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6045)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6046)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6046)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6047)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6047)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6172)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6172)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6173)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6173)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6174)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6174)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6175)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((((int)threadIdx.x) >> 6) * 896) + (((((int)threadIdx.x) & 63) >> 4) * 32)) + 6175)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc[(((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2))] = conv2d_nhwc_local[0];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 1)] = conv2d_nhwc_local[1];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 64)] = conv2d_nhwc_local[2];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 65)] = conv2d_nhwc_local[3];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3584)] = conv2d_nhwc_local[4];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3585)] = conv2d_nhwc_local[5];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3648)] = conv2d_nhwc_local[6];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3649)] = conv2d_nhwc_local[7];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 7168)] = conv2d_nhwc_local[8];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 7169)] = conv2d_nhwc_local[9];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 7232)] = conv2d_nhwc_local[10];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 7233)] = conv2d_nhwc_local[11];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 10752)] = conv2d_nhwc_local[12];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 10753)] = conv2d_nhwc_local[13];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 10816)] = conv2d_nhwc_local[14];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 10817)] = conv2d_nhwc_local[15];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 14336)] = conv2d_nhwc_local[16];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 14337)] = conv2d_nhwc_local[17];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 14400)] = conv2d_nhwc_local[18];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 14401)] = conv2d_nhwc_local[19];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 17920)] = conv2d_nhwc_local[20];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 17921)] = conv2d_nhwc_local[21];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 17984)] = conv2d_nhwc_local[22];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 17985)] = conv2d_nhwc_local[23];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 21504)] = conv2d_nhwc_local[24];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 21505)] = conv2d_nhwc_local[25];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 21568)] = conv2d_nhwc_local[26];
  conv2d_nhwc[((((((((((int)blockIdx.x) / 14) * 50176) + ((((int)threadIdx.x) >> 6) * 25088)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + (((((int)threadIdx.x) & 63) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 21569)] = conv2d_nhwc_local[27];
}


