
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[7];
  __shared__ float PadInput_shared[1152];
  __shared__ float weight_shared[9216];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((int)threadIdx.x)))
    );
    int src_bytes = ((16 <= ((int)blockIdx.x)) && (1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((int)threadIdx.x) >> 3)))) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)threadIdx.x) >> 3) * 256)) + (((int)threadIdx.x) & 7)) - 3840))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 64)))
    );
    int src_bytes = (((1 <= (((((int)blockIdx.x) >> 4) * 2) + ((((int)threadIdx.x) + 64) / 72))) && (1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 8) % 9)))) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 8) % 9)) < 15)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 64) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((((int)threadIdx.x) >> 3) + 8) % 9) * 256)) + (((int)threadIdx.x) & 7)) - 3840))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 128)))
    );
    int src_bytes = ((1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 7) % 9))) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 7) % 9)) < 15)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 128) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((((int)threadIdx.x) >> 3) + 7) % 9) * 256)) + (((int)threadIdx.x) & 7)) - 3840))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 192)))
    );
    int src_bytes = ((((((((int)blockIdx.x) >> 4) * 2) + ((((int)threadIdx.x) + 192) / 72)) < 15) && (1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 6) % 9)))) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 6) % 9)) < 15)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 192) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((((int)threadIdx.x) >> 3) + 6) % 9) * 256)) + (((int)threadIdx.x) & 7)) - 3840))), "n"(4), "r"(src_bytes)
    );
  }
  if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 256)))
    );
    int src_bytes = (((((((int)blockIdx.x) >> 4) * 2) + ((((int)threadIdx.x) + 256) / 72)) < 15) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((int)threadIdx.x) >> 3)) < 10)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 256) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)threadIdx.x) >> 3) * 256)) + (((int)threadIdx.x) & 7)) - 2560))), "n"(4), "r"(src_bytes)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 65536))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 131072))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 196608))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 262144))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 327680))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 393216))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 458752))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 524288))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 288)))
    );
    int src_bytes = ((16 <= ((int)blockIdx.x)) && (1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((int)threadIdx.x) >> 3)))) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)threadIdx.x) >> 3) * 256)) + (((int)threadIdx.x) & 7)) - 3832))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 352)))
    );
    int src_bytes = (((1 <= (((((int)blockIdx.x) >> 4) * 2) + ((((int)threadIdx.x) + 64) / 72))) && (1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 8) % 9)))) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 8) % 9)) < 15)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 64) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((((int)threadIdx.x) >> 3) + 8) % 9) * 256)) + (((int)threadIdx.x) & 7)) - 3832))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 416)))
    );
    int src_bytes = ((1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 7) % 9))) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 7) % 9)) < 15)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 128) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((((int)threadIdx.x) >> 3) + 7) % 9) * 256)) + (((int)threadIdx.x) & 7)) - 3832))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 480)))
    );
    int src_bytes = ((((((((int)blockIdx.x) >> 4) * 2) + ((((int)threadIdx.x) + 192) / 72)) < 15) && (1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 6) % 9)))) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 6) % 9)) < 15)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 192) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((((int)threadIdx.x) >> 3) + 6) % 9) * 256)) + (((int)threadIdx.x) & 7)) - 3832))), "n"(4), "r"(src_bytes)
    );
  }
  if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 544)))
    );
    int src_bytes = (((((((int)blockIdx.x) >> 4) * 2) + ((((int)threadIdx.x) + 256) / 72)) < 15) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((int)threadIdx.x) >> 3)) < 10)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 256) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)threadIdx.x) >> 3) * 256)) + (((int)threadIdx.x) & 7)) - 2552))), "n"(4), "r"(src_bytes)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2304)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2048))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2560)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 67584))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2816)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 133120))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3072)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 198656))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3328)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 264192))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3584)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 329728))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3840)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 395264))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4096)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 460800))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4352)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 526336))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 576)))
    );
    int src_bytes = ((16 <= ((int)blockIdx.x)) && (1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((int)threadIdx.x) >> 3)))) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)threadIdx.x) >> 3) * 256)) + (((int)threadIdx.x) & 7)) - 3824))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 640)))
    );
    int src_bytes = (((1 <= (((((int)blockIdx.x) >> 4) * 2) + ((((int)threadIdx.x) + 64) / 72))) && (1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 8) % 9)))) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 8) % 9)) < 15)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 64) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((((int)threadIdx.x) >> 3) + 8) % 9) * 256)) + (((int)threadIdx.x) & 7)) - 3824))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 704)))
    );
    int src_bytes = ((1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 7) % 9))) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 7) % 9)) < 15)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 128) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((((int)threadIdx.x) >> 3) + 7) % 9) * 256)) + (((int)threadIdx.x) & 7)) - 3824))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 768)))
    );
    int src_bytes = ((((((((int)blockIdx.x) >> 4) * 2) + ((((int)threadIdx.x) + 192) / 72)) < 15) && (1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 6) % 9)))) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 6) % 9)) < 15)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 192) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((((int)threadIdx.x) >> 3) + 6) % 9) * 256)) + (((int)threadIdx.x) & 7)) - 3824))), "n"(4), "r"(src_bytes)
    );
  }
  if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 832)))
    );
    int src_bytes = (((((((int)blockIdx.x) >> 4) * 2) + ((((int)threadIdx.x) + 256) / 72)) < 15) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((int)threadIdx.x) >> 3)) < 10)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 256) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)threadIdx.x) >> 3) * 256)) + (((int)threadIdx.x) & 7)) - 2544))), "n"(4), "r"(src_bytes)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4608)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 4096))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 4864)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 69632))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5120)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 135168))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5376)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 200704))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5632)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 266240))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 5888)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 331776))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 6144)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 397312))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 6400)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 462848))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 6656)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 528384))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int rh_0_rw_0_rc_0_fused = 0; rh_0_rw_0_rc_0_fused < 29; ++rh_0_rw_0_rc_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 288) + ((int)threadIdx.x))))
    );
    int src_bytes = ((16 <= ((int)blockIdx.x)) && (1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((int)threadIdx.x) >> 3)))) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)threadIdx.x) >> 3) * 256)) + (rh_0_rw_0_rc_0_fused * 8)) + (((int)threadIdx.x) & 7)) - 3816))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 288) + ((int)threadIdx.x)) + 64)))
    );
    int src_bytes = (((1 <= (((((int)blockIdx.x) >> 4) * 2) + ((((int)threadIdx.x) + 64) / 72))) && (1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 8) % 9)))) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 8) % 9)) < 15)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 64) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((((int)threadIdx.x) >> 3) + 8) % 9) * 256)) + (rh_0_rw_0_rc_0_fused * 8)) + (((int)threadIdx.x) & 7)) - 3816))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 288) + ((int)threadIdx.x)) + 128)))
    );
    int src_bytes = ((1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 7) % 9))) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 7) % 9)) < 15)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 128) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((((int)threadIdx.x) >> 3) + 7) % 9) * 256)) + (rh_0_rw_0_rc_0_fused * 8)) + (((int)threadIdx.x) & 7)) - 3816))), "n"(4), "r"(src_bytes)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 288) + ((int)threadIdx.x)) + 192)))
    );
    int src_bytes = ((((((((int)blockIdx.x) >> 4) * 2) + ((((int)threadIdx.x) + 192) / 72)) < 15) && (1 <= ((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 6) % 9)))) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((((int)threadIdx.x) >> 3) + 6) % 9)) < 15)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 192) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((((int)threadIdx.x) >> 3) + 6) % 9) * 256)) + (rh_0_rw_0_rc_0_fused * 8)) + (((int)threadIdx.x) & 7)) - 3816))), "n"(4), "r"(src_bytes)
    );
  }
    if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 288) + ((int)threadIdx.x)) + 256)))
    );
    int src_bytes = (((((((int)blockIdx.x) >> 4) * 2) + ((((int)threadIdx.x) + 256) / 72)) < 15) && (((((((int)blockIdx.x) & 15) >> 3) * 7) + (((int)threadIdx.x) >> 3)) < 10)) ? 4 : 0;
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2, %3;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((((int)blockIdx.x) >> 4) * 7168) + (((((int)threadIdx.x) + 256) / 72) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)threadIdx.x) >> 3) * 256)) + (rh_0_rw_0_rc_0_fused * 8)) + (((int)threadIdx.x) & 7)) - 2536))), "n"(4), "r"(src_bytes)
    );
  }
    }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 2048) + ((((int)threadIdx.x) >> 3) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 6144))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 2048) + ((((int)threadIdx.x) >> 3) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 71680))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 2048) + ((((int)threadIdx.x) >> 3) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 137216))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 2048) + ((((int)threadIdx.x) >> 3) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 202752))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 2048) + ((((int)threadIdx.x) >> 3) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 268288))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 2048) + ((((int)threadIdx.x) >> 3) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 333824))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 2048) + ((((int)threadIdx.x) >> 3) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 399360))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 2048) + ((((int)threadIdx.x) >> 3) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 464896))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2304) + (((int)threadIdx.x) * 4)) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 2048) + ((((int)threadIdx.x) >> 3) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 530432))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

    __syncthreads();
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72))] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31))]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 32)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 256)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 288)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 512)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 544)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 8)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31))]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 32)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 256)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 288)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 512)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 544)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 16)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31))]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 32)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 256)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 288)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 32)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 512)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 544)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 24)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31))]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 32)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 32)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 256)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 288)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 512)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 544)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 32)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31))]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 32)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 256)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 288)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 512)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 544)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 40)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31))]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 32)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 256)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 288)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 512)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 544)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 48)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31))]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 32)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 256)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 288)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 512)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 544)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 64)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 96)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 320)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 352)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 576)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 608)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 64)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 96)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 320)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 352)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 576)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 608)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 64)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 96)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 320)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 352)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 576)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 608)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 64)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 96)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 320)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 352)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 576)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 608)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 64)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 96)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 320)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 352)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 576)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 608)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 64)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 96)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 320)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 352)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 576)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 608)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 64)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 96)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 320)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 352)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 576)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 608)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 128)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 160)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 384)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 416)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 640)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 672)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 128)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 160)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 384)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 416)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 640)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 672)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 128)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 160)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 384)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 416)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 640)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 672)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 128)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 160)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 384)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 416)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 640)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 672)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 128)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 160)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 384)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 416)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 640)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 672)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 128)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 160)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 384)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 416)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 640)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 672)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 128)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 160)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 384)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 416)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 640)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 672)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 192)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 224)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 448)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 480)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 704)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 736)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 192)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 224)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 448)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 480)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 704)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 736)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 192)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 224)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 448)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 480)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 704)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 736)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 192)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 224)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 448)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 480)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 704)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 736)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 192)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 224)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 448)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 480)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 704)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 736)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 192)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 224)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 448)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 480)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 704)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 736)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 192)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 224)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 448)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 480)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 704)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 736)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 768)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 800)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1024)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1056)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1280)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1312)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 768)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 800)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1024)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1056)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1280)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1312)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 768)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 800)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1024)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1056)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1280)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1312)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 768)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 800)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1024)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1056)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 112)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1280)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 113)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1312)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 768)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 800)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 112)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1024)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 113)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1056)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 120)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1280)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 121)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1312)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 112)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 768)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 113)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 800)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 120)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1024)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 121)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1056)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1280)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1312)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 120)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 768)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 121)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 800)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1024)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1056)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1280)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1312)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 832)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 864)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1088)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1120)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1344)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1376)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 832)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 864)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1088)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1120)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1344)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1376)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 832)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 864)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1088)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1120)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1344)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1376)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 832)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 864)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1088)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1120)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 114)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1344)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 115)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1376)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 832)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 864)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 114)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1088)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 115)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1120)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 122)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1344)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 123)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1376)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 114)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 832)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 115)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 864)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 122)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1088)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 123)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1120)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1344)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1376)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 122)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 832)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 123)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 864)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1088)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1120)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1344)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1376)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 896)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 928)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1152)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1184)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1408)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1440)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 896)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 928)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1152)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1184)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1408)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1440)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 896)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 928)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1152)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1184)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1408)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1440)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 896)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 928)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1152)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1184)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 116)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1408)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 117)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1440)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 896)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 928)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 116)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1152)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 117)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1184)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 124)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1408)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 125)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1440)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 116)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 896)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 117)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 928)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 124)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1152)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 125)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1184)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1408)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1440)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 124)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 896)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 125)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 928)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1152)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1184)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1408)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1440)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 960)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 992)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1216)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1248)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1472)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1504)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 960)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 992)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1216)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1248)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1472)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1504)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 960)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 992)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1216)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1248)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1472)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1504)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 960)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 992)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1216)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1248)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 118)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1472)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 119)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1504)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 960)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 992)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 118)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1216)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 119)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1248)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 126)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1472)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 127)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1504)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 118)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 960)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 119)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 992)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 126)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1216)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 127)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1248)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1472)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1504)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 126)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 960)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 127)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 992)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1216)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1248)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1472)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1504)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 144)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1536)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1568)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1792)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1824)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 160)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2048)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 161)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2080)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1536)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1568)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 160)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1792)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 161)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1824)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 168)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2048)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 169)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2080)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 160)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1536)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 161)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1568)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 168)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1792)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 169)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1824)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 176)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2048)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 177)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2080)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 168)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1536)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 169)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1568)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 176)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1792)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 177)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1824)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 184)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2048)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 185)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2080)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 176)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1536)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 177)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1568)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 184)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1792)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 185)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1824)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2048)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2080)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 184)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1536)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 185)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1568)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1792)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1824)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2048)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2080)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1536)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1568)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1792)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1824)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 208)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2048)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 209)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2080)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1600)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1632)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1856)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1888)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 162)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2112)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 163)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2144)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1600)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1632)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 162)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1856)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 163)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1888)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 170)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2112)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 171)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2144)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 162)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1600)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 163)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1632)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 170)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1856)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 171)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1888)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 178)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2112)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 179)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2144)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 170)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1600)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 171)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1632)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 178)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1856)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 179)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1888)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 186)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2112)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 187)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2144)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 178)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1600)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 179)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1632)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 186)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1856)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 187)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1888)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2112)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2144)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 186)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1600)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 187)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1632)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1856)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1888)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2112)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2144)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1600)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1632)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1856)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1888)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 210)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2112)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 211)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2144)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1664)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1696)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1920)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1952)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 164)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2176)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 165)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2208)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1664)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1696)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 164)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1920)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 165)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1952)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 172)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2176)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 173)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2208)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 164)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1664)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 165)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1696)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 172)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1920)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 173)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1952)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 180)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2176)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 181)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2208)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 172)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1664)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 173)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1696)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 180)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1920)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 181)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1952)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 188)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2176)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 189)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2208)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 180)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1664)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 181)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1696)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 188)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1920)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 189)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1952)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2176)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2208)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 188)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1664)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 189)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1696)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1920)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1952)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2176)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2208)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1664)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1696)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1920)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1952)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 212)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2176)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 213)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2208)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1728)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1760)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1984)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2016)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 166)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2240)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 167)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2272)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1728)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1760)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 166)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1984)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 167)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2016)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 174)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2240)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 175)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2272)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 166)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1728)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 167)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1760)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 174)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1984)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 175)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2016)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 182)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2240)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 183)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2272)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 174)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1728)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 175)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1760)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 182)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1984)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 183)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2016)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 190)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2240)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 191)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2272)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 182)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1728)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 183)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1760)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 190)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1984)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 191)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2016)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2240)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2272)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 190)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1728)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 191)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1760)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1984)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2016)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2240)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2272)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1728)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1760)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 1984)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2016)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 214)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2240)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 288) + ((((int)threadIdx.x) >> 5) * 72)) + 215)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2304) + (((int)threadIdx.x) & 31)) + 2272)]));
  }
__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 288)] * weight_shared[((((int)threadIdx.x) & 31) + 2304)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 289)] * weight_shared[((((int)threadIdx.x) & 31) + 2336)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 296)] * weight_shared[((((int)threadIdx.x) & 31) + 2560)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 297)] * weight_shared[((((int)threadIdx.x) & 31) + 2592)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 304)] * weight_shared[((((int)threadIdx.x) & 31) + 2816)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 305)] * weight_shared[((((int)threadIdx.x) & 31) + 2848)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 296)] * weight_shared[((((int)threadIdx.x) & 31) + 2304)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 297)] * weight_shared[((((int)threadIdx.x) & 31) + 2336)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 304)] * weight_shared[((((int)threadIdx.x) & 31) + 2560)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 305)] * weight_shared[((((int)threadIdx.x) & 31) + 2592)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 312)] * weight_shared[((((int)threadIdx.x) & 31) + 2816)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 313)] * weight_shared[((((int)threadIdx.x) & 31) + 2848)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 304)] * weight_shared[((((int)threadIdx.x) & 31) + 2304)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 305)] * weight_shared[((((int)threadIdx.x) & 31) + 2336)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 312)] * weight_shared[((((int)threadIdx.x) & 31) + 2560)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 313)] * weight_shared[((((int)threadIdx.x) & 31) + 2592)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 320)] * weight_shared[((((int)threadIdx.x) & 31) + 2816)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 321)] * weight_shared[((((int)threadIdx.x) & 31) + 2848)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 312)] * weight_shared[((((int)threadIdx.x) & 31) + 2304)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 313)] * weight_shared[((((int)threadIdx.x) & 31) + 2336)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 320)] * weight_shared[((((int)threadIdx.x) & 31) + 2560)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 321)] * weight_shared[((((int)threadIdx.x) & 31) + 2592)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 328)] * weight_shared[((((int)threadIdx.x) & 31) + 2816)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 329)] * weight_shared[((((int)threadIdx.x) & 31) + 2848)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 320)] * weight_shared[((((int)threadIdx.x) & 31) + 2304)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 321)] * weight_shared[((((int)threadIdx.x) & 31) + 2336)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 328)] * weight_shared[((((int)threadIdx.x) & 31) + 2560)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 329)] * weight_shared[((((int)threadIdx.x) & 31) + 2592)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 336)] * weight_shared[((((int)threadIdx.x) & 31) + 2816)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 337)] * weight_shared[((((int)threadIdx.x) & 31) + 2848)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 328)] * weight_shared[((((int)threadIdx.x) & 31) + 2304)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 329)] * weight_shared[((((int)threadIdx.x) & 31) + 2336)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 336)] * weight_shared[((((int)threadIdx.x) & 31) + 2560)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 337)] * weight_shared[((((int)threadIdx.x) & 31) + 2592)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 344)] * weight_shared[((((int)threadIdx.x) & 31) + 2816)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 345)] * weight_shared[((((int)threadIdx.x) & 31) + 2848)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 336)] * weight_shared[((((int)threadIdx.x) & 31) + 2304)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 337)] * weight_shared[((((int)threadIdx.x) & 31) + 2336)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 344)] * weight_shared[((((int)threadIdx.x) & 31) + 2560)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 345)] * weight_shared[((((int)threadIdx.x) & 31) + 2592)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 352)] * weight_shared[((((int)threadIdx.x) & 31) + 2816)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 353)] * weight_shared[((((int)threadIdx.x) & 31) + 2848)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 290)] * weight_shared[((((int)threadIdx.x) & 31) + 2368)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 291)] * weight_shared[((((int)threadIdx.x) & 31) + 2400)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 298)] * weight_shared[((((int)threadIdx.x) & 31) + 2624)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 299)] * weight_shared[((((int)threadIdx.x) & 31) + 2656)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 306)] * weight_shared[((((int)threadIdx.x) & 31) + 2880)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 307)] * weight_shared[((((int)threadIdx.x) & 31) + 2912)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 298)] * weight_shared[((((int)threadIdx.x) & 31) + 2368)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 299)] * weight_shared[((((int)threadIdx.x) & 31) + 2400)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 306)] * weight_shared[((((int)threadIdx.x) & 31) + 2624)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 307)] * weight_shared[((((int)threadIdx.x) & 31) + 2656)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 314)] * weight_shared[((((int)threadIdx.x) & 31) + 2880)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 315)] * weight_shared[((((int)threadIdx.x) & 31) + 2912)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 306)] * weight_shared[((((int)threadIdx.x) & 31) + 2368)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 307)] * weight_shared[((((int)threadIdx.x) & 31) + 2400)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 314)] * weight_shared[((((int)threadIdx.x) & 31) + 2624)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 315)] * weight_shared[((((int)threadIdx.x) & 31) + 2656)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 322)] * weight_shared[((((int)threadIdx.x) & 31) + 2880)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 323)] * weight_shared[((((int)threadIdx.x) & 31) + 2912)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 314)] * weight_shared[((((int)threadIdx.x) & 31) + 2368)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 315)] * weight_shared[((((int)threadIdx.x) & 31) + 2400)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 322)] * weight_shared[((((int)threadIdx.x) & 31) + 2624)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 323)] * weight_shared[((((int)threadIdx.x) & 31) + 2656)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 330)] * weight_shared[((((int)threadIdx.x) & 31) + 2880)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 331)] * weight_shared[((((int)threadIdx.x) & 31) + 2912)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 322)] * weight_shared[((((int)threadIdx.x) & 31) + 2368)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 323)] * weight_shared[((((int)threadIdx.x) & 31) + 2400)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 330)] * weight_shared[((((int)threadIdx.x) & 31) + 2624)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 331)] * weight_shared[((((int)threadIdx.x) & 31) + 2656)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 338)] * weight_shared[((((int)threadIdx.x) & 31) + 2880)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 339)] * weight_shared[((((int)threadIdx.x) & 31) + 2912)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 330)] * weight_shared[((((int)threadIdx.x) & 31) + 2368)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 331)] * weight_shared[((((int)threadIdx.x) & 31) + 2400)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 338)] * weight_shared[((((int)threadIdx.x) & 31) + 2624)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 339)] * weight_shared[((((int)threadIdx.x) & 31) + 2656)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 346)] * weight_shared[((((int)threadIdx.x) & 31) + 2880)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 347)] * weight_shared[((((int)threadIdx.x) & 31) + 2912)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 338)] * weight_shared[((((int)threadIdx.x) & 31) + 2368)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 339)] * weight_shared[((((int)threadIdx.x) & 31) + 2400)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 346)] * weight_shared[((((int)threadIdx.x) & 31) + 2624)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 347)] * weight_shared[((((int)threadIdx.x) & 31) + 2656)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 354)] * weight_shared[((((int)threadIdx.x) & 31) + 2880)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 355)] * weight_shared[((((int)threadIdx.x) & 31) + 2912)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 292)] * weight_shared[((((int)threadIdx.x) & 31) + 2432)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 293)] * weight_shared[((((int)threadIdx.x) & 31) + 2464)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 300)] * weight_shared[((((int)threadIdx.x) & 31) + 2688)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 301)] * weight_shared[((((int)threadIdx.x) & 31) + 2720)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 308)] * weight_shared[((((int)threadIdx.x) & 31) + 2944)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 309)] * weight_shared[((((int)threadIdx.x) & 31) + 2976)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 300)] * weight_shared[((((int)threadIdx.x) & 31) + 2432)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 301)] * weight_shared[((((int)threadIdx.x) & 31) + 2464)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 308)] * weight_shared[((((int)threadIdx.x) & 31) + 2688)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 309)] * weight_shared[((((int)threadIdx.x) & 31) + 2720)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 316)] * weight_shared[((((int)threadIdx.x) & 31) + 2944)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 317)] * weight_shared[((((int)threadIdx.x) & 31) + 2976)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 308)] * weight_shared[((((int)threadIdx.x) & 31) + 2432)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 309)] * weight_shared[((((int)threadIdx.x) & 31) + 2464)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 316)] * weight_shared[((((int)threadIdx.x) & 31) + 2688)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 317)] * weight_shared[((((int)threadIdx.x) & 31) + 2720)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 324)] * weight_shared[((((int)threadIdx.x) & 31) + 2944)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 325)] * weight_shared[((((int)threadIdx.x) & 31) + 2976)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 316)] * weight_shared[((((int)threadIdx.x) & 31) + 2432)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 317)] * weight_shared[((((int)threadIdx.x) & 31) + 2464)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 324)] * weight_shared[((((int)threadIdx.x) & 31) + 2688)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 325)] * weight_shared[((((int)threadIdx.x) & 31) + 2720)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 332)] * weight_shared[((((int)threadIdx.x) & 31) + 2944)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 333)] * weight_shared[((((int)threadIdx.x) & 31) + 2976)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 324)] * weight_shared[((((int)threadIdx.x) & 31) + 2432)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 325)] * weight_shared[((((int)threadIdx.x) & 31) + 2464)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 332)] * weight_shared[((((int)threadIdx.x) & 31) + 2688)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 333)] * weight_shared[((((int)threadIdx.x) & 31) + 2720)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 340)] * weight_shared[((((int)threadIdx.x) & 31) + 2944)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 341)] * weight_shared[((((int)threadIdx.x) & 31) + 2976)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 332)] * weight_shared[((((int)threadIdx.x) & 31) + 2432)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 333)] * weight_shared[((((int)threadIdx.x) & 31) + 2464)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 340)] * weight_shared[((((int)threadIdx.x) & 31) + 2688)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 341)] * weight_shared[((((int)threadIdx.x) & 31) + 2720)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 348)] * weight_shared[((((int)threadIdx.x) & 31) + 2944)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 349)] * weight_shared[((((int)threadIdx.x) & 31) + 2976)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 340)] * weight_shared[((((int)threadIdx.x) & 31) + 2432)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 341)] * weight_shared[((((int)threadIdx.x) & 31) + 2464)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 348)] * weight_shared[((((int)threadIdx.x) & 31) + 2688)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 349)] * weight_shared[((((int)threadIdx.x) & 31) + 2720)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 356)] * weight_shared[((((int)threadIdx.x) & 31) + 2944)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 357)] * weight_shared[((((int)threadIdx.x) & 31) + 2976)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 294)] * weight_shared[((((int)threadIdx.x) & 31) + 2496)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 295)] * weight_shared[((((int)threadIdx.x) & 31) + 2528)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 302)] * weight_shared[((((int)threadIdx.x) & 31) + 2752)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 303)] * weight_shared[((((int)threadIdx.x) & 31) + 2784)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 310)] * weight_shared[((((int)threadIdx.x) & 31) + 3008)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 311)] * weight_shared[((((int)threadIdx.x) & 31) + 3040)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 302)] * weight_shared[((((int)threadIdx.x) & 31) + 2496)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 303)] * weight_shared[((((int)threadIdx.x) & 31) + 2528)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 310)] * weight_shared[((((int)threadIdx.x) & 31) + 2752)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 311)] * weight_shared[((((int)threadIdx.x) & 31) + 2784)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 318)] * weight_shared[((((int)threadIdx.x) & 31) + 3008)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 319)] * weight_shared[((((int)threadIdx.x) & 31) + 3040)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 310)] * weight_shared[((((int)threadIdx.x) & 31) + 2496)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 311)] * weight_shared[((((int)threadIdx.x) & 31) + 2528)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 318)] * weight_shared[((((int)threadIdx.x) & 31) + 2752)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 319)] * weight_shared[((((int)threadIdx.x) & 31) + 2784)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 326)] * weight_shared[((((int)threadIdx.x) & 31) + 3008)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 327)] * weight_shared[((((int)threadIdx.x) & 31) + 3040)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 318)] * weight_shared[((((int)threadIdx.x) & 31) + 2496)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 319)] * weight_shared[((((int)threadIdx.x) & 31) + 2528)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 326)] * weight_shared[((((int)threadIdx.x) & 31) + 2752)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 327)] * weight_shared[((((int)threadIdx.x) & 31) + 2784)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 334)] * weight_shared[((((int)threadIdx.x) & 31) + 3008)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 335)] * weight_shared[((((int)threadIdx.x) & 31) + 3040)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 326)] * weight_shared[((((int)threadIdx.x) & 31) + 2496)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 327)] * weight_shared[((((int)threadIdx.x) & 31) + 2528)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 334)] * weight_shared[((((int)threadIdx.x) & 31) + 2752)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 335)] * weight_shared[((((int)threadIdx.x) & 31) + 2784)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 342)] * weight_shared[((((int)threadIdx.x) & 31) + 3008)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 343)] * weight_shared[((((int)threadIdx.x) & 31) + 3040)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 334)] * weight_shared[((((int)threadIdx.x) & 31) + 2496)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 335)] * weight_shared[((((int)threadIdx.x) & 31) + 2528)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 342)] * weight_shared[((((int)threadIdx.x) & 31) + 2752)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 343)] * weight_shared[((((int)threadIdx.x) & 31) + 2784)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 350)] * weight_shared[((((int)threadIdx.x) & 31) + 3008)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 351)] * weight_shared[((((int)threadIdx.x) & 31) + 3040)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 342)] * weight_shared[((((int)threadIdx.x) & 31) + 2496)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 343)] * weight_shared[((((int)threadIdx.x) & 31) + 2528)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 350)] * weight_shared[((((int)threadIdx.x) & 31) + 2752)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 351)] * weight_shared[((((int)threadIdx.x) & 31) + 2784)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 358)] * weight_shared[((((int)threadIdx.x) & 31) + 3008)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 359)] * weight_shared[((((int)threadIdx.x) & 31) + 3040)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 360)] * weight_shared[((((int)threadIdx.x) & 31) + 3072)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 361)] * weight_shared[((((int)threadIdx.x) & 31) + 3104)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 368)] * weight_shared[((((int)threadIdx.x) & 31) + 3328)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 369)] * weight_shared[((((int)threadIdx.x) & 31) + 3360)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 376)] * weight_shared[((((int)threadIdx.x) & 31) + 3584)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 377)] * weight_shared[((((int)threadIdx.x) & 31) + 3616)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 368)] * weight_shared[((((int)threadIdx.x) & 31) + 3072)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 369)] * weight_shared[((((int)threadIdx.x) & 31) + 3104)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 376)] * weight_shared[((((int)threadIdx.x) & 31) + 3328)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 377)] * weight_shared[((((int)threadIdx.x) & 31) + 3360)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 384)] * weight_shared[((((int)threadIdx.x) & 31) + 3584)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 385)] * weight_shared[((((int)threadIdx.x) & 31) + 3616)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 376)] * weight_shared[((((int)threadIdx.x) & 31) + 3072)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 377)] * weight_shared[((((int)threadIdx.x) & 31) + 3104)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 384)] * weight_shared[((((int)threadIdx.x) & 31) + 3328)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 385)] * weight_shared[((((int)threadIdx.x) & 31) + 3360)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 392)] * weight_shared[((((int)threadIdx.x) & 31) + 3584)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 393)] * weight_shared[((((int)threadIdx.x) & 31) + 3616)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 384)] * weight_shared[((((int)threadIdx.x) & 31) + 3072)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 385)] * weight_shared[((((int)threadIdx.x) & 31) + 3104)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 392)] * weight_shared[((((int)threadIdx.x) & 31) + 3328)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 393)] * weight_shared[((((int)threadIdx.x) & 31) + 3360)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 400)] * weight_shared[((((int)threadIdx.x) & 31) + 3584)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 401)] * weight_shared[((((int)threadIdx.x) & 31) + 3616)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 392)] * weight_shared[((((int)threadIdx.x) & 31) + 3072)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 393)] * weight_shared[((((int)threadIdx.x) & 31) + 3104)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 400)] * weight_shared[((((int)threadIdx.x) & 31) + 3328)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 401)] * weight_shared[((((int)threadIdx.x) & 31) + 3360)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 408)] * weight_shared[((((int)threadIdx.x) & 31) + 3584)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 409)] * weight_shared[((((int)threadIdx.x) & 31) + 3616)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 400)] * weight_shared[((((int)threadIdx.x) & 31) + 3072)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 401)] * weight_shared[((((int)threadIdx.x) & 31) + 3104)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 408)] * weight_shared[((((int)threadIdx.x) & 31) + 3328)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 409)] * weight_shared[((((int)threadIdx.x) & 31) + 3360)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 416)] * weight_shared[((((int)threadIdx.x) & 31) + 3584)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 417)] * weight_shared[((((int)threadIdx.x) & 31) + 3616)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 408)] * weight_shared[((((int)threadIdx.x) & 31) + 3072)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 409)] * weight_shared[((((int)threadIdx.x) & 31) + 3104)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 416)] * weight_shared[((((int)threadIdx.x) & 31) + 3328)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 417)] * weight_shared[((((int)threadIdx.x) & 31) + 3360)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 424)] * weight_shared[((((int)threadIdx.x) & 31) + 3584)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 425)] * weight_shared[((((int)threadIdx.x) & 31) + 3616)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 362)] * weight_shared[((((int)threadIdx.x) & 31) + 3136)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 363)] * weight_shared[((((int)threadIdx.x) & 31) + 3168)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 370)] * weight_shared[((((int)threadIdx.x) & 31) + 3392)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 371)] * weight_shared[((((int)threadIdx.x) & 31) + 3424)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 378)] * weight_shared[((((int)threadIdx.x) & 31) + 3648)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 379)] * weight_shared[((((int)threadIdx.x) & 31) + 3680)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 370)] * weight_shared[((((int)threadIdx.x) & 31) + 3136)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 371)] * weight_shared[((((int)threadIdx.x) & 31) + 3168)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 378)] * weight_shared[((((int)threadIdx.x) & 31) + 3392)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 379)] * weight_shared[((((int)threadIdx.x) & 31) + 3424)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 386)] * weight_shared[((((int)threadIdx.x) & 31) + 3648)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 387)] * weight_shared[((((int)threadIdx.x) & 31) + 3680)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 378)] * weight_shared[((((int)threadIdx.x) & 31) + 3136)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 379)] * weight_shared[((((int)threadIdx.x) & 31) + 3168)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 386)] * weight_shared[((((int)threadIdx.x) & 31) + 3392)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 387)] * weight_shared[((((int)threadIdx.x) & 31) + 3424)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 394)] * weight_shared[((((int)threadIdx.x) & 31) + 3648)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 395)] * weight_shared[((((int)threadIdx.x) & 31) + 3680)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 386)] * weight_shared[((((int)threadIdx.x) & 31) + 3136)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 387)] * weight_shared[((((int)threadIdx.x) & 31) + 3168)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 394)] * weight_shared[((((int)threadIdx.x) & 31) + 3392)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 395)] * weight_shared[((((int)threadIdx.x) & 31) + 3424)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 402)] * weight_shared[((((int)threadIdx.x) & 31) + 3648)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 403)] * weight_shared[((((int)threadIdx.x) & 31) + 3680)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 394)] * weight_shared[((((int)threadIdx.x) & 31) + 3136)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 395)] * weight_shared[((((int)threadIdx.x) & 31) + 3168)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 402)] * weight_shared[((((int)threadIdx.x) & 31) + 3392)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 403)] * weight_shared[((((int)threadIdx.x) & 31) + 3424)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 410)] * weight_shared[((((int)threadIdx.x) & 31) + 3648)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 411)] * weight_shared[((((int)threadIdx.x) & 31) + 3680)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 402)] * weight_shared[((((int)threadIdx.x) & 31) + 3136)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 403)] * weight_shared[((((int)threadIdx.x) & 31) + 3168)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 410)] * weight_shared[((((int)threadIdx.x) & 31) + 3392)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 411)] * weight_shared[((((int)threadIdx.x) & 31) + 3424)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 418)] * weight_shared[((((int)threadIdx.x) & 31) + 3648)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 419)] * weight_shared[((((int)threadIdx.x) & 31) + 3680)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 410)] * weight_shared[((((int)threadIdx.x) & 31) + 3136)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 411)] * weight_shared[((((int)threadIdx.x) & 31) + 3168)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 418)] * weight_shared[((((int)threadIdx.x) & 31) + 3392)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 419)] * weight_shared[((((int)threadIdx.x) & 31) + 3424)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 426)] * weight_shared[((((int)threadIdx.x) & 31) + 3648)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 427)] * weight_shared[((((int)threadIdx.x) & 31) + 3680)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 364)] * weight_shared[((((int)threadIdx.x) & 31) + 3200)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 365)] * weight_shared[((((int)threadIdx.x) & 31) + 3232)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 372)] * weight_shared[((((int)threadIdx.x) & 31) + 3456)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 373)] * weight_shared[((((int)threadIdx.x) & 31) + 3488)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 380)] * weight_shared[((((int)threadIdx.x) & 31) + 3712)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 381)] * weight_shared[((((int)threadIdx.x) & 31) + 3744)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 372)] * weight_shared[((((int)threadIdx.x) & 31) + 3200)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 373)] * weight_shared[((((int)threadIdx.x) & 31) + 3232)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 380)] * weight_shared[((((int)threadIdx.x) & 31) + 3456)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 381)] * weight_shared[((((int)threadIdx.x) & 31) + 3488)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 388)] * weight_shared[((((int)threadIdx.x) & 31) + 3712)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 389)] * weight_shared[((((int)threadIdx.x) & 31) + 3744)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 380)] * weight_shared[((((int)threadIdx.x) & 31) + 3200)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 381)] * weight_shared[((((int)threadIdx.x) & 31) + 3232)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 388)] * weight_shared[((((int)threadIdx.x) & 31) + 3456)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 389)] * weight_shared[((((int)threadIdx.x) & 31) + 3488)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 396)] * weight_shared[((((int)threadIdx.x) & 31) + 3712)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 397)] * weight_shared[((((int)threadIdx.x) & 31) + 3744)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 388)] * weight_shared[((((int)threadIdx.x) & 31) + 3200)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 389)] * weight_shared[((((int)threadIdx.x) & 31) + 3232)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 396)] * weight_shared[((((int)threadIdx.x) & 31) + 3456)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 397)] * weight_shared[((((int)threadIdx.x) & 31) + 3488)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 404)] * weight_shared[((((int)threadIdx.x) & 31) + 3712)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 405)] * weight_shared[((((int)threadIdx.x) & 31) + 3744)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 396)] * weight_shared[((((int)threadIdx.x) & 31) + 3200)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 397)] * weight_shared[((((int)threadIdx.x) & 31) + 3232)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 404)] * weight_shared[((((int)threadIdx.x) & 31) + 3456)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 405)] * weight_shared[((((int)threadIdx.x) & 31) + 3488)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 412)] * weight_shared[((((int)threadIdx.x) & 31) + 3712)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 413)] * weight_shared[((((int)threadIdx.x) & 31) + 3744)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 404)] * weight_shared[((((int)threadIdx.x) & 31) + 3200)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 405)] * weight_shared[((((int)threadIdx.x) & 31) + 3232)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 412)] * weight_shared[((((int)threadIdx.x) & 31) + 3456)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 413)] * weight_shared[((((int)threadIdx.x) & 31) + 3488)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 420)] * weight_shared[((((int)threadIdx.x) & 31) + 3712)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 421)] * weight_shared[((((int)threadIdx.x) & 31) + 3744)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 412)] * weight_shared[((((int)threadIdx.x) & 31) + 3200)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 413)] * weight_shared[((((int)threadIdx.x) & 31) + 3232)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 420)] * weight_shared[((((int)threadIdx.x) & 31) + 3456)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 421)] * weight_shared[((((int)threadIdx.x) & 31) + 3488)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 428)] * weight_shared[((((int)threadIdx.x) & 31) + 3712)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 429)] * weight_shared[((((int)threadIdx.x) & 31) + 3744)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 366)] * weight_shared[((((int)threadIdx.x) & 31) + 3264)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 367)] * weight_shared[((((int)threadIdx.x) & 31) + 3296)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 374)] * weight_shared[((((int)threadIdx.x) & 31) + 3520)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 375)] * weight_shared[((((int)threadIdx.x) & 31) + 3552)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 382)] * weight_shared[((((int)threadIdx.x) & 31) + 3776)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 383)] * weight_shared[((((int)threadIdx.x) & 31) + 3808)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 374)] * weight_shared[((((int)threadIdx.x) & 31) + 3264)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 375)] * weight_shared[((((int)threadIdx.x) & 31) + 3296)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 382)] * weight_shared[((((int)threadIdx.x) & 31) + 3520)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 383)] * weight_shared[((((int)threadIdx.x) & 31) + 3552)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 390)] * weight_shared[((((int)threadIdx.x) & 31) + 3776)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 391)] * weight_shared[((((int)threadIdx.x) & 31) + 3808)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 382)] * weight_shared[((((int)threadIdx.x) & 31) + 3264)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 383)] * weight_shared[((((int)threadIdx.x) & 31) + 3296)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 390)] * weight_shared[((((int)threadIdx.x) & 31) + 3520)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 391)] * weight_shared[((((int)threadIdx.x) & 31) + 3552)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 398)] * weight_shared[((((int)threadIdx.x) & 31) + 3776)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 399)] * weight_shared[((((int)threadIdx.x) & 31) + 3808)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 390)] * weight_shared[((((int)threadIdx.x) & 31) + 3264)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 391)] * weight_shared[((((int)threadIdx.x) & 31) + 3296)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 398)] * weight_shared[((((int)threadIdx.x) & 31) + 3520)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 399)] * weight_shared[((((int)threadIdx.x) & 31) + 3552)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 406)] * weight_shared[((((int)threadIdx.x) & 31) + 3776)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 407)] * weight_shared[((((int)threadIdx.x) & 31) + 3808)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 398)] * weight_shared[((((int)threadIdx.x) & 31) + 3264)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 399)] * weight_shared[((((int)threadIdx.x) & 31) + 3296)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 406)] * weight_shared[((((int)threadIdx.x) & 31) + 3520)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 407)] * weight_shared[((((int)threadIdx.x) & 31) + 3552)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 414)] * weight_shared[((((int)threadIdx.x) & 31) + 3776)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 415)] * weight_shared[((((int)threadIdx.x) & 31) + 3808)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 406)] * weight_shared[((((int)threadIdx.x) & 31) + 3264)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 407)] * weight_shared[((((int)threadIdx.x) & 31) + 3296)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 414)] * weight_shared[((((int)threadIdx.x) & 31) + 3520)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 415)] * weight_shared[((((int)threadIdx.x) & 31) + 3552)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 422)] * weight_shared[((((int)threadIdx.x) & 31) + 3776)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 423)] * weight_shared[((((int)threadIdx.x) & 31) + 3808)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 414)] * weight_shared[((((int)threadIdx.x) & 31) + 3264)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 415)] * weight_shared[((((int)threadIdx.x) & 31) + 3296)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 422)] * weight_shared[((((int)threadIdx.x) & 31) + 3520)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 423)] * weight_shared[((((int)threadIdx.x) & 31) + 3552)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 430)] * weight_shared[((((int)threadIdx.x) & 31) + 3776)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 431)] * weight_shared[((((int)threadIdx.x) & 31) + 3808)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 432)] * weight_shared[((((int)threadIdx.x) & 31) + 3840)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 433)] * weight_shared[((((int)threadIdx.x) & 31) + 3872)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 440)] * weight_shared[((((int)threadIdx.x) & 31) + 4096)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 441)] * weight_shared[((((int)threadIdx.x) & 31) + 4128)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 448)] * weight_shared[((((int)threadIdx.x) & 31) + 4352)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 449)] * weight_shared[((((int)threadIdx.x) & 31) + 4384)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 440)] * weight_shared[((((int)threadIdx.x) & 31) + 3840)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 441)] * weight_shared[((((int)threadIdx.x) & 31) + 3872)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 448)] * weight_shared[((((int)threadIdx.x) & 31) + 4096)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 449)] * weight_shared[((((int)threadIdx.x) & 31) + 4128)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 456)] * weight_shared[((((int)threadIdx.x) & 31) + 4352)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 457)] * weight_shared[((((int)threadIdx.x) & 31) + 4384)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 448)] * weight_shared[((((int)threadIdx.x) & 31) + 3840)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 449)] * weight_shared[((((int)threadIdx.x) & 31) + 3872)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 456)] * weight_shared[((((int)threadIdx.x) & 31) + 4096)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 457)] * weight_shared[((((int)threadIdx.x) & 31) + 4128)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 464)] * weight_shared[((((int)threadIdx.x) & 31) + 4352)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 465)] * weight_shared[((((int)threadIdx.x) & 31) + 4384)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 456)] * weight_shared[((((int)threadIdx.x) & 31) + 3840)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 457)] * weight_shared[((((int)threadIdx.x) & 31) + 3872)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 464)] * weight_shared[((((int)threadIdx.x) & 31) + 4096)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 465)] * weight_shared[((((int)threadIdx.x) & 31) + 4128)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 472)] * weight_shared[((((int)threadIdx.x) & 31) + 4352)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 473)] * weight_shared[((((int)threadIdx.x) & 31) + 4384)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 464)] * weight_shared[((((int)threadIdx.x) & 31) + 3840)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 465)] * weight_shared[((((int)threadIdx.x) & 31) + 3872)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 472)] * weight_shared[((((int)threadIdx.x) & 31) + 4096)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 473)] * weight_shared[((((int)threadIdx.x) & 31) + 4128)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 480)] * weight_shared[((((int)threadIdx.x) & 31) + 4352)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 481)] * weight_shared[((((int)threadIdx.x) & 31) + 4384)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 472)] * weight_shared[((((int)threadIdx.x) & 31) + 3840)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 473)] * weight_shared[((((int)threadIdx.x) & 31) + 3872)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 480)] * weight_shared[((((int)threadIdx.x) & 31) + 4096)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 481)] * weight_shared[((((int)threadIdx.x) & 31) + 4128)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 488)] * weight_shared[((((int)threadIdx.x) & 31) + 4352)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 489)] * weight_shared[((((int)threadIdx.x) & 31) + 4384)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 480)] * weight_shared[((((int)threadIdx.x) & 31) + 3840)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 481)] * weight_shared[((((int)threadIdx.x) & 31) + 3872)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 488)] * weight_shared[((((int)threadIdx.x) & 31) + 4096)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 489)] * weight_shared[((((int)threadIdx.x) & 31) + 4128)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 496)] * weight_shared[((((int)threadIdx.x) & 31) + 4352)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 497)] * weight_shared[((((int)threadIdx.x) & 31) + 4384)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 434)] * weight_shared[((((int)threadIdx.x) & 31) + 3904)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 435)] * weight_shared[((((int)threadIdx.x) & 31) + 3936)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 442)] * weight_shared[((((int)threadIdx.x) & 31) + 4160)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 443)] * weight_shared[((((int)threadIdx.x) & 31) + 4192)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 450)] * weight_shared[((((int)threadIdx.x) & 31) + 4416)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 451)] * weight_shared[((((int)threadIdx.x) & 31) + 4448)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 442)] * weight_shared[((((int)threadIdx.x) & 31) + 3904)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 443)] * weight_shared[((((int)threadIdx.x) & 31) + 3936)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 450)] * weight_shared[((((int)threadIdx.x) & 31) + 4160)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 451)] * weight_shared[((((int)threadIdx.x) & 31) + 4192)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 458)] * weight_shared[((((int)threadIdx.x) & 31) + 4416)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 459)] * weight_shared[((((int)threadIdx.x) & 31) + 4448)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 450)] * weight_shared[((((int)threadIdx.x) & 31) + 3904)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 451)] * weight_shared[((((int)threadIdx.x) & 31) + 3936)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 458)] * weight_shared[((((int)threadIdx.x) & 31) + 4160)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 459)] * weight_shared[((((int)threadIdx.x) & 31) + 4192)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 466)] * weight_shared[((((int)threadIdx.x) & 31) + 4416)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 467)] * weight_shared[((((int)threadIdx.x) & 31) + 4448)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 458)] * weight_shared[((((int)threadIdx.x) & 31) + 3904)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 459)] * weight_shared[((((int)threadIdx.x) & 31) + 3936)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 466)] * weight_shared[((((int)threadIdx.x) & 31) + 4160)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 467)] * weight_shared[((((int)threadIdx.x) & 31) + 4192)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 474)] * weight_shared[((((int)threadIdx.x) & 31) + 4416)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 475)] * weight_shared[((((int)threadIdx.x) & 31) + 4448)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 466)] * weight_shared[((((int)threadIdx.x) & 31) + 3904)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 467)] * weight_shared[((((int)threadIdx.x) & 31) + 3936)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 474)] * weight_shared[((((int)threadIdx.x) & 31) + 4160)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 475)] * weight_shared[((((int)threadIdx.x) & 31) + 4192)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 482)] * weight_shared[((((int)threadIdx.x) & 31) + 4416)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 483)] * weight_shared[((((int)threadIdx.x) & 31) + 4448)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 474)] * weight_shared[((((int)threadIdx.x) & 31) + 3904)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 475)] * weight_shared[((((int)threadIdx.x) & 31) + 3936)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 482)] * weight_shared[((((int)threadIdx.x) & 31) + 4160)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 483)] * weight_shared[((((int)threadIdx.x) & 31) + 4192)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 490)] * weight_shared[((((int)threadIdx.x) & 31) + 4416)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 491)] * weight_shared[((((int)threadIdx.x) & 31) + 4448)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 482)] * weight_shared[((((int)threadIdx.x) & 31) + 3904)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 483)] * weight_shared[((((int)threadIdx.x) & 31) + 3936)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 490)] * weight_shared[((((int)threadIdx.x) & 31) + 4160)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 491)] * weight_shared[((((int)threadIdx.x) & 31) + 4192)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 498)] * weight_shared[((((int)threadIdx.x) & 31) + 4416)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 499)] * weight_shared[((((int)threadIdx.x) & 31) + 4448)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 436)] * weight_shared[((((int)threadIdx.x) & 31) + 3968)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 437)] * weight_shared[((((int)threadIdx.x) & 31) + 4000)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 444)] * weight_shared[((((int)threadIdx.x) & 31) + 4224)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 445)] * weight_shared[((((int)threadIdx.x) & 31) + 4256)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 452)] * weight_shared[((((int)threadIdx.x) & 31) + 4480)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 453)] * weight_shared[((((int)threadIdx.x) & 31) + 4512)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 444)] * weight_shared[((((int)threadIdx.x) & 31) + 3968)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 445)] * weight_shared[((((int)threadIdx.x) & 31) + 4000)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 452)] * weight_shared[((((int)threadIdx.x) & 31) + 4224)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 453)] * weight_shared[((((int)threadIdx.x) & 31) + 4256)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 460)] * weight_shared[((((int)threadIdx.x) & 31) + 4480)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 461)] * weight_shared[((((int)threadIdx.x) & 31) + 4512)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 452)] * weight_shared[((((int)threadIdx.x) & 31) + 3968)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 453)] * weight_shared[((((int)threadIdx.x) & 31) + 4000)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 460)] * weight_shared[((((int)threadIdx.x) & 31) + 4224)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 461)] * weight_shared[((((int)threadIdx.x) & 31) + 4256)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 468)] * weight_shared[((((int)threadIdx.x) & 31) + 4480)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 469)] * weight_shared[((((int)threadIdx.x) & 31) + 4512)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 460)] * weight_shared[((((int)threadIdx.x) & 31) + 3968)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 461)] * weight_shared[((((int)threadIdx.x) & 31) + 4000)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 468)] * weight_shared[((((int)threadIdx.x) & 31) + 4224)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 469)] * weight_shared[((((int)threadIdx.x) & 31) + 4256)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 476)] * weight_shared[((((int)threadIdx.x) & 31) + 4480)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 477)] * weight_shared[((((int)threadIdx.x) & 31) + 4512)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 468)] * weight_shared[((((int)threadIdx.x) & 31) + 3968)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 469)] * weight_shared[((((int)threadIdx.x) & 31) + 4000)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 476)] * weight_shared[((((int)threadIdx.x) & 31) + 4224)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 477)] * weight_shared[((((int)threadIdx.x) & 31) + 4256)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 484)] * weight_shared[((((int)threadIdx.x) & 31) + 4480)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 485)] * weight_shared[((((int)threadIdx.x) & 31) + 4512)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 476)] * weight_shared[((((int)threadIdx.x) & 31) + 3968)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 477)] * weight_shared[((((int)threadIdx.x) & 31) + 4000)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 484)] * weight_shared[((((int)threadIdx.x) & 31) + 4224)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 485)] * weight_shared[((((int)threadIdx.x) & 31) + 4256)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 492)] * weight_shared[((((int)threadIdx.x) & 31) + 4480)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 493)] * weight_shared[((((int)threadIdx.x) & 31) + 4512)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 484)] * weight_shared[((((int)threadIdx.x) & 31) + 3968)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 485)] * weight_shared[((((int)threadIdx.x) & 31) + 4000)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 492)] * weight_shared[((((int)threadIdx.x) & 31) + 4224)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 493)] * weight_shared[((((int)threadIdx.x) & 31) + 4256)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 500)] * weight_shared[((((int)threadIdx.x) & 31) + 4480)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 501)] * weight_shared[((((int)threadIdx.x) & 31) + 4512)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 438)] * weight_shared[((((int)threadIdx.x) & 31) + 4032)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 439)] * weight_shared[((((int)threadIdx.x) & 31) + 4064)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 446)] * weight_shared[((((int)threadIdx.x) & 31) + 4288)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 447)] * weight_shared[((((int)threadIdx.x) & 31) + 4320)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 454)] * weight_shared[((((int)threadIdx.x) & 31) + 4544)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 455)] * weight_shared[((((int)threadIdx.x) & 31) + 4576)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 446)] * weight_shared[((((int)threadIdx.x) & 31) + 4032)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 447)] * weight_shared[((((int)threadIdx.x) & 31) + 4064)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 454)] * weight_shared[((((int)threadIdx.x) & 31) + 4288)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 455)] * weight_shared[((((int)threadIdx.x) & 31) + 4320)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 462)] * weight_shared[((((int)threadIdx.x) & 31) + 4544)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 463)] * weight_shared[((((int)threadIdx.x) & 31) + 4576)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 454)] * weight_shared[((((int)threadIdx.x) & 31) + 4032)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 455)] * weight_shared[((((int)threadIdx.x) & 31) + 4064)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 462)] * weight_shared[((((int)threadIdx.x) & 31) + 4288)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 463)] * weight_shared[((((int)threadIdx.x) & 31) + 4320)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 470)] * weight_shared[((((int)threadIdx.x) & 31) + 4544)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 471)] * weight_shared[((((int)threadIdx.x) & 31) + 4576)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 462)] * weight_shared[((((int)threadIdx.x) & 31) + 4032)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 463)] * weight_shared[((((int)threadIdx.x) & 31) + 4064)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 470)] * weight_shared[((((int)threadIdx.x) & 31) + 4288)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 471)] * weight_shared[((((int)threadIdx.x) & 31) + 4320)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 478)] * weight_shared[((((int)threadIdx.x) & 31) + 4544)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 479)] * weight_shared[((((int)threadIdx.x) & 31) + 4576)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 470)] * weight_shared[((((int)threadIdx.x) & 31) + 4032)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 471)] * weight_shared[((((int)threadIdx.x) & 31) + 4064)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 478)] * weight_shared[((((int)threadIdx.x) & 31) + 4288)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 479)] * weight_shared[((((int)threadIdx.x) & 31) + 4320)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 486)] * weight_shared[((((int)threadIdx.x) & 31) + 4544)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 487)] * weight_shared[((((int)threadIdx.x) & 31) + 4576)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 478)] * weight_shared[((((int)threadIdx.x) & 31) + 4032)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 479)] * weight_shared[((((int)threadIdx.x) & 31) + 4064)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 486)] * weight_shared[((((int)threadIdx.x) & 31) + 4288)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 487)] * weight_shared[((((int)threadIdx.x) & 31) + 4320)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 494)] * weight_shared[((((int)threadIdx.x) & 31) + 4544)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 495)] * weight_shared[((((int)threadIdx.x) & 31) + 4576)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 486)] * weight_shared[((((int)threadIdx.x) & 31) + 4032)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 487)] * weight_shared[((((int)threadIdx.x) & 31) + 4064)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 494)] * weight_shared[((((int)threadIdx.x) & 31) + 4288)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 495)] * weight_shared[((((int)threadIdx.x) & 31) + 4320)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 502)] * weight_shared[((((int)threadIdx.x) & 31) + 4544)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 503)] * weight_shared[((((int)threadIdx.x) & 31) + 4576)]));
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 576)] * weight_shared[((((int)threadIdx.x) & 31) + 4608)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 577)] * weight_shared[((((int)threadIdx.x) & 31) + 4640)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 584)] * weight_shared[((((int)threadIdx.x) & 31) + 4864)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 585)] * weight_shared[((((int)threadIdx.x) & 31) + 4896)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 592)] * weight_shared[((((int)threadIdx.x) & 31) + 5120)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 593)] * weight_shared[((((int)threadIdx.x) & 31) + 5152)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 584)] * weight_shared[((((int)threadIdx.x) & 31) + 4608)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 585)] * weight_shared[((((int)threadIdx.x) & 31) + 4640)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 592)] * weight_shared[((((int)threadIdx.x) & 31) + 4864)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 593)] * weight_shared[((((int)threadIdx.x) & 31) + 4896)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 600)] * weight_shared[((((int)threadIdx.x) & 31) + 5120)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 601)] * weight_shared[((((int)threadIdx.x) & 31) + 5152)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 592)] * weight_shared[((((int)threadIdx.x) & 31) + 4608)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 593)] * weight_shared[((((int)threadIdx.x) & 31) + 4640)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 600)] * weight_shared[((((int)threadIdx.x) & 31) + 4864)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 601)] * weight_shared[((((int)threadIdx.x) & 31) + 4896)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 608)] * weight_shared[((((int)threadIdx.x) & 31) + 5120)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 609)] * weight_shared[((((int)threadIdx.x) & 31) + 5152)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 600)] * weight_shared[((((int)threadIdx.x) & 31) + 4608)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 601)] * weight_shared[((((int)threadIdx.x) & 31) + 4640)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 608)] * weight_shared[((((int)threadIdx.x) & 31) + 4864)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 609)] * weight_shared[((((int)threadIdx.x) & 31) + 4896)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 616)] * weight_shared[((((int)threadIdx.x) & 31) + 5120)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 617)] * weight_shared[((((int)threadIdx.x) & 31) + 5152)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 608)] * weight_shared[((((int)threadIdx.x) & 31) + 4608)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 609)] * weight_shared[((((int)threadIdx.x) & 31) + 4640)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 616)] * weight_shared[((((int)threadIdx.x) & 31) + 4864)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 617)] * weight_shared[((((int)threadIdx.x) & 31) + 4896)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 624)] * weight_shared[((((int)threadIdx.x) & 31) + 5120)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 625)] * weight_shared[((((int)threadIdx.x) & 31) + 5152)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 616)] * weight_shared[((((int)threadIdx.x) & 31) + 4608)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 617)] * weight_shared[((((int)threadIdx.x) & 31) + 4640)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 624)] * weight_shared[((((int)threadIdx.x) & 31) + 4864)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 625)] * weight_shared[((((int)threadIdx.x) & 31) + 4896)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 632)] * weight_shared[((((int)threadIdx.x) & 31) + 5120)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 633)] * weight_shared[((((int)threadIdx.x) & 31) + 5152)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 624)] * weight_shared[((((int)threadIdx.x) & 31) + 4608)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 625)] * weight_shared[((((int)threadIdx.x) & 31) + 4640)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 632)] * weight_shared[((((int)threadIdx.x) & 31) + 4864)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 633)] * weight_shared[((((int)threadIdx.x) & 31) + 4896)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 640)] * weight_shared[((((int)threadIdx.x) & 31) + 5120)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 641)] * weight_shared[((((int)threadIdx.x) & 31) + 5152)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 578)] * weight_shared[((((int)threadIdx.x) & 31) + 4672)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 579)] * weight_shared[((((int)threadIdx.x) & 31) + 4704)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 586)] * weight_shared[((((int)threadIdx.x) & 31) + 4928)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 587)] * weight_shared[((((int)threadIdx.x) & 31) + 4960)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 594)] * weight_shared[((((int)threadIdx.x) & 31) + 5184)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 595)] * weight_shared[((((int)threadIdx.x) & 31) + 5216)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 586)] * weight_shared[((((int)threadIdx.x) & 31) + 4672)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 587)] * weight_shared[((((int)threadIdx.x) & 31) + 4704)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 594)] * weight_shared[((((int)threadIdx.x) & 31) + 4928)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 595)] * weight_shared[((((int)threadIdx.x) & 31) + 4960)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 602)] * weight_shared[((((int)threadIdx.x) & 31) + 5184)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 603)] * weight_shared[((((int)threadIdx.x) & 31) + 5216)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 594)] * weight_shared[((((int)threadIdx.x) & 31) + 4672)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 595)] * weight_shared[((((int)threadIdx.x) & 31) + 4704)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 602)] * weight_shared[((((int)threadIdx.x) & 31) + 4928)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 603)] * weight_shared[((((int)threadIdx.x) & 31) + 4960)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 610)] * weight_shared[((((int)threadIdx.x) & 31) + 5184)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 611)] * weight_shared[((((int)threadIdx.x) & 31) + 5216)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 602)] * weight_shared[((((int)threadIdx.x) & 31) + 4672)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 603)] * weight_shared[((((int)threadIdx.x) & 31) + 4704)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 610)] * weight_shared[((((int)threadIdx.x) & 31) + 4928)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 611)] * weight_shared[((((int)threadIdx.x) & 31) + 4960)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 618)] * weight_shared[((((int)threadIdx.x) & 31) + 5184)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 619)] * weight_shared[((((int)threadIdx.x) & 31) + 5216)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 610)] * weight_shared[((((int)threadIdx.x) & 31) + 4672)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 611)] * weight_shared[((((int)threadIdx.x) & 31) + 4704)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 618)] * weight_shared[((((int)threadIdx.x) & 31) + 4928)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 619)] * weight_shared[((((int)threadIdx.x) & 31) + 4960)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 626)] * weight_shared[((((int)threadIdx.x) & 31) + 5184)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 627)] * weight_shared[((((int)threadIdx.x) & 31) + 5216)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 618)] * weight_shared[((((int)threadIdx.x) & 31) + 4672)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 619)] * weight_shared[((((int)threadIdx.x) & 31) + 4704)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 626)] * weight_shared[((((int)threadIdx.x) & 31) + 4928)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 627)] * weight_shared[((((int)threadIdx.x) & 31) + 4960)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 634)] * weight_shared[((((int)threadIdx.x) & 31) + 5184)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 635)] * weight_shared[((((int)threadIdx.x) & 31) + 5216)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 626)] * weight_shared[((((int)threadIdx.x) & 31) + 4672)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 627)] * weight_shared[((((int)threadIdx.x) & 31) + 4704)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 634)] * weight_shared[((((int)threadIdx.x) & 31) + 4928)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 635)] * weight_shared[((((int)threadIdx.x) & 31) + 4960)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 642)] * weight_shared[((((int)threadIdx.x) & 31) + 5184)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 643)] * weight_shared[((((int)threadIdx.x) & 31) + 5216)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 580)] * weight_shared[((((int)threadIdx.x) & 31) + 4736)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 581)] * weight_shared[((((int)threadIdx.x) & 31) + 4768)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 588)] * weight_shared[((((int)threadIdx.x) & 31) + 4992)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 589)] * weight_shared[((((int)threadIdx.x) & 31) + 5024)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 596)] * weight_shared[((((int)threadIdx.x) & 31) + 5248)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 597)] * weight_shared[((((int)threadIdx.x) & 31) + 5280)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 588)] * weight_shared[((((int)threadIdx.x) & 31) + 4736)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 589)] * weight_shared[((((int)threadIdx.x) & 31) + 4768)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 596)] * weight_shared[((((int)threadIdx.x) & 31) + 4992)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 597)] * weight_shared[((((int)threadIdx.x) & 31) + 5024)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 604)] * weight_shared[((((int)threadIdx.x) & 31) + 5248)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 605)] * weight_shared[((((int)threadIdx.x) & 31) + 5280)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 596)] * weight_shared[((((int)threadIdx.x) & 31) + 4736)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 597)] * weight_shared[((((int)threadIdx.x) & 31) + 4768)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 604)] * weight_shared[((((int)threadIdx.x) & 31) + 4992)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 605)] * weight_shared[((((int)threadIdx.x) & 31) + 5024)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 612)] * weight_shared[((((int)threadIdx.x) & 31) + 5248)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 613)] * weight_shared[((((int)threadIdx.x) & 31) + 5280)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 604)] * weight_shared[((((int)threadIdx.x) & 31) + 4736)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 605)] * weight_shared[((((int)threadIdx.x) & 31) + 4768)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 612)] * weight_shared[((((int)threadIdx.x) & 31) + 4992)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 613)] * weight_shared[((((int)threadIdx.x) & 31) + 5024)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 620)] * weight_shared[((((int)threadIdx.x) & 31) + 5248)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 621)] * weight_shared[((((int)threadIdx.x) & 31) + 5280)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 612)] * weight_shared[((((int)threadIdx.x) & 31) + 4736)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 613)] * weight_shared[((((int)threadIdx.x) & 31) + 4768)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 620)] * weight_shared[((((int)threadIdx.x) & 31) + 4992)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 621)] * weight_shared[((((int)threadIdx.x) & 31) + 5024)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 628)] * weight_shared[((((int)threadIdx.x) & 31) + 5248)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 629)] * weight_shared[((((int)threadIdx.x) & 31) + 5280)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 620)] * weight_shared[((((int)threadIdx.x) & 31) + 4736)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 621)] * weight_shared[((((int)threadIdx.x) & 31) + 4768)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 628)] * weight_shared[((((int)threadIdx.x) & 31) + 4992)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 629)] * weight_shared[((((int)threadIdx.x) & 31) + 5024)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 636)] * weight_shared[((((int)threadIdx.x) & 31) + 5248)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 637)] * weight_shared[((((int)threadIdx.x) & 31) + 5280)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 628)] * weight_shared[((((int)threadIdx.x) & 31) + 4736)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 629)] * weight_shared[((((int)threadIdx.x) & 31) + 4768)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 636)] * weight_shared[((((int)threadIdx.x) & 31) + 4992)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 637)] * weight_shared[((((int)threadIdx.x) & 31) + 5024)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 644)] * weight_shared[((((int)threadIdx.x) & 31) + 5248)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 645)] * weight_shared[((((int)threadIdx.x) & 31) + 5280)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 582)] * weight_shared[((((int)threadIdx.x) & 31) + 4800)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 583)] * weight_shared[((((int)threadIdx.x) & 31) + 4832)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 590)] * weight_shared[((((int)threadIdx.x) & 31) + 5056)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 591)] * weight_shared[((((int)threadIdx.x) & 31) + 5088)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 598)] * weight_shared[((((int)threadIdx.x) & 31) + 5312)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 599)] * weight_shared[((((int)threadIdx.x) & 31) + 5344)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 590)] * weight_shared[((((int)threadIdx.x) & 31) + 4800)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 591)] * weight_shared[((((int)threadIdx.x) & 31) + 4832)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 598)] * weight_shared[((((int)threadIdx.x) & 31) + 5056)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 599)] * weight_shared[((((int)threadIdx.x) & 31) + 5088)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 606)] * weight_shared[((((int)threadIdx.x) & 31) + 5312)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 607)] * weight_shared[((((int)threadIdx.x) & 31) + 5344)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 598)] * weight_shared[((((int)threadIdx.x) & 31) + 4800)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 599)] * weight_shared[((((int)threadIdx.x) & 31) + 4832)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 606)] * weight_shared[((((int)threadIdx.x) & 31) + 5056)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 607)] * weight_shared[((((int)threadIdx.x) & 31) + 5088)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 614)] * weight_shared[((((int)threadIdx.x) & 31) + 5312)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 615)] * weight_shared[((((int)threadIdx.x) & 31) + 5344)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 606)] * weight_shared[((((int)threadIdx.x) & 31) + 4800)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 607)] * weight_shared[((((int)threadIdx.x) & 31) + 4832)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 614)] * weight_shared[((((int)threadIdx.x) & 31) + 5056)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 615)] * weight_shared[((((int)threadIdx.x) & 31) + 5088)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 622)] * weight_shared[((((int)threadIdx.x) & 31) + 5312)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 623)] * weight_shared[((((int)threadIdx.x) & 31) + 5344)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 614)] * weight_shared[((((int)threadIdx.x) & 31) + 4800)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 615)] * weight_shared[((((int)threadIdx.x) & 31) + 4832)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 622)] * weight_shared[((((int)threadIdx.x) & 31) + 5056)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 623)] * weight_shared[((((int)threadIdx.x) & 31) + 5088)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 630)] * weight_shared[((((int)threadIdx.x) & 31) + 5312)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 631)] * weight_shared[((((int)threadIdx.x) & 31) + 5344)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 622)] * weight_shared[((((int)threadIdx.x) & 31) + 4800)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 623)] * weight_shared[((((int)threadIdx.x) & 31) + 4832)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 630)] * weight_shared[((((int)threadIdx.x) & 31) + 5056)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 631)] * weight_shared[((((int)threadIdx.x) & 31) + 5088)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 638)] * weight_shared[((((int)threadIdx.x) & 31) + 5312)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 639)] * weight_shared[((((int)threadIdx.x) & 31) + 5344)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 630)] * weight_shared[((((int)threadIdx.x) & 31) + 4800)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 631)] * weight_shared[((((int)threadIdx.x) & 31) + 4832)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 638)] * weight_shared[((((int)threadIdx.x) & 31) + 5056)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 639)] * weight_shared[((((int)threadIdx.x) & 31) + 5088)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 646)] * weight_shared[((((int)threadIdx.x) & 31) + 5312)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 647)] * weight_shared[((((int)threadIdx.x) & 31) + 5344)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 648)] * weight_shared[((((int)threadIdx.x) & 31) + 5376)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 649)] * weight_shared[((((int)threadIdx.x) & 31) + 5408)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 656)] * weight_shared[((((int)threadIdx.x) & 31) + 5632)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 657)] * weight_shared[((((int)threadIdx.x) & 31) + 5664)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 664)] * weight_shared[((((int)threadIdx.x) & 31) + 5888)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 665)] * weight_shared[((((int)threadIdx.x) & 31) + 5920)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 656)] * weight_shared[((((int)threadIdx.x) & 31) + 5376)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 657)] * weight_shared[((((int)threadIdx.x) & 31) + 5408)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 664)] * weight_shared[((((int)threadIdx.x) & 31) + 5632)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 665)] * weight_shared[((((int)threadIdx.x) & 31) + 5664)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 672)] * weight_shared[((((int)threadIdx.x) & 31) + 5888)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 673)] * weight_shared[((((int)threadIdx.x) & 31) + 5920)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 664)] * weight_shared[((((int)threadIdx.x) & 31) + 5376)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 665)] * weight_shared[((((int)threadIdx.x) & 31) + 5408)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 672)] * weight_shared[((((int)threadIdx.x) & 31) + 5632)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 673)] * weight_shared[((((int)threadIdx.x) & 31) + 5664)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 680)] * weight_shared[((((int)threadIdx.x) & 31) + 5888)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 681)] * weight_shared[((((int)threadIdx.x) & 31) + 5920)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 672)] * weight_shared[((((int)threadIdx.x) & 31) + 5376)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 673)] * weight_shared[((((int)threadIdx.x) & 31) + 5408)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 680)] * weight_shared[((((int)threadIdx.x) & 31) + 5632)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 681)] * weight_shared[((((int)threadIdx.x) & 31) + 5664)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 688)] * weight_shared[((((int)threadIdx.x) & 31) + 5888)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 689)] * weight_shared[((((int)threadIdx.x) & 31) + 5920)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 680)] * weight_shared[((((int)threadIdx.x) & 31) + 5376)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 681)] * weight_shared[((((int)threadIdx.x) & 31) + 5408)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 688)] * weight_shared[((((int)threadIdx.x) & 31) + 5632)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 689)] * weight_shared[((((int)threadIdx.x) & 31) + 5664)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 696)] * weight_shared[((((int)threadIdx.x) & 31) + 5888)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 697)] * weight_shared[((((int)threadIdx.x) & 31) + 5920)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 688)] * weight_shared[((((int)threadIdx.x) & 31) + 5376)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 689)] * weight_shared[((((int)threadIdx.x) & 31) + 5408)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 696)] * weight_shared[((((int)threadIdx.x) & 31) + 5632)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 697)] * weight_shared[((((int)threadIdx.x) & 31) + 5664)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 704)] * weight_shared[((((int)threadIdx.x) & 31) + 5888)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 705)] * weight_shared[((((int)threadIdx.x) & 31) + 5920)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 696)] * weight_shared[((((int)threadIdx.x) & 31) + 5376)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 697)] * weight_shared[((((int)threadIdx.x) & 31) + 5408)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 704)] * weight_shared[((((int)threadIdx.x) & 31) + 5632)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 705)] * weight_shared[((((int)threadIdx.x) & 31) + 5664)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 712)] * weight_shared[((((int)threadIdx.x) & 31) + 5888)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 713)] * weight_shared[((((int)threadIdx.x) & 31) + 5920)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 650)] * weight_shared[((((int)threadIdx.x) & 31) + 5440)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 651)] * weight_shared[((((int)threadIdx.x) & 31) + 5472)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 658)] * weight_shared[((((int)threadIdx.x) & 31) + 5696)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 659)] * weight_shared[((((int)threadIdx.x) & 31) + 5728)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 666)] * weight_shared[((((int)threadIdx.x) & 31) + 5952)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 667)] * weight_shared[((((int)threadIdx.x) & 31) + 5984)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 658)] * weight_shared[((((int)threadIdx.x) & 31) + 5440)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 659)] * weight_shared[((((int)threadIdx.x) & 31) + 5472)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 666)] * weight_shared[((((int)threadIdx.x) & 31) + 5696)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 667)] * weight_shared[((((int)threadIdx.x) & 31) + 5728)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 674)] * weight_shared[((((int)threadIdx.x) & 31) + 5952)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 675)] * weight_shared[((((int)threadIdx.x) & 31) + 5984)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 666)] * weight_shared[((((int)threadIdx.x) & 31) + 5440)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 667)] * weight_shared[((((int)threadIdx.x) & 31) + 5472)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 674)] * weight_shared[((((int)threadIdx.x) & 31) + 5696)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 675)] * weight_shared[((((int)threadIdx.x) & 31) + 5728)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 682)] * weight_shared[((((int)threadIdx.x) & 31) + 5952)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 683)] * weight_shared[((((int)threadIdx.x) & 31) + 5984)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 674)] * weight_shared[((((int)threadIdx.x) & 31) + 5440)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 675)] * weight_shared[((((int)threadIdx.x) & 31) + 5472)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 682)] * weight_shared[((((int)threadIdx.x) & 31) + 5696)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 683)] * weight_shared[((((int)threadIdx.x) & 31) + 5728)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 690)] * weight_shared[((((int)threadIdx.x) & 31) + 5952)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 691)] * weight_shared[((((int)threadIdx.x) & 31) + 5984)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 682)] * weight_shared[((((int)threadIdx.x) & 31) + 5440)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 683)] * weight_shared[((((int)threadIdx.x) & 31) + 5472)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 690)] * weight_shared[((((int)threadIdx.x) & 31) + 5696)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 691)] * weight_shared[((((int)threadIdx.x) & 31) + 5728)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 698)] * weight_shared[((((int)threadIdx.x) & 31) + 5952)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 699)] * weight_shared[((((int)threadIdx.x) & 31) + 5984)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 690)] * weight_shared[((((int)threadIdx.x) & 31) + 5440)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 691)] * weight_shared[((((int)threadIdx.x) & 31) + 5472)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 698)] * weight_shared[((((int)threadIdx.x) & 31) + 5696)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 699)] * weight_shared[((((int)threadIdx.x) & 31) + 5728)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 706)] * weight_shared[((((int)threadIdx.x) & 31) + 5952)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 707)] * weight_shared[((((int)threadIdx.x) & 31) + 5984)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 698)] * weight_shared[((((int)threadIdx.x) & 31) + 5440)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 699)] * weight_shared[((((int)threadIdx.x) & 31) + 5472)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 706)] * weight_shared[((((int)threadIdx.x) & 31) + 5696)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 707)] * weight_shared[((((int)threadIdx.x) & 31) + 5728)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 714)] * weight_shared[((((int)threadIdx.x) & 31) + 5952)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 715)] * weight_shared[((((int)threadIdx.x) & 31) + 5984)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 652)] * weight_shared[((((int)threadIdx.x) & 31) + 5504)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 653)] * weight_shared[((((int)threadIdx.x) & 31) + 5536)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 660)] * weight_shared[((((int)threadIdx.x) & 31) + 5760)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 661)] * weight_shared[((((int)threadIdx.x) & 31) + 5792)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 668)] * weight_shared[((((int)threadIdx.x) & 31) + 6016)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 669)] * weight_shared[((((int)threadIdx.x) & 31) + 6048)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 660)] * weight_shared[((((int)threadIdx.x) & 31) + 5504)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 661)] * weight_shared[((((int)threadIdx.x) & 31) + 5536)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 668)] * weight_shared[((((int)threadIdx.x) & 31) + 5760)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 669)] * weight_shared[((((int)threadIdx.x) & 31) + 5792)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 676)] * weight_shared[((((int)threadIdx.x) & 31) + 6016)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 677)] * weight_shared[((((int)threadIdx.x) & 31) + 6048)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 668)] * weight_shared[((((int)threadIdx.x) & 31) + 5504)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 669)] * weight_shared[((((int)threadIdx.x) & 31) + 5536)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 676)] * weight_shared[((((int)threadIdx.x) & 31) + 5760)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 677)] * weight_shared[((((int)threadIdx.x) & 31) + 5792)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 684)] * weight_shared[((((int)threadIdx.x) & 31) + 6016)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 685)] * weight_shared[((((int)threadIdx.x) & 31) + 6048)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 676)] * weight_shared[((((int)threadIdx.x) & 31) + 5504)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 677)] * weight_shared[((((int)threadIdx.x) & 31) + 5536)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 684)] * weight_shared[((((int)threadIdx.x) & 31) + 5760)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 685)] * weight_shared[((((int)threadIdx.x) & 31) + 5792)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 692)] * weight_shared[((((int)threadIdx.x) & 31) + 6016)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 693)] * weight_shared[((((int)threadIdx.x) & 31) + 6048)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 684)] * weight_shared[((((int)threadIdx.x) & 31) + 5504)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 685)] * weight_shared[((((int)threadIdx.x) & 31) + 5536)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 692)] * weight_shared[((((int)threadIdx.x) & 31) + 5760)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 693)] * weight_shared[((((int)threadIdx.x) & 31) + 5792)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 700)] * weight_shared[((((int)threadIdx.x) & 31) + 6016)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 701)] * weight_shared[((((int)threadIdx.x) & 31) + 6048)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 692)] * weight_shared[((((int)threadIdx.x) & 31) + 5504)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 693)] * weight_shared[((((int)threadIdx.x) & 31) + 5536)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 700)] * weight_shared[((((int)threadIdx.x) & 31) + 5760)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 701)] * weight_shared[((((int)threadIdx.x) & 31) + 5792)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 708)] * weight_shared[((((int)threadIdx.x) & 31) + 6016)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 709)] * weight_shared[((((int)threadIdx.x) & 31) + 6048)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 700)] * weight_shared[((((int)threadIdx.x) & 31) + 5504)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 701)] * weight_shared[((((int)threadIdx.x) & 31) + 5536)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 708)] * weight_shared[((((int)threadIdx.x) & 31) + 5760)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 709)] * weight_shared[((((int)threadIdx.x) & 31) + 5792)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 716)] * weight_shared[((((int)threadIdx.x) & 31) + 6016)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 717)] * weight_shared[((((int)threadIdx.x) & 31) + 6048)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 654)] * weight_shared[((((int)threadIdx.x) & 31) + 5568)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 655)] * weight_shared[((((int)threadIdx.x) & 31) + 5600)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 662)] * weight_shared[((((int)threadIdx.x) & 31) + 5824)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 663)] * weight_shared[((((int)threadIdx.x) & 31) + 5856)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 670)] * weight_shared[((((int)threadIdx.x) & 31) + 6080)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 671)] * weight_shared[((((int)threadIdx.x) & 31) + 6112)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 662)] * weight_shared[((((int)threadIdx.x) & 31) + 5568)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 663)] * weight_shared[((((int)threadIdx.x) & 31) + 5600)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 670)] * weight_shared[((((int)threadIdx.x) & 31) + 5824)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 671)] * weight_shared[((((int)threadIdx.x) & 31) + 5856)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 678)] * weight_shared[((((int)threadIdx.x) & 31) + 6080)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 679)] * weight_shared[((((int)threadIdx.x) & 31) + 6112)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 670)] * weight_shared[((((int)threadIdx.x) & 31) + 5568)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 671)] * weight_shared[((((int)threadIdx.x) & 31) + 5600)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 678)] * weight_shared[((((int)threadIdx.x) & 31) + 5824)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 679)] * weight_shared[((((int)threadIdx.x) & 31) + 5856)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 686)] * weight_shared[((((int)threadIdx.x) & 31) + 6080)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 687)] * weight_shared[((((int)threadIdx.x) & 31) + 6112)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 678)] * weight_shared[((((int)threadIdx.x) & 31) + 5568)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 679)] * weight_shared[((((int)threadIdx.x) & 31) + 5600)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 686)] * weight_shared[((((int)threadIdx.x) & 31) + 5824)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 687)] * weight_shared[((((int)threadIdx.x) & 31) + 5856)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 694)] * weight_shared[((((int)threadIdx.x) & 31) + 6080)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 695)] * weight_shared[((((int)threadIdx.x) & 31) + 6112)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 686)] * weight_shared[((((int)threadIdx.x) & 31) + 5568)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 687)] * weight_shared[((((int)threadIdx.x) & 31) + 5600)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 694)] * weight_shared[((((int)threadIdx.x) & 31) + 5824)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 695)] * weight_shared[((((int)threadIdx.x) & 31) + 5856)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 702)] * weight_shared[((((int)threadIdx.x) & 31) + 6080)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 703)] * weight_shared[((((int)threadIdx.x) & 31) + 6112)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 694)] * weight_shared[((((int)threadIdx.x) & 31) + 5568)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 695)] * weight_shared[((((int)threadIdx.x) & 31) + 5600)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 702)] * weight_shared[((((int)threadIdx.x) & 31) + 5824)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 703)] * weight_shared[((((int)threadIdx.x) & 31) + 5856)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 710)] * weight_shared[((((int)threadIdx.x) & 31) + 6080)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 711)] * weight_shared[((((int)threadIdx.x) & 31) + 6112)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 702)] * weight_shared[((((int)threadIdx.x) & 31) + 5568)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 703)] * weight_shared[((((int)threadIdx.x) & 31) + 5600)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 710)] * weight_shared[((((int)threadIdx.x) & 31) + 5824)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 711)] * weight_shared[((((int)threadIdx.x) & 31) + 5856)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 718)] * weight_shared[((((int)threadIdx.x) & 31) + 6080)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 719)] * weight_shared[((((int)threadIdx.x) & 31) + 6112)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 720)] * weight_shared[((((int)threadIdx.x) & 31) + 6144)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 721)] * weight_shared[((((int)threadIdx.x) & 31) + 6176)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 728)] * weight_shared[((((int)threadIdx.x) & 31) + 6400)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 729)] * weight_shared[((((int)threadIdx.x) & 31) + 6432)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 736)] * weight_shared[((((int)threadIdx.x) & 31) + 6656)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 737)] * weight_shared[((((int)threadIdx.x) & 31) + 6688)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 728)] * weight_shared[((((int)threadIdx.x) & 31) + 6144)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 729)] * weight_shared[((((int)threadIdx.x) & 31) + 6176)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 736)] * weight_shared[((((int)threadIdx.x) & 31) + 6400)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 737)] * weight_shared[((((int)threadIdx.x) & 31) + 6432)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 744)] * weight_shared[((((int)threadIdx.x) & 31) + 6656)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 745)] * weight_shared[((((int)threadIdx.x) & 31) + 6688)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 736)] * weight_shared[((((int)threadIdx.x) & 31) + 6144)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 737)] * weight_shared[((((int)threadIdx.x) & 31) + 6176)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 744)] * weight_shared[((((int)threadIdx.x) & 31) + 6400)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 745)] * weight_shared[((((int)threadIdx.x) & 31) + 6432)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 752)] * weight_shared[((((int)threadIdx.x) & 31) + 6656)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 753)] * weight_shared[((((int)threadIdx.x) & 31) + 6688)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 744)] * weight_shared[((((int)threadIdx.x) & 31) + 6144)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 745)] * weight_shared[((((int)threadIdx.x) & 31) + 6176)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 752)] * weight_shared[((((int)threadIdx.x) & 31) + 6400)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 753)] * weight_shared[((((int)threadIdx.x) & 31) + 6432)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 760)] * weight_shared[((((int)threadIdx.x) & 31) + 6656)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 761)] * weight_shared[((((int)threadIdx.x) & 31) + 6688)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 752)] * weight_shared[((((int)threadIdx.x) & 31) + 6144)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 753)] * weight_shared[((((int)threadIdx.x) & 31) + 6176)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 760)] * weight_shared[((((int)threadIdx.x) & 31) + 6400)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 761)] * weight_shared[((((int)threadIdx.x) & 31) + 6432)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 768)] * weight_shared[((((int)threadIdx.x) & 31) + 6656)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 769)] * weight_shared[((((int)threadIdx.x) & 31) + 6688)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 760)] * weight_shared[((((int)threadIdx.x) & 31) + 6144)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 761)] * weight_shared[((((int)threadIdx.x) & 31) + 6176)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 768)] * weight_shared[((((int)threadIdx.x) & 31) + 6400)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 769)] * weight_shared[((((int)threadIdx.x) & 31) + 6432)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 776)] * weight_shared[((((int)threadIdx.x) & 31) + 6656)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 777)] * weight_shared[((((int)threadIdx.x) & 31) + 6688)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 768)] * weight_shared[((((int)threadIdx.x) & 31) + 6144)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 769)] * weight_shared[((((int)threadIdx.x) & 31) + 6176)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 776)] * weight_shared[((((int)threadIdx.x) & 31) + 6400)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 777)] * weight_shared[((((int)threadIdx.x) & 31) + 6432)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 784)] * weight_shared[((((int)threadIdx.x) & 31) + 6656)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 785)] * weight_shared[((((int)threadIdx.x) & 31) + 6688)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 722)] * weight_shared[((((int)threadIdx.x) & 31) + 6208)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 723)] * weight_shared[((((int)threadIdx.x) & 31) + 6240)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 730)] * weight_shared[((((int)threadIdx.x) & 31) + 6464)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 731)] * weight_shared[((((int)threadIdx.x) & 31) + 6496)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 738)] * weight_shared[((((int)threadIdx.x) & 31) + 6720)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 739)] * weight_shared[((((int)threadIdx.x) & 31) + 6752)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 730)] * weight_shared[((((int)threadIdx.x) & 31) + 6208)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 731)] * weight_shared[((((int)threadIdx.x) & 31) + 6240)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 738)] * weight_shared[((((int)threadIdx.x) & 31) + 6464)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 739)] * weight_shared[((((int)threadIdx.x) & 31) + 6496)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 746)] * weight_shared[((((int)threadIdx.x) & 31) + 6720)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 747)] * weight_shared[((((int)threadIdx.x) & 31) + 6752)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 738)] * weight_shared[((((int)threadIdx.x) & 31) + 6208)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 739)] * weight_shared[((((int)threadIdx.x) & 31) + 6240)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 746)] * weight_shared[((((int)threadIdx.x) & 31) + 6464)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 747)] * weight_shared[((((int)threadIdx.x) & 31) + 6496)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 754)] * weight_shared[((((int)threadIdx.x) & 31) + 6720)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 755)] * weight_shared[((((int)threadIdx.x) & 31) + 6752)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 746)] * weight_shared[((((int)threadIdx.x) & 31) + 6208)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 747)] * weight_shared[((((int)threadIdx.x) & 31) + 6240)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 754)] * weight_shared[((((int)threadIdx.x) & 31) + 6464)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 755)] * weight_shared[((((int)threadIdx.x) & 31) + 6496)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 762)] * weight_shared[((((int)threadIdx.x) & 31) + 6720)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 763)] * weight_shared[((((int)threadIdx.x) & 31) + 6752)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 754)] * weight_shared[((((int)threadIdx.x) & 31) + 6208)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 755)] * weight_shared[((((int)threadIdx.x) & 31) + 6240)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 762)] * weight_shared[((((int)threadIdx.x) & 31) + 6464)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 763)] * weight_shared[((((int)threadIdx.x) & 31) + 6496)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 770)] * weight_shared[((((int)threadIdx.x) & 31) + 6720)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 771)] * weight_shared[((((int)threadIdx.x) & 31) + 6752)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 762)] * weight_shared[((((int)threadIdx.x) & 31) + 6208)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 763)] * weight_shared[((((int)threadIdx.x) & 31) + 6240)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 770)] * weight_shared[((((int)threadIdx.x) & 31) + 6464)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 771)] * weight_shared[((((int)threadIdx.x) & 31) + 6496)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 778)] * weight_shared[((((int)threadIdx.x) & 31) + 6720)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 779)] * weight_shared[((((int)threadIdx.x) & 31) + 6752)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 770)] * weight_shared[((((int)threadIdx.x) & 31) + 6208)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 771)] * weight_shared[((((int)threadIdx.x) & 31) + 6240)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 778)] * weight_shared[((((int)threadIdx.x) & 31) + 6464)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 779)] * weight_shared[((((int)threadIdx.x) & 31) + 6496)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 786)] * weight_shared[((((int)threadIdx.x) & 31) + 6720)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 787)] * weight_shared[((((int)threadIdx.x) & 31) + 6752)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 724)] * weight_shared[((((int)threadIdx.x) & 31) + 6272)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 725)] * weight_shared[((((int)threadIdx.x) & 31) + 6304)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 732)] * weight_shared[((((int)threadIdx.x) & 31) + 6528)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 733)] * weight_shared[((((int)threadIdx.x) & 31) + 6560)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 740)] * weight_shared[((((int)threadIdx.x) & 31) + 6784)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 741)] * weight_shared[((((int)threadIdx.x) & 31) + 6816)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 732)] * weight_shared[((((int)threadIdx.x) & 31) + 6272)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 733)] * weight_shared[((((int)threadIdx.x) & 31) + 6304)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 740)] * weight_shared[((((int)threadIdx.x) & 31) + 6528)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 741)] * weight_shared[((((int)threadIdx.x) & 31) + 6560)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 748)] * weight_shared[((((int)threadIdx.x) & 31) + 6784)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 749)] * weight_shared[((((int)threadIdx.x) & 31) + 6816)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 740)] * weight_shared[((((int)threadIdx.x) & 31) + 6272)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 741)] * weight_shared[((((int)threadIdx.x) & 31) + 6304)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 748)] * weight_shared[((((int)threadIdx.x) & 31) + 6528)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 749)] * weight_shared[((((int)threadIdx.x) & 31) + 6560)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 756)] * weight_shared[((((int)threadIdx.x) & 31) + 6784)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 757)] * weight_shared[((((int)threadIdx.x) & 31) + 6816)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 748)] * weight_shared[((((int)threadIdx.x) & 31) + 6272)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 749)] * weight_shared[((((int)threadIdx.x) & 31) + 6304)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 756)] * weight_shared[((((int)threadIdx.x) & 31) + 6528)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 757)] * weight_shared[((((int)threadIdx.x) & 31) + 6560)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 764)] * weight_shared[((((int)threadIdx.x) & 31) + 6784)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 765)] * weight_shared[((((int)threadIdx.x) & 31) + 6816)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 756)] * weight_shared[((((int)threadIdx.x) & 31) + 6272)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 757)] * weight_shared[((((int)threadIdx.x) & 31) + 6304)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 764)] * weight_shared[((((int)threadIdx.x) & 31) + 6528)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 765)] * weight_shared[((((int)threadIdx.x) & 31) + 6560)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 772)] * weight_shared[((((int)threadIdx.x) & 31) + 6784)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 773)] * weight_shared[((((int)threadIdx.x) & 31) + 6816)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 764)] * weight_shared[((((int)threadIdx.x) & 31) + 6272)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 765)] * weight_shared[((((int)threadIdx.x) & 31) + 6304)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 772)] * weight_shared[((((int)threadIdx.x) & 31) + 6528)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 773)] * weight_shared[((((int)threadIdx.x) & 31) + 6560)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 780)] * weight_shared[((((int)threadIdx.x) & 31) + 6784)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 781)] * weight_shared[((((int)threadIdx.x) & 31) + 6816)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 772)] * weight_shared[((((int)threadIdx.x) & 31) + 6272)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 773)] * weight_shared[((((int)threadIdx.x) & 31) + 6304)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 780)] * weight_shared[((((int)threadIdx.x) & 31) + 6528)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 781)] * weight_shared[((((int)threadIdx.x) & 31) + 6560)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 788)] * weight_shared[((((int)threadIdx.x) & 31) + 6784)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 789)] * weight_shared[((((int)threadIdx.x) & 31) + 6816)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 726)] * weight_shared[((((int)threadIdx.x) & 31) + 6336)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 727)] * weight_shared[((((int)threadIdx.x) & 31) + 6368)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 734)] * weight_shared[((((int)threadIdx.x) & 31) + 6592)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 735)] * weight_shared[((((int)threadIdx.x) & 31) + 6624)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 742)] * weight_shared[((((int)threadIdx.x) & 31) + 6848)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 743)] * weight_shared[((((int)threadIdx.x) & 31) + 6880)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 734)] * weight_shared[((((int)threadIdx.x) & 31) + 6336)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 735)] * weight_shared[((((int)threadIdx.x) & 31) + 6368)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 742)] * weight_shared[((((int)threadIdx.x) & 31) + 6592)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 743)] * weight_shared[((((int)threadIdx.x) & 31) + 6624)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 750)] * weight_shared[((((int)threadIdx.x) & 31) + 6848)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 751)] * weight_shared[((((int)threadIdx.x) & 31) + 6880)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 742)] * weight_shared[((((int)threadIdx.x) & 31) + 6336)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 743)] * weight_shared[((((int)threadIdx.x) & 31) + 6368)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 750)] * weight_shared[((((int)threadIdx.x) & 31) + 6592)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 751)] * weight_shared[((((int)threadIdx.x) & 31) + 6624)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 758)] * weight_shared[((((int)threadIdx.x) & 31) + 6848)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 759)] * weight_shared[((((int)threadIdx.x) & 31) + 6880)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 750)] * weight_shared[((((int)threadIdx.x) & 31) + 6336)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 751)] * weight_shared[((((int)threadIdx.x) & 31) + 6368)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 758)] * weight_shared[((((int)threadIdx.x) & 31) + 6592)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 759)] * weight_shared[((((int)threadIdx.x) & 31) + 6624)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 766)] * weight_shared[((((int)threadIdx.x) & 31) + 6848)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 767)] * weight_shared[((((int)threadIdx.x) & 31) + 6880)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 758)] * weight_shared[((((int)threadIdx.x) & 31) + 6336)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 759)] * weight_shared[((((int)threadIdx.x) & 31) + 6368)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 766)] * weight_shared[((((int)threadIdx.x) & 31) + 6592)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 767)] * weight_shared[((((int)threadIdx.x) & 31) + 6624)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 774)] * weight_shared[((((int)threadIdx.x) & 31) + 6848)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 775)] * weight_shared[((((int)threadIdx.x) & 31) + 6880)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 766)] * weight_shared[((((int)threadIdx.x) & 31) + 6336)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 767)] * weight_shared[((((int)threadIdx.x) & 31) + 6368)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 774)] * weight_shared[((((int)threadIdx.x) & 31) + 6592)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 775)] * weight_shared[((((int)threadIdx.x) & 31) + 6624)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 782)] * weight_shared[((((int)threadIdx.x) & 31) + 6848)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 783)] * weight_shared[((((int)threadIdx.x) & 31) + 6880)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 774)] * weight_shared[((((int)threadIdx.x) & 31) + 6336)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 775)] * weight_shared[((((int)threadIdx.x) & 31) + 6368)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 782)] * weight_shared[((((int)threadIdx.x) & 31) + 6592)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 783)] * weight_shared[((((int)threadIdx.x) & 31) + 6624)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 790)] * weight_shared[((((int)threadIdx.x) & 31) + 6848)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 791)] * weight_shared[((((int)threadIdx.x) & 31) + 6880)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 864)] * weight_shared[((((int)threadIdx.x) & 31) + 6912)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 865)] * weight_shared[((((int)threadIdx.x) & 31) + 6944)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 872)] * weight_shared[((((int)threadIdx.x) & 31) + 7168)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 873)] * weight_shared[((((int)threadIdx.x) & 31) + 7200)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 880)] * weight_shared[((((int)threadIdx.x) & 31) + 7424)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 881)] * weight_shared[((((int)threadIdx.x) & 31) + 7456)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 872)] * weight_shared[((((int)threadIdx.x) & 31) + 6912)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 873)] * weight_shared[((((int)threadIdx.x) & 31) + 6944)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 880)] * weight_shared[((((int)threadIdx.x) & 31) + 7168)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 881)] * weight_shared[((((int)threadIdx.x) & 31) + 7200)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 888)] * weight_shared[((((int)threadIdx.x) & 31) + 7424)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 889)] * weight_shared[((((int)threadIdx.x) & 31) + 7456)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 880)] * weight_shared[((((int)threadIdx.x) & 31) + 6912)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 881)] * weight_shared[((((int)threadIdx.x) & 31) + 6944)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 888)] * weight_shared[((((int)threadIdx.x) & 31) + 7168)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 889)] * weight_shared[((((int)threadIdx.x) & 31) + 7200)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 896)] * weight_shared[((((int)threadIdx.x) & 31) + 7424)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 897)] * weight_shared[((((int)threadIdx.x) & 31) + 7456)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 888)] * weight_shared[((((int)threadIdx.x) & 31) + 6912)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 889)] * weight_shared[((((int)threadIdx.x) & 31) + 6944)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 896)] * weight_shared[((((int)threadIdx.x) & 31) + 7168)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 897)] * weight_shared[((((int)threadIdx.x) & 31) + 7200)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 904)] * weight_shared[((((int)threadIdx.x) & 31) + 7424)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 905)] * weight_shared[((((int)threadIdx.x) & 31) + 7456)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 896)] * weight_shared[((((int)threadIdx.x) & 31) + 6912)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 897)] * weight_shared[((((int)threadIdx.x) & 31) + 6944)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 904)] * weight_shared[((((int)threadIdx.x) & 31) + 7168)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 905)] * weight_shared[((((int)threadIdx.x) & 31) + 7200)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 912)] * weight_shared[((((int)threadIdx.x) & 31) + 7424)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 913)] * weight_shared[((((int)threadIdx.x) & 31) + 7456)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 904)] * weight_shared[((((int)threadIdx.x) & 31) + 6912)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 905)] * weight_shared[((((int)threadIdx.x) & 31) + 6944)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 912)] * weight_shared[((((int)threadIdx.x) & 31) + 7168)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 913)] * weight_shared[((((int)threadIdx.x) & 31) + 7200)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 920)] * weight_shared[((((int)threadIdx.x) & 31) + 7424)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 921)] * weight_shared[((((int)threadIdx.x) & 31) + 7456)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 912)] * weight_shared[((((int)threadIdx.x) & 31) + 6912)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 913)] * weight_shared[((((int)threadIdx.x) & 31) + 6944)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 920)] * weight_shared[((((int)threadIdx.x) & 31) + 7168)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 921)] * weight_shared[((((int)threadIdx.x) & 31) + 7200)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 928)] * weight_shared[((((int)threadIdx.x) & 31) + 7424)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 929)] * weight_shared[((((int)threadIdx.x) & 31) + 7456)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 866)] * weight_shared[((((int)threadIdx.x) & 31) + 6976)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 867)] * weight_shared[((((int)threadIdx.x) & 31) + 7008)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 874)] * weight_shared[((((int)threadIdx.x) & 31) + 7232)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 875)] * weight_shared[((((int)threadIdx.x) & 31) + 7264)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 882)] * weight_shared[((((int)threadIdx.x) & 31) + 7488)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 883)] * weight_shared[((((int)threadIdx.x) & 31) + 7520)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 874)] * weight_shared[((((int)threadIdx.x) & 31) + 6976)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 875)] * weight_shared[((((int)threadIdx.x) & 31) + 7008)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 882)] * weight_shared[((((int)threadIdx.x) & 31) + 7232)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 883)] * weight_shared[((((int)threadIdx.x) & 31) + 7264)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 890)] * weight_shared[((((int)threadIdx.x) & 31) + 7488)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 891)] * weight_shared[((((int)threadIdx.x) & 31) + 7520)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 882)] * weight_shared[((((int)threadIdx.x) & 31) + 6976)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 883)] * weight_shared[((((int)threadIdx.x) & 31) + 7008)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 890)] * weight_shared[((((int)threadIdx.x) & 31) + 7232)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 891)] * weight_shared[((((int)threadIdx.x) & 31) + 7264)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 898)] * weight_shared[((((int)threadIdx.x) & 31) + 7488)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 899)] * weight_shared[((((int)threadIdx.x) & 31) + 7520)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 890)] * weight_shared[((((int)threadIdx.x) & 31) + 6976)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 891)] * weight_shared[((((int)threadIdx.x) & 31) + 7008)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 898)] * weight_shared[((((int)threadIdx.x) & 31) + 7232)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 899)] * weight_shared[((((int)threadIdx.x) & 31) + 7264)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 906)] * weight_shared[((((int)threadIdx.x) & 31) + 7488)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 907)] * weight_shared[((((int)threadIdx.x) & 31) + 7520)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 898)] * weight_shared[((((int)threadIdx.x) & 31) + 6976)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 899)] * weight_shared[((((int)threadIdx.x) & 31) + 7008)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 906)] * weight_shared[((((int)threadIdx.x) & 31) + 7232)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 907)] * weight_shared[((((int)threadIdx.x) & 31) + 7264)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 914)] * weight_shared[((((int)threadIdx.x) & 31) + 7488)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 915)] * weight_shared[((((int)threadIdx.x) & 31) + 7520)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 906)] * weight_shared[((((int)threadIdx.x) & 31) + 6976)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 907)] * weight_shared[((((int)threadIdx.x) & 31) + 7008)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 914)] * weight_shared[((((int)threadIdx.x) & 31) + 7232)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 915)] * weight_shared[((((int)threadIdx.x) & 31) + 7264)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 922)] * weight_shared[((((int)threadIdx.x) & 31) + 7488)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 923)] * weight_shared[((((int)threadIdx.x) & 31) + 7520)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 914)] * weight_shared[((((int)threadIdx.x) & 31) + 6976)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 915)] * weight_shared[((((int)threadIdx.x) & 31) + 7008)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 922)] * weight_shared[((((int)threadIdx.x) & 31) + 7232)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 923)] * weight_shared[((((int)threadIdx.x) & 31) + 7264)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 930)] * weight_shared[((((int)threadIdx.x) & 31) + 7488)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 931)] * weight_shared[((((int)threadIdx.x) & 31) + 7520)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 868)] * weight_shared[((((int)threadIdx.x) & 31) + 7040)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 869)] * weight_shared[((((int)threadIdx.x) & 31) + 7072)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 876)] * weight_shared[((((int)threadIdx.x) & 31) + 7296)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 877)] * weight_shared[((((int)threadIdx.x) & 31) + 7328)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 884)] * weight_shared[((((int)threadIdx.x) & 31) + 7552)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 885)] * weight_shared[((((int)threadIdx.x) & 31) + 7584)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 876)] * weight_shared[((((int)threadIdx.x) & 31) + 7040)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 877)] * weight_shared[((((int)threadIdx.x) & 31) + 7072)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 884)] * weight_shared[((((int)threadIdx.x) & 31) + 7296)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 885)] * weight_shared[((((int)threadIdx.x) & 31) + 7328)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 892)] * weight_shared[((((int)threadIdx.x) & 31) + 7552)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 893)] * weight_shared[((((int)threadIdx.x) & 31) + 7584)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 884)] * weight_shared[((((int)threadIdx.x) & 31) + 7040)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 885)] * weight_shared[((((int)threadIdx.x) & 31) + 7072)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 892)] * weight_shared[((((int)threadIdx.x) & 31) + 7296)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 893)] * weight_shared[((((int)threadIdx.x) & 31) + 7328)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 900)] * weight_shared[((((int)threadIdx.x) & 31) + 7552)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 901)] * weight_shared[((((int)threadIdx.x) & 31) + 7584)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 892)] * weight_shared[((((int)threadIdx.x) & 31) + 7040)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 893)] * weight_shared[((((int)threadIdx.x) & 31) + 7072)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 900)] * weight_shared[((((int)threadIdx.x) & 31) + 7296)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 901)] * weight_shared[((((int)threadIdx.x) & 31) + 7328)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 908)] * weight_shared[((((int)threadIdx.x) & 31) + 7552)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 909)] * weight_shared[((((int)threadIdx.x) & 31) + 7584)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 900)] * weight_shared[((((int)threadIdx.x) & 31) + 7040)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 901)] * weight_shared[((((int)threadIdx.x) & 31) + 7072)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 908)] * weight_shared[((((int)threadIdx.x) & 31) + 7296)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 909)] * weight_shared[((((int)threadIdx.x) & 31) + 7328)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 916)] * weight_shared[((((int)threadIdx.x) & 31) + 7552)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 917)] * weight_shared[((((int)threadIdx.x) & 31) + 7584)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 908)] * weight_shared[((((int)threadIdx.x) & 31) + 7040)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 909)] * weight_shared[((((int)threadIdx.x) & 31) + 7072)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 916)] * weight_shared[((((int)threadIdx.x) & 31) + 7296)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 917)] * weight_shared[((((int)threadIdx.x) & 31) + 7328)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 924)] * weight_shared[((((int)threadIdx.x) & 31) + 7552)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 925)] * weight_shared[((((int)threadIdx.x) & 31) + 7584)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 916)] * weight_shared[((((int)threadIdx.x) & 31) + 7040)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 917)] * weight_shared[((((int)threadIdx.x) & 31) + 7072)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 924)] * weight_shared[((((int)threadIdx.x) & 31) + 7296)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 925)] * weight_shared[((((int)threadIdx.x) & 31) + 7328)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 932)] * weight_shared[((((int)threadIdx.x) & 31) + 7552)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 933)] * weight_shared[((((int)threadIdx.x) & 31) + 7584)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 870)] * weight_shared[((((int)threadIdx.x) & 31) + 7104)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 871)] * weight_shared[((((int)threadIdx.x) & 31) + 7136)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 878)] * weight_shared[((((int)threadIdx.x) & 31) + 7360)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 879)] * weight_shared[((((int)threadIdx.x) & 31) + 7392)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 886)] * weight_shared[((((int)threadIdx.x) & 31) + 7616)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 887)] * weight_shared[((((int)threadIdx.x) & 31) + 7648)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 878)] * weight_shared[((((int)threadIdx.x) & 31) + 7104)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 879)] * weight_shared[((((int)threadIdx.x) & 31) + 7136)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 886)] * weight_shared[((((int)threadIdx.x) & 31) + 7360)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 887)] * weight_shared[((((int)threadIdx.x) & 31) + 7392)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 894)] * weight_shared[((((int)threadIdx.x) & 31) + 7616)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 895)] * weight_shared[((((int)threadIdx.x) & 31) + 7648)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 886)] * weight_shared[((((int)threadIdx.x) & 31) + 7104)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 887)] * weight_shared[((((int)threadIdx.x) & 31) + 7136)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 894)] * weight_shared[((((int)threadIdx.x) & 31) + 7360)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 895)] * weight_shared[((((int)threadIdx.x) & 31) + 7392)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 902)] * weight_shared[((((int)threadIdx.x) & 31) + 7616)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 903)] * weight_shared[((((int)threadIdx.x) & 31) + 7648)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 894)] * weight_shared[((((int)threadIdx.x) & 31) + 7104)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 895)] * weight_shared[((((int)threadIdx.x) & 31) + 7136)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 902)] * weight_shared[((((int)threadIdx.x) & 31) + 7360)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 903)] * weight_shared[((((int)threadIdx.x) & 31) + 7392)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 910)] * weight_shared[((((int)threadIdx.x) & 31) + 7616)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 911)] * weight_shared[((((int)threadIdx.x) & 31) + 7648)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 902)] * weight_shared[((((int)threadIdx.x) & 31) + 7104)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 903)] * weight_shared[((((int)threadIdx.x) & 31) + 7136)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 910)] * weight_shared[((((int)threadIdx.x) & 31) + 7360)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 911)] * weight_shared[((((int)threadIdx.x) & 31) + 7392)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 918)] * weight_shared[((((int)threadIdx.x) & 31) + 7616)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 919)] * weight_shared[((((int)threadIdx.x) & 31) + 7648)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 910)] * weight_shared[((((int)threadIdx.x) & 31) + 7104)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 911)] * weight_shared[((((int)threadIdx.x) & 31) + 7136)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 918)] * weight_shared[((((int)threadIdx.x) & 31) + 7360)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 919)] * weight_shared[((((int)threadIdx.x) & 31) + 7392)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 926)] * weight_shared[((((int)threadIdx.x) & 31) + 7616)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 927)] * weight_shared[((((int)threadIdx.x) & 31) + 7648)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 918)] * weight_shared[((((int)threadIdx.x) & 31) + 7104)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 919)] * weight_shared[((((int)threadIdx.x) & 31) + 7136)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 926)] * weight_shared[((((int)threadIdx.x) & 31) + 7360)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 927)] * weight_shared[((((int)threadIdx.x) & 31) + 7392)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 934)] * weight_shared[((((int)threadIdx.x) & 31) + 7616)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 935)] * weight_shared[((((int)threadIdx.x) & 31) + 7648)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 936)] * weight_shared[((((int)threadIdx.x) & 31) + 7680)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 937)] * weight_shared[((((int)threadIdx.x) & 31) + 7712)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 944)] * weight_shared[((((int)threadIdx.x) & 31) + 7936)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 945)] * weight_shared[((((int)threadIdx.x) & 31) + 7968)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 952)] * weight_shared[((((int)threadIdx.x) & 31) + 8192)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 953)] * weight_shared[((((int)threadIdx.x) & 31) + 8224)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 944)] * weight_shared[((((int)threadIdx.x) & 31) + 7680)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 945)] * weight_shared[((((int)threadIdx.x) & 31) + 7712)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 952)] * weight_shared[((((int)threadIdx.x) & 31) + 7936)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 953)] * weight_shared[((((int)threadIdx.x) & 31) + 7968)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 960)] * weight_shared[((((int)threadIdx.x) & 31) + 8192)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 961)] * weight_shared[((((int)threadIdx.x) & 31) + 8224)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 952)] * weight_shared[((((int)threadIdx.x) & 31) + 7680)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 953)] * weight_shared[((((int)threadIdx.x) & 31) + 7712)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 960)] * weight_shared[((((int)threadIdx.x) & 31) + 7936)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 961)] * weight_shared[((((int)threadIdx.x) & 31) + 7968)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 968)] * weight_shared[((((int)threadIdx.x) & 31) + 8192)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 969)] * weight_shared[((((int)threadIdx.x) & 31) + 8224)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 960)] * weight_shared[((((int)threadIdx.x) & 31) + 7680)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 961)] * weight_shared[((((int)threadIdx.x) & 31) + 7712)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 968)] * weight_shared[((((int)threadIdx.x) & 31) + 7936)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 969)] * weight_shared[((((int)threadIdx.x) & 31) + 7968)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 976)] * weight_shared[((((int)threadIdx.x) & 31) + 8192)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 977)] * weight_shared[((((int)threadIdx.x) & 31) + 8224)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 968)] * weight_shared[((((int)threadIdx.x) & 31) + 7680)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 969)] * weight_shared[((((int)threadIdx.x) & 31) + 7712)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 976)] * weight_shared[((((int)threadIdx.x) & 31) + 7936)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 977)] * weight_shared[((((int)threadIdx.x) & 31) + 7968)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 984)] * weight_shared[((((int)threadIdx.x) & 31) + 8192)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 985)] * weight_shared[((((int)threadIdx.x) & 31) + 8224)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 976)] * weight_shared[((((int)threadIdx.x) & 31) + 7680)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 977)] * weight_shared[((((int)threadIdx.x) & 31) + 7712)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 984)] * weight_shared[((((int)threadIdx.x) & 31) + 7936)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 985)] * weight_shared[((((int)threadIdx.x) & 31) + 7968)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 992)] * weight_shared[((((int)threadIdx.x) & 31) + 8192)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 993)] * weight_shared[((((int)threadIdx.x) & 31) + 8224)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 984)] * weight_shared[((((int)threadIdx.x) & 31) + 7680)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 985)] * weight_shared[((((int)threadIdx.x) & 31) + 7712)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 992)] * weight_shared[((((int)threadIdx.x) & 31) + 7936)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 993)] * weight_shared[((((int)threadIdx.x) & 31) + 7968)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1000)] * weight_shared[((((int)threadIdx.x) & 31) + 8192)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1001)] * weight_shared[((((int)threadIdx.x) & 31) + 8224)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 938)] * weight_shared[((((int)threadIdx.x) & 31) + 7744)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 939)] * weight_shared[((((int)threadIdx.x) & 31) + 7776)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 946)] * weight_shared[((((int)threadIdx.x) & 31) + 8000)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 947)] * weight_shared[((((int)threadIdx.x) & 31) + 8032)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 954)] * weight_shared[((((int)threadIdx.x) & 31) + 8256)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 955)] * weight_shared[((((int)threadIdx.x) & 31) + 8288)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 946)] * weight_shared[((((int)threadIdx.x) & 31) + 7744)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 947)] * weight_shared[((((int)threadIdx.x) & 31) + 7776)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 954)] * weight_shared[((((int)threadIdx.x) & 31) + 8000)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 955)] * weight_shared[((((int)threadIdx.x) & 31) + 8032)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 962)] * weight_shared[((((int)threadIdx.x) & 31) + 8256)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 963)] * weight_shared[((((int)threadIdx.x) & 31) + 8288)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 954)] * weight_shared[((((int)threadIdx.x) & 31) + 7744)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 955)] * weight_shared[((((int)threadIdx.x) & 31) + 7776)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 962)] * weight_shared[((((int)threadIdx.x) & 31) + 8000)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 963)] * weight_shared[((((int)threadIdx.x) & 31) + 8032)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 970)] * weight_shared[((((int)threadIdx.x) & 31) + 8256)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 971)] * weight_shared[((((int)threadIdx.x) & 31) + 8288)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 962)] * weight_shared[((((int)threadIdx.x) & 31) + 7744)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 963)] * weight_shared[((((int)threadIdx.x) & 31) + 7776)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 970)] * weight_shared[((((int)threadIdx.x) & 31) + 8000)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 971)] * weight_shared[((((int)threadIdx.x) & 31) + 8032)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 978)] * weight_shared[((((int)threadIdx.x) & 31) + 8256)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 979)] * weight_shared[((((int)threadIdx.x) & 31) + 8288)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 970)] * weight_shared[((((int)threadIdx.x) & 31) + 7744)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 971)] * weight_shared[((((int)threadIdx.x) & 31) + 7776)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 978)] * weight_shared[((((int)threadIdx.x) & 31) + 8000)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 979)] * weight_shared[((((int)threadIdx.x) & 31) + 8032)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 986)] * weight_shared[((((int)threadIdx.x) & 31) + 8256)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 987)] * weight_shared[((((int)threadIdx.x) & 31) + 8288)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 978)] * weight_shared[((((int)threadIdx.x) & 31) + 7744)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 979)] * weight_shared[((((int)threadIdx.x) & 31) + 7776)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 986)] * weight_shared[((((int)threadIdx.x) & 31) + 8000)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 987)] * weight_shared[((((int)threadIdx.x) & 31) + 8032)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 994)] * weight_shared[((((int)threadIdx.x) & 31) + 8256)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 995)] * weight_shared[((((int)threadIdx.x) & 31) + 8288)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 986)] * weight_shared[((((int)threadIdx.x) & 31) + 7744)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 987)] * weight_shared[((((int)threadIdx.x) & 31) + 7776)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 994)] * weight_shared[((((int)threadIdx.x) & 31) + 8000)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 995)] * weight_shared[((((int)threadIdx.x) & 31) + 8032)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1002)] * weight_shared[((((int)threadIdx.x) & 31) + 8256)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1003)] * weight_shared[((((int)threadIdx.x) & 31) + 8288)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 940)] * weight_shared[((((int)threadIdx.x) & 31) + 7808)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 941)] * weight_shared[((((int)threadIdx.x) & 31) + 7840)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 948)] * weight_shared[((((int)threadIdx.x) & 31) + 8064)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 949)] * weight_shared[((((int)threadIdx.x) & 31) + 8096)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 956)] * weight_shared[((((int)threadIdx.x) & 31) + 8320)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 957)] * weight_shared[((((int)threadIdx.x) & 31) + 8352)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 948)] * weight_shared[((((int)threadIdx.x) & 31) + 7808)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 949)] * weight_shared[((((int)threadIdx.x) & 31) + 7840)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 956)] * weight_shared[((((int)threadIdx.x) & 31) + 8064)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 957)] * weight_shared[((((int)threadIdx.x) & 31) + 8096)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 964)] * weight_shared[((((int)threadIdx.x) & 31) + 8320)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 965)] * weight_shared[((((int)threadIdx.x) & 31) + 8352)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 956)] * weight_shared[((((int)threadIdx.x) & 31) + 7808)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 957)] * weight_shared[((((int)threadIdx.x) & 31) + 7840)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 964)] * weight_shared[((((int)threadIdx.x) & 31) + 8064)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 965)] * weight_shared[((((int)threadIdx.x) & 31) + 8096)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 972)] * weight_shared[((((int)threadIdx.x) & 31) + 8320)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 973)] * weight_shared[((((int)threadIdx.x) & 31) + 8352)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 964)] * weight_shared[((((int)threadIdx.x) & 31) + 7808)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 965)] * weight_shared[((((int)threadIdx.x) & 31) + 7840)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 972)] * weight_shared[((((int)threadIdx.x) & 31) + 8064)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 973)] * weight_shared[((((int)threadIdx.x) & 31) + 8096)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 980)] * weight_shared[((((int)threadIdx.x) & 31) + 8320)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 981)] * weight_shared[((((int)threadIdx.x) & 31) + 8352)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 972)] * weight_shared[((((int)threadIdx.x) & 31) + 7808)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 973)] * weight_shared[((((int)threadIdx.x) & 31) + 7840)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 980)] * weight_shared[((((int)threadIdx.x) & 31) + 8064)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 981)] * weight_shared[((((int)threadIdx.x) & 31) + 8096)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 988)] * weight_shared[((((int)threadIdx.x) & 31) + 8320)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 989)] * weight_shared[((((int)threadIdx.x) & 31) + 8352)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 980)] * weight_shared[((((int)threadIdx.x) & 31) + 7808)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 981)] * weight_shared[((((int)threadIdx.x) & 31) + 7840)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 988)] * weight_shared[((((int)threadIdx.x) & 31) + 8064)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 989)] * weight_shared[((((int)threadIdx.x) & 31) + 8096)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 996)] * weight_shared[((((int)threadIdx.x) & 31) + 8320)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 997)] * weight_shared[((((int)threadIdx.x) & 31) + 8352)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 988)] * weight_shared[((((int)threadIdx.x) & 31) + 7808)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 989)] * weight_shared[((((int)threadIdx.x) & 31) + 7840)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 996)] * weight_shared[((((int)threadIdx.x) & 31) + 8064)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 997)] * weight_shared[((((int)threadIdx.x) & 31) + 8096)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1004)] * weight_shared[((((int)threadIdx.x) & 31) + 8320)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1005)] * weight_shared[((((int)threadIdx.x) & 31) + 8352)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 942)] * weight_shared[((((int)threadIdx.x) & 31) + 7872)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 943)] * weight_shared[((((int)threadIdx.x) & 31) + 7904)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 950)] * weight_shared[((((int)threadIdx.x) & 31) + 8128)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 951)] * weight_shared[((((int)threadIdx.x) & 31) + 8160)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 958)] * weight_shared[((((int)threadIdx.x) & 31) + 8384)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 959)] * weight_shared[((((int)threadIdx.x) & 31) + 8416)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 950)] * weight_shared[((((int)threadIdx.x) & 31) + 7872)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 951)] * weight_shared[((((int)threadIdx.x) & 31) + 7904)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 958)] * weight_shared[((((int)threadIdx.x) & 31) + 8128)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 959)] * weight_shared[((((int)threadIdx.x) & 31) + 8160)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 966)] * weight_shared[((((int)threadIdx.x) & 31) + 8384)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 967)] * weight_shared[((((int)threadIdx.x) & 31) + 8416)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 958)] * weight_shared[((((int)threadIdx.x) & 31) + 7872)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 959)] * weight_shared[((((int)threadIdx.x) & 31) + 7904)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 966)] * weight_shared[((((int)threadIdx.x) & 31) + 8128)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 967)] * weight_shared[((((int)threadIdx.x) & 31) + 8160)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 974)] * weight_shared[((((int)threadIdx.x) & 31) + 8384)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 975)] * weight_shared[((((int)threadIdx.x) & 31) + 8416)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 966)] * weight_shared[((((int)threadIdx.x) & 31) + 7872)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 967)] * weight_shared[((((int)threadIdx.x) & 31) + 7904)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 974)] * weight_shared[((((int)threadIdx.x) & 31) + 8128)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 975)] * weight_shared[((((int)threadIdx.x) & 31) + 8160)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 982)] * weight_shared[((((int)threadIdx.x) & 31) + 8384)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 983)] * weight_shared[((((int)threadIdx.x) & 31) + 8416)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 974)] * weight_shared[((((int)threadIdx.x) & 31) + 7872)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 975)] * weight_shared[((((int)threadIdx.x) & 31) + 7904)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 982)] * weight_shared[((((int)threadIdx.x) & 31) + 8128)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 983)] * weight_shared[((((int)threadIdx.x) & 31) + 8160)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 990)] * weight_shared[((((int)threadIdx.x) & 31) + 8384)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 991)] * weight_shared[((((int)threadIdx.x) & 31) + 8416)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 982)] * weight_shared[((((int)threadIdx.x) & 31) + 7872)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 983)] * weight_shared[((((int)threadIdx.x) & 31) + 7904)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 990)] * weight_shared[((((int)threadIdx.x) & 31) + 8128)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 991)] * weight_shared[((((int)threadIdx.x) & 31) + 8160)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 998)] * weight_shared[((((int)threadIdx.x) & 31) + 8384)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 999)] * weight_shared[((((int)threadIdx.x) & 31) + 8416)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 990)] * weight_shared[((((int)threadIdx.x) & 31) + 7872)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 991)] * weight_shared[((((int)threadIdx.x) & 31) + 7904)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 998)] * weight_shared[((((int)threadIdx.x) & 31) + 8128)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 999)] * weight_shared[((((int)threadIdx.x) & 31) + 8160)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1006)] * weight_shared[((((int)threadIdx.x) & 31) + 8384)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1007)] * weight_shared[((((int)threadIdx.x) & 31) + 8416)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1008)] * weight_shared[((((int)threadIdx.x) & 31) + 8448)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1009)] * weight_shared[((((int)threadIdx.x) & 31) + 8480)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1016)] * weight_shared[((((int)threadIdx.x) & 31) + 8704)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1017)] * weight_shared[((((int)threadIdx.x) & 31) + 8736)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1024)] * weight_shared[((((int)threadIdx.x) & 31) + 8960)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1025)] * weight_shared[((((int)threadIdx.x) & 31) + 8992)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1016)] * weight_shared[((((int)threadIdx.x) & 31) + 8448)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1017)] * weight_shared[((((int)threadIdx.x) & 31) + 8480)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1024)] * weight_shared[((((int)threadIdx.x) & 31) + 8704)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1025)] * weight_shared[((((int)threadIdx.x) & 31) + 8736)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1032)] * weight_shared[((((int)threadIdx.x) & 31) + 8960)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1033)] * weight_shared[((((int)threadIdx.x) & 31) + 8992)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1024)] * weight_shared[((((int)threadIdx.x) & 31) + 8448)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1025)] * weight_shared[((((int)threadIdx.x) & 31) + 8480)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1032)] * weight_shared[((((int)threadIdx.x) & 31) + 8704)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1033)] * weight_shared[((((int)threadIdx.x) & 31) + 8736)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1040)] * weight_shared[((((int)threadIdx.x) & 31) + 8960)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1041)] * weight_shared[((((int)threadIdx.x) & 31) + 8992)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1032)] * weight_shared[((((int)threadIdx.x) & 31) + 8448)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1033)] * weight_shared[((((int)threadIdx.x) & 31) + 8480)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1040)] * weight_shared[((((int)threadIdx.x) & 31) + 8704)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1041)] * weight_shared[((((int)threadIdx.x) & 31) + 8736)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1048)] * weight_shared[((((int)threadIdx.x) & 31) + 8960)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1049)] * weight_shared[((((int)threadIdx.x) & 31) + 8992)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1040)] * weight_shared[((((int)threadIdx.x) & 31) + 8448)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1041)] * weight_shared[((((int)threadIdx.x) & 31) + 8480)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1048)] * weight_shared[((((int)threadIdx.x) & 31) + 8704)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1049)] * weight_shared[((((int)threadIdx.x) & 31) + 8736)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1056)] * weight_shared[((((int)threadIdx.x) & 31) + 8960)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1057)] * weight_shared[((((int)threadIdx.x) & 31) + 8992)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1048)] * weight_shared[((((int)threadIdx.x) & 31) + 8448)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1049)] * weight_shared[((((int)threadIdx.x) & 31) + 8480)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1056)] * weight_shared[((((int)threadIdx.x) & 31) + 8704)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1057)] * weight_shared[((((int)threadIdx.x) & 31) + 8736)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1064)] * weight_shared[((((int)threadIdx.x) & 31) + 8960)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1065)] * weight_shared[((((int)threadIdx.x) & 31) + 8992)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1056)] * weight_shared[((((int)threadIdx.x) & 31) + 8448)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1057)] * weight_shared[((((int)threadIdx.x) & 31) + 8480)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1064)] * weight_shared[((((int)threadIdx.x) & 31) + 8704)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1065)] * weight_shared[((((int)threadIdx.x) & 31) + 8736)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1072)] * weight_shared[((((int)threadIdx.x) & 31) + 8960)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1073)] * weight_shared[((((int)threadIdx.x) & 31) + 8992)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1010)] * weight_shared[((((int)threadIdx.x) & 31) + 8512)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1011)] * weight_shared[((((int)threadIdx.x) & 31) + 8544)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1018)] * weight_shared[((((int)threadIdx.x) & 31) + 8768)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1019)] * weight_shared[((((int)threadIdx.x) & 31) + 8800)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1026)] * weight_shared[((((int)threadIdx.x) & 31) + 9024)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1027)] * weight_shared[((((int)threadIdx.x) & 31) + 9056)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1018)] * weight_shared[((((int)threadIdx.x) & 31) + 8512)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1019)] * weight_shared[((((int)threadIdx.x) & 31) + 8544)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1026)] * weight_shared[((((int)threadIdx.x) & 31) + 8768)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1027)] * weight_shared[((((int)threadIdx.x) & 31) + 8800)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1034)] * weight_shared[((((int)threadIdx.x) & 31) + 9024)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1035)] * weight_shared[((((int)threadIdx.x) & 31) + 9056)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1026)] * weight_shared[((((int)threadIdx.x) & 31) + 8512)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1027)] * weight_shared[((((int)threadIdx.x) & 31) + 8544)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1034)] * weight_shared[((((int)threadIdx.x) & 31) + 8768)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1035)] * weight_shared[((((int)threadIdx.x) & 31) + 8800)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1042)] * weight_shared[((((int)threadIdx.x) & 31) + 9024)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1043)] * weight_shared[((((int)threadIdx.x) & 31) + 9056)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1034)] * weight_shared[((((int)threadIdx.x) & 31) + 8512)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1035)] * weight_shared[((((int)threadIdx.x) & 31) + 8544)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1042)] * weight_shared[((((int)threadIdx.x) & 31) + 8768)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1043)] * weight_shared[((((int)threadIdx.x) & 31) + 8800)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1050)] * weight_shared[((((int)threadIdx.x) & 31) + 9024)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1051)] * weight_shared[((((int)threadIdx.x) & 31) + 9056)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1042)] * weight_shared[((((int)threadIdx.x) & 31) + 8512)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1043)] * weight_shared[((((int)threadIdx.x) & 31) + 8544)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1050)] * weight_shared[((((int)threadIdx.x) & 31) + 8768)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1051)] * weight_shared[((((int)threadIdx.x) & 31) + 8800)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1058)] * weight_shared[((((int)threadIdx.x) & 31) + 9024)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1059)] * weight_shared[((((int)threadIdx.x) & 31) + 9056)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1050)] * weight_shared[((((int)threadIdx.x) & 31) + 8512)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1051)] * weight_shared[((((int)threadIdx.x) & 31) + 8544)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1058)] * weight_shared[((((int)threadIdx.x) & 31) + 8768)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1059)] * weight_shared[((((int)threadIdx.x) & 31) + 8800)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1066)] * weight_shared[((((int)threadIdx.x) & 31) + 9024)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1067)] * weight_shared[((((int)threadIdx.x) & 31) + 9056)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1058)] * weight_shared[((((int)threadIdx.x) & 31) + 8512)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1059)] * weight_shared[((((int)threadIdx.x) & 31) + 8544)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1066)] * weight_shared[((((int)threadIdx.x) & 31) + 8768)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1067)] * weight_shared[((((int)threadIdx.x) & 31) + 8800)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1074)] * weight_shared[((((int)threadIdx.x) & 31) + 9024)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1075)] * weight_shared[((((int)threadIdx.x) & 31) + 9056)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1012)] * weight_shared[((((int)threadIdx.x) & 31) + 8576)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1013)] * weight_shared[((((int)threadIdx.x) & 31) + 8608)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1020)] * weight_shared[((((int)threadIdx.x) & 31) + 8832)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1021)] * weight_shared[((((int)threadIdx.x) & 31) + 8864)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1028)] * weight_shared[((((int)threadIdx.x) & 31) + 9088)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1029)] * weight_shared[((((int)threadIdx.x) & 31) + 9120)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1020)] * weight_shared[((((int)threadIdx.x) & 31) + 8576)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1021)] * weight_shared[((((int)threadIdx.x) & 31) + 8608)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1028)] * weight_shared[((((int)threadIdx.x) & 31) + 8832)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1029)] * weight_shared[((((int)threadIdx.x) & 31) + 8864)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1036)] * weight_shared[((((int)threadIdx.x) & 31) + 9088)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1037)] * weight_shared[((((int)threadIdx.x) & 31) + 9120)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1028)] * weight_shared[((((int)threadIdx.x) & 31) + 8576)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1029)] * weight_shared[((((int)threadIdx.x) & 31) + 8608)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1036)] * weight_shared[((((int)threadIdx.x) & 31) + 8832)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1037)] * weight_shared[((((int)threadIdx.x) & 31) + 8864)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1044)] * weight_shared[((((int)threadIdx.x) & 31) + 9088)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1045)] * weight_shared[((((int)threadIdx.x) & 31) + 9120)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1036)] * weight_shared[((((int)threadIdx.x) & 31) + 8576)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1037)] * weight_shared[((((int)threadIdx.x) & 31) + 8608)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1044)] * weight_shared[((((int)threadIdx.x) & 31) + 8832)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1045)] * weight_shared[((((int)threadIdx.x) & 31) + 8864)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1052)] * weight_shared[((((int)threadIdx.x) & 31) + 9088)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1053)] * weight_shared[((((int)threadIdx.x) & 31) + 9120)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1044)] * weight_shared[((((int)threadIdx.x) & 31) + 8576)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1045)] * weight_shared[((((int)threadIdx.x) & 31) + 8608)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1052)] * weight_shared[((((int)threadIdx.x) & 31) + 8832)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1053)] * weight_shared[((((int)threadIdx.x) & 31) + 8864)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1060)] * weight_shared[((((int)threadIdx.x) & 31) + 9088)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1061)] * weight_shared[((((int)threadIdx.x) & 31) + 9120)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1052)] * weight_shared[((((int)threadIdx.x) & 31) + 8576)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1053)] * weight_shared[((((int)threadIdx.x) & 31) + 8608)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1060)] * weight_shared[((((int)threadIdx.x) & 31) + 8832)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1061)] * weight_shared[((((int)threadIdx.x) & 31) + 8864)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1068)] * weight_shared[((((int)threadIdx.x) & 31) + 9088)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1069)] * weight_shared[((((int)threadIdx.x) & 31) + 9120)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1060)] * weight_shared[((((int)threadIdx.x) & 31) + 8576)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1061)] * weight_shared[((((int)threadIdx.x) & 31) + 8608)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1068)] * weight_shared[((((int)threadIdx.x) & 31) + 8832)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1069)] * weight_shared[((((int)threadIdx.x) & 31) + 8864)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1076)] * weight_shared[((((int)threadIdx.x) & 31) + 9088)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1077)] * weight_shared[((((int)threadIdx.x) & 31) + 9120)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1014)] * weight_shared[((((int)threadIdx.x) & 31) + 8640)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1015)] * weight_shared[((((int)threadIdx.x) & 31) + 8672)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1022)] * weight_shared[((((int)threadIdx.x) & 31) + 8896)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1023)] * weight_shared[((((int)threadIdx.x) & 31) + 8928)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1030)] * weight_shared[((((int)threadIdx.x) & 31) + 9152)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1031)] * weight_shared[((((int)threadIdx.x) & 31) + 9184)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1022)] * weight_shared[((((int)threadIdx.x) & 31) + 8640)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1023)] * weight_shared[((((int)threadIdx.x) & 31) + 8672)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1030)] * weight_shared[((((int)threadIdx.x) & 31) + 8896)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1031)] * weight_shared[((((int)threadIdx.x) & 31) + 8928)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1038)] * weight_shared[((((int)threadIdx.x) & 31) + 9152)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1039)] * weight_shared[((((int)threadIdx.x) & 31) + 9184)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1030)] * weight_shared[((((int)threadIdx.x) & 31) + 8640)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1031)] * weight_shared[((((int)threadIdx.x) & 31) + 8672)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1038)] * weight_shared[((((int)threadIdx.x) & 31) + 8896)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1039)] * weight_shared[((((int)threadIdx.x) & 31) + 8928)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1046)] * weight_shared[((((int)threadIdx.x) & 31) + 9152)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1047)] * weight_shared[((((int)threadIdx.x) & 31) + 9184)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1038)] * weight_shared[((((int)threadIdx.x) & 31) + 8640)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1039)] * weight_shared[((((int)threadIdx.x) & 31) + 8672)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1046)] * weight_shared[((((int)threadIdx.x) & 31) + 8896)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1047)] * weight_shared[((((int)threadIdx.x) & 31) + 8928)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1054)] * weight_shared[((((int)threadIdx.x) & 31) + 9152)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1055)] * weight_shared[((((int)threadIdx.x) & 31) + 9184)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1046)] * weight_shared[((((int)threadIdx.x) & 31) + 8640)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1047)] * weight_shared[((((int)threadIdx.x) & 31) + 8672)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1054)] * weight_shared[((((int)threadIdx.x) & 31) + 8896)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1055)] * weight_shared[((((int)threadIdx.x) & 31) + 8928)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1062)] * weight_shared[((((int)threadIdx.x) & 31) + 9152)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1063)] * weight_shared[((((int)threadIdx.x) & 31) + 9184)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1054)] * weight_shared[((((int)threadIdx.x) & 31) + 8640)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1055)] * weight_shared[((((int)threadIdx.x) & 31) + 8672)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1062)] * weight_shared[((((int)threadIdx.x) & 31) + 8896)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1063)] * weight_shared[((((int)threadIdx.x) & 31) + 8928)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1070)] * weight_shared[((((int)threadIdx.x) & 31) + 9152)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1071)] * weight_shared[((((int)threadIdx.x) & 31) + 9184)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1062)] * weight_shared[((((int)threadIdx.x) & 31) + 8640)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1063)] * weight_shared[((((int)threadIdx.x) & 31) + 8672)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1070)] * weight_shared[((((int)threadIdx.x) & 31) + 8896)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1071)] * weight_shared[((((int)threadIdx.x) & 31) + 8928)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1078)] * weight_shared[((((int)threadIdx.x) & 31) + 9152)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 72) + 1079)] * weight_shared[((((int)threadIdx.x) & 31) + 9184)]));
  conv2d_nhwc[((((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) & 31))] = conv2d_nhwc_local[0];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) & 31)) + 256)] = conv2d_nhwc_local[1];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) & 31)) + 512)] = conv2d_nhwc_local[2];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) & 31)) + 768)] = conv2d_nhwc_local[3];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) & 31)) + 1024)] = conv2d_nhwc_local[4];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) & 31)) + 1280)] = conv2d_nhwc_local[5];
  conv2d_nhwc[(((((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 5) * 3584)) + (((((int)blockIdx.x) & 15) >> 3) * 1792)) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) & 31)) + 1536)] = conv2d_nhwc_local[6];
}


