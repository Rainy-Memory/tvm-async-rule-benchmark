
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(56) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[7];
  __shared__ float PadInput_shared[5184];
  __shared__ float weight_shared[4608];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  for (int rc_0 = 0; rc_0 < 8; ++rc_0) {
    __syncthreads();
    PadInput_shared[(((int)threadIdx.x) * 4)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 1)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 2)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 3)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 224)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 225)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 226)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 227)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 448)] = ((((32 <= ((int)threadIdx.x)) && (1 <= (((((int)threadIdx.x) >> 4) + 7) % 9))) && ((((((int)threadIdx.x) * 4) + 448) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 112) / 144) * 3584) + ((((((int)threadIdx.x) >> 4) + 7) % 9) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 449)] = ((((32 <= ((int)threadIdx.x)) && (1 <= (((((int)threadIdx.x) >> 4) + 7) % 9))) && ((((((int)threadIdx.x) * 4) + 449) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 112) / 144) * 3584) + ((((((int)threadIdx.x) >> 4) + 7) % 9) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 4095)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 450)] = ((((32 <= ((int)threadIdx.x)) && (1 <= (((((int)threadIdx.x) >> 4) + 7) % 9))) && ((((((int)threadIdx.x) * 4) + 450) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 112) / 144) * 3584) + ((((((int)threadIdx.x) >> 4) + 7) % 9) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 4094)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 451)] = ((((32 <= ((int)threadIdx.x)) && (1 <= (((((int)threadIdx.x) >> 4) + 7) % 9))) && ((((((int)threadIdx.x) * 4) + 451) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 112) / 144) * 3584) + ((((((int)threadIdx.x) >> 4) + 7) % 9) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 4093)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 672)] = inputs[(((((((((int)threadIdx.x) + 168) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 3) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 32) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 673)] = inputs[(((((((((int)threadIdx.x) + 168) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 3) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 33) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 674)] = inputs[(((((((((int)threadIdx.x) + 168) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 3) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 34) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 675)] = inputs[(((((((((int)threadIdx.x) + 168) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 3) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 35) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 896)] = ((((int)threadIdx.x) < 48) ? inputs[(((((((((int)threadIdx.x) + 224) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 1536)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 897)] = ((((int)threadIdx.x) < 48) ? inputs[(((((((((int)threadIdx.x) + 224) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 1535)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 898)] = ((((int)threadIdx.x) < 48) ? inputs[(((((((((int)threadIdx.x) + 224) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 1534)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 899)] = ((((int)threadIdx.x) < 48) ? inputs[(((((((((int)threadIdx.x) + 224) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 1533)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 1120)] = (((2 <= (((((int)threadIdx.x) >> 3) + 17) % 18)) && ((((((int)threadIdx.x) * 4) + 544) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 280) / 144) * 3584) + (((((((int)threadIdx.x) >> 3) + 17) % 18) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 32) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 1121)] = (((2 <= (((((int)threadIdx.x) >> 3) + 17) % 18)) && ((((((int)threadIdx.x) * 4) + 545) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 280) / 144) * 3584) + (((((((int)threadIdx.x) >> 3) + 17) % 18) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 33) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 1122)] = (((2 <= (((((int)threadIdx.x) >> 3) + 17) % 18)) && ((((((int)threadIdx.x) * 4) + 546) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 280) / 144) * 3584) + (((((((int)threadIdx.x) >> 3) + 17) % 18) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 34) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 1123)] = (((2 <= (((((int)threadIdx.x) >> 3) + 17) % 18)) && ((((((int)threadIdx.x) * 4) + 547) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 280) / 144) * 3584) + (((((((int)threadIdx.x) >> 3) + 17) % 18) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 35) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 1344)] = inputs[(((((((((int)threadIdx.x) + 336) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 2560)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 1345)] = inputs[(((((((((int)threadIdx.x) + 336) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 2559)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 1346)] = inputs[(((((((((int)threadIdx.x) + 336) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 2558)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 1347)] = inputs[(((((((((int)threadIdx.x) + 336) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 2557)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 1568)] = (((2 <= (((((int)threadIdx.x) >> 3) + 13) % 18)) && ((((((int)threadIdx.x) * 4) + 416) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 392) / 144) * 3584) + (((((((int)threadIdx.x) >> 3) + 13) % 18) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 32) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 1569)] = (((2 <= (((((int)threadIdx.x) >> 3) + 13) % 18)) && ((((((int)threadIdx.x) * 4) + 417) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 392) / 144) * 3584) + (((((((int)threadIdx.x) >> 3) + 13) % 18) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 33) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 1570)] = (((2 <= (((((int)threadIdx.x) >> 3) + 13) % 18)) && ((((((int)threadIdx.x) * 4) + 418) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 392) / 144) * 3584) + (((((((int)threadIdx.x) >> 3) + 13) % 18) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 34) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 1571)] = (((2 <= (((((int)threadIdx.x) >> 3) + 13) % 18)) && ((((((int)threadIdx.x) * 4) + 419) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 392) / 144) * 3584) + (((((((int)threadIdx.x) >> 3) + 13) % 18) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 35) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 1792)] = inputs[(((((((((int)threadIdx.x) + 448) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 3584)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 1793)] = inputs[(((((((((int)threadIdx.x) + 448) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 3583)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 1794)] = inputs[(((((((((int)threadIdx.x) + 448) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 3582)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 1795)] = inputs[(((((((((int)threadIdx.x) + 448) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 3581)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 2016)] = inputs[(((((((((int)threadIdx.x) + 504) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 9) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 32) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 2017)] = inputs[(((((((((int)threadIdx.x) + 504) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 9) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 33) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 2018)] = inputs[(((((((((int)threadIdx.x) + 504) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 9) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 34) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 2019)] = inputs[(((((((((int)threadIdx.x) + 504) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 9) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 35) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 2240)] = (((1 <= (((((int)threadIdx.x) >> 4) + 8) % 9)) && ((((((int)threadIdx.x) * 4) + 512) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 560) / 144) * 3584) + ((((((int)threadIdx.x) >> 4) + 8) % 9) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 2241)] = (((1 <= (((((int)threadIdx.x) >> 4) + 8) % 9)) && ((((((int)threadIdx.x) * 4) + 513) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 560) / 144) * 3584) + ((((((int)threadIdx.x) >> 4) + 8) % 9) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 4095)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 2242)] = (((1 <= (((((int)threadIdx.x) >> 4) + 8) % 9)) && ((((((int)threadIdx.x) * 4) + 514) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 560) / 144) * 3584) + ((((((int)threadIdx.x) >> 4) + 8) % 9) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 4094)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 2243)] = (((1 <= (((((int)threadIdx.x) >> 4) + 8) % 9)) && ((((((int)threadIdx.x) * 4) + 515) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 560) / 144) * 3584) + ((((((int)threadIdx.x) >> 4) + 8) % 9) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 4093)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 2464)] = inputs[(((((((((int)threadIdx.x) + 616) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 5) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 32) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 2465)] = inputs[(((((((((int)threadIdx.x) + 616) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 5) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 33) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 2466)] = inputs[(((((((((int)threadIdx.x) + 616) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 5) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 34) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 2467)] = inputs[(((((((((int)threadIdx.x) + 616) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 5) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 35) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 2688)] = (((1 <= (((((int)threadIdx.x) >> 4) + 6) % 9)) && ((((((int)threadIdx.x) * 4) + 384) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 672) / 144) * 3584) + ((((((int)threadIdx.x) >> 4) + 6) % 9) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 2689)] = (((1 <= (((((int)threadIdx.x) >> 4) + 6) % 9)) && ((((((int)threadIdx.x) * 4) + 385) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 672) / 144) * 3584) + ((((((int)threadIdx.x) >> 4) + 6) % 9) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 4095)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 2690)] = (((1 <= (((((int)threadIdx.x) >> 4) + 6) % 9)) && ((((((int)threadIdx.x) * 4) + 386) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 672) / 144) * 3584) + ((((((int)threadIdx.x) >> 4) + 6) % 9) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 4094)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 2691)] = (((1 <= (((((int)threadIdx.x) >> 4) + 6) % 9)) && ((((((int)threadIdx.x) * 4) + 387) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 672) / 144) * 3584) + ((((((int)threadIdx.x) >> 4) + 6) % 9) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 4093)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 2912)] = ((8 <= ((int)threadIdx.x)) ? inputs[(((((((((int)threadIdx.x) + 728) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 1) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 32) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 2913)] = ((8 <= ((int)threadIdx.x)) ? inputs[(((((((((int)threadIdx.x) + 728) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 1) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 33) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 2914)] = ((8 <= ((int)threadIdx.x)) ? inputs[(((((((((int)threadIdx.x) + 728) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 1) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 34) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 2915)] = ((8 <= ((int)threadIdx.x)) ? inputs[(((((((((int)threadIdx.x) + 728) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 1) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 35) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 3136)] = inputs[(((((((((int)threadIdx.x) + 784) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 2048)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 3137)] = inputs[(((((((((int)threadIdx.x) + 784) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 2047)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 3138)] = inputs[(((((((((int)threadIdx.x) + 784) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 2046)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 3139)] = inputs[(((((((((int)threadIdx.x) + 784) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 2045)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 3360)] = (((2 <= (((((int)threadIdx.x) >> 3) + 15) % 18)) && ((((((int)threadIdx.x) * 4) + 480) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 840) / 144) * 3584) + (((((((int)threadIdx.x) >> 3) + 15) % 18) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 32) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 3361)] = (((2 <= (((((int)threadIdx.x) >> 3) + 15) % 18)) && ((((((int)threadIdx.x) * 4) + 481) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 840) / 144) * 3584) + (((((((int)threadIdx.x) >> 3) + 15) % 18) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 33) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 3362)] = (((2 <= (((((int)threadIdx.x) >> 3) + 15) % 18)) && ((((((int)threadIdx.x) * 4) + 482) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 840) / 144) * 3584) + (((((((int)threadIdx.x) >> 3) + 15) % 18) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 34) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 3363)] = (((2 <= (((((int)threadIdx.x) >> 3) + 15) % 18)) && ((((((int)threadIdx.x) * 4) + 483) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 840) / 144) * 3584) + (((((((int)threadIdx.x) >> 3) + 15) % 18) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 35) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 3584)] = inputs[(((((((((int)threadIdx.x) + 896) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 3072)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 3585)] = inputs[(((((((((int)threadIdx.x) + 896) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 3071)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 3586)] = inputs[(((((((((int)threadIdx.x) + 896) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 3070)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 3587)] = inputs[(((((((((int)threadIdx.x) + 896) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 3069)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 3808)] = ((((int)threadIdx.x) < 40) ? inputs[(((((((((int)threadIdx.x) + 952) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 11) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 32) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 3809)] = ((((int)threadIdx.x) < 40) ? inputs[(((((((((int)threadIdx.x) + 952) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 11) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 33) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 3810)] = ((((int)threadIdx.x) < 40) ? inputs[(((((((((int)threadIdx.x) + 952) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 11) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 34) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 3811)] = ((((int)threadIdx.x) < 40) ? inputs[(((((((((int)threadIdx.x) + 952) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 11) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 35) & 63)) - 4096)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 4032)] = ((16 <= ((int)threadIdx.x)) ? inputs[(((((((int)threadIdx.x) >> 4) * 512) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 20992)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 4033)] = ((16 <= ((int)threadIdx.x)) ? inputs[(((((((int)threadIdx.x) >> 4) * 512) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 20993)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 4034)] = ((16 <= ((int)threadIdx.x)) ? inputs[(((((((int)threadIdx.x) >> 4) * 512) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 20994)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 4035)] = ((16 <= ((int)threadIdx.x)) ? inputs[(((((((int)threadIdx.x) >> 4) * 512) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 20995)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 4256)] = inputs[(((((((((int)threadIdx.x) + 1064) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 7) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 32) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 4257)] = inputs[(((((((((int)threadIdx.x) + 1064) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 7) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 33) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 4258)] = inputs[(((((((((int)threadIdx.x) + 1064) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 7) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 34) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 4259)] = inputs[(((((((((int)threadIdx.x) + 1064) / 144) * 3584) + ((((((int)threadIdx.x) >> 3) + 7) >> 1) * 512)) + (rc_0 * 64)) + (((((int)threadIdx.x) * 4) + 35) & 63)) - 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 4480)] = ((((((int)threadIdx.x) < 32) && (1 <= (((((int)threadIdx.x) >> 4) + 7) % 9))) && ((((((int)threadIdx.x) * 4) + 448) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 1120) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 512)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 4481)] = ((((((int)threadIdx.x) < 32) && (1 <= (((((int)threadIdx.x) >> 4) + 7) % 9))) && ((((((int)threadIdx.x) * 4) + 449) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 1120) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 511)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 4482)] = ((((((int)threadIdx.x) < 32) && (1 <= (((((int)threadIdx.x) >> 4) + 7) % 9))) && ((((((int)threadIdx.x) * 4) + 450) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 1120) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 510)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 4483)] = ((((((int)threadIdx.x) < 32) && (1 <= (((((int)threadIdx.x) >> 4) + 7) % 9))) && ((((((int)threadIdx.x) * 4) + 451) % 576) < 512)) ? inputs[(((((((((int)threadIdx.x) + 1120) / 144) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) - 509)] : 0.000000e+00f);
    PadInput_shared[((((int)threadIdx.x) * 4) + 4704)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 4705)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 4706)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 4707)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 4928)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 4929)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 4930)] = 0.000000e+00f;
    PadInput_shared[((((int)threadIdx.x) * 4) + 4931)] = 0.000000e+00f;
    if (((int)threadIdx.x) < 8) {
      PadInput_shared[((((int)threadIdx.x) * 4) + 5152)] = 0.000000e+00f;
    }
    if (((int)threadIdx.x) < 8) {
      PadInput_shared[((((int)threadIdx.x) * 4) + 5153)] = 0.000000e+00f;
    }
    if (((int)threadIdx.x) < 8) {
      PadInput_shared[((((int)threadIdx.x) * 4) + 5154)] = 0.000000e+00f;
    }
    if (((int)threadIdx.x) < 8) {
      PadInput_shared[((((int)threadIdx.x) * 4) + 5155)] = 0.000000e+00f;
    }
    weight_shared[(((int)threadIdx.x) * 2)] = weight[((((rc_0 * 32768) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2))];
    weight_shared[((((int)threadIdx.x) * 2) + 1)] = weight[(((((rc_0 * 32768) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1)];
    weight_shared[((((int)threadIdx.x) * 2) + 112)] = weight[(((((rc_0 * 32768) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 7168)];
    weight_shared[((((int)threadIdx.x) * 2) + 113)] = weight[(((((rc_0 * 32768) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 7169)];
    weight_shared[((((int)threadIdx.x) * 2) + 224)] = weight[(((((rc_0 * 32768) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 14336)];
    weight_shared[((((int)threadIdx.x) * 2) + 225)] = weight[(((((rc_0 * 32768) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 14337)];
    weight_shared[((((int)threadIdx.x) * 2) + 336)] = weight[(((((rc_0 * 32768) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 21504)];
    weight_shared[((((int)threadIdx.x) * 2) + 337)] = weight[(((((rc_0 * 32768) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 21505)];
    weight_shared[((((int)threadIdx.x) * 2) + 448)] = weight[(((((((((int)threadIdx.x) + 224) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 56) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2))];
    weight_shared[((((int)threadIdx.x) * 2) + 449)] = weight[((((((((((int)threadIdx.x) + 224) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 56) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1)];
    weight_shared[((((int)threadIdx.x) * 2) + 560)] = weight[((((((((((int)threadIdx.x) + 280) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 3072)];
    weight_shared[((((int)threadIdx.x) * 2) + 561)] = weight[((((((((((int)threadIdx.x) + 280) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 3073)];
    weight_shared[((((int)threadIdx.x) * 2) + 672)] = weight[((((((((((int)threadIdx.x) + 336) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 10240)];
    weight_shared[((((int)threadIdx.x) * 2) + 673)] = weight[((((((((((int)threadIdx.x) + 336) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 10241)];
    weight_shared[((((int)threadIdx.x) * 2) + 784)] = weight[((((((((((int)threadIdx.x) + 392) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 17408)];
    weight_shared[((((int)threadIdx.x) * 2) + 785)] = weight[((((((((((int)threadIdx.x) + 392) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 17409)];
    weight_shared[((((int)threadIdx.x) * 2) + 896)] = weight[((((((((((int)threadIdx.x) + 448) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 24576)];
    weight_shared[((((int)threadIdx.x) * 2) + 897)] = weight[((((((((((int)threadIdx.x) + 448) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 24577)];
    weight_shared[((((int)threadIdx.x) * 2) + 1008)] = weight[(((((((((int)threadIdx.x) + 504) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 62) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2))];
    weight_shared[((((int)threadIdx.x) * 2) + 1009)] = weight[((((((((((int)threadIdx.x) + 504) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 62) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1)];
    weight_shared[((((int)threadIdx.x) * 2) + 1120)] = weight[((((((((((int)threadIdx.x) + 560) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 6144)];
    weight_shared[((((int)threadIdx.x) * 2) + 1121)] = weight[((((((((((int)threadIdx.x) + 560) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 6145)];
    weight_shared[((((int)threadIdx.x) * 2) + 1232)] = weight[((((((((((int)threadIdx.x) + 616) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 13312)];
    weight_shared[((((int)threadIdx.x) * 2) + 1233)] = weight[((((((((((int)threadIdx.x) + 616) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 13313)];
    weight_shared[((((int)threadIdx.x) * 2) + 1344)] = weight[((((((((((int)threadIdx.x) + 672) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 20480)];
    weight_shared[((((int)threadIdx.x) * 2) + 1345)] = weight[((((((((((int)threadIdx.x) + 672) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 20481)];
    weight_shared[((((int)threadIdx.x) * 2) + 1456)] = weight[(((((((((int)threadIdx.x) + 728) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 54) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2))];
    weight_shared[((((int)threadIdx.x) * 2) + 1457)] = weight[((((((((((int)threadIdx.x) + 728) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 54) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1)];
    weight_shared[((((int)threadIdx.x) * 2) + 1568)] = weight[((((((((((int)threadIdx.x) + 784) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 2048)];
    weight_shared[((((int)threadIdx.x) * 2) + 1569)] = weight[((((((((((int)threadIdx.x) + 784) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 2049)];
    weight_shared[((((int)threadIdx.x) * 2) + 1680)] = weight[((((((((((int)threadIdx.x) + 840) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 9216)];
    weight_shared[((((int)threadIdx.x) * 2) + 1681)] = weight[((((((((((int)threadIdx.x) + 840) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 9217)];
    weight_shared[((((int)threadIdx.x) * 2) + 1792)] = weight[((((((((((int)threadIdx.x) + 896) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 16384)];
    weight_shared[((((int)threadIdx.x) * 2) + 1793)] = weight[((((((((((int)threadIdx.x) + 896) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 16385)];
    weight_shared[((((int)threadIdx.x) * 2) + 1904)] = weight[((((((((((int)threadIdx.x) + 952) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 23552)];
    weight_shared[((((int)threadIdx.x) * 2) + 1905)] = weight[((((((((((int)threadIdx.x) + 952) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 23553)];
    weight_shared[((((int)threadIdx.x) * 2) + 2016)] = weight[(((((((((int)threadIdx.x) + 1008) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 60) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2))];
    weight_shared[((((int)threadIdx.x) * 2) + 2017)] = weight[((((((((((int)threadIdx.x) + 1008) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 60) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1)];
    weight_shared[((((int)threadIdx.x) * 2) + 2128)] = weight[((((((((((int)threadIdx.x) + 1064) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 5120)];
    weight_shared[((((int)threadIdx.x) * 2) + 2129)] = weight[((((((((((int)threadIdx.x) + 1064) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 5121)];
    weight_shared[((((int)threadIdx.x) * 2) + 2240)] = weight[((((((((((int)threadIdx.x) + 1120) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 12288)];
    weight_shared[((((int)threadIdx.x) * 2) + 2241)] = weight[((((((((((int)threadIdx.x) + 1120) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 12289)];
    weight_shared[((((int)threadIdx.x) * 2) + 2352)] = weight[((((((((((int)threadIdx.x) + 1176) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 19456)];
    weight_shared[((((int)threadIdx.x) * 2) + 2353)] = weight[((((((((((int)threadIdx.x) + 1176) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 19457)];
    weight_shared[((((int)threadIdx.x) * 2) + 2464)] = weight[(((((((((int)threadIdx.x) + 1232) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 52) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2))];
    weight_shared[((((int)threadIdx.x) * 2) + 2465)] = weight[((((((((((int)threadIdx.x) + 1232) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 52) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1)];
    weight_shared[((((int)threadIdx.x) * 2) + 2576)] = weight[((((((((((int)threadIdx.x) + 1288) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1024)];
    weight_shared[((((int)threadIdx.x) * 2) + 2577)] = weight[((((((((((int)threadIdx.x) + 1288) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1025)];
    weight_shared[((((int)threadIdx.x) * 2) + 2688)] = weight[((((((((((int)threadIdx.x) + 1344) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 8192)];
    weight_shared[((((int)threadIdx.x) * 2) + 2689)] = weight[((((((((((int)threadIdx.x) + 1344) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 8193)];
    weight_shared[((((int)threadIdx.x) * 2) + 2800)] = weight[((((((((((int)threadIdx.x) + 1400) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 15360)];
    weight_shared[((((int)threadIdx.x) * 2) + 2801)] = weight[((((((((((int)threadIdx.x) + 1400) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 15361)];
    weight_shared[((((int)threadIdx.x) * 2) + 2912)] = weight[((((((((((int)threadIdx.x) + 1456) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 22528)];
    weight_shared[((((int)threadIdx.x) * 2) + 2913)] = weight[((((((((((int)threadIdx.x) + 1456) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 22529)];
    weight_shared[((((int)threadIdx.x) * 2) + 3024)] = weight[(((((((((int)threadIdx.x) + 1512) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 58) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2))];
    weight_shared[((((int)threadIdx.x) * 2) + 3025)] = weight[((((((((((int)threadIdx.x) + 1512) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 58) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1)];
    weight_shared[((((int)threadIdx.x) * 2) + 3136)] = weight[((((((((((int)threadIdx.x) + 1568) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 4096)];
    weight_shared[((((int)threadIdx.x) * 2) + 3137)] = weight[((((((((((int)threadIdx.x) + 1568) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 4097)];
    weight_shared[((((int)threadIdx.x) * 2) + 3248)] = weight[((((((((((int)threadIdx.x) + 1624) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 11264)];
    weight_shared[((((int)threadIdx.x) * 2) + 3249)] = weight[((((((((((int)threadIdx.x) + 1624) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 11265)];
    weight_shared[((((int)threadIdx.x) * 2) + 3360)] = weight[((((((((((int)threadIdx.x) + 1680) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 18432)];
    weight_shared[((((int)threadIdx.x) * 2) + 3361)] = weight[((((((((((int)threadIdx.x) + 1680) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 18433)];
    weight_shared[((((int)threadIdx.x) * 2) + 3472)] = weight[((((((((((int)threadIdx.x) + 1736) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 25600)];
    weight_shared[((((int)threadIdx.x) * 2) + 3473)] = weight[((((((((((int)threadIdx.x) + 1736) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 25601)];
    weight_shared[((((int)threadIdx.x) * 2) + 3584)] = weight[(((((rc_0 * 32768) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1835008)];
    weight_shared[((((int)threadIdx.x) * 2) + 3585)] = weight[(((((rc_0 * 32768) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1835009)];
    weight_shared[((((int)threadIdx.x) * 2) + 3696)] = weight[((((((((((int)threadIdx.x) + 1848) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 7168)];
    weight_shared[((((int)threadIdx.x) * 2) + 3697)] = weight[((((((((((int)threadIdx.x) + 1848) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 7169)];
    weight_shared[((((int)threadIdx.x) * 2) + 3808)] = weight[((((((((((int)threadIdx.x) + 1904) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 14336)];
    weight_shared[((((int)threadIdx.x) * 2) + 3809)] = weight[((((((((((int)threadIdx.x) + 1904) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 14337)];
    weight_shared[((((int)threadIdx.x) * 2) + 3920)] = weight[((((((((((int)threadIdx.x) + 1960) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 21504)];
    weight_shared[((((int)threadIdx.x) * 2) + 3921)] = weight[((((((((((int)threadIdx.x) + 1960) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 21505)];
    weight_shared[((((int)threadIdx.x) * 2) + 4032)] = weight[(((((((((int)threadIdx.x) + 2016) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 56) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2))];
    weight_shared[((((int)threadIdx.x) * 2) + 4033)] = weight[((((((((((int)threadIdx.x) + 2016) >> 8) * 262144) + (rc_0 * 32768)) + ((((((int)threadIdx.x) >> 2) + 56) & 63) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1)];
    weight_shared[((((int)threadIdx.x) * 2) + 4144)] = weight[((((((((((int)threadIdx.x) + 2072) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 3072)];
    weight_shared[((((int)threadIdx.x) * 2) + 4145)] = weight[((((((((((int)threadIdx.x) + 2072) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 3073)];
    weight_shared[((((int)threadIdx.x) * 2) + 4256)] = weight[((((((((((int)threadIdx.x) + 2128) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 10240)];
    weight_shared[((((int)threadIdx.x) * 2) + 4257)] = weight[((((((((((int)threadIdx.x) + 2128) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 10241)];
    weight_shared[((((int)threadIdx.x) * 2) + 4368)] = weight[((((((((((int)threadIdx.x) + 2184) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 17408)];
    weight_shared[((((int)threadIdx.x) * 2) + 4369)] = weight[((((((((((int)threadIdx.x) + 2184) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 17409)];
    weight_shared[((((int)threadIdx.x) * 2) + 4480)] = weight[((((((((((int)threadIdx.x) + 2240) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 24576)];
    weight_shared[((((int)threadIdx.x) * 2) + 4481)] = weight[((((((((((int)threadIdx.x) + 2240) >> 8) * 262144) + (rc_0 * 32768)) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 24577)];
    if (((int)threadIdx.x) < 8) {
      weight_shared[((((int)threadIdx.x) * 2) + 4592)] = weight[(((((rc_0 * 32768) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 2128896)];
    }
    if (((int)threadIdx.x) < 8) {
      weight_shared[((((int)threadIdx.x) * 2) + 4593)] = weight[(((((rc_0 * 32768) + ((((int)threadIdx.x) >> 2) * 512)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 2128897)];
    }
    __syncthreads();
    for (int rw_1 = 0; rw_1 < 3; ++rw_1) {
      for (int rc_1 = 0; rc_1 < 32; ++rc_1) {
        conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2))] * weight_shared[(((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7))]));
        conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 576)] * weight_shared[(((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7))]));
        conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 1152)] * weight_shared[(((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7))]));
        conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 1728)] * weight_shared[(((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7))]));
        conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 2304)] * weight_shared[(((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7))]));
        conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 2880)] * weight_shared[(((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7))]));
        conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 3456)] * weight_shared[(((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7))]));
        conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 1)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 8)]));
        conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 577)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 8)]));
        conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 1153)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 8)]));
        conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 1729)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 8)]));
        conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 2305)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 8)]));
        conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 2881)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 8)]));
        conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 3457)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 8)]));
        conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 576)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1536)]));
        conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 1152)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1536)]));
        conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 1728)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1536)]));
        conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 2304)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1536)]));
        conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 2880)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1536)]));
        conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 3456)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1536)]));
        conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 4032)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1536)]));
        conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 577)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1544)]));
        conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 1153)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1544)]));
        conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 1729)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1544)]));
        conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 2305)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1544)]));
        conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 2881)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1544)]));
        conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 3457)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1544)]));
        conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 4033)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 1544)]));
        conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 1152)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3072)]));
        conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 1728)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3072)]));
        conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 2304)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3072)]));
        conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 2880)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3072)]));
        conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 3456)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3072)]));
        conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 4032)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3072)]));
        conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 4608)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3072)]));
        conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 1153)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3080)]));
        conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 1729)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3080)]));
        conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 2305)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3080)]));
        conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 2881)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3080)]));
        conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 3457)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3080)]));
        conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 4033)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3080)]));
        conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((((int)threadIdx.x) >> 3) * 64) + (rw_1 * 64)) + (rc_1 * 2)) + 4609)] * weight_shared[((((rw_1 * 512) + (rc_1 * 16)) + (((int)threadIdx.x) & 7)) + 3080)]));
      }
    }
  }
  conv2d_nhwc[((((((int)threadIdx.x) >> 3) * 512) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 7))] = conv2d_nhwc_local[0];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 3) * 512) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 7)) + 3584)] = conv2d_nhwc_local[1];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 3) * 512) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 7)) + 7168)] = conv2d_nhwc_local[2];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 3) * 512) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 7)) + 10752)] = conv2d_nhwc_local[3];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 3) * 512) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 7)) + 14336)] = conv2d_nhwc_local[4];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 3) * 512) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 7)) + 17920)] = conv2d_nhwc_local[5];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 3) * 512) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 7)) + 21504)] = conv2d_nhwc_local[6];
}


