
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[7];
  __shared__ float PadInput_shared[832];
  __shared__ float weight_shared[4096];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  for (int rc_0 = 0; rc_0 < 16; ++rc_0) {
    __syncthreads();
    PadInput_shared[(((int)threadIdx.x) * 4)] = inputs[(((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4))];
    PadInput_shared[((((int)threadIdx.x) * 4) + 1)] = inputs[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 1)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 2)] = inputs[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 2)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 3)] = inputs[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 3)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 256)] = inputs[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 4096)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 257)] = inputs[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 4097)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 258)] = inputs[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 4098)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 259)] = inputs[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 4099)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 512)] = inputs[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 8192)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 513)] = inputs[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 8193)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 514)] = inputs[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 8194)];
    PadInput_shared[((((int)threadIdx.x) * 4) + 515)] = inputs[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rc_0 * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 8195)];
    if (((int)threadIdx.x) < 16) {
      PadInput_shared[((((int)threadIdx.x) * 4) + 768)] = inputs[(((((((int)blockIdx.x) >> 3) * 28672) + (rc_0 * 64)) + (((int)threadIdx.x) * 4)) + 12288)];
    }
    if (((int)threadIdx.x) < 16) {
      PadInput_shared[((((int)threadIdx.x) * 4) + 769)] = inputs[(((((((int)blockIdx.x) >> 3) * 28672) + (rc_0 * 64)) + (((int)threadIdx.x) * 4)) + 12289)];
    }
    if (((int)threadIdx.x) < 16) {
      PadInput_shared[((((int)threadIdx.x) * 4) + 770)] = inputs[(((((((int)blockIdx.x) >> 3) * 28672) + (rc_0 * 64)) + (((int)threadIdx.x) * 4)) + 12290)];
    }
    if (((int)threadIdx.x) < 16) {
      PadInput_shared[((((int)threadIdx.x) * 4) + 771)] = inputs[(((((((int)blockIdx.x) >> 3) * 28672) + (rc_0 * 64)) + (((int)threadIdx.x) * 4)) + 12291)];
    }
    *(float4*)(weight_shared + (((int)threadIdx.x) * 4)) = *(float4*)(weight + ((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 256)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 2048));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 4096));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 768)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 6144));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 8192));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1280)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 10240));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 12288));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 1792)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 14336));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2048)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 16384));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2304)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 18432));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2560)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 20480));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 2816)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 22528));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3072)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 24576));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3328)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 26624));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3584)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 28672));
    *(float4*)(weight_shared + ((((int)threadIdx.x) * 4) + 3840)) = *(float4*)(weight + (((((rc_0 * 32768) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 30720));
    __syncthreads();
    for (int w_3 = 0; w_3 < 7; ++w_3) {
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[(w_3 * 128)] * weight_shared[((int)threadIdx.x)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 1)] * weight_shared[(((int)threadIdx.x) + 64)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 2)] * weight_shared[(((int)threadIdx.x) + 128)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 3)] * weight_shared[(((int)threadIdx.x) + 192)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 4)] * weight_shared[(((int)threadIdx.x) + 256)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 5)] * weight_shared[(((int)threadIdx.x) + 320)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 6)] * weight_shared[(((int)threadIdx.x) + 384)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 7)] * weight_shared[(((int)threadIdx.x) + 448)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 8)] * weight_shared[(((int)threadIdx.x) + 512)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 9)] * weight_shared[(((int)threadIdx.x) + 576)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 10)] * weight_shared[(((int)threadIdx.x) + 640)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 11)] * weight_shared[(((int)threadIdx.x) + 704)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 12)] * weight_shared[(((int)threadIdx.x) + 768)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 13)] * weight_shared[(((int)threadIdx.x) + 832)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 14)] * weight_shared[(((int)threadIdx.x) + 896)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 15)] * weight_shared[(((int)threadIdx.x) + 960)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 16)] * weight_shared[(((int)threadIdx.x) + 1024)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 17)] * weight_shared[(((int)threadIdx.x) + 1088)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 18)] * weight_shared[(((int)threadIdx.x) + 1152)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 19)] * weight_shared[(((int)threadIdx.x) + 1216)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 20)] * weight_shared[(((int)threadIdx.x) + 1280)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 21)] * weight_shared[(((int)threadIdx.x) + 1344)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 22)] * weight_shared[(((int)threadIdx.x) + 1408)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 23)] * weight_shared[(((int)threadIdx.x) + 1472)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 24)] * weight_shared[(((int)threadIdx.x) + 1536)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 25)] * weight_shared[(((int)threadIdx.x) + 1600)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 26)] * weight_shared[(((int)threadIdx.x) + 1664)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 27)] * weight_shared[(((int)threadIdx.x) + 1728)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 28)] * weight_shared[(((int)threadIdx.x) + 1792)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 29)] * weight_shared[(((int)threadIdx.x) + 1856)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 30)] * weight_shared[(((int)threadIdx.x) + 1920)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 31)] * weight_shared[(((int)threadIdx.x) + 1984)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 32)] * weight_shared[(((int)threadIdx.x) + 2048)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 33)] * weight_shared[(((int)threadIdx.x) + 2112)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 34)] * weight_shared[(((int)threadIdx.x) + 2176)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 35)] * weight_shared[(((int)threadIdx.x) + 2240)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 36)] * weight_shared[(((int)threadIdx.x) + 2304)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 37)] * weight_shared[(((int)threadIdx.x) + 2368)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 38)] * weight_shared[(((int)threadIdx.x) + 2432)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 39)] * weight_shared[(((int)threadIdx.x) + 2496)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 40)] * weight_shared[(((int)threadIdx.x) + 2560)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 41)] * weight_shared[(((int)threadIdx.x) + 2624)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 42)] * weight_shared[(((int)threadIdx.x) + 2688)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 43)] * weight_shared[(((int)threadIdx.x) + 2752)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 44)] * weight_shared[(((int)threadIdx.x) + 2816)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 45)] * weight_shared[(((int)threadIdx.x) + 2880)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 46)] * weight_shared[(((int)threadIdx.x) + 2944)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 47)] * weight_shared[(((int)threadIdx.x) + 3008)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 48)] * weight_shared[(((int)threadIdx.x) + 3072)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 49)] * weight_shared[(((int)threadIdx.x) + 3136)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 50)] * weight_shared[(((int)threadIdx.x) + 3200)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 51)] * weight_shared[(((int)threadIdx.x) + 3264)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 52)] * weight_shared[(((int)threadIdx.x) + 3328)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 53)] * weight_shared[(((int)threadIdx.x) + 3392)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 54)] * weight_shared[(((int)threadIdx.x) + 3456)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 55)] * weight_shared[(((int)threadIdx.x) + 3520)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 56)] * weight_shared[(((int)threadIdx.x) + 3584)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 57)] * weight_shared[(((int)threadIdx.x) + 3648)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 58)] * weight_shared[(((int)threadIdx.x) + 3712)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 59)] * weight_shared[(((int)threadIdx.x) + 3776)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 60)] * weight_shared[(((int)threadIdx.x) + 3840)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 61)] * weight_shared[(((int)threadIdx.x) + 3904)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 62)] * weight_shared[(((int)threadIdx.x) + 3968)]));
      conv2d_nhwc_local[w_3] = (conv2d_nhwc_local[w_3] + (PadInput_shared[((w_3 * 128) + 63)] * weight_shared[(((int)threadIdx.x) + 4032)]));
    }
  }
  conv2d_nhwc[((((((int)blockIdx.x) >> 3) * 3584) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x))] = conv2d_nhwc_local[0];
  conv2d_nhwc[(((((((int)blockIdx.x) >> 3) * 3584) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 512)] = conv2d_nhwc_local[1];
  conv2d_nhwc[(((((((int)blockIdx.x) >> 3) * 3584) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 1024)] = conv2d_nhwc_local[2];
  conv2d_nhwc[(((((((int)blockIdx.x) >> 3) * 3584) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 1536)] = conv2d_nhwc_local[3];
  conv2d_nhwc[(((((((int)blockIdx.x) >> 3) * 3584) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 2048)] = conv2d_nhwc_local[4];
  conv2d_nhwc[(((((((int)blockIdx.x) >> 3) * 3584) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 2560)] = conv2d_nhwc_local[5];
  conv2d_nhwc[(((((((int)blockIdx.x) >> 3) * 3584) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 3072)] = conv2d_nhwc_local[6];
}


