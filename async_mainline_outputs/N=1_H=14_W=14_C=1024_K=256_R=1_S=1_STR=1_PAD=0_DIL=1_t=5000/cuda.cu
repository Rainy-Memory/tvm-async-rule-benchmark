
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[7];
  __shared__ float PadInput_shared[5376];
  __shared__ float weight_shared[3072];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)) + 4096))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)) + 8192))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)) + 12288))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)) + 16384))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)) + 20480))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)) + 24576))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 2) * 256) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 2) * 256) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 4096))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 2) * 256) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 8192))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 2) * 256) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 12288))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)) + 64))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)) + 4160))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 2304)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)) + 8256))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 2560)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)) + 12352))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 2816)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)) + 16448))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 3072)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)) + 20544))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 3328)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)threadIdx.x) & 15) * 4)) + 24640))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 2) * 256) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 16384))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 2) * 256) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 20480))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 2) * 256) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 24576))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 2) * 256) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 28672))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int rh_0_rw_0_rc_0_fused = 0; rh_0_rw_0_rc_0_fused < 14; ++rh_0_rw_0_rc_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1792) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rh_0_rw_0_rc_0_fused * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 128))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1792) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rh_0_rw_0_rc_0_fused * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 4224))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1792) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rh_0_rw_0_rc_0_fused * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 8320))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1792) + (((int)threadIdx.x) * 4)) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rh_0_rw_0_rc_0_fused * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 12416))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1792) + (((int)threadIdx.x) * 4)) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rh_0_rw_0_rc_0_fused * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 16512))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1792) + (((int)threadIdx.x) * 4)) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rh_0_rw_0_rc_0_fused * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 20608))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1792) + (((int)threadIdx.x) * 4)) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 4) * 28672) + ((((int)threadIdx.x) >> 4) * 1024)) + (rh_0_rw_0_rc_0_fused * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 24704))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 16384) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 32768))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 16384) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 36864))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 16384) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 40960))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + (((int)threadIdx.x) * 4)) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 16384) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.x) & 15) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 45056))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448))] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15))]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 16)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 32)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 48)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 64)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 80)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 96)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 112)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 128)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 144)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 160)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 176)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 192)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 208)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 224)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 240)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 256)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 272)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 288)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 304)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 320)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 336)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 352)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 368)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 384)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 400)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 416)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 432)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 448)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 464)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 480)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 496)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 64)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15))]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 16)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 32)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 48)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 64)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 80)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 96)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 112)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 128)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 144)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 160)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 176)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 192)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 208)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 224)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 240)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 256)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 272)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 288)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 304)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 320)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 336)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 352)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 368)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 384)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 400)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 416)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 432)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 448)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 464)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 480)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 496)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 128)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15))]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 16)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 32)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 48)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 64)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 80)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 96)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 112)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 128)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 144)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 160)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 176)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 192)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 208)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 224)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 240)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 144)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 256)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 272)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 288)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 304)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 320)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 336)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 352)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 368)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 384)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 400)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 416)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 432)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 448)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 464)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 480)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 496)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 192)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15))]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 16)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 32)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 48)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 64)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 80)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 96)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 112)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 128)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 144)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 160)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 176)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 192)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 208)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 224)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 240)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 208)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 256)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 209)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 272)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 210)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 288)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 211)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 304)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 212)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 320)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 213)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 336)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 214)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 352)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 215)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 368)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 216)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 384)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 217)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 400)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 218)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 416)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 219)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 432)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 220)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 448)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 221)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 464)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 222)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 480)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 223)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 496)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 256)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15))]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 257)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 16)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 258)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 32)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 259)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 48)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 260)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 64)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 261)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 80)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 262)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 96)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 263)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 112)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 264)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 128)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 265)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 144)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 266)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 160)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 267)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 176)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 268)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 192)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 269)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 208)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 270)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 224)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 271)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 240)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 272)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 256)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 273)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 272)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 274)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 288)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 275)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 304)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 276)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 320)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 277)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 336)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 278)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 352)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 279)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 368)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 280)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 384)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 281)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 400)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 282)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 416)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 283)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 432)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 284)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 448)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 285)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 464)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 286)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 480)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 287)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 496)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 320)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15))]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 321)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 16)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 322)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 32)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 323)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 48)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 324)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 64)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 325)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 80)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 326)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 96)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 327)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 112)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 328)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 128)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 329)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 144)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 330)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 160)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 331)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 176)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 332)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 192)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 333)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 208)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 334)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 224)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 335)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 240)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 336)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 256)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 337)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 272)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 338)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 288)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 339)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 304)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 340)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 320)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 341)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 336)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 342)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 352)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 343)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 368)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 344)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 384)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 345)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 400)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 346)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 416)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 347)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 432)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 348)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 448)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 349)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 464)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 350)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 480)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 351)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 496)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 384)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15))]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 385)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 16)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 386)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 32)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 387)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 48)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 388)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 64)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 389)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 80)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 390)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 96)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 391)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 112)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 392)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 128)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 393)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 144)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 394)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 160)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 395)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 176)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 396)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 192)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 397)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 208)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 398)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 224)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 399)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 240)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 400)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 256)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 401)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 272)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 402)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 288)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 403)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 304)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 404)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 320)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 405)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 336)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 406)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 352)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 407)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 368)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 408)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 384)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 409)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 400)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 410)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 416)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 411)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 432)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 412)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 448)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 413)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 464)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 414)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 480)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 415)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 496)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 32)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 512)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 528)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 544)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 560)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 576)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 592)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 608)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 624)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 640)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 656)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 672)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 688)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 704)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 720)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 736)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 752)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 768)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 784)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 800)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 816)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 832)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 848)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 864)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 880)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 896)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 912)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 928)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 944)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 960)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 976)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 992)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 1008)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 512)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 528)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 544)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 560)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 576)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 592)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 608)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 624)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 640)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 656)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 672)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 688)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 704)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 720)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 736)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 752)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 112)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 768)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 113)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 784)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 114)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 800)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 115)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 816)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 116)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 832)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 117)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 848)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 118)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 864)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 119)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 880)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 120)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 896)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 121)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 912)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 122)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 928)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 123)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 944)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 124)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 960)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 125)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 976)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 126)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 992)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 127)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 1008)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 160)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 512)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 161)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 528)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 162)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 544)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 163)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 560)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 164)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 576)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 165)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 592)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 166)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 608)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 167)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 624)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 168)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 640)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 169)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 656)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 170)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 672)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 171)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 688)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 172)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 704)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 173)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 720)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 174)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 736)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 175)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 752)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 176)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 768)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 177)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 784)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 178)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 800)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 179)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 816)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 180)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 832)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 181)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 848)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 182)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 864)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 183)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 880)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 184)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 896)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 185)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 912)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 186)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 928)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 187)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 944)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 188)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 960)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 189)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 976)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 190)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 992)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 191)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 1008)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 224)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 512)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 225)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 528)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 226)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 544)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 227)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 560)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 228)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 576)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 229)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 592)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 230)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 608)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 231)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 624)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 232)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 640)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 233)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 656)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 234)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 672)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 235)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 688)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 236)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 704)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 237)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 720)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 238)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 736)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 239)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 752)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 240)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 768)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 241)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 784)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 242)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 800)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 243)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 816)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 244)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 832)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 245)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 848)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 246)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 864)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 247)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 880)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 248)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 896)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 249)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 912)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 250)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 928)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 251)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 944)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 252)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 960)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 253)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 976)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 254)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 992)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 255)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 1008)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 288)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 512)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 289)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 528)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 290)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 544)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 291)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 560)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 292)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 576)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 293)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 592)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 294)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 608)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 295)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 624)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 296)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 640)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 297)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 656)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 298)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 672)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 299)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 688)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 300)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 704)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 301)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 720)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 302)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 736)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 303)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 752)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 304)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 768)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 305)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 784)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 306)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 800)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 307)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 816)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 308)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 832)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 309)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 848)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 310)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 864)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 311)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 880)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 312)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 896)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 313)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 912)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 314)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 928)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 315)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 944)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 316)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 960)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 317)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 976)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 318)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 992)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 319)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 1008)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 352)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 512)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 353)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 528)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 354)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 544)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 355)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 560)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 356)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 576)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 357)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 592)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 358)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 608)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 359)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 624)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 360)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 640)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 361)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 656)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 362)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 672)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 363)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 688)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 364)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 704)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 365)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 720)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 366)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 736)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 367)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 752)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 368)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 768)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 369)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 784)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 370)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 800)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 371)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 816)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 372)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 832)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 373)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 848)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 374)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 864)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 375)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 880)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 376)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 896)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 377)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 912)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 378)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 928)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 379)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 944)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 380)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 960)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 381)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 976)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 382)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 992)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 383)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 1008)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 416)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 512)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 417)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 528)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 418)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 544)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 419)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 560)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 420)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 576)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 421)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 592)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 422)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 608)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 423)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 624)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 424)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 640)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 425)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 656)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 426)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 672)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 427)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 688)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 428)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 704)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 429)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 720)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 430)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 736)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 431)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 752)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 432)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 768)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 433)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 784)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 434)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 800)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 435)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 816)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 436)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 832)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 437)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 848)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 438)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 864)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 439)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 880)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 440)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 896)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 441)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 912)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 442)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 928)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 443)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 944)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 444)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 960)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 445)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 976)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 446)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 992)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1792) + ((((int)threadIdx.x) >> 4) * 448)) + 447)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + (((int)threadIdx.x) & 15)) + 1008)]));
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3584)] * weight_shared[((((int)threadIdx.x) & 15) + 2048)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3585)] * weight_shared[((((int)threadIdx.x) & 15) + 2064)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3586)] * weight_shared[((((int)threadIdx.x) & 15) + 2080)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3587)] * weight_shared[((((int)threadIdx.x) & 15) + 2096)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3588)] * weight_shared[((((int)threadIdx.x) & 15) + 2112)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3589)] * weight_shared[((((int)threadIdx.x) & 15) + 2128)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3590)] * weight_shared[((((int)threadIdx.x) & 15) + 2144)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3591)] * weight_shared[((((int)threadIdx.x) & 15) + 2160)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3592)] * weight_shared[((((int)threadIdx.x) & 15) + 2176)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3593)] * weight_shared[((((int)threadIdx.x) & 15) + 2192)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3594)] * weight_shared[((((int)threadIdx.x) & 15) + 2208)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3595)] * weight_shared[((((int)threadIdx.x) & 15) + 2224)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3596)] * weight_shared[((((int)threadIdx.x) & 15) + 2240)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3597)] * weight_shared[((((int)threadIdx.x) & 15) + 2256)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3598)] * weight_shared[((((int)threadIdx.x) & 15) + 2272)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3599)] * weight_shared[((((int)threadIdx.x) & 15) + 2288)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3600)] * weight_shared[((((int)threadIdx.x) & 15) + 2304)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3601)] * weight_shared[((((int)threadIdx.x) & 15) + 2320)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3602)] * weight_shared[((((int)threadIdx.x) & 15) + 2336)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3603)] * weight_shared[((((int)threadIdx.x) & 15) + 2352)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3604)] * weight_shared[((((int)threadIdx.x) & 15) + 2368)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3605)] * weight_shared[((((int)threadIdx.x) & 15) + 2384)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3606)] * weight_shared[((((int)threadIdx.x) & 15) + 2400)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3607)] * weight_shared[((((int)threadIdx.x) & 15) + 2416)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3608)] * weight_shared[((((int)threadIdx.x) & 15) + 2432)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3609)] * weight_shared[((((int)threadIdx.x) & 15) + 2448)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3610)] * weight_shared[((((int)threadIdx.x) & 15) + 2464)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3611)] * weight_shared[((((int)threadIdx.x) & 15) + 2480)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3612)] * weight_shared[((((int)threadIdx.x) & 15) + 2496)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3613)] * weight_shared[((((int)threadIdx.x) & 15) + 2512)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3614)] * weight_shared[((((int)threadIdx.x) & 15) + 2528)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3615)] * weight_shared[((((int)threadIdx.x) & 15) + 2544)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3648)] * weight_shared[((((int)threadIdx.x) & 15) + 2048)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3649)] * weight_shared[((((int)threadIdx.x) & 15) + 2064)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3650)] * weight_shared[((((int)threadIdx.x) & 15) + 2080)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3651)] * weight_shared[((((int)threadIdx.x) & 15) + 2096)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3652)] * weight_shared[((((int)threadIdx.x) & 15) + 2112)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3653)] * weight_shared[((((int)threadIdx.x) & 15) + 2128)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3654)] * weight_shared[((((int)threadIdx.x) & 15) + 2144)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3655)] * weight_shared[((((int)threadIdx.x) & 15) + 2160)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3656)] * weight_shared[((((int)threadIdx.x) & 15) + 2176)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3657)] * weight_shared[((((int)threadIdx.x) & 15) + 2192)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3658)] * weight_shared[((((int)threadIdx.x) & 15) + 2208)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3659)] * weight_shared[((((int)threadIdx.x) & 15) + 2224)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3660)] * weight_shared[((((int)threadIdx.x) & 15) + 2240)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3661)] * weight_shared[((((int)threadIdx.x) & 15) + 2256)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3662)] * weight_shared[((((int)threadIdx.x) & 15) + 2272)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3663)] * weight_shared[((((int)threadIdx.x) & 15) + 2288)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3664)] * weight_shared[((((int)threadIdx.x) & 15) + 2304)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3665)] * weight_shared[((((int)threadIdx.x) & 15) + 2320)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3666)] * weight_shared[((((int)threadIdx.x) & 15) + 2336)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3667)] * weight_shared[((((int)threadIdx.x) & 15) + 2352)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3668)] * weight_shared[((((int)threadIdx.x) & 15) + 2368)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3669)] * weight_shared[((((int)threadIdx.x) & 15) + 2384)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3670)] * weight_shared[((((int)threadIdx.x) & 15) + 2400)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3671)] * weight_shared[((((int)threadIdx.x) & 15) + 2416)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3672)] * weight_shared[((((int)threadIdx.x) & 15) + 2432)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3673)] * weight_shared[((((int)threadIdx.x) & 15) + 2448)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3674)] * weight_shared[((((int)threadIdx.x) & 15) + 2464)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3675)] * weight_shared[((((int)threadIdx.x) & 15) + 2480)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3676)] * weight_shared[((((int)threadIdx.x) & 15) + 2496)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3677)] * weight_shared[((((int)threadIdx.x) & 15) + 2512)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3678)] * weight_shared[((((int)threadIdx.x) & 15) + 2528)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3679)] * weight_shared[((((int)threadIdx.x) & 15) + 2544)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3712)] * weight_shared[((((int)threadIdx.x) & 15) + 2048)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3713)] * weight_shared[((((int)threadIdx.x) & 15) + 2064)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3714)] * weight_shared[((((int)threadIdx.x) & 15) + 2080)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3715)] * weight_shared[((((int)threadIdx.x) & 15) + 2096)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3716)] * weight_shared[((((int)threadIdx.x) & 15) + 2112)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3717)] * weight_shared[((((int)threadIdx.x) & 15) + 2128)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3718)] * weight_shared[((((int)threadIdx.x) & 15) + 2144)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3719)] * weight_shared[((((int)threadIdx.x) & 15) + 2160)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3720)] * weight_shared[((((int)threadIdx.x) & 15) + 2176)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3721)] * weight_shared[((((int)threadIdx.x) & 15) + 2192)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3722)] * weight_shared[((((int)threadIdx.x) & 15) + 2208)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3723)] * weight_shared[((((int)threadIdx.x) & 15) + 2224)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3724)] * weight_shared[((((int)threadIdx.x) & 15) + 2240)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3725)] * weight_shared[((((int)threadIdx.x) & 15) + 2256)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3726)] * weight_shared[((((int)threadIdx.x) & 15) + 2272)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3727)] * weight_shared[((((int)threadIdx.x) & 15) + 2288)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3728)] * weight_shared[((((int)threadIdx.x) & 15) + 2304)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3729)] * weight_shared[((((int)threadIdx.x) & 15) + 2320)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3730)] * weight_shared[((((int)threadIdx.x) & 15) + 2336)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3731)] * weight_shared[((((int)threadIdx.x) & 15) + 2352)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3732)] * weight_shared[((((int)threadIdx.x) & 15) + 2368)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3733)] * weight_shared[((((int)threadIdx.x) & 15) + 2384)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3734)] * weight_shared[((((int)threadIdx.x) & 15) + 2400)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3735)] * weight_shared[((((int)threadIdx.x) & 15) + 2416)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3736)] * weight_shared[((((int)threadIdx.x) & 15) + 2432)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3737)] * weight_shared[((((int)threadIdx.x) & 15) + 2448)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3738)] * weight_shared[((((int)threadIdx.x) & 15) + 2464)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3739)] * weight_shared[((((int)threadIdx.x) & 15) + 2480)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3740)] * weight_shared[((((int)threadIdx.x) & 15) + 2496)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3741)] * weight_shared[((((int)threadIdx.x) & 15) + 2512)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3742)] * weight_shared[((((int)threadIdx.x) & 15) + 2528)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3743)] * weight_shared[((((int)threadIdx.x) & 15) + 2544)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3776)] * weight_shared[((((int)threadIdx.x) & 15) + 2048)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3777)] * weight_shared[((((int)threadIdx.x) & 15) + 2064)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3778)] * weight_shared[((((int)threadIdx.x) & 15) + 2080)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3779)] * weight_shared[((((int)threadIdx.x) & 15) + 2096)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3780)] * weight_shared[((((int)threadIdx.x) & 15) + 2112)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3781)] * weight_shared[((((int)threadIdx.x) & 15) + 2128)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3782)] * weight_shared[((((int)threadIdx.x) & 15) + 2144)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3783)] * weight_shared[((((int)threadIdx.x) & 15) + 2160)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3784)] * weight_shared[((((int)threadIdx.x) & 15) + 2176)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3785)] * weight_shared[((((int)threadIdx.x) & 15) + 2192)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3786)] * weight_shared[((((int)threadIdx.x) & 15) + 2208)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3787)] * weight_shared[((((int)threadIdx.x) & 15) + 2224)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3788)] * weight_shared[((((int)threadIdx.x) & 15) + 2240)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3789)] * weight_shared[((((int)threadIdx.x) & 15) + 2256)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3790)] * weight_shared[((((int)threadIdx.x) & 15) + 2272)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3791)] * weight_shared[((((int)threadIdx.x) & 15) + 2288)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3792)] * weight_shared[((((int)threadIdx.x) & 15) + 2304)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3793)] * weight_shared[((((int)threadIdx.x) & 15) + 2320)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3794)] * weight_shared[((((int)threadIdx.x) & 15) + 2336)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3795)] * weight_shared[((((int)threadIdx.x) & 15) + 2352)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3796)] * weight_shared[((((int)threadIdx.x) & 15) + 2368)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3797)] * weight_shared[((((int)threadIdx.x) & 15) + 2384)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3798)] * weight_shared[((((int)threadIdx.x) & 15) + 2400)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3799)] * weight_shared[((((int)threadIdx.x) & 15) + 2416)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3800)] * weight_shared[((((int)threadIdx.x) & 15) + 2432)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3801)] * weight_shared[((((int)threadIdx.x) & 15) + 2448)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3802)] * weight_shared[((((int)threadIdx.x) & 15) + 2464)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3803)] * weight_shared[((((int)threadIdx.x) & 15) + 2480)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3804)] * weight_shared[((((int)threadIdx.x) & 15) + 2496)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3805)] * weight_shared[((((int)threadIdx.x) & 15) + 2512)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3806)] * weight_shared[((((int)threadIdx.x) & 15) + 2528)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3807)] * weight_shared[((((int)threadIdx.x) & 15) + 2544)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3840)] * weight_shared[((((int)threadIdx.x) & 15) + 2048)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3841)] * weight_shared[((((int)threadIdx.x) & 15) + 2064)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3842)] * weight_shared[((((int)threadIdx.x) & 15) + 2080)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3843)] * weight_shared[((((int)threadIdx.x) & 15) + 2096)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3844)] * weight_shared[((((int)threadIdx.x) & 15) + 2112)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3845)] * weight_shared[((((int)threadIdx.x) & 15) + 2128)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3846)] * weight_shared[((((int)threadIdx.x) & 15) + 2144)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3847)] * weight_shared[((((int)threadIdx.x) & 15) + 2160)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3848)] * weight_shared[((((int)threadIdx.x) & 15) + 2176)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3849)] * weight_shared[((((int)threadIdx.x) & 15) + 2192)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3850)] * weight_shared[((((int)threadIdx.x) & 15) + 2208)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3851)] * weight_shared[((((int)threadIdx.x) & 15) + 2224)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3852)] * weight_shared[((((int)threadIdx.x) & 15) + 2240)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3853)] * weight_shared[((((int)threadIdx.x) & 15) + 2256)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3854)] * weight_shared[((((int)threadIdx.x) & 15) + 2272)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3855)] * weight_shared[((((int)threadIdx.x) & 15) + 2288)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3856)] * weight_shared[((((int)threadIdx.x) & 15) + 2304)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3857)] * weight_shared[((((int)threadIdx.x) & 15) + 2320)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3858)] * weight_shared[((((int)threadIdx.x) & 15) + 2336)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3859)] * weight_shared[((((int)threadIdx.x) & 15) + 2352)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3860)] * weight_shared[((((int)threadIdx.x) & 15) + 2368)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3861)] * weight_shared[((((int)threadIdx.x) & 15) + 2384)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3862)] * weight_shared[((((int)threadIdx.x) & 15) + 2400)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3863)] * weight_shared[((((int)threadIdx.x) & 15) + 2416)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3864)] * weight_shared[((((int)threadIdx.x) & 15) + 2432)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3865)] * weight_shared[((((int)threadIdx.x) & 15) + 2448)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3866)] * weight_shared[((((int)threadIdx.x) & 15) + 2464)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3867)] * weight_shared[((((int)threadIdx.x) & 15) + 2480)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3868)] * weight_shared[((((int)threadIdx.x) & 15) + 2496)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3869)] * weight_shared[((((int)threadIdx.x) & 15) + 2512)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3870)] * weight_shared[((((int)threadIdx.x) & 15) + 2528)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3871)] * weight_shared[((((int)threadIdx.x) & 15) + 2544)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3904)] * weight_shared[((((int)threadIdx.x) & 15) + 2048)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3905)] * weight_shared[((((int)threadIdx.x) & 15) + 2064)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3906)] * weight_shared[((((int)threadIdx.x) & 15) + 2080)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3907)] * weight_shared[((((int)threadIdx.x) & 15) + 2096)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3908)] * weight_shared[((((int)threadIdx.x) & 15) + 2112)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3909)] * weight_shared[((((int)threadIdx.x) & 15) + 2128)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3910)] * weight_shared[((((int)threadIdx.x) & 15) + 2144)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3911)] * weight_shared[((((int)threadIdx.x) & 15) + 2160)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3912)] * weight_shared[((((int)threadIdx.x) & 15) + 2176)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3913)] * weight_shared[((((int)threadIdx.x) & 15) + 2192)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3914)] * weight_shared[((((int)threadIdx.x) & 15) + 2208)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3915)] * weight_shared[((((int)threadIdx.x) & 15) + 2224)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3916)] * weight_shared[((((int)threadIdx.x) & 15) + 2240)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3917)] * weight_shared[((((int)threadIdx.x) & 15) + 2256)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3918)] * weight_shared[((((int)threadIdx.x) & 15) + 2272)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3919)] * weight_shared[((((int)threadIdx.x) & 15) + 2288)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3920)] * weight_shared[((((int)threadIdx.x) & 15) + 2304)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3921)] * weight_shared[((((int)threadIdx.x) & 15) + 2320)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3922)] * weight_shared[((((int)threadIdx.x) & 15) + 2336)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3923)] * weight_shared[((((int)threadIdx.x) & 15) + 2352)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3924)] * weight_shared[((((int)threadIdx.x) & 15) + 2368)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3925)] * weight_shared[((((int)threadIdx.x) & 15) + 2384)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3926)] * weight_shared[((((int)threadIdx.x) & 15) + 2400)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3927)] * weight_shared[((((int)threadIdx.x) & 15) + 2416)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3928)] * weight_shared[((((int)threadIdx.x) & 15) + 2432)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3929)] * weight_shared[((((int)threadIdx.x) & 15) + 2448)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3930)] * weight_shared[((((int)threadIdx.x) & 15) + 2464)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3931)] * weight_shared[((((int)threadIdx.x) & 15) + 2480)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3932)] * weight_shared[((((int)threadIdx.x) & 15) + 2496)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3933)] * weight_shared[((((int)threadIdx.x) & 15) + 2512)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3934)] * weight_shared[((((int)threadIdx.x) & 15) + 2528)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3935)] * weight_shared[((((int)threadIdx.x) & 15) + 2544)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3968)] * weight_shared[((((int)threadIdx.x) & 15) + 2048)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3969)] * weight_shared[((((int)threadIdx.x) & 15) + 2064)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3970)] * weight_shared[((((int)threadIdx.x) & 15) + 2080)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3971)] * weight_shared[((((int)threadIdx.x) & 15) + 2096)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3972)] * weight_shared[((((int)threadIdx.x) & 15) + 2112)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3973)] * weight_shared[((((int)threadIdx.x) & 15) + 2128)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3974)] * weight_shared[((((int)threadIdx.x) & 15) + 2144)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3975)] * weight_shared[((((int)threadIdx.x) & 15) + 2160)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3976)] * weight_shared[((((int)threadIdx.x) & 15) + 2176)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3977)] * weight_shared[((((int)threadIdx.x) & 15) + 2192)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3978)] * weight_shared[((((int)threadIdx.x) & 15) + 2208)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3979)] * weight_shared[((((int)threadIdx.x) & 15) + 2224)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3980)] * weight_shared[((((int)threadIdx.x) & 15) + 2240)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3981)] * weight_shared[((((int)threadIdx.x) & 15) + 2256)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3982)] * weight_shared[((((int)threadIdx.x) & 15) + 2272)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3983)] * weight_shared[((((int)threadIdx.x) & 15) + 2288)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3984)] * weight_shared[((((int)threadIdx.x) & 15) + 2304)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3985)] * weight_shared[((((int)threadIdx.x) & 15) + 2320)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3986)] * weight_shared[((((int)threadIdx.x) & 15) + 2336)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3987)] * weight_shared[((((int)threadIdx.x) & 15) + 2352)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3988)] * weight_shared[((((int)threadIdx.x) & 15) + 2368)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3989)] * weight_shared[((((int)threadIdx.x) & 15) + 2384)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3990)] * weight_shared[((((int)threadIdx.x) & 15) + 2400)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3991)] * weight_shared[((((int)threadIdx.x) & 15) + 2416)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3992)] * weight_shared[((((int)threadIdx.x) & 15) + 2432)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3993)] * weight_shared[((((int)threadIdx.x) & 15) + 2448)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3994)] * weight_shared[((((int)threadIdx.x) & 15) + 2464)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3995)] * weight_shared[((((int)threadIdx.x) & 15) + 2480)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3996)] * weight_shared[((((int)threadIdx.x) & 15) + 2496)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3997)] * weight_shared[((((int)threadIdx.x) & 15) + 2512)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3998)] * weight_shared[((((int)threadIdx.x) & 15) + 2528)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3999)] * weight_shared[((((int)threadIdx.x) & 15) + 2544)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3616)] * weight_shared[((((int)threadIdx.x) & 15) + 2560)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3617)] * weight_shared[((((int)threadIdx.x) & 15) + 2576)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3618)] * weight_shared[((((int)threadIdx.x) & 15) + 2592)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3619)] * weight_shared[((((int)threadIdx.x) & 15) + 2608)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3620)] * weight_shared[((((int)threadIdx.x) & 15) + 2624)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3621)] * weight_shared[((((int)threadIdx.x) & 15) + 2640)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3622)] * weight_shared[((((int)threadIdx.x) & 15) + 2656)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3623)] * weight_shared[((((int)threadIdx.x) & 15) + 2672)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3624)] * weight_shared[((((int)threadIdx.x) & 15) + 2688)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3625)] * weight_shared[((((int)threadIdx.x) & 15) + 2704)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3626)] * weight_shared[((((int)threadIdx.x) & 15) + 2720)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3627)] * weight_shared[((((int)threadIdx.x) & 15) + 2736)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3628)] * weight_shared[((((int)threadIdx.x) & 15) + 2752)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3629)] * weight_shared[((((int)threadIdx.x) & 15) + 2768)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3630)] * weight_shared[((((int)threadIdx.x) & 15) + 2784)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3631)] * weight_shared[((((int)threadIdx.x) & 15) + 2800)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3632)] * weight_shared[((((int)threadIdx.x) & 15) + 2816)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3633)] * weight_shared[((((int)threadIdx.x) & 15) + 2832)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3634)] * weight_shared[((((int)threadIdx.x) & 15) + 2848)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3635)] * weight_shared[((((int)threadIdx.x) & 15) + 2864)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3636)] * weight_shared[((((int)threadIdx.x) & 15) + 2880)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3637)] * weight_shared[((((int)threadIdx.x) & 15) + 2896)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3638)] * weight_shared[((((int)threadIdx.x) & 15) + 2912)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3639)] * weight_shared[((((int)threadIdx.x) & 15) + 2928)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3640)] * weight_shared[((((int)threadIdx.x) & 15) + 2944)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3641)] * weight_shared[((((int)threadIdx.x) & 15) + 2960)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3642)] * weight_shared[((((int)threadIdx.x) & 15) + 2976)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3643)] * weight_shared[((((int)threadIdx.x) & 15) + 2992)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3644)] * weight_shared[((((int)threadIdx.x) & 15) + 3008)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3645)] * weight_shared[((((int)threadIdx.x) & 15) + 3024)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3646)] * weight_shared[((((int)threadIdx.x) & 15) + 3040)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3647)] * weight_shared[((((int)threadIdx.x) & 15) + 3056)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3680)] * weight_shared[((((int)threadIdx.x) & 15) + 2560)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3681)] * weight_shared[((((int)threadIdx.x) & 15) + 2576)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3682)] * weight_shared[((((int)threadIdx.x) & 15) + 2592)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3683)] * weight_shared[((((int)threadIdx.x) & 15) + 2608)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3684)] * weight_shared[((((int)threadIdx.x) & 15) + 2624)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3685)] * weight_shared[((((int)threadIdx.x) & 15) + 2640)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3686)] * weight_shared[((((int)threadIdx.x) & 15) + 2656)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3687)] * weight_shared[((((int)threadIdx.x) & 15) + 2672)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3688)] * weight_shared[((((int)threadIdx.x) & 15) + 2688)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3689)] * weight_shared[((((int)threadIdx.x) & 15) + 2704)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3690)] * weight_shared[((((int)threadIdx.x) & 15) + 2720)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3691)] * weight_shared[((((int)threadIdx.x) & 15) + 2736)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3692)] * weight_shared[((((int)threadIdx.x) & 15) + 2752)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3693)] * weight_shared[((((int)threadIdx.x) & 15) + 2768)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3694)] * weight_shared[((((int)threadIdx.x) & 15) + 2784)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3695)] * weight_shared[((((int)threadIdx.x) & 15) + 2800)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3696)] * weight_shared[((((int)threadIdx.x) & 15) + 2816)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3697)] * weight_shared[((((int)threadIdx.x) & 15) + 2832)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3698)] * weight_shared[((((int)threadIdx.x) & 15) + 2848)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3699)] * weight_shared[((((int)threadIdx.x) & 15) + 2864)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3700)] * weight_shared[((((int)threadIdx.x) & 15) + 2880)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3701)] * weight_shared[((((int)threadIdx.x) & 15) + 2896)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3702)] * weight_shared[((((int)threadIdx.x) & 15) + 2912)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3703)] * weight_shared[((((int)threadIdx.x) & 15) + 2928)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3704)] * weight_shared[((((int)threadIdx.x) & 15) + 2944)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3705)] * weight_shared[((((int)threadIdx.x) & 15) + 2960)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3706)] * weight_shared[((((int)threadIdx.x) & 15) + 2976)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3707)] * weight_shared[((((int)threadIdx.x) & 15) + 2992)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3708)] * weight_shared[((((int)threadIdx.x) & 15) + 3008)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3709)] * weight_shared[((((int)threadIdx.x) & 15) + 3024)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3710)] * weight_shared[((((int)threadIdx.x) & 15) + 3040)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3711)] * weight_shared[((((int)threadIdx.x) & 15) + 3056)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3744)] * weight_shared[((((int)threadIdx.x) & 15) + 2560)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3745)] * weight_shared[((((int)threadIdx.x) & 15) + 2576)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3746)] * weight_shared[((((int)threadIdx.x) & 15) + 2592)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3747)] * weight_shared[((((int)threadIdx.x) & 15) + 2608)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3748)] * weight_shared[((((int)threadIdx.x) & 15) + 2624)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3749)] * weight_shared[((((int)threadIdx.x) & 15) + 2640)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3750)] * weight_shared[((((int)threadIdx.x) & 15) + 2656)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3751)] * weight_shared[((((int)threadIdx.x) & 15) + 2672)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3752)] * weight_shared[((((int)threadIdx.x) & 15) + 2688)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3753)] * weight_shared[((((int)threadIdx.x) & 15) + 2704)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3754)] * weight_shared[((((int)threadIdx.x) & 15) + 2720)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3755)] * weight_shared[((((int)threadIdx.x) & 15) + 2736)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3756)] * weight_shared[((((int)threadIdx.x) & 15) + 2752)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3757)] * weight_shared[((((int)threadIdx.x) & 15) + 2768)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3758)] * weight_shared[((((int)threadIdx.x) & 15) + 2784)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3759)] * weight_shared[((((int)threadIdx.x) & 15) + 2800)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3760)] * weight_shared[((((int)threadIdx.x) & 15) + 2816)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3761)] * weight_shared[((((int)threadIdx.x) & 15) + 2832)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3762)] * weight_shared[((((int)threadIdx.x) & 15) + 2848)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3763)] * weight_shared[((((int)threadIdx.x) & 15) + 2864)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3764)] * weight_shared[((((int)threadIdx.x) & 15) + 2880)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3765)] * weight_shared[((((int)threadIdx.x) & 15) + 2896)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3766)] * weight_shared[((((int)threadIdx.x) & 15) + 2912)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3767)] * weight_shared[((((int)threadIdx.x) & 15) + 2928)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3768)] * weight_shared[((((int)threadIdx.x) & 15) + 2944)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3769)] * weight_shared[((((int)threadIdx.x) & 15) + 2960)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3770)] * weight_shared[((((int)threadIdx.x) & 15) + 2976)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3771)] * weight_shared[((((int)threadIdx.x) & 15) + 2992)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3772)] * weight_shared[((((int)threadIdx.x) & 15) + 3008)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3773)] * weight_shared[((((int)threadIdx.x) & 15) + 3024)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3774)] * weight_shared[((((int)threadIdx.x) & 15) + 3040)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3775)] * weight_shared[((((int)threadIdx.x) & 15) + 3056)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3808)] * weight_shared[((((int)threadIdx.x) & 15) + 2560)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3809)] * weight_shared[((((int)threadIdx.x) & 15) + 2576)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3810)] * weight_shared[((((int)threadIdx.x) & 15) + 2592)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3811)] * weight_shared[((((int)threadIdx.x) & 15) + 2608)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3812)] * weight_shared[((((int)threadIdx.x) & 15) + 2624)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3813)] * weight_shared[((((int)threadIdx.x) & 15) + 2640)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3814)] * weight_shared[((((int)threadIdx.x) & 15) + 2656)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3815)] * weight_shared[((((int)threadIdx.x) & 15) + 2672)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3816)] * weight_shared[((((int)threadIdx.x) & 15) + 2688)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3817)] * weight_shared[((((int)threadIdx.x) & 15) + 2704)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3818)] * weight_shared[((((int)threadIdx.x) & 15) + 2720)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3819)] * weight_shared[((((int)threadIdx.x) & 15) + 2736)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3820)] * weight_shared[((((int)threadIdx.x) & 15) + 2752)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3821)] * weight_shared[((((int)threadIdx.x) & 15) + 2768)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3822)] * weight_shared[((((int)threadIdx.x) & 15) + 2784)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3823)] * weight_shared[((((int)threadIdx.x) & 15) + 2800)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3824)] * weight_shared[((((int)threadIdx.x) & 15) + 2816)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3825)] * weight_shared[((((int)threadIdx.x) & 15) + 2832)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3826)] * weight_shared[((((int)threadIdx.x) & 15) + 2848)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3827)] * weight_shared[((((int)threadIdx.x) & 15) + 2864)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3828)] * weight_shared[((((int)threadIdx.x) & 15) + 2880)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3829)] * weight_shared[((((int)threadIdx.x) & 15) + 2896)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3830)] * weight_shared[((((int)threadIdx.x) & 15) + 2912)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3831)] * weight_shared[((((int)threadIdx.x) & 15) + 2928)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3832)] * weight_shared[((((int)threadIdx.x) & 15) + 2944)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3833)] * weight_shared[((((int)threadIdx.x) & 15) + 2960)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3834)] * weight_shared[((((int)threadIdx.x) & 15) + 2976)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3835)] * weight_shared[((((int)threadIdx.x) & 15) + 2992)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3836)] * weight_shared[((((int)threadIdx.x) & 15) + 3008)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3837)] * weight_shared[((((int)threadIdx.x) & 15) + 3024)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3838)] * weight_shared[((((int)threadIdx.x) & 15) + 3040)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3839)] * weight_shared[((((int)threadIdx.x) & 15) + 3056)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3872)] * weight_shared[((((int)threadIdx.x) & 15) + 2560)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3873)] * weight_shared[((((int)threadIdx.x) & 15) + 2576)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3874)] * weight_shared[((((int)threadIdx.x) & 15) + 2592)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3875)] * weight_shared[((((int)threadIdx.x) & 15) + 2608)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3876)] * weight_shared[((((int)threadIdx.x) & 15) + 2624)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3877)] * weight_shared[((((int)threadIdx.x) & 15) + 2640)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3878)] * weight_shared[((((int)threadIdx.x) & 15) + 2656)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3879)] * weight_shared[((((int)threadIdx.x) & 15) + 2672)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3880)] * weight_shared[((((int)threadIdx.x) & 15) + 2688)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3881)] * weight_shared[((((int)threadIdx.x) & 15) + 2704)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3882)] * weight_shared[((((int)threadIdx.x) & 15) + 2720)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3883)] * weight_shared[((((int)threadIdx.x) & 15) + 2736)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3884)] * weight_shared[((((int)threadIdx.x) & 15) + 2752)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3885)] * weight_shared[((((int)threadIdx.x) & 15) + 2768)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3886)] * weight_shared[((((int)threadIdx.x) & 15) + 2784)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3887)] * weight_shared[((((int)threadIdx.x) & 15) + 2800)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3888)] * weight_shared[((((int)threadIdx.x) & 15) + 2816)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3889)] * weight_shared[((((int)threadIdx.x) & 15) + 2832)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3890)] * weight_shared[((((int)threadIdx.x) & 15) + 2848)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3891)] * weight_shared[((((int)threadIdx.x) & 15) + 2864)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3892)] * weight_shared[((((int)threadIdx.x) & 15) + 2880)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3893)] * weight_shared[((((int)threadIdx.x) & 15) + 2896)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3894)] * weight_shared[((((int)threadIdx.x) & 15) + 2912)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3895)] * weight_shared[((((int)threadIdx.x) & 15) + 2928)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3896)] * weight_shared[((((int)threadIdx.x) & 15) + 2944)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3897)] * weight_shared[((((int)threadIdx.x) & 15) + 2960)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3898)] * weight_shared[((((int)threadIdx.x) & 15) + 2976)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3899)] * weight_shared[((((int)threadIdx.x) & 15) + 2992)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3900)] * weight_shared[((((int)threadIdx.x) & 15) + 3008)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3901)] * weight_shared[((((int)threadIdx.x) & 15) + 3024)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3902)] * weight_shared[((((int)threadIdx.x) & 15) + 3040)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3903)] * weight_shared[((((int)threadIdx.x) & 15) + 3056)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3936)] * weight_shared[((((int)threadIdx.x) & 15) + 2560)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3937)] * weight_shared[((((int)threadIdx.x) & 15) + 2576)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3938)] * weight_shared[((((int)threadIdx.x) & 15) + 2592)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3939)] * weight_shared[((((int)threadIdx.x) & 15) + 2608)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3940)] * weight_shared[((((int)threadIdx.x) & 15) + 2624)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3941)] * weight_shared[((((int)threadIdx.x) & 15) + 2640)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3942)] * weight_shared[((((int)threadIdx.x) & 15) + 2656)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3943)] * weight_shared[((((int)threadIdx.x) & 15) + 2672)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3944)] * weight_shared[((((int)threadIdx.x) & 15) + 2688)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3945)] * weight_shared[((((int)threadIdx.x) & 15) + 2704)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3946)] * weight_shared[((((int)threadIdx.x) & 15) + 2720)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3947)] * weight_shared[((((int)threadIdx.x) & 15) + 2736)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3948)] * weight_shared[((((int)threadIdx.x) & 15) + 2752)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3949)] * weight_shared[((((int)threadIdx.x) & 15) + 2768)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3950)] * weight_shared[((((int)threadIdx.x) & 15) + 2784)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3951)] * weight_shared[((((int)threadIdx.x) & 15) + 2800)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3952)] * weight_shared[((((int)threadIdx.x) & 15) + 2816)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3953)] * weight_shared[((((int)threadIdx.x) & 15) + 2832)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3954)] * weight_shared[((((int)threadIdx.x) & 15) + 2848)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3955)] * weight_shared[((((int)threadIdx.x) & 15) + 2864)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3956)] * weight_shared[((((int)threadIdx.x) & 15) + 2880)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3957)] * weight_shared[((((int)threadIdx.x) & 15) + 2896)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3958)] * weight_shared[((((int)threadIdx.x) & 15) + 2912)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3959)] * weight_shared[((((int)threadIdx.x) & 15) + 2928)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3960)] * weight_shared[((((int)threadIdx.x) & 15) + 2944)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3961)] * weight_shared[((((int)threadIdx.x) & 15) + 2960)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3962)] * weight_shared[((((int)threadIdx.x) & 15) + 2976)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3963)] * weight_shared[((((int)threadIdx.x) & 15) + 2992)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3964)] * weight_shared[((((int)threadIdx.x) & 15) + 3008)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3965)] * weight_shared[((((int)threadIdx.x) & 15) + 3024)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3966)] * weight_shared[((((int)threadIdx.x) & 15) + 3040)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3967)] * weight_shared[((((int)threadIdx.x) & 15) + 3056)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4000)] * weight_shared[((((int)threadIdx.x) & 15) + 2560)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4001)] * weight_shared[((((int)threadIdx.x) & 15) + 2576)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4002)] * weight_shared[((((int)threadIdx.x) & 15) + 2592)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4003)] * weight_shared[((((int)threadIdx.x) & 15) + 2608)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4004)] * weight_shared[((((int)threadIdx.x) & 15) + 2624)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4005)] * weight_shared[((((int)threadIdx.x) & 15) + 2640)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4006)] * weight_shared[((((int)threadIdx.x) & 15) + 2656)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4007)] * weight_shared[((((int)threadIdx.x) & 15) + 2672)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4008)] * weight_shared[((((int)threadIdx.x) & 15) + 2688)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4009)] * weight_shared[((((int)threadIdx.x) & 15) + 2704)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4010)] * weight_shared[((((int)threadIdx.x) & 15) + 2720)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4011)] * weight_shared[((((int)threadIdx.x) & 15) + 2736)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4012)] * weight_shared[((((int)threadIdx.x) & 15) + 2752)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4013)] * weight_shared[((((int)threadIdx.x) & 15) + 2768)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4014)] * weight_shared[((((int)threadIdx.x) & 15) + 2784)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4015)] * weight_shared[((((int)threadIdx.x) & 15) + 2800)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4016)] * weight_shared[((((int)threadIdx.x) & 15) + 2816)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4017)] * weight_shared[((((int)threadIdx.x) & 15) + 2832)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4018)] * weight_shared[((((int)threadIdx.x) & 15) + 2848)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4019)] * weight_shared[((((int)threadIdx.x) & 15) + 2864)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4020)] * weight_shared[((((int)threadIdx.x) & 15) + 2880)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4021)] * weight_shared[((((int)threadIdx.x) & 15) + 2896)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4022)] * weight_shared[((((int)threadIdx.x) & 15) + 2912)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4023)] * weight_shared[((((int)threadIdx.x) & 15) + 2928)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4024)] * weight_shared[((((int)threadIdx.x) & 15) + 2944)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4025)] * weight_shared[((((int)threadIdx.x) & 15) + 2960)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4026)] * weight_shared[((((int)threadIdx.x) & 15) + 2976)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4027)] * weight_shared[((((int)threadIdx.x) & 15) + 2992)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4028)] * weight_shared[((((int)threadIdx.x) & 15) + 3008)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4029)] * weight_shared[((((int)threadIdx.x) & 15) + 3024)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4030)] * weight_shared[((((int)threadIdx.x) & 15) + 3040)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4031)] * weight_shared[((((int)threadIdx.x) & 15) + 3056)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((int)threadIdx.x) >> 4) * 448)] * weight_shared[(((int)threadIdx.x) & 15)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 1)] * weight_shared[((((int)threadIdx.x) & 15) + 16)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 2)] * weight_shared[((((int)threadIdx.x) & 15) + 32)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 3)] * weight_shared[((((int)threadIdx.x) & 15) + 48)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 4)] * weight_shared[((((int)threadIdx.x) & 15) + 64)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 5)] * weight_shared[((((int)threadIdx.x) & 15) + 80)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 6)] * weight_shared[((((int)threadIdx.x) & 15) + 96)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 7)] * weight_shared[((((int)threadIdx.x) & 15) + 112)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 8)] * weight_shared[((((int)threadIdx.x) & 15) + 128)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 9)] * weight_shared[((((int)threadIdx.x) & 15) + 144)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 10)] * weight_shared[((((int)threadIdx.x) & 15) + 160)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 11)] * weight_shared[((((int)threadIdx.x) & 15) + 176)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 12)] * weight_shared[((((int)threadIdx.x) & 15) + 192)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 13)] * weight_shared[((((int)threadIdx.x) & 15) + 208)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 14)] * weight_shared[((((int)threadIdx.x) & 15) + 224)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 15)] * weight_shared[((((int)threadIdx.x) & 15) + 240)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 16)] * weight_shared[((((int)threadIdx.x) & 15) + 256)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 17)] * weight_shared[((((int)threadIdx.x) & 15) + 272)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 18)] * weight_shared[((((int)threadIdx.x) & 15) + 288)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 19)] * weight_shared[((((int)threadIdx.x) & 15) + 304)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 20)] * weight_shared[((((int)threadIdx.x) & 15) + 320)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 21)] * weight_shared[((((int)threadIdx.x) & 15) + 336)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 22)] * weight_shared[((((int)threadIdx.x) & 15) + 352)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 23)] * weight_shared[((((int)threadIdx.x) & 15) + 368)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 24)] * weight_shared[((((int)threadIdx.x) & 15) + 384)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 25)] * weight_shared[((((int)threadIdx.x) & 15) + 400)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 26)] * weight_shared[((((int)threadIdx.x) & 15) + 416)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 27)] * weight_shared[((((int)threadIdx.x) & 15) + 432)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 28)] * weight_shared[((((int)threadIdx.x) & 15) + 448)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 29)] * weight_shared[((((int)threadIdx.x) & 15) + 464)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 30)] * weight_shared[((((int)threadIdx.x) & 15) + 480)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 31)] * weight_shared[((((int)threadIdx.x) & 15) + 496)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 64)] * weight_shared[(((int)threadIdx.x) & 15)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 65)] * weight_shared[((((int)threadIdx.x) & 15) + 16)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 66)] * weight_shared[((((int)threadIdx.x) & 15) + 32)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 67)] * weight_shared[((((int)threadIdx.x) & 15) + 48)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 68)] * weight_shared[((((int)threadIdx.x) & 15) + 64)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 69)] * weight_shared[((((int)threadIdx.x) & 15) + 80)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 70)] * weight_shared[((((int)threadIdx.x) & 15) + 96)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 71)] * weight_shared[((((int)threadIdx.x) & 15) + 112)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 72)] * weight_shared[((((int)threadIdx.x) & 15) + 128)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 73)] * weight_shared[((((int)threadIdx.x) & 15) + 144)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 74)] * weight_shared[((((int)threadIdx.x) & 15) + 160)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 75)] * weight_shared[((((int)threadIdx.x) & 15) + 176)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 76)] * weight_shared[((((int)threadIdx.x) & 15) + 192)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 77)] * weight_shared[((((int)threadIdx.x) & 15) + 208)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 78)] * weight_shared[((((int)threadIdx.x) & 15) + 224)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 79)] * weight_shared[((((int)threadIdx.x) & 15) + 240)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 80)] * weight_shared[((((int)threadIdx.x) & 15) + 256)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 81)] * weight_shared[((((int)threadIdx.x) & 15) + 272)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 82)] * weight_shared[((((int)threadIdx.x) & 15) + 288)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 83)] * weight_shared[((((int)threadIdx.x) & 15) + 304)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 84)] * weight_shared[((((int)threadIdx.x) & 15) + 320)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 85)] * weight_shared[((((int)threadIdx.x) & 15) + 336)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 86)] * weight_shared[((((int)threadIdx.x) & 15) + 352)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 87)] * weight_shared[((((int)threadIdx.x) & 15) + 368)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 88)] * weight_shared[((((int)threadIdx.x) & 15) + 384)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 89)] * weight_shared[((((int)threadIdx.x) & 15) + 400)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 90)] * weight_shared[((((int)threadIdx.x) & 15) + 416)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 91)] * weight_shared[((((int)threadIdx.x) & 15) + 432)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 92)] * weight_shared[((((int)threadIdx.x) & 15) + 448)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 93)] * weight_shared[((((int)threadIdx.x) & 15) + 464)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 94)] * weight_shared[((((int)threadIdx.x) & 15) + 480)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 95)] * weight_shared[((((int)threadIdx.x) & 15) + 496)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 128)] * weight_shared[(((int)threadIdx.x) & 15)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 129)] * weight_shared[((((int)threadIdx.x) & 15) + 16)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 130)] * weight_shared[((((int)threadIdx.x) & 15) + 32)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 131)] * weight_shared[((((int)threadIdx.x) & 15) + 48)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 132)] * weight_shared[((((int)threadIdx.x) & 15) + 64)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 133)] * weight_shared[((((int)threadIdx.x) & 15) + 80)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 134)] * weight_shared[((((int)threadIdx.x) & 15) + 96)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 135)] * weight_shared[((((int)threadIdx.x) & 15) + 112)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 136)] * weight_shared[((((int)threadIdx.x) & 15) + 128)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 137)] * weight_shared[((((int)threadIdx.x) & 15) + 144)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 138)] * weight_shared[((((int)threadIdx.x) & 15) + 160)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 139)] * weight_shared[((((int)threadIdx.x) & 15) + 176)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 140)] * weight_shared[((((int)threadIdx.x) & 15) + 192)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 141)] * weight_shared[((((int)threadIdx.x) & 15) + 208)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 142)] * weight_shared[((((int)threadIdx.x) & 15) + 224)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 143)] * weight_shared[((((int)threadIdx.x) & 15) + 240)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 144)] * weight_shared[((((int)threadIdx.x) & 15) + 256)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 145)] * weight_shared[((((int)threadIdx.x) & 15) + 272)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 146)] * weight_shared[((((int)threadIdx.x) & 15) + 288)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 147)] * weight_shared[((((int)threadIdx.x) & 15) + 304)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 148)] * weight_shared[((((int)threadIdx.x) & 15) + 320)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 149)] * weight_shared[((((int)threadIdx.x) & 15) + 336)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 150)] * weight_shared[((((int)threadIdx.x) & 15) + 352)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 151)] * weight_shared[((((int)threadIdx.x) & 15) + 368)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 152)] * weight_shared[((((int)threadIdx.x) & 15) + 384)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 153)] * weight_shared[((((int)threadIdx.x) & 15) + 400)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 154)] * weight_shared[((((int)threadIdx.x) & 15) + 416)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 155)] * weight_shared[((((int)threadIdx.x) & 15) + 432)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 156)] * weight_shared[((((int)threadIdx.x) & 15) + 448)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 157)] * weight_shared[((((int)threadIdx.x) & 15) + 464)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 158)] * weight_shared[((((int)threadIdx.x) & 15) + 480)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 159)] * weight_shared[((((int)threadIdx.x) & 15) + 496)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 192)] * weight_shared[(((int)threadIdx.x) & 15)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 193)] * weight_shared[((((int)threadIdx.x) & 15) + 16)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 194)] * weight_shared[((((int)threadIdx.x) & 15) + 32)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 195)] * weight_shared[((((int)threadIdx.x) & 15) + 48)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 196)] * weight_shared[((((int)threadIdx.x) & 15) + 64)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 197)] * weight_shared[((((int)threadIdx.x) & 15) + 80)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 198)] * weight_shared[((((int)threadIdx.x) & 15) + 96)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 199)] * weight_shared[((((int)threadIdx.x) & 15) + 112)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 200)] * weight_shared[((((int)threadIdx.x) & 15) + 128)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 201)] * weight_shared[((((int)threadIdx.x) & 15) + 144)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 202)] * weight_shared[((((int)threadIdx.x) & 15) + 160)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 203)] * weight_shared[((((int)threadIdx.x) & 15) + 176)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 204)] * weight_shared[((((int)threadIdx.x) & 15) + 192)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 205)] * weight_shared[((((int)threadIdx.x) & 15) + 208)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 206)] * weight_shared[((((int)threadIdx.x) & 15) + 224)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 207)] * weight_shared[((((int)threadIdx.x) & 15) + 240)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 208)] * weight_shared[((((int)threadIdx.x) & 15) + 256)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 209)] * weight_shared[((((int)threadIdx.x) & 15) + 272)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 210)] * weight_shared[((((int)threadIdx.x) & 15) + 288)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 211)] * weight_shared[((((int)threadIdx.x) & 15) + 304)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 212)] * weight_shared[((((int)threadIdx.x) & 15) + 320)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 213)] * weight_shared[((((int)threadIdx.x) & 15) + 336)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 214)] * weight_shared[((((int)threadIdx.x) & 15) + 352)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 215)] * weight_shared[((((int)threadIdx.x) & 15) + 368)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 216)] * weight_shared[((((int)threadIdx.x) & 15) + 384)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 217)] * weight_shared[((((int)threadIdx.x) & 15) + 400)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 218)] * weight_shared[((((int)threadIdx.x) & 15) + 416)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 219)] * weight_shared[((((int)threadIdx.x) & 15) + 432)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 220)] * weight_shared[((((int)threadIdx.x) & 15) + 448)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 221)] * weight_shared[((((int)threadIdx.x) & 15) + 464)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 222)] * weight_shared[((((int)threadIdx.x) & 15) + 480)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 223)] * weight_shared[((((int)threadIdx.x) & 15) + 496)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 256)] * weight_shared[(((int)threadIdx.x) & 15)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 257)] * weight_shared[((((int)threadIdx.x) & 15) + 16)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 258)] * weight_shared[((((int)threadIdx.x) & 15) + 32)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 259)] * weight_shared[((((int)threadIdx.x) & 15) + 48)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 260)] * weight_shared[((((int)threadIdx.x) & 15) + 64)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 261)] * weight_shared[((((int)threadIdx.x) & 15) + 80)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 262)] * weight_shared[((((int)threadIdx.x) & 15) + 96)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 263)] * weight_shared[((((int)threadIdx.x) & 15) + 112)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 264)] * weight_shared[((((int)threadIdx.x) & 15) + 128)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 265)] * weight_shared[((((int)threadIdx.x) & 15) + 144)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 266)] * weight_shared[((((int)threadIdx.x) & 15) + 160)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 267)] * weight_shared[((((int)threadIdx.x) & 15) + 176)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 268)] * weight_shared[((((int)threadIdx.x) & 15) + 192)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 269)] * weight_shared[((((int)threadIdx.x) & 15) + 208)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 270)] * weight_shared[((((int)threadIdx.x) & 15) + 224)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 271)] * weight_shared[((((int)threadIdx.x) & 15) + 240)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 272)] * weight_shared[((((int)threadIdx.x) & 15) + 256)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 273)] * weight_shared[((((int)threadIdx.x) & 15) + 272)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 274)] * weight_shared[((((int)threadIdx.x) & 15) + 288)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 275)] * weight_shared[((((int)threadIdx.x) & 15) + 304)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 276)] * weight_shared[((((int)threadIdx.x) & 15) + 320)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 277)] * weight_shared[((((int)threadIdx.x) & 15) + 336)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 278)] * weight_shared[((((int)threadIdx.x) & 15) + 352)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 279)] * weight_shared[((((int)threadIdx.x) & 15) + 368)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 280)] * weight_shared[((((int)threadIdx.x) & 15) + 384)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 281)] * weight_shared[((((int)threadIdx.x) & 15) + 400)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 282)] * weight_shared[((((int)threadIdx.x) & 15) + 416)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 283)] * weight_shared[((((int)threadIdx.x) & 15) + 432)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 284)] * weight_shared[((((int)threadIdx.x) & 15) + 448)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 285)] * weight_shared[((((int)threadIdx.x) & 15) + 464)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 286)] * weight_shared[((((int)threadIdx.x) & 15) + 480)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 287)] * weight_shared[((((int)threadIdx.x) & 15) + 496)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 320)] * weight_shared[(((int)threadIdx.x) & 15)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 321)] * weight_shared[((((int)threadIdx.x) & 15) + 16)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 322)] * weight_shared[((((int)threadIdx.x) & 15) + 32)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 323)] * weight_shared[((((int)threadIdx.x) & 15) + 48)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 324)] * weight_shared[((((int)threadIdx.x) & 15) + 64)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 325)] * weight_shared[((((int)threadIdx.x) & 15) + 80)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 326)] * weight_shared[((((int)threadIdx.x) & 15) + 96)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 327)] * weight_shared[((((int)threadIdx.x) & 15) + 112)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 328)] * weight_shared[((((int)threadIdx.x) & 15) + 128)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 329)] * weight_shared[((((int)threadIdx.x) & 15) + 144)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 330)] * weight_shared[((((int)threadIdx.x) & 15) + 160)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 331)] * weight_shared[((((int)threadIdx.x) & 15) + 176)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 332)] * weight_shared[((((int)threadIdx.x) & 15) + 192)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 333)] * weight_shared[((((int)threadIdx.x) & 15) + 208)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 334)] * weight_shared[((((int)threadIdx.x) & 15) + 224)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 335)] * weight_shared[((((int)threadIdx.x) & 15) + 240)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 336)] * weight_shared[((((int)threadIdx.x) & 15) + 256)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 337)] * weight_shared[((((int)threadIdx.x) & 15) + 272)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 338)] * weight_shared[((((int)threadIdx.x) & 15) + 288)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 339)] * weight_shared[((((int)threadIdx.x) & 15) + 304)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 340)] * weight_shared[((((int)threadIdx.x) & 15) + 320)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 341)] * weight_shared[((((int)threadIdx.x) & 15) + 336)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 342)] * weight_shared[((((int)threadIdx.x) & 15) + 352)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 343)] * weight_shared[((((int)threadIdx.x) & 15) + 368)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 344)] * weight_shared[((((int)threadIdx.x) & 15) + 384)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 345)] * weight_shared[((((int)threadIdx.x) & 15) + 400)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 346)] * weight_shared[((((int)threadIdx.x) & 15) + 416)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 347)] * weight_shared[((((int)threadIdx.x) & 15) + 432)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 348)] * weight_shared[((((int)threadIdx.x) & 15) + 448)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 349)] * weight_shared[((((int)threadIdx.x) & 15) + 464)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 350)] * weight_shared[((((int)threadIdx.x) & 15) + 480)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 351)] * weight_shared[((((int)threadIdx.x) & 15) + 496)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 384)] * weight_shared[(((int)threadIdx.x) & 15)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 385)] * weight_shared[((((int)threadIdx.x) & 15) + 16)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 386)] * weight_shared[((((int)threadIdx.x) & 15) + 32)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 387)] * weight_shared[((((int)threadIdx.x) & 15) + 48)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 388)] * weight_shared[((((int)threadIdx.x) & 15) + 64)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 389)] * weight_shared[((((int)threadIdx.x) & 15) + 80)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 390)] * weight_shared[((((int)threadIdx.x) & 15) + 96)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 391)] * weight_shared[((((int)threadIdx.x) & 15) + 112)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 392)] * weight_shared[((((int)threadIdx.x) & 15) + 128)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 393)] * weight_shared[((((int)threadIdx.x) & 15) + 144)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 394)] * weight_shared[((((int)threadIdx.x) & 15) + 160)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 395)] * weight_shared[((((int)threadIdx.x) & 15) + 176)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 396)] * weight_shared[((((int)threadIdx.x) & 15) + 192)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 397)] * weight_shared[((((int)threadIdx.x) & 15) + 208)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 398)] * weight_shared[((((int)threadIdx.x) & 15) + 224)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 399)] * weight_shared[((((int)threadIdx.x) & 15) + 240)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 400)] * weight_shared[((((int)threadIdx.x) & 15) + 256)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 401)] * weight_shared[((((int)threadIdx.x) & 15) + 272)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 402)] * weight_shared[((((int)threadIdx.x) & 15) + 288)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 403)] * weight_shared[((((int)threadIdx.x) & 15) + 304)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 404)] * weight_shared[((((int)threadIdx.x) & 15) + 320)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 405)] * weight_shared[((((int)threadIdx.x) & 15) + 336)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 406)] * weight_shared[((((int)threadIdx.x) & 15) + 352)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 407)] * weight_shared[((((int)threadIdx.x) & 15) + 368)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 408)] * weight_shared[((((int)threadIdx.x) & 15) + 384)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 409)] * weight_shared[((((int)threadIdx.x) & 15) + 400)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 410)] * weight_shared[((((int)threadIdx.x) & 15) + 416)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 411)] * weight_shared[((((int)threadIdx.x) & 15) + 432)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 412)] * weight_shared[((((int)threadIdx.x) & 15) + 448)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 413)] * weight_shared[((((int)threadIdx.x) & 15) + 464)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 414)] * weight_shared[((((int)threadIdx.x) & 15) + 480)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 415)] * weight_shared[((((int)threadIdx.x) & 15) + 496)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 32)] * weight_shared[((((int)threadIdx.x) & 15) + 512)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 33)] * weight_shared[((((int)threadIdx.x) & 15) + 528)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 34)] * weight_shared[((((int)threadIdx.x) & 15) + 544)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 35)] * weight_shared[((((int)threadIdx.x) & 15) + 560)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 36)] * weight_shared[((((int)threadIdx.x) & 15) + 576)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 37)] * weight_shared[((((int)threadIdx.x) & 15) + 592)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 38)] * weight_shared[((((int)threadIdx.x) & 15) + 608)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 39)] * weight_shared[((((int)threadIdx.x) & 15) + 624)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 40)] * weight_shared[((((int)threadIdx.x) & 15) + 640)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 41)] * weight_shared[((((int)threadIdx.x) & 15) + 656)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 42)] * weight_shared[((((int)threadIdx.x) & 15) + 672)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 43)] * weight_shared[((((int)threadIdx.x) & 15) + 688)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 44)] * weight_shared[((((int)threadIdx.x) & 15) + 704)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 45)] * weight_shared[((((int)threadIdx.x) & 15) + 720)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 46)] * weight_shared[((((int)threadIdx.x) & 15) + 736)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 47)] * weight_shared[((((int)threadIdx.x) & 15) + 752)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 48)] * weight_shared[((((int)threadIdx.x) & 15) + 768)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 49)] * weight_shared[((((int)threadIdx.x) & 15) + 784)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 50)] * weight_shared[((((int)threadIdx.x) & 15) + 800)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 51)] * weight_shared[((((int)threadIdx.x) & 15) + 816)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 52)] * weight_shared[((((int)threadIdx.x) & 15) + 832)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 53)] * weight_shared[((((int)threadIdx.x) & 15) + 848)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 54)] * weight_shared[((((int)threadIdx.x) & 15) + 864)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 55)] * weight_shared[((((int)threadIdx.x) & 15) + 880)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 56)] * weight_shared[((((int)threadIdx.x) & 15) + 896)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 57)] * weight_shared[((((int)threadIdx.x) & 15) + 912)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 58)] * weight_shared[((((int)threadIdx.x) & 15) + 928)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 59)] * weight_shared[((((int)threadIdx.x) & 15) + 944)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 60)] * weight_shared[((((int)threadIdx.x) & 15) + 960)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 61)] * weight_shared[((((int)threadIdx.x) & 15) + 976)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 62)] * weight_shared[((((int)threadIdx.x) & 15) + 992)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 63)] * weight_shared[((((int)threadIdx.x) & 15) + 1008)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 96)] * weight_shared[((((int)threadIdx.x) & 15) + 512)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 97)] * weight_shared[((((int)threadIdx.x) & 15) + 528)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 98)] * weight_shared[((((int)threadIdx.x) & 15) + 544)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 99)] * weight_shared[((((int)threadIdx.x) & 15) + 560)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 100)] * weight_shared[((((int)threadIdx.x) & 15) + 576)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 101)] * weight_shared[((((int)threadIdx.x) & 15) + 592)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 102)] * weight_shared[((((int)threadIdx.x) & 15) + 608)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 103)] * weight_shared[((((int)threadIdx.x) & 15) + 624)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 104)] * weight_shared[((((int)threadIdx.x) & 15) + 640)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 105)] * weight_shared[((((int)threadIdx.x) & 15) + 656)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 106)] * weight_shared[((((int)threadIdx.x) & 15) + 672)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 107)] * weight_shared[((((int)threadIdx.x) & 15) + 688)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 108)] * weight_shared[((((int)threadIdx.x) & 15) + 704)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 109)] * weight_shared[((((int)threadIdx.x) & 15) + 720)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 110)] * weight_shared[((((int)threadIdx.x) & 15) + 736)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 111)] * weight_shared[((((int)threadIdx.x) & 15) + 752)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 112)] * weight_shared[((((int)threadIdx.x) & 15) + 768)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 113)] * weight_shared[((((int)threadIdx.x) & 15) + 784)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 114)] * weight_shared[((((int)threadIdx.x) & 15) + 800)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 115)] * weight_shared[((((int)threadIdx.x) & 15) + 816)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 116)] * weight_shared[((((int)threadIdx.x) & 15) + 832)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 117)] * weight_shared[((((int)threadIdx.x) & 15) + 848)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 118)] * weight_shared[((((int)threadIdx.x) & 15) + 864)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 119)] * weight_shared[((((int)threadIdx.x) & 15) + 880)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 120)] * weight_shared[((((int)threadIdx.x) & 15) + 896)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 121)] * weight_shared[((((int)threadIdx.x) & 15) + 912)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 122)] * weight_shared[((((int)threadIdx.x) & 15) + 928)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 123)] * weight_shared[((((int)threadIdx.x) & 15) + 944)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 124)] * weight_shared[((((int)threadIdx.x) & 15) + 960)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 125)] * weight_shared[((((int)threadIdx.x) & 15) + 976)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 126)] * weight_shared[((((int)threadIdx.x) & 15) + 992)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 127)] * weight_shared[((((int)threadIdx.x) & 15) + 1008)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 160)] * weight_shared[((((int)threadIdx.x) & 15) + 512)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 161)] * weight_shared[((((int)threadIdx.x) & 15) + 528)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 162)] * weight_shared[((((int)threadIdx.x) & 15) + 544)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 163)] * weight_shared[((((int)threadIdx.x) & 15) + 560)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 164)] * weight_shared[((((int)threadIdx.x) & 15) + 576)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 165)] * weight_shared[((((int)threadIdx.x) & 15) + 592)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 166)] * weight_shared[((((int)threadIdx.x) & 15) + 608)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 167)] * weight_shared[((((int)threadIdx.x) & 15) + 624)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 168)] * weight_shared[((((int)threadIdx.x) & 15) + 640)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 169)] * weight_shared[((((int)threadIdx.x) & 15) + 656)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 170)] * weight_shared[((((int)threadIdx.x) & 15) + 672)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 171)] * weight_shared[((((int)threadIdx.x) & 15) + 688)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 172)] * weight_shared[((((int)threadIdx.x) & 15) + 704)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 173)] * weight_shared[((((int)threadIdx.x) & 15) + 720)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 174)] * weight_shared[((((int)threadIdx.x) & 15) + 736)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 175)] * weight_shared[((((int)threadIdx.x) & 15) + 752)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 176)] * weight_shared[((((int)threadIdx.x) & 15) + 768)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 177)] * weight_shared[((((int)threadIdx.x) & 15) + 784)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 178)] * weight_shared[((((int)threadIdx.x) & 15) + 800)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 179)] * weight_shared[((((int)threadIdx.x) & 15) + 816)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 180)] * weight_shared[((((int)threadIdx.x) & 15) + 832)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 181)] * weight_shared[((((int)threadIdx.x) & 15) + 848)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 182)] * weight_shared[((((int)threadIdx.x) & 15) + 864)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 183)] * weight_shared[((((int)threadIdx.x) & 15) + 880)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 184)] * weight_shared[((((int)threadIdx.x) & 15) + 896)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 185)] * weight_shared[((((int)threadIdx.x) & 15) + 912)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 186)] * weight_shared[((((int)threadIdx.x) & 15) + 928)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 187)] * weight_shared[((((int)threadIdx.x) & 15) + 944)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 188)] * weight_shared[((((int)threadIdx.x) & 15) + 960)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 189)] * weight_shared[((((int)threadIdx.x) & 15) + 976)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 190)] * weight_shared[((((int)threadIdx.x) & 15) + 992)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 191)] * weight_shared[((((int)threadIdx.x) & 15) + 1008)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 224)] * weight_shared[((((int)threadIdx.x) & 15) + 512)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 225)] * weight_shared[((((int)threadIdx.x) & 15) + 528)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 226)] * weight_shared[((((int)threadIdx.x) & 15) + 544)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 227)] * weight_shared[((((int)threadIdx.x) & 15) + 560)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 228)] * weight_shared[((((int)threadIdx.x) & 15) + 576)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 229)] * weight_shared[((((int)threadIdx.x) & 15) + 592)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 230)] * weight_shared[((((int)threadIdx.x) & 15) + 608)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 231)] * weight_shared[((((int)threadIdx.x) & 15) + 624)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 232)] * weight_shared[((((int)threadIdx.x) & 15) + 640)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 233)] * weight_shared[((((int)threadIdx.x) & 15) + 656)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 234)] * weight_shared[((((int)threadIdx.x) & 15) + 672)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 235)] * weight_shared[((((int)threadIdx.x) & 15) + 688)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 236)] * weight_shared[((((int)threadIdx.x) & 15) + 704)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 237)] * weight_shared[((((int)threadIdx.x) & 15) + 720)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 238)] * weight_shared[((((int)threadIdx.x) & 15) + 736)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 239)] * weight_shared[((((int)threadIdx.x) & 15) + 752)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 240)] * weight_shared[((((int)threadIdx.x) & 15) + 768)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 241)] * weight_shared[((((int)threadIdx.x) & 15) + 784)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 242)] * weight_shared[((((int)threadIdx.x) & 15) + 800)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 243)] * weight_shared[((((int)threadIdx.x) & 15) + 816)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 244)] * weight_shared[((((int)threadIdx.x) & 15) + 832)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 245)] * weight_shared[((((int)threadIdx.x) & 15) + 848)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 246)] * weight_shared[((((int)threadIdx.x) & 15) + 864)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 247)] * weight_shared[((((int)threadIdx.x) & 15) + 880)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 248)] * weight_shared[((((int)threadIdx.x) & 15) + 896)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 249)] * weight_shared[((((int)threadIdx.x) & 15) + 912)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 250)] * weight_shared[((((int)threadIdx.x) & 15) + 928)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 251)] * weight_shared[((((int)threadIdx.x) & 15) + 944)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 252)] * weight_shared[((((int)threadIdx.x) & 15) + 960)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 253)] * weight_shared[((((int)threadIdx.x) & 15) + 976)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 254)] * weight_shared[((((int)threadIdx.x) & 15) + 992)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 255)] * weight_shared[((((int)threadIdx.x) & 15) + 1008)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 288)] * weight_shared[((((int)threadIdx.x) & 15) + 512)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 289)] * weight_shared[((((int)threadIdx.x) & 15) + 528)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 290)] * weight_shared[((((int)threadIdx.x) & 15) + 544)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 291)] * weight_shared[((((int)threadIdx.x) & 15) + 560)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 292)] * weight_shared[((((int)threadIdx.x) & 15) + 576)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 293)] * weight_shared[((((int)threadIdx.x) & 15) + 592)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 294)] * weight_shared[((((int)threadIdx.x) & 15) + 608)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 295)] * weight_shared[((((int)threadIdx.x) & 15) + 624)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 296)] * weight_shared[((((int)threadIdx.x) & 15) + 640)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 297)] * weight_shared[((((int)threadIdx.x) & 15) + 656)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 298)] * weight_shared[((((int)threadIdx.x) & 15) + 672)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 299)] * weight_shared[((((int)threadIdx.x) & 15) + 688)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 300)] * weight_shared[((((int)threadIdx.x) & 15) + 704)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 301)] * weight_shared[((((int)threadIdx.x) & 15) + 720)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 302)] * weight_shared[((((int)threadIdx.x) & 15) + 736)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 303)] * weight_shared[((((int)threadIdx.x) & 15) + 752)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 304)] * weight_shared[((((int)threadIdx.x) & 15) + 768)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 305)] * weight_shared[((((int)threadIdx.x) & 15) + 784)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 306)] * weight_shared[((((int)threadIdx.x) & 15) + 800)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 307)] * weight_shared[((((int)threadIdx.x) & 15) + 816)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 308)] * weight_shared[((((int)threadIdx.x) & 15) + 832)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 309)] * weight_shared[((((int)threadIdx.x) & 15) + 848)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 310)] * weight_shared[((((int)threadIdx.x) & 15) + 864)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 311)] * weight_shared[((((int)threadIdx.x) & 15) + 880)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 312)] * weight_shared[((((int)threadIdx.x) & 15) + 896)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 313)] * weight_shared[((((int)threadIdx.x) & 15) + 912)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 314)] * weight_shared[((((int)threadIdx.x) & 15) + 928)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 315)] * weight_shared[((((int)threadIdx.x) & 15) + 944)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 316)] * weight_shared[((((int)threadIdx.x) & 15) + 960)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 317)] * weight_shared[((((int)threadIdx.x) & 15) + 976)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 318)] * weight_shared[((((int)threadIdx.x) & 15) + 992)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 319)] * weight_shared[((((int)threadIdx.x) & 15) + 1008)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 352)] * weight_shared[((((int)threadIdx.x) & 15) + 512)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 353)] * weight_shared[((((int)threadIdx.x) & 15) + 528)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 354)] * weight_shared[((((int)threadIdx.x) & 15) + 544)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 355)] * weight_shared[((((int)threadIdx.x) & 15) + 560)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 356)] * weight_shared[((((int)threadIdx.x) & 15) + 576)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 357)] * weight_shared[((((int)threadIdx.x) & 15) + 592)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 358)] * weight_shared[((((int)threadIdx.x) & 15) + 608)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 359)] * weight_shared[((((int)threadIdx.x) & 15) + 624)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 360)] * weight_shared[((((int)threadIdx.x) & 15) + 640)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 361)] * weight_shared[((((int)threadIdx.x) & 15) + 656)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 362)] * weight_shared[((((int)threadIdx.x) & 15) + 672)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 363)] * weight_shared[((((int)threadIdx.x) & 15) + 688)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 364)] * weight_shared[((((int)threadIdx.x) & 15) + 704)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 365)] * weight_shared[((((int)threadIdx.x) & 15) + 720)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 366)] * weight_shared[((((int)threadIdx.x) & 15) + 736)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 367)] * weight_shared[((((int)threadIdx.x) & 15) + 752)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 368)] * weight_shared[((((int)threadIdx.x) & 15) + 768)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 369)] * weight_shared[((((int)threadIdx.x) & 15) + 784)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 370)] * weight_shared[((((int)threadIdx.x) & 15) + 800)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 371)] * weight_shared[((((int)threadIdx.x) & 15) + 816)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 372)] * weight_shared[((((int)threadIdx.x) & 15) + 832)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 373)] * weight_shared[((((int)threadIdx.x) & 15) + 848)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 374)] * weight_shared[((((int)threadIdx.x) & 15) + 864)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 375)] * weight_shared[((((int)threadIdx.x) & 15) + 880)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 376)] * weight_shared[((((int)threadIdx.x) & 15) + 896)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 377)] * weight_shared[((((int)threadIdx.x) & 15) + 912)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 378)] * weight_shared[((((int)threadIdx.x) & 15) + 928)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 379)] * weight_shared[((((int)threadIdx.x) & 15) + 944)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 380)] * weight_shared[((((int)threadIdx.x) & 15) + 960)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 381)] * weight_shared[((((int)threadIdx.x) & 15) + 976)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 382)] * weight_shared[((((int)threadIdx.x) & 15) + 992)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 383)] * weight_shared[((((int)threadIdx.x) & 15) + 1008)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 416)] * weight_shared[((((int)threadIdx.x) & 15) + 512)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 417)] * weight_shared[((((int)threadIdx.x) & 15) + 528)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 418)] * weight_shared[((((int)threadIdx.x) & 15) + 544)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 419)] * weight_shared[((((int)threadIdx.x) & 15) + 560)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 420)] * weight_shared[((((int)threadIdx.x) & 15) + 576)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 421)] * weight_shared[((((int)threadIdx.x) & 15) + 592)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 422)] * weight_shared[((((int)threadIdx.x) & 15) + 608)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 423)] * weight_shared[((((int)threadIdx.x) & 15) + 624)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 424)] * weight_shared[((((int)threadIdx.x) & 15) + 640)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 425)] * weight_shared[((((int)threadIdx.x) & 15) + 656)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 426)] * weight_shared[((((int)threadIdx.x) & 15) + 672)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 427)] * weight_shared[((((int)threadIdx.x) & 15) + 688)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 428)] * weight_shared[((((int)threadIdx.x) & 15) + 704)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 429)] * weight_shared[((((int)threadIdx.x) & 15) + 720)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 430)] * weight_shared[((((int)threadIdx.x) & 15) + 736)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 431)] * weight_shared[((((int)threadIdx.x) & 15) + 752)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 432)] * weight_shared[((((int)threadIdx.x) & 15) + 768)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 433)] * weight_shared[((((int)threadIdx.x) & 15) + 784)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 434)] * weight_shared[((((int)threadIdx.x) & 15) + 800)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 435)] * weight_shared[((((int)threadIdx.x) & 15) + 816)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 436)] * weight_shared[((((int)threadIdx.x) & 15) + 832)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 437)] * weight_shared[((((int)threadIdx.x) & 15) + 848)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 438)] * weight_shared[((((int)threadIdx.x) & 15) + 864)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 439)] * weight_shared[((((int)threadIdx.x) & 15) + 880)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 440)] * weight_shared[((((int)threadIdx.x) & 15) + 896)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 441)] * weight_shared[((((int)threadIdx.x) & 15) + 912)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 442)] * weight_shared[((((int)threadIdx.x) & 15) + 928)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 443)] * weight_shared[((((int)threadIdx.x) & 15) + 944)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 444)] * weight_shared[((((int)threadIdx.x) & 15) + 960)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 445)] * weight_shared[((((int)threadIdx.x) & 15) + 976)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 446)] * weight_shared[((((int)threadIdx.x) & 15) + 992)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 448) + 447)] * weight_shared[((((int)threadIdx.x) & 15) + 1008)]));
  conv2d_nhwc[(((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 4) * 1792)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15))] = conv2d_nhwc_local[0];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 4) * 1792)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15)) + 256)] = conv2d_nhwc_local[1];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 4) * 1792)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15)) + 512)] = conv2d_nhwc_local[2];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 4) * 1792)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15)) + 768)] = conv2d_nhwc_local[3];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 4) * 1792)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15)) + 1024)] = conv2d_nhwc_local[4];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 4) * 1792)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15)) + 1280)] = conv2d_nhwc_local[5];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 4) * 7168) + ((((int)threadIdx.x) >> 4) * 1792)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15)) + 1536)] = conv2d_nhwc_local[6];
}


