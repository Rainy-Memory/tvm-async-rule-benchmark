
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[7];
  __shared__ float PadInput_shared[2496];
  __shared__ float weight_shared[6144];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  for (int ax0_ax1_ax2_ax3_fused_0 = 0; ax0_ax1_ax2_ax3_fused_0 < 26; ++ax0_ax1_ax2_ax3_fused_0) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((ax0_ax1_ax2_ax3_fused_0 * 32) + ((int)threadIdx.x))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((ax0_ax1_ax2_ax3_fused_0 >> 1) * 14336) + ((((int)blockIdx.x) >> 4) * 2048)) + ((ax0_ax1_ax2_ax3_fused_0 & 1) * 32)) + ((int)threadIdx.x)))), "n"(4)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2048))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 4096))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 6144))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 8192))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 10240))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 12288))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14336))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 16384))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1152)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 18432))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 20480))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1408)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 22528))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 24576))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1664)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 26624))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 28672))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1920)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 30720))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int ax0_ax1_ax2_ax3_fused_0_1 = 0; ax0_ax1_ax2_ax3_fused_0_1 < 26; ++ax0_ax1_ax2_ax3_fused_0_1) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((ax0_ax1_ax2_ax3_fused_0_1 * 32) + ((int)threadIdx.x)) + 832)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((ax0_ax1_ax2_ax3_fused_0_1 >> 1) * 14336) + ((((int)blockIdx.x) >> 4) * 2048)) + ((ax0_ax1_ax2_ax3_fused_0_1 & 1) * 32)) + ((int)threadIdx.x)) + 64))), "n"(4)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 32768))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2176)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 34816))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2304)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 36864))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2432)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 38912))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2560)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 40960))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2688)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 43008))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2816)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 45056))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2944)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 47104))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3072)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 49152))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3200)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 51200))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3328)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 53248))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3456)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 55296))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3584)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 57344))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3712)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 59392))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3840)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 61440))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3968)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 63488))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int rh_0_rw_0_rc_0_fused = 0; rh_0_rw_0_rc_0_fused < 14; ++rh_0_rw_0_rc_0_fused) {
    __syncthreads();
    for (int ax0_ax1_ax2_ax3_fused_0_2 = 0; ax0_ax1_ax2_ax3_fused_0_2 < 26; ++ax0_ax1_ax2_ax3_fused_0_2) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 832) + (ax0_ax1_ax2_ax3_fused_0_2 * 32)) + ((int)threadIdx.x))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((ax0_ax1_ax2_ax3_fused_0_2 >> 1) * 14336) + ((((int)blockIdx.x) >> 4) * 2048)) + (rh_0_rw_0_rc_0_fused * 64)) + ((ax0_ax1_ax2_ax3_fused_0_2 & 1) * 32)) + ((int)threadIdx.x)) + 128))), "n"(4)
    );
  }
    }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 65536))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 67584))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 69632))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 71680))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 73728))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 75776))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 77824))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 79872))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 81920))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1152)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 83968))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 86016))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1408)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 88064))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 90112))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1664)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 92160))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 94208))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1920)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 32768) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 15) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 96256))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    for (int h_3 = 0; h_3 < 7; ++h_3) {
      for (int rc_2 = 0; rc_2 < 64; ++rc_2) {
        conv2d_nhwc_local[h_3] = (conv2d_nhwc_local[h_3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 832) + (h_3 * 128)) + rc_2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + (rc_2 * 32)) + ((int)threadIdx.x))]));
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  for (int h_3_1 = 0; h_3_1 < 7; ++h_3_1) {
    for (int rc_2_1 = 0; rc_2_1 < 64; ++rc_2_1) {
      conv2d_nhwc_local[h_3_1] = (conv2d_nhwc_local[h_3_1] + (PadInput_shared[(((h_3_1 * 128) + rc_2_1) + 1664)] * weight_shared[(((rc_2_1 * 32) + ((int)threadIdx.x)) + 4096)]));
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int h_3_2 = 0; h_3_2 < 7; ++h_3_2) {
    for (int rc_2_2 = 0; rc_2_2 < 64; ++rc_2_2) {
      conv2d_nhwc_local[h_3_2] = (conv2d_nhwc_local[h_3_2] + (PadInput_shared[((h_3_2 * 128) + rc_2_2)] * weight_shared[((rc_2_2 * 32) + ((int)threadIdx.x))]));
    }
  }
  conv2d_nhwc[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = conv2d_nhwc_local[0];
  conv2d_nhwc[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 3584)] = conv2d_nhwc_local[1];
  conv2d_nhwc[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 7168)] = conv2d_nhwc_local[2];
  conv2d_nhwc[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 10752)] = conv2d_nhwc_local[3];
  conv2d_nhwc[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 14336)] = conv2d_nhwc_local[4];
  conv2d_nhwc[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 17920)] = conv2d_nhwc_local[5];
  conv2d_nhwc[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 21504)] = conv2d_nhwc_local[6];
}


