
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[8];
  __shared__ float PadInput_shared[3456];
  __shared__ float weight_shared[8192];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[7] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((int)threadIdx.x)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (((int)threadIdx.x) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 112)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 112) >> 5) * 512)) + ((((int)threadIdx.x) + 16) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (((int)threadIdx.x) & 31)) + 3584))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 336)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 336) >> 5) * 512)) + ((((int)threadIdx.x) + 16) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (((int)threadIdx.x) & 31)) + 7168))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 560)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 560) >> 5) * 512)) + ((((int)threadIdx.x) + 16) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (((int)threadIdx.x) & 31)) + 10752))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 80) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 784)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 784) >> 5) * 512)) + ((((int)threadIdx.x) + 16) & 31)))), "n"(4)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((int)threadIdx.x)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 112)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 112) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 224) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 336)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 336) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 1792))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 560)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 560) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 672) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 784)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 784) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 3584))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1008)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1008) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1120)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1120) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1232)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1232) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1344)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 5376))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1456)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1456) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1568)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1568) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1680)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1680) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 7168))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1904)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1904) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2016)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 2016) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((int)threadIdx.x)) + 32))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 864)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (((int)threadIdx.x) & 31)) + 32))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 976)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 112) >> 5) * 512)) + ((((int)threadIdx.x) + 16) & 31)) + 32))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 1088)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (((int)threadIdx.x) & 31)) + 3616))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 1200)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 336) >> 5) * 512)) + ((((int)threadIdx.x) + 16) & 31)) + 32))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 1312)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (((int)threadIdx.x) & 31)) + 7200))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 1424)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 560) >> 5) * 512)) + ((((int)threadIdx.x) + 16) & 31)) + 32))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (((int)threadIdx.x) & 31)) + 10784))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 80) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 1648)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 784) >> 5) * 512)) + ((((int)threadIdx.x) + 16) & 31)) + 32))), "n"(4)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2160)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 112) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2272)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 224) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2384)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 336) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2496)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 9984))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2608)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 560) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2720)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 672) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2832)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 784) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2944)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 11776))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3056)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1008) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3168)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1120) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3280)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1232) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3392)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 13568))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3504)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1456) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3616)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1568) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3728)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1680) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3840)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 15360))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3952)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1904) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 8192))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 4064)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 2016) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((int)threadIdx.x)) + 8224))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 1728)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (((int)threadIdx.x) & 31)) + 64))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 1840)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 112) >> 5) * 512)) + ((((int)threadIdx.x) + 16) & 31)) + 64))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 1952)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (((int)threadIdx.x) & 31)) + 3648))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 2064)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 336) >> 5) * 512)) + ((((int)threadIdx.x) + 16) & 31)) + 64))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 2176)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (((int)threadIdx.x) & 31)) + 7232))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 2288)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 560) >> 5) * 512)) + ((((int)threadIdx.x) + 16) & 31)) + 64))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 2400)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (((int)threadIdx.x) & 31)) + 10816))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 80) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 2512)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 784) >> 5) * 512)) + ((((int)threadIdx.x) + 16) & 31)) + 64))), "n"(4)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 4096)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 16384))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 4208)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 112) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 16384))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 4320)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 224) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 16384))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 4432)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 336) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 16384))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 4544)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 18176))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 4656)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 560) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 16384))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 4768)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 672) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 16384))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 4880)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 784) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 16384))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 4992)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 19968))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 5104)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1008) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 16384))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 5216)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1120) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 16384))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 5328)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1232) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 16384))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 5440)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 21760))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 5552)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1456) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 16384))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 5664)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1568) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 16384))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 5776)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1680) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 16384))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 5888)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 23552))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 6000)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1904) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 16384))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 6112)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 2016) >> 6) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((int)threadIdx.x)) + 16416))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int rh_0_rw_0_rc_0_fused = 0; rh_0_rw_0_rc_0_fused < 13; ++rh_0_rw_0_rc_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 864) + ((int)threadIdx.x))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (rh_0_rw_0_rc_0_fused * 32)) + (((int)threadIdx.x) & 31)) + 96))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 864) + ((int)threadIdx.x)) + 112)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 112) >> 5) * 512)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 96))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 864) + ((int)threadIdx.x)) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (rh_0_rw_0_rc_0_fused * 32)) + (((int)threadIdx.x) & 31)) + 3680))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 864) + ((int)threadIdx.x)) + 336)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 336) >> 5) * 512)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 96))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 864) + ((int)threadIdx.x)) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (rh_0_rw_0_rc_0_fused * 32)) + (((int)threadIdx.x) & 31)) + 7264))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 864) + ((int)threadIdx.x)) + 560)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 560) >> 5) * 512)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 96))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 864) + ((int)threadIdx.x)) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 2) * 28672) + ((((int)threadIdx.x) >> 5) * 512)) + (rh_0_rw_0_rc_0_fused * 32)) + (((int)threadIdx.x) & 31)) + 10848))), "n"(4)
    );
  }
    if (((int)threadIdx.x) < 80) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 864) + ((int)threadIdx.x)) + 784)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 2) * 28672) + (((((int)threadIdx.x) + 784) >> 5) * 512)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 96))), "n"(4)
    );
  }
    }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 24576))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 112)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 112) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 24576))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 224) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 24576))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 336)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 336) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 24576))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 26368))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 560)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 560) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 24576))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 672) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 24576))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 784)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 784) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 24576))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 28160))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 1008)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 1008) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 24576))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 1120)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 1120) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 24576))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 1232)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 1232) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 24576))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 1344)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 29952))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 1456)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 1456) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 24576))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 1568)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 1568) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 24576))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 1680)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 1680) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 24576))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + ((((int)threadIdx.x) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((int)threadIdx.x) & 63)) + 31744))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 1904)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 1904) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 24576))), "n"(4)
    );
  }
    if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 2048) + ((int)threadIdx.x)) + 2016)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 8192) + (((((int)threadIdx.x) + 2016) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((int)threadIdx.x)) + 24608))), "n"(4)
    );
  }
    }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

    __syncthreads();
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64))] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64))] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 448)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 448)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 449)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 449)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 450)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 450)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 451)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 451)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 452)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 452)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 453)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 453)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 454)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 454)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 455)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 455)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 456)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 456)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 457)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 457)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 458)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 458)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 459)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 459)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 460)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 460)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 461)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 461)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 462)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 462)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 463)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 463)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64))] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64))] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 448)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 448)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 449)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 449)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 450)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 450)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 451)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 451)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 452)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 452)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 453)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 453)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 454)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 454)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 455)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 455)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 456)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 456)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 457)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 457)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 458)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 458)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 459)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 459)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 460)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 460)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 461)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 461)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 462)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 462)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 463)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 463)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1024)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1056)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 464)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1024)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 464)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1056)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1088)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1120)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 465)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1088)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 465)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1120)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1152)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1184)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 466)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1152)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 466)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1184)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1216)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1248)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 467)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1216)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 467)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1248)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1280)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1312)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 468)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1280)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 468)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1312)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1344)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1376)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 469)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1344)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 469)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1376)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1408)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1440)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 470)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1408)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 470)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1440)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1472)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1504)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 471)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1472)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 471)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1504)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1536)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1568)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 472)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1536)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 472)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1568)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1600)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1632)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 473)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1600)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 473)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1632)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1664)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1696)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 474)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1664)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 474)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1696)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1728)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1760)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 475)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1728)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 475)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1760)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1792)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1824)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 476)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1792)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 476)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1824)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1856)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1888)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 477)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1856)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 477)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1888)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1920)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1952)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 478)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1920)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 478)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1952)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1984)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 2016)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 479)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1984)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 479)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 2016)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1025)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1057)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 464)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1025)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 464)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1057)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1089)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1121)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 465)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1089)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 465)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1121)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1153)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1185)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 466)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1153)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 466)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1185)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1217)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1249)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 467)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1217)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 467)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1249)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1281)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1313)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 468)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1281)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 468)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1313)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1345)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1377)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 469)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1345)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 469)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1377)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1409)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1441)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 470)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1409)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 470)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1441)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1473)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1505)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 471)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1473)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 471)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1505)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1537)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1569)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 472)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1537)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 472)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1569)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1601)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1633)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 473)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1601)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 473)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1633)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1665)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1697)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 474)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1665)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 474)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1697)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1729)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1761)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 475)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1729)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 475)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1761)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1793)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1825)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 476)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1793)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 476)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1825)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1857)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1889)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 477)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1857)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 477)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1889)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1921)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1953)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 478)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1921)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 478)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1953)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1985)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 2017)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 479)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1985)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 864) + ((((int)threadIdx.x) >> 4) * 64)) + 479)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 2017)]));
  }
__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 864)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 864)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1312)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1312)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 865)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 865)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1313)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1313)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 866)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 866)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1314)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1314)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 867)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 867)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1315)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1315)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 868)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 868)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1316)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1316)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 869)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 869)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1317)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1317)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 870)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 870)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1318)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1318)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 871)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 871)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1319)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1319)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 872)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 872)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1320)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1320)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 873)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 873)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1321)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1321)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 874)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 874)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1322)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1322)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 875)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 875)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1323)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1323)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 876)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 876)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1324)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1324)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 877)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 877)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1325)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1325)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 878)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 878)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1326)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1326)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 879)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 879)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1327)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1327)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 864)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 864)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1312)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1312)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 865)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 865)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1313)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1313)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 866)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 866)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1314)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1314)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 867)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 867)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1315)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1315)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 868)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 868)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1316)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1316)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 869)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 869)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1317)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1317)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 870)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 870)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1318)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1318)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 871)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 871)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1319)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1319)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 872)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 872)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1320)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1320)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 873)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 873)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1321)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1321)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 874)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 874)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1322)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1322)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 875)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 875)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1323)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1323)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 876)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 876)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1324)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1324)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 877)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 877)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1325)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1325)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 878)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 878)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1326)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1326)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 879)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 879)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1327)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1327)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 880)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 880)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1328)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1328)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 881)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 881)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1329)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1329)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 882)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 882)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1330)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1330)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 883)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 883)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1331)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1331)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 884)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 884)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 885)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 885)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 886)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 886)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 887)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 887)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 888)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 888)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1336)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1336)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 889)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 889)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1337)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1337)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 890)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 890)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1338)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1338)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 891)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 891)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1339)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1339)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 892)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 892)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1340)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1340)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 893)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 893)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1341)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1341)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 894)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 894)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1342)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1342)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 895)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 895)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1343)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1343)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 880)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 880)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1328)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1328)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 881)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 881)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1329)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1329)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 882)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 882)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1330)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1330)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 883)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 883)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1331)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1331)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 884)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 884)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 885)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 885)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 886)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 886)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 887)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 887)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 888)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 888)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1336)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1336)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 889)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 889)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1337)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1337)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 890)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 890)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1338)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1338)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 891)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 891)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1339)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1339)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 892)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 892)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1340)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1340)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 893)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 893)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1341)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1341)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 894)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 894)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1342)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1342)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 895)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 895)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1343)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1343)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4096)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4128)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2176)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4096)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2176)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4128)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4160)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4192)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2177)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4160)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2177)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4192)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4224)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4256)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2178)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4224)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2178)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4256)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4288)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4320)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2179)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4288)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2179)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4320)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4352)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4384)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2180)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4352)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2180)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4384)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4416)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4448)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2181)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4416)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2181)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4448)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4480)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4512)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2182)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4480)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2182)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4512)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4544)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4576)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2183)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4544)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2183)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4576)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4608)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4640)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2184)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4608)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2184)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4640)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4672)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4704)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2185)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4672)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2185)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4704)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4736)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4768)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2186)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4736)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2186)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4768)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4800)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4832)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2187)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4800)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2187)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4832)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4864)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4896)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2188)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4864)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2188)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4896)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4928)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4960)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2189)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4928)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2189)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4960)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4992)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5024)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2190)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4992)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2190)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5024)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5056)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5088)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2191)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5056)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2191)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5088)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4097)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4129)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2176)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4097)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2176)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4129)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4161)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4193)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2177)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4161)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2177)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4193)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4225)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4257)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2178)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4225)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2178)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4257)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4289)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4321)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2179)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4289)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2179)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4321)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4353)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4385)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2180)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4353)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2180)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4385)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4417)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4449)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2181)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4417)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2181)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4449)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4481)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4513)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2182)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4481)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2182)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4513)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4545)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4577)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2183)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4545)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2183)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4577)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4609)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4641)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2184)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4609)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2184)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4641)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4673)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4705)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2185)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4673)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2185)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4705)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4737)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4769)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2186)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4737)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2186)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4769)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4801)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4833)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2187)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4801)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2187)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4833)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4865)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4897)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2188)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4865)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2188)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4897)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4929)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4961)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2189)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4929)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2189)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4961)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4993)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5025)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2190)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4993)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2190)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5025)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5057)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5089)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2191)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5057)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2191)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5089)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5120)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5152)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5120)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5152)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5184)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5216)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5184)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5216)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5248)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5280)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5248)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5280)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5312)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5344)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5312)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5344)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1748)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5376)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1748)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5408)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5376)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5408)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1749)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5440)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1749)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5472)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5440)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5472)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1750)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5504)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1750)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5536)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5504)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5536)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1751)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5568)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1751)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5600)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5568)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5600)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1752)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5632)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1752)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5664)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5632)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5664)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1753)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5696)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1753)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5728)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5696)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5728)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1754)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5760)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1754)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5792)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5760)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5792)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1755)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5824)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1755)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5856)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5824)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5856)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1756)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5888)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1756)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5920)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5888)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5920)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1757)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5952)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1757)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5984)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5952)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5984)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1758)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6016)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1758)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6048)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6016)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6048)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1759)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6080)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1759)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6112)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6080)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6112)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5121)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5153)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5121)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5153)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5185)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5217)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5185)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5217)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5249)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5281)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5249)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5281)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5313)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5345)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5313)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5345)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1748)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5377)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1748)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5409)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5377)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5409)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1749)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5441)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1749)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5473)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5441)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5473)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1750)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5505)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1750)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5537)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5505)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5537)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1751)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5569)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1751)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5601)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5569)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5601)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1752)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5633)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1752)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5665)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5633)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5665)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1753)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5697)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1753)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5729)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5697)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5729)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1754)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5761)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1754)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5793)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5761)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5793)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1755)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5825)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1755)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5857)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5825)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5857)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1756)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5889)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1756)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5921)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5889)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5921)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1757)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5953)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1757)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5985)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5953)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 5985)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1758)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6017)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1758)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6049)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6017)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6049)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1759)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6081)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 1759)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6113)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6081)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6113)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2592)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6144)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2592)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6176)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3040)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6144)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3040)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6176)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2593)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6208)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2593)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6240)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3041)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6208)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3041)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6240)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2594)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6272)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2594)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6304)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3042)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6272)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3042)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6304)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2595)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6336)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2595)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6368)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3043)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6336)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3043)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6368)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2596)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6400)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2596)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6432)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3044)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6400)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3044)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6432)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2597)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6464)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2597)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6496)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3045)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6464)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3045)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6496)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2598)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6528)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2598)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6560)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3046)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6528)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3046)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6560)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2599)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6592)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2599)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6624)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3047)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6592)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3047)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6624)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2600)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6656)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2600)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6688)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3048)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6656)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3048)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6688)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2601)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6720)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2601)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6752)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3049)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6720)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3049)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6752)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2602)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6784)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2602)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6816)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3050)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6784)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3050)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6816)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2603)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6848)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2603)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6880)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3051)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6848)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3051)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6880)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2604)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6912)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2604)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6944)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6912)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6944)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2605)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6976)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2605)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7008)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6976)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7008)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2606)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7040)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2606)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7072)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7040)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7072)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2607)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7104)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2607)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7136)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7104)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7136)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2592)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6145)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2592)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6177)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3040)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6145)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3040)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6177)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2593)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6209)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2593)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6241)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3041)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6209)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3041)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6241)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2594)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6273)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2594)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6305)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3042)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6273)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3042)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6305)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2595)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6337)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2595)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6369)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3043)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6337)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3043)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6369)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2596)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6401)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2596)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6433)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3044)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6401)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3044)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6433)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2597)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6465)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2597)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6497)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3045)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6465)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3045)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6497)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2598)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6529)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2598)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6561)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3046)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6529)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3046)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6561)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2599)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6593)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2599)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6625)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3047)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6593)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3047)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6625)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2600)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6657)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2600)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6689)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3048)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6657)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3048)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6689)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2601)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6721)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2601)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6753)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3049)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6721)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3049)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6753)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2602)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6785)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2602)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6817)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3050)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6785)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3050)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6817)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2603)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6849)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2603)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6881)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3051)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6849)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3051)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6881)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2604)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6913)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2604)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6945)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6913)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6945)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2605)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6977)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2605)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7009)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 6977)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7009)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2606)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7041)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2606)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7073)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7041)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7073)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2607)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7105)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2607)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7137)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7105)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7137)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2608)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7168)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2608)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7200)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7168)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7200)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2609)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7232)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2609)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7264)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7232)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7264)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2610)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7296)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2610)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7328)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7296)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7328)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2611)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7360)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2611)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7392)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7360)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7392)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2612)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7424)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2612)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7456)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7424)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7456)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2613)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7488)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2613)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7520)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7488)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7520)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2614)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7552)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2614)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7584)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7552)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7584)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2615)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7616)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2615)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7648)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7616)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7648)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2616)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7680)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2616)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7712)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7680)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7712)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2617)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7744)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2617)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7776)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7744)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7776)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2618)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7808)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2618)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7840)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7808)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7840)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2619)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7872)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2619)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7904)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7872)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7904)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2620)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7936)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2620)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7968)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7936)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7968)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2621)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8000)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2621)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8032)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8000)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8032)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2622)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8064)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2622)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8096)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8064)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8096)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2623)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8128)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2623)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8160)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8128)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8160)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2608)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7169)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2608)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7201)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7169)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7201)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2609)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7233)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2609)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7265)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7233)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7265)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2610)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7297)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2610)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7329)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7297)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7329)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2611)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7361)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2611)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7393)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7361)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7393)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2612)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7425)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2612)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7457)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7425)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7457)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2613)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7489)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2613)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7521)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7489)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7521)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2614)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7553)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2614)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7585)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7553)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7585)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2615)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7617)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2615)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7649)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7617)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7649)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2616)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7681)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2616)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7713)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7681)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7713)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2617)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7745)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2617)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7777)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7745)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7777)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2618)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7809)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2618)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7841)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7809)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7841)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2619)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7873)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2619)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7905)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7873)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7905)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2620)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7937)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2620)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7969)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7937)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 7969)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2621)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8001)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2621)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8033)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8001)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8033)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2622)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8065)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2622)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8097)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8065)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8097)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2623)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8129)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 2623)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8161)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8129)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 64) + 3071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 8161)]));
  conv2d_nhwc[(((((((int)blockIdx.x) >> 2) * 3584) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 2))] = conv2d_nhwc_local[0];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 2) * 3584) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)] = conv2d_nhwc_local[2];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 2) * 3584) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1792)] = conv2d_nhwc_local[4];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 2) * 3584) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1824)] = conv2d_nhwc_local[6];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 2) * 3584) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)] = conv2d_nhwc_local[1];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 2) * 3584) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)] = conv2d_nhwc_local[3];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 2) * 3584) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1793)] = conv2d_nhwc_local[5];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 2) * 3584) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1825)] = conv2d_nhwc_local[7];
}


