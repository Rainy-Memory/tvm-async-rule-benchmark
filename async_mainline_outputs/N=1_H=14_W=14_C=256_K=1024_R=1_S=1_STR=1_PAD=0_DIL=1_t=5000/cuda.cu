
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[28];
  __shared__ float PadInput_shared[1344];
  __shared__ float weight_shared[6144];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[14] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[15] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[16] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[17] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[18] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[19] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  conv2d_nhwc_local[20] = 0.000000e+00f;
  conv2d_nhwc_local[7] = 0.000000e+00f;
  conv2d_nhwc_local[21] = 0.000000e+00f;
  conv2d_nhwc_local[8] = 0.000000e+00f;
  conv2d_nhwc_local[22] = 0.000000e+00f;
  conv2d_nhwc_local[9] = 0.000000e+00f;
  conv2d_nhwc_local[23] = 0.000000e+00f;
  conv2d_nhwc_local[10] = 0.000000e+00f;
  conv2d_nhwc_local[24] = 0.000000e+00f;
  conv2d_nhwc_local[11] = 0.000000e+00f;
  conv2d_nhwc_local[25] = 0.000000e+00f;
  conv2d_nhwc_local[12] = 0.000000e+00f;
  conv2d_nhwc_local[26] = 0.000000e+00f;
  conv2d_nhwc_local[13] = 0.000000e+00f;
  conv2d_nhwc_local[27] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((int)threadIdx.x)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 3) * 7168) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 3) * 7168) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 2048))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 3) * 7168) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 4096))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 64) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 3) * 7168) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 6144))), "n"(4)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 5) * 1024) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 1024) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 4096))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 1024) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 8192))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 1024) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 12288))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 3) * 7168) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 16))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 576)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 3) * 7168) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 2064))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 704)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 3) * 7168) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 4112))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 64) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) + 832)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 3) * 7168) + ((((int)threadIdx.x) >> 4) * 256)) + (((int)threadIdx.x) & 15)) + 6160))), "n"(4)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 1024) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 16384))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2560)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 1024) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 20480))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3072)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 1024) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 24576))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 3584)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 1024) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 28672))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int rh_0_rw_0_rc_0_fused = 0; rh_0_rw_0_rc_0_fused < 14; ++rh_0_rw_0_rc_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((rh_0_rw_0_rc_0_fused + 2) % 3) * 448) + ((int)threadIdx.x))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 3) * 7168) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 16)) + (((int)threadIdx.x) & 15)) + 32))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 448) + ((int)threadIdx.x)) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 3) * 7168) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 16)) + (((int)threadIdx.x) & 15)) + 2080))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 448) + ((int)threadIdx.x)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 3) * 7168) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 16)) + (((int)threadIdx.x) & 15)) + 4128))), "n"(4)
    );
  }
    if (((int)threadIdx.x) < 64) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 448) + ((int)threadIdx.x)) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 3) * 7168) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 16)) + (((int)threadIdx.x) & 15)) + 6176))), "n"(4)
    );
  }
    }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 16384) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 32768))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 16384) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 36864))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 16384) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 40960))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + (((int)threadIdx.x) * 4)) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 16384) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 45056))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112))] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112))] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112))] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112))] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 16)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 32)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 32)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 32)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 32)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 48)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 64)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 80)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 96)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2))]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 64)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 128)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 192)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 65)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 129)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 193)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 256)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 320)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 384)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 448)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 257)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 321)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 385)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 449)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 512)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 576)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 640)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 704)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 513)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 577)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 641)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 705)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 768)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 832)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 896)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 960)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 769)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 833)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 897)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 961)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1024)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1088)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1152)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1216)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1025)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1089)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1153)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1217)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1280)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1344)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1408)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1472)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1281)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1345)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1409)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1473)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1536)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1600)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1664)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1728)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1537)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1601)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1665)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1729)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1792)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1856)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1920)]));
    conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1984)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1793)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1857)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1921)]));
    conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 448) + ((((int)threadIdx.x) >> 5) * 112)) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 31) * 2)) + 1985)]));
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 896)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4096)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 896)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4160)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 897)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4224)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 897)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4288)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 896)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4097)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 896)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4161)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 897)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4225)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 897)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4289)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 912)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4096)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 912)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4160)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 913)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4224)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 913)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4288)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 912)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4097)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 912)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4161)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 913)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4225)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 913)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4289)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 928)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4096)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 928)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4160)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 929)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4224)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 929)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4288)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 928)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4097)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 928)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4161)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 929)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4225)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 929)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4289)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 944)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4096)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 944)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4160)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 945)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4224)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 945)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4288)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 944)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4097)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 944)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4161)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 945)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4225)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 945)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4289)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 960)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4096)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 960)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4160)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 961)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4224)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 961)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4288)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 960)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4097)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 960)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4161)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 961)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4225)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 961)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4289)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 976)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4096)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 976)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4160)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 977)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4224)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 977)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4288)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 976)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4097)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 976)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4161)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 977)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4225)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 977)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4289)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 992)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4096)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 992)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4160)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 993)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4224)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 993)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4288)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 992)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4097)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 992)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4161)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 993)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4225)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 993)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4289)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 898)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4352)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 898)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4416)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 899)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4480)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 899)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4544)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 898)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4353)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 898)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4417)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 899)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4481)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 899)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4545)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 914)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4352)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 914)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4416)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 915)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4480)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 915)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4544)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 914)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4353)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 914)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4417)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 915)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4481)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 915)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4545)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 930)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4352)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 930)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4416)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 931)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4480)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 931)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4544)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 930)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4353)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 930)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4417)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 931)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4481)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 931)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4545)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 946)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4352)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 946)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4416)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 947)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4480)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 947)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4544)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 946)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4353)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 946)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4417)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 947)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4481)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 947)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4545)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 962)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4352)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 962)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4416)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 963)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4480)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 963)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4544)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 962)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4353)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 962)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4417)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 963)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4481)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 963)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4545)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 978)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4352)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 978)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4416)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 979)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4480)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 979)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4544)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 978)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4353)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 978)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4417)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 979)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4481)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 979)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4545)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 994)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4352)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 994)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4416)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 995)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4480)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 995)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4544)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 994)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4353)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 994)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4417)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 995)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4481)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 995)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4545)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 900)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4608)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 900)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4672)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 901)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4736)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 901)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4800)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 900)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4609)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 900)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4673)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 901)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4737)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 901)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4801)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 916)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4608)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 916)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4672)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 917)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4736)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 917)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4800)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 916)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4609)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 916)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4673)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 917)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4737)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 917)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4801)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 932)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4608)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 932)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4672)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 933)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4736)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 933)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4800)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 932)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4609)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 932)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4673)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 933)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4737)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 933)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4801)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 948)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4608)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 948)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4672)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 949)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4736)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 949)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4800)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 948)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4609)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 948)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4673)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 949)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4737)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 949)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4801)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 964)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4608)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 964)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4672)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 965)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4736)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 965)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4800)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 964)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4609)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 964)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4673)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 965)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4737)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 965)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4801)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 980)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4608)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 980)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4672)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 981)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4736)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 981)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4800)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 980)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4609)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 980)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4673)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 981)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4737)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 981)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4801)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 996)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4608)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 996)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4672)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 997)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4736)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 997)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4800)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 996)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4609)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 996)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4673)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 997)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4737)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 997)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4801)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 902)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4864)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 902)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4928)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 903)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4992)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 903)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5056)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 902)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4865)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 902)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4929)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 903)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4993)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 903)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5057)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 918)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4864)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 918)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4928)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 919)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4992)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 919)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5056)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 918)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4865)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 918)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4929)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 919)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4993)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 919)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5057)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 934)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4864)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 934)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4928)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 935)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4992)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 935)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5056)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 934)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4865)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 934)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4929)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 935)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4993)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 935)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5057)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 950)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4864)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 950)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4928)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 951)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4992)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 951)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5056)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 950)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4865)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 950)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4929)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 951)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4993)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 951)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5057)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 966)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4864)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 966)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4928)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 967)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4992)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 967)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5056)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 966)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4865)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 966)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4929)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 967)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4993)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 967)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5057)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 982)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4864)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 982)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4928)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 983)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4992)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 983)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5056)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 982)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4865)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 982)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4929)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 983)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4993)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 983)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5057)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 998)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4864)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 998)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4928)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 999)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4992)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 999)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5056)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 998)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4865)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 998)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4929)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 999)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 4993)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 999)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5057)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 904)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5120)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 904)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5184)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 905)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5248)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 905)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5312)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 904)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5121)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 904)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5185)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 905)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5249)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 905)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5313)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 920)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5120)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 920)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5184)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 921)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5248)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 921)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5312)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 920)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5121)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 920)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5185)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 921)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5249)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 921)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5313)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 936)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5120)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 936)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5184)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 937)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5248)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 937)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5312)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 936)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5121)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 936)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5185)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 937)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5249)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 937)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5313)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 952)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5120)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 952)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5184)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 953)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5248)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 953)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5312)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 952)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5121)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 952)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5185)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 953)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5249)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 953)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5313)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 968)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5120)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 968)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5184)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 969)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5248)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 969)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5312)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 968)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5121)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 968)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5185)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 969)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5249)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 969)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5313)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 984)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5120)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 984)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5184)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 985)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5248)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 985)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5312)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 984)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5121)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 984)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5185)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 985)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5249)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 985)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5313)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1000)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5120)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1000)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5184)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1001)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5248)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1001)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5312)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1000)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5121)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1000)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5185)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1001)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5249)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1001)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5313)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 906)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5376)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 906)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5440)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 907)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5504)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 907)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5568)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 906)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5377)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 906)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5441)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 907)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5505)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 907)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5569)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 922)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5376)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 922)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5440)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 923)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5504)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 923)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5568)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 922)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5377)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 922)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5441)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 923)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5505)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 923)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5569)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 938)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5376)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 938)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5440)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 939)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5504)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 939)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5568)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 938)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5377)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 938)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5441)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 939)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5505)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 939)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5569)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 954)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5376)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 954)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5440)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 955)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5504)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 955)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5568)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 954)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5377)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 954)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5441)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 955)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5505)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 955)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5569)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 970)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5376)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 970)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5440)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 971)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5504)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 971)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5568)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 970)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5377)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 970)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5441)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 971)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5505)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 971)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5569)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 986)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5376)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 986)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5440)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 987)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5504)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 987)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5568)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 986)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5377)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 986)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5441)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 987)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5505)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 987)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5569)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1002)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5376)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1002)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5440)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1003)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5504)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1003)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5568)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1002)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5377)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1002)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5441)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1003)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5505)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1003)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5569)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 908)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5632)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 908)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5696)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 909)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5760)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 909)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5824)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 908)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5633)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 908)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5697)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 909)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5761)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 909)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5825)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 924)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5632)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 924)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5696)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 925)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5760)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 925)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5824)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 924)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5633)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 924)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5697)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 925)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5761)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 925)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5825)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 940)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5632)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 940)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5696)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 941)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5760)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 941)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5824)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 940)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5633)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 940)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5697)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 941)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5761)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 941)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5825)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 956)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5632)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 956)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5696)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 957)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5760)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 957)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5824)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 956)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5633)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 956)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5697)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 957)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5761)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 957)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5825)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 972)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5632)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 972)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5696)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 973)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5760)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 973)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5824)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 972)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5633)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 972)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5697)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 973)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5761)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 973)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5825)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 988)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5632)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 988)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5696)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 989)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5760)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 989)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5824)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 988)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5633)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 988)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5697)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 989)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5761)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 989)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5825)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1004)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5632)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1004)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5696)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1005)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5760)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1005)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5824)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1004)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5633)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1004)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5697)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1005)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5761)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1005)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5825)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 910)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5888)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 910)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5952)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 911)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6016)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 911)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6080)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 910)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5889)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 910)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5953)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 911)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6017)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 911)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6081)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 926)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5888)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 926)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5952)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 927)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6016)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 927)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6080)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 926)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5889)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 926)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5953)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 927)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6017)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 927)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6081)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 942)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5888)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 942)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5952)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 943)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6016)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 943)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6080)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 942)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5889)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 942)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5953)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 943)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6017)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 943)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6081)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 958)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5888)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 958)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5952)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 959)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6016)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 959)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6080)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 958)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5889)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 958)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5953)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 959)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6017)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 959)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6081)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 974)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5888)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 974)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5952)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 975)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6016)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 975)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6080)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 974)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5889)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 974)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5953)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 975)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6017)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 975)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6081)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 990)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5888)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 990)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5952)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 991)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6016)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 991)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6080)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 990)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5889)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 990)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5953)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 991)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6017)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 991)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6081)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1006)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5888)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1006)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5952)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1007)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6016)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1007)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6080)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1006)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5889)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1006)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 5953)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1007)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6017)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1007)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 6081)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((int)threadIdx.x) >> 5) * 112)] * weight_shared[((((int)threadIdx.x) & 31) * 2)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((int)threadIdx.x) >> 5) * 112)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((int)threadIdx.x) >> 5) * 112)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((int)threadIdx.x) >> 5) * 112)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 1)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 16)] * weight_shared[((((int)threadIdx.x) & 31) * 2)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 16)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 17)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 17)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 16)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 16)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 17)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 17)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 32)] * weight_shared[((((int)threadIdx.x) & 31) * 2)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 32)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 33)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 33)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 32)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 32)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 33)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 33)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 48)] * weight_shared[((((int)threadIdx.x) & 31) * 2)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 48)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 49)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 49)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 48)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 48)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 49)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 49)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 64)] * weight_shared[((((int)threadIdx.x) & 31) * 2)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 64)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 65)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 65)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 64)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 64)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 65)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 65)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 80)] * weight_shared[((((int)threadIdx.x) & 31) * 2)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 80)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 81)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 81)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 80)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 80)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 81)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 81)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 96)] * weight_shared[((((int)threadIdx.x) & 31) * 2)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 96)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 64)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 97)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 128)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 97)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 192)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 96)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 96)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 65)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 97)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 129)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 97)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 193)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 2)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 2)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 3)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 3)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 2)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 2)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 3)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 3)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 18)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 18)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 19)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 19)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 18)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 18)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 19)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 19)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 34)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 34)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 35)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 35)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 34)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 34)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 35)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 35)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 50)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 50)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 51)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 51)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 50)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 50)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 51)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 51)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 66)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 66)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 67)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 67)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 66)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 66)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 67)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 67)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 82)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 82)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 83)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 83)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 82)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 82)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 83)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 83)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 98)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 256)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 98)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 320)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 99)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 384)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 99)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 448)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 98)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 257)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 98)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 321)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 99)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 385)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 99)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 449)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 4)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 4)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 5)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 5)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 4)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 4)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 5)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 5)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 20)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 20)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 21)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 21)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 20)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 20)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 21)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 21)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 36)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 36)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 37)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 37)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 36)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 36)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 37)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 37)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 52)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 52)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 53)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 53)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 52)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 52)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 53)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 53)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 68)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 68)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 69)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 69)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 68)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 68)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 69)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 69)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 84)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 84)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 85)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 85)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 84)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 84)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 85)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 85)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 100)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 512)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 100)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 576)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 101)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 640)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 101)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 704)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 100)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 513)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 100)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 577)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 101)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 641)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 101)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 705)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 6)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 6)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 7)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 7)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 6)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 6)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 7)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 7)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 22)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 22)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 23)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 23)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 22)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 22)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 23)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 23)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 38)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 38)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 39)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 39)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 38)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 38)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 39)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 39)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 54)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 54)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 55)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 55)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 54)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 54)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 55)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 55)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 70)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 70)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 71)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 71)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 70)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 70)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 71)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 71)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 86)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 86)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 87)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 87)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 86)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 86)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 87)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 87)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 102)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 768)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 102)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 832)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 103)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 896)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 103)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 960)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 102)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 769)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 102)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 833)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 103)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 897)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 103)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 961)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 8)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 8)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 9)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 9)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 8)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 8)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 9)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 9)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 24)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 24)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 25)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 25)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 24)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 24)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 25)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 25)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 40)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 40)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 41)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 41)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 40)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 40)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 41)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 41)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 56)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 56)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 57)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 57)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 56)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 56)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 57)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 57)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 72)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 72)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 73)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 73)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 72)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 72)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 73)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 73)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 88)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 88)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 89)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 89)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 88)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 88)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 89)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 89)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 104)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1024)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 104)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1088)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 105)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1152)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 105)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1216)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 104)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1025)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 104)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1089)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 105)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1153)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 105)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1217)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 10)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 10)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 11)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 11)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 10)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 10)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 11)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 11)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 26)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 26)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 27)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 27)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 26)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 26)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 27)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 27)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 42)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 42)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 43)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 43)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 42)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 42)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 43)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 43)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 58)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 58)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 59)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 59)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 58)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 58)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 59)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 59)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 74)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 74)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 75)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 75)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 74)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 74)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 75)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 75)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 90)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 90)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 91)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 91)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 90)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 90)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 91)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 91)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 106)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1280)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 106)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1344)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 107)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1408)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 107)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1472)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 106)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1281)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 106)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1345)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 107)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1409)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 107)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1473)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 12)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 12)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 13)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 13)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 12)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 12)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 13)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 13)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 28)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 28)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 29)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 29)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 28)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 28)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 29)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 29)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 44)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 44)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 45)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 45)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 44)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 44)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 45)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 45)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 60)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 60)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 61)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 61)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 60)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 60)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 61)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 61)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 76)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 76)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 77)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 77)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 76)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 76)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 77)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 77)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 92)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 92)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 93)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 93)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 92)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 92)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 93)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 93)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 108)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1536)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 108)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1600)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 109)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1664)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 109)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1728)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 108)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1537)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 108)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1601)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 109)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1665)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 109)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1729)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 14)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 14)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 15)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 15)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 14)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 14)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 15)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 15)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 30)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 30)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 31)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  conv2d_nhwc_local[16] = (conv2d_nhwc_local[16] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 31)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 30)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 30)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 31)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  conv2d_nhwc_local[17] = (conv2d_nhwc_local[17] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 31)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 46)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 46)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 47)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  conv2d_nhwc_local[18] = (conv2d_nhwc_local[18] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 47)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 46)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 46)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 47)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  conv2d_nhwc_local[19] = (conv2d_nhwc_local[19] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 47)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 62)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 62)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 63)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  conv2d_nhwc_local[20] = (conv2d_nhwc_local[20] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 63)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 62)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 62)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 63)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  conv2d_nhwc_local[21] = (conv2d_nhwc_local[21] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 63)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 78)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 78)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 79)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  conv2d_nhwc_local[22] = (conv2d_nhwc_local[22] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 79)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 78)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 78)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 79)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  conv2d_nhwc_local[23] = (conv2d_nhwc_local[23] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 79)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 94)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 94)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 95)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  conv2d_nhwc_local[24] = (conv2d_nhwc_local[24] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 95)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 94)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 94)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 95)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  conv2d_nhwc_local[25] = (conv2d_nhwc_local[25] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 95)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 110)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1792)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 110)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1856)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 111)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1920)]));
  conv2d_nhwc_local[26] = (conv2d_nhwc_local[26] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 111)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1984)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 110)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1793)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 110)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1857)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 111)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1921)]));
  conv2d_nhwc_local[27] = (conv2d_nhwc_local[27] + (PadInput_shared[(((((int)threadIdx.x) >> 5) * 112) + 111)] * weight_shared[(((((int)threadIdx.x) & 31) * 2) + 1985)]));
  conv2d_nhwc[(((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2))] = conv2d_nhwc_local[0];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 64)] = conv2d_nhwc_local[14];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 1)] = conv2d_nhwc_local[1];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 65)] = conv2d_nhwc_local[15];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 1024)] = conv2d_nhwc_local[2];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 1088)] = conv2d_nhwc_local[16];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 1025)] = conv2d_nhwc_local[3];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 1089)] = conv2d_nhwc_local[17];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 2048)] = conv2d_nhwc_local[4];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 2112)] = conv2d_nhwc_local[18];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 2049)] = conv2d_nhwc_local[5];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 2113)] = conv2d_nhwc_local[19];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 3072)] = conv2d_nhwc_local[6];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 3136)] = conv2d_nhwc_local[20];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 3073)] = conv2d_nhwc_local[7];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 3137)] = conv2d_nhwc_local[21];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 4096)] = conv2d_nhwc_local[8];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 4160)] = conv2d_nhwc_local[22];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 4097)] = conv2d_nhwc_local[9];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 4161)] = conv2d_nhwc_local[23];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 5120)] = conv2d_nhwc_local[10];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 5184)] = conv2d_nhwc_local[24];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 5121)] = conv2d_nhwc_local[11];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 5185)] = conv2d_nhwc_local[25];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 6144)] = conv2d_nhwc_local[12];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 6208)] = conv2d_nhwc_local[26];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 6145)] = conv2d_nhwc_local[13];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + ((((int)blockIdx.x) & 7) * 128)) + ((((int)threadIdx.x) & 31) * 2)) + 6209)] = conv2d_nhwc_local[27];
}


