
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[16];
  __shared__ float PadInput_shared[2048];
  __shared__ float weight_shared[4096];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[8] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[9] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[10] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[11] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[12] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[13] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  conv2d_nhwc_local[14] = 0.000000e+00f;
  conv2d_nhwc_local[7] = 0.000000e+00f;
  conv2d_nhwc_local[15] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)threadIdx.x) & 7) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)threadIdx.x) & 7) * 4)) + 14336))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)threadIdx.x) & 7) * 4)) + 28672))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)threadIdx.x) & 7) * 4)) + 43008))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) * 4)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 512))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1024))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1536))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2048))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2560))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3072))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3584))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)threadIdx.x) & 7) * 4)) + 32))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)threadIdx.x) & 7) * 4)) + 14368))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)threadIdx.x) & 7) * 4)) + 28704))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)threadIdx.x) & 7) * 4)) + 43040))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 4096))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1152)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 4608))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 5120))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1408)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 5632))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1536)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 6144))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1664)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 6656))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 7168))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 1920)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 7680))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)threadIdx.x) & 7) * 4)) + 64))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 1152)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)threadIdx.x) & 7) * 4)) + 14400))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 1280)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)threadIdx.x) & 7) * 4)) + 28736))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 4) + 1408)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)threadIdx.x) & 7) * 4)) + 43072))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 8192))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2176)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 8704))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2304)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 9216))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2432)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 9728))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2560)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 10240))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2688)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 10752))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2816)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 11264))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((int)threadIdx.x) * 4) + 2944)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 11776))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int rh_0_rw_0_rc_0_fused = 0; rh_0_rw_0_rc_0_fused < 13; ++rh_0_rw_0_rc_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 512) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 96))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 512) + (((int)threadIdx.x) * 4)) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14432))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 512) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 28768))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 512) + (((int)threadIdx.x) * 4)) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((((int)blockIdx.x) / 28) * 57344) + (((((int)blockIdx.x) % 28) >> 2) * 2048)) + ((((int)threadIdx.x) >> 3) * 512)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 43104))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1024) + (((int)threadIdx.x) * 4))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 12288))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1024) + (((int)threadIdx.x) * 4)) + 128)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 12800))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1024) + (((int)threadIdx.x) * 4)) + 256)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 13312))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1024) + (((int)threadIdx.x) * 4)) + 384)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 13824))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1024) + (((int)threadIdx.x) * 4)) + 512)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14336))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1024) + (((int)threadIdx.x) * 4)) + 640)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 14848))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1024) + (((int)threadIdx.x) * 4)) + 768)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 15360))), "n"(16)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 3) & 3) * 1024) + (((int)threadIdx.x) * 4)) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 15872))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

    __syncthreads();
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256))] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 64)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256))] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 32)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 96)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 32)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 128)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 192)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 160)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 224)] * weight_shared[(((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 160)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 224)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 161)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 225)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 161)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 225)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 162)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 226)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 162)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 226)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 163)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 227)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 163)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 227)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 164)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 228)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 164)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 228)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 165)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 229)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 165)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 229)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 166)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 230)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 166)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 230)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 167)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 231)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 167)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 231)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 168)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 232)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 168)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 232)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 169)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 233)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 169)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 233)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 170)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 234)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 170)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 234)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 171)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 235)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 171)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 235)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 172)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 236)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 172)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 236)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 173)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 237)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 173)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 237)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 174)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 238)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 174)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 238)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 175)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 239)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 175)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 239)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 112)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 112)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 113)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 113)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 114)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 114)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 115)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 115)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 144)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 208)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 144)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 208)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 176)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 240)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 176)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 240)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 209)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 209)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 177)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 241)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 177)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 241)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 210)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 210)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 178)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 242)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 178)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 242)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 211)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 211)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 179)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 243)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 179)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 243)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 116)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 116)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 117)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 117)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 118)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 118)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 119)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 119)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 212)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 212)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 180)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 244)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 180)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 244)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 213)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 213)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 181)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 245)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 181)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 245)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 214)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 214)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 182)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 246)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 182)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 246)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 215)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 215)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 183)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 247)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 183)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 247)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 120)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 120)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 121)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 121)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 122)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 122)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 123)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 123)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 216)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 216)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 184)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 248)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 184)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 248)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 217)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 217)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 185)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 249)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 185)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 249)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 218)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 218)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 186)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 250)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 186)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 250)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 219)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 219)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 187)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 251)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 187)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 251)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 124)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 124)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 125)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 125)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 126)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 126)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 127)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 127)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 220)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 220)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 188)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 252)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 188)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 252)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 221)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 221)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 189)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 253)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 189)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 253)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 222)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 222)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 190)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 254)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 190)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 254)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 223)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 223)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 191)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 255)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 191)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused & 3) * 512) + ((((int)threadIdx.x) >> 4) * 256)) + 255)] * weight_shared[((((rh_0_rw_0_rc_0_fused & 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
  }
__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 512)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 576)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 512)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 576)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 544)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 608)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 544)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 608)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 513)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 577)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 513)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 577)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 545)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 609)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 545)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 609)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 514)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 578)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 514)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 578)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 546)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 610)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 546)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 610)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 515)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 579)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 515)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 579)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 547)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 611)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 547)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 611)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 640)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 704)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 640)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 704)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 672)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 672)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 641)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 705)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 641)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 705)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 673)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 673)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 642)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 706)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 642)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 706)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 674)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 674)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 643)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 707)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 643)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 707)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 675)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 675)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 516)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 580)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 516)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 580)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 548)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 612)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 548)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 612)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 517)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 581)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 517)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 581)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 549)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 613)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 549)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 613)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 518)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 582)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 518)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 582)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 550)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 614)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 550)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 614)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 519)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 583)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 519)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 583)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 551)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 615)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 551)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 615)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 644)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 708)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 644)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 708)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 676)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 676)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 645)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 709)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 645)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 709)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 677)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 677)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 646)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 710)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 646)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 710)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 678)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 678)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 647)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 711)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 647)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 711)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 679)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 679)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 520)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 584)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 520)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 584)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 552)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 616)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 552)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 616)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 521)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 585)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 521)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 585)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 553)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 617)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 553)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 617)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 522)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 586)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 522)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 586)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 554)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 618)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 554)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 618)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 523)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 587)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 523)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 587)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 555)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 619)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 555)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 619)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 648)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 712)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 648)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 712)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 680)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 680)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 649)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 713)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 649)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 713)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 681)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 681)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 650)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 714)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 650)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 714)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 682)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 682)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 651)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 715)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 651)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 715)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 683)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 683)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 524)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 588)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 524)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 588)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 556)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 620)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 556)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 620)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 525)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 589)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 525)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 589)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 557)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 621)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 557)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 621)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 526)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 590)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 526)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 590)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 558)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 622)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 558)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 622)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 527)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 591)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 527)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 591)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 559)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 623)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 559)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 623)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 652)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 716)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 652)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 716)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 684)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 748)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 684)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 748)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 653)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 717)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 653)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 717)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 685)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 749)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 685)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 749)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 654)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 718)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 654)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 718)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 686)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 750)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 686)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 750)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 655)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 719)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 655)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 719)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 687)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 751)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 687)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 751)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 528)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 592)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 528)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 592)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 560)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 624)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 560)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 624)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 529)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 593)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 529)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 593)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 561)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 625)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 561)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 625)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 530)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 594)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 530)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 594)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 562)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 626)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 562)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 626)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 531)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 595)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 531)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 595)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 563)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 627)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 563)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 627)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 656)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 720)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 656)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 720)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 688)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 752)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 688)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 752)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 721)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 721)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 689)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 753)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 689)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 753)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 722)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 722)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 690)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 754)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 690)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 754)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 723)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 723)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 691)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 755)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 691)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 755)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 532)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 596)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 532)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 596)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 564)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 628)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 564)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 628)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 533)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 597)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 533)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 597)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 565)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 629)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 565)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 629)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 534)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 598)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 534)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 598)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 566)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 630)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 566)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 630)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 535)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 599)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 535)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 599)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 567)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 631)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 567)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 631)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 724)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 724)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 692)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 756)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 692)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 756)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 725)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 725)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 693)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 757)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 693)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 757)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 726)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 726)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 694)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 758)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 694)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 758)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 727)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 727)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 695)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 759)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 695)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 759)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 536)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 600)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 536)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 600)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 568)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 632)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 568)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 632)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 537)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 601)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 537)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 601)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 569)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 633)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 569)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 633)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 538)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 602)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 538)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 602)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 570)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 634)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 570)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 634)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 539)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 603)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 539)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 603)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 571)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 635)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 571)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 635)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 664)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 664)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 696)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 760)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 696)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 760)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 665)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 665)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 697)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 761)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 697)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 761)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 666)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 666)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 698)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 762)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 698)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 762)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 667)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 667)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 699)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 763)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 699)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 763)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 540)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 604)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 540)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 604)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 572)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 636)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 572)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 636)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 541)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 605)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 541)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 605)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 573)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 637)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 573)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 637)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 542)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 606)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 542)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 606)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 574)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 638)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 574)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 638)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 543)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 607)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 543)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 607)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 575)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 639)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 575)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 639)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 668)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 668)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 700)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 764)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 700)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 764)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 669)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 669)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 701)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 765)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 701)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 765)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 670)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 670)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 702)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 766)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 702)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 766)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 671)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 671)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 703)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 767)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 703)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 767)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1024)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1088)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1024)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1088)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1120)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1056)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1120)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1025)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1089)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1025)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1089)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1121)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1057)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1121)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1026)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1090)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1026)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1090)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1122)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1058)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1122)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1027)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1091)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1027)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1091)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1123)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1059)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1123)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1216)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1216)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1184)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1248)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1184)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1248)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1217)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1217)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1185)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1249)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1185)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1249)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1218)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1218)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1186)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1250)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1186)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1250)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1219)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1219)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1187)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1251)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1187)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1251)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1028)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1092)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1028)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1092)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1124)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1060)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1124)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1029)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1093)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1029)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1093)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1125)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1061)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1125)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1030)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1094)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1030)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1094)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1126)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1062)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1126)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1031)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1095)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1031)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1095)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1127)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1063)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1127)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1220)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1220)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1188)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1252)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1188)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1252)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1221)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1221)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1189)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1253)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1189)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1253)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1222)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1222)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1190)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1254)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1190)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1254)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1223)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1223)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1191)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1255)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1191)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1255)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1032)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1096)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1032)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1096)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1128)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1064)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1128)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1033)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1097)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1033)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1097)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1129)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1065)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1129)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1034)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1098)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1034)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1098)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1130)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1066)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1130)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1035)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1099)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1035)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1099)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1131)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1067)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1131)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1160)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1224)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1160)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1224)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1256)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1256)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1161)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1225)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1161)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1225)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1257)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1257)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1162)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1226)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1162)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1226)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1258)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1258)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1163)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1227)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1163)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1227)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1259)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1259)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1036)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1100)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1036)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1100)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1132)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1068)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1132)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1037)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1101)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1037)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1101)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1133)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1069)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1133)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1038)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1102)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1038)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1102)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1134)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1070)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1134)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1039)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1103)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1039)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1103)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1135)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1071)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1135)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1164)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1228)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1164)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1228)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1260)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1260)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1165)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1229)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1165)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1229)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1261)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1261)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1166)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1230)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1166)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1230)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1262)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1262)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1167)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1231)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1167)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1231)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1263)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1263)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1040)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1104)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1040)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1104)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1072)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1136)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1072)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1136)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1041)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1105)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1041)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1105)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1073)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1137)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1073)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1137)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1042)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1106)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1042)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1106)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1074)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1138)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1074)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1138)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1043)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1107)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1043)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1107)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1075)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1139)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1075)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1139)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1168)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1232)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1168)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1232)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1264)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1264)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1169)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1233)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1169)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1233)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1265)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1265)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1170)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1234)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1170)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1234)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1266)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1266)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1171)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1235)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1171)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1235)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1267)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1267)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1044)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1108)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1044)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1108)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1076)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1140)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1076)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1140)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1045)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1109)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1045)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1109)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1077)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1141)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1077)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1141)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1046)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1110)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1046)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1110)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1078)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1142)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1078)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1142)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1047)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1111)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1047)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1111)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1079)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1143)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1079)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1143)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1172)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1236)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1172)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1236)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1268)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1268)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1173)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1237)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1173)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1237)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1269)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1269)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1174)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1238)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1174)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1238)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1270)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1270)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1175)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1239)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1175)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1239)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1271)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1271)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1048)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1112)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1048)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1112)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1080)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1144)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1080)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1144)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1049)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1113)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1049)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1113)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1081)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1081)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1050)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1114)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1050)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1114)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1082)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1082)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1051)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1115)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1051)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1115)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1083)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1083)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1176)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1240)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1176)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1240)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1208)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1272)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1208)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1272)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1177)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1241)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1177)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1241)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1209)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1273)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1209)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1273)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1178)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1242)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1178)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1242)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1210)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1274)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1210)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1274)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1179)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1243)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1179)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1243)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1211)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1275)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1211)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1275)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1116)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1052)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1116)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1084)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1084)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1117)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1053)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1117)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1085)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1085)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1118)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1054)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1118)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1086)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1086)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1119)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1055)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1119)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1087)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1087)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1180)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1244)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1180)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1244)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1212)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1276)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1212)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1276)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1181)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1245)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1181)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1245)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1213)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1277)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1213)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1277)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1182)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1246)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1182)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1246)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1214)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1278)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1214)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1278)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1183)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1247)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1183)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1247)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1215)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1279)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1215)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1279)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1536)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1600)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1536)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1600)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1568)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1632)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1568)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1632)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1537)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1601)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1537)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1601)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1569)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1633)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1569)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1633)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1538)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1602)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1538)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1602)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1570)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1634)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1570)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1634)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1539)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1603)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1539)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1603)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1571)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1635)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1571)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1635)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1664)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1664)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1728)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1696)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1760)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1696)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1760)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1665)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1665)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1729)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1697)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1761)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1697)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1761)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1666)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1666)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1730)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1698)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1762)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1698)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1762)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1667)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1667)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1731)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1699)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1763)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1699)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1763)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1540)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1604)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1540)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1604)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1572)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1636)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1572)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1636)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1541)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1605)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1541)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1605)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1573)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1637)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1573)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1637)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1542)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1606)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1542)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1606)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1574)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1638)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1574)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1638)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1543)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1607)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1543)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1607)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1575)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1639)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1575)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1639)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1668)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1668)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1732)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1700)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1764)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1700)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1764)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1669)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1669)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1733)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1701)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1765)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1701)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1765)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1670)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1670)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1734)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1702)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1766)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1702)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1766)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1671)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1671)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1735)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1703)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1767)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1703)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1767)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1544)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1608)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1544)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1608)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1576)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1640)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1576)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1640)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1545)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1609)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1545)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1609)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1577)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1641)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1577)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1641)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1546)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1610)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1546)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1610)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1578)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1642)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1578)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1642)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1547)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1611)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1547)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1611)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1579)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1643)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1579)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1643)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1672)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1672)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1736)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1704)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1768)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1704)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1768)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1673)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1673)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1737)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1705)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1769)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1705)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1769)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1674)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1674)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1738)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1706)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1770)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1706)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1770)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1675)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1675)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1739)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1707)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1771)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1707)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1771)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1548)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1612)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1548)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1612)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1580)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1644)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1580)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1644)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1549)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1613)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1549)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1613)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1581)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1645)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1581)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1645)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1550)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1614)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1550)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1614)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1582)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1646)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1582)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1646)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1551)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1615)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1551)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1615)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1583)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1647)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1583)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1647)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1676)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1676)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1740)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1708)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1772)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1708)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1772)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1677)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1677)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1741)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1709)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1773)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1709)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1773)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1678)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1678)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1742)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1710)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1774)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1710)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1774)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1679)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1679)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1743)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1711)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1775)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1711)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1775)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1552)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1616)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1552)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1616)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1584)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1648)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1584)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1648)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1553)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1617)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1553)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1617)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1585)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1649)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1585)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1649)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1554)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1618)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1554)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1618)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1586)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1650)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1586)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1650)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1555)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1619)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1555)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1619)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1587)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1651)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1587)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1651)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1680)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1680)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1744)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1712)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1776)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1712)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1776)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1681)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1681)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1745)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1713)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1777)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1713)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1777)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1682)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1682)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1746)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1714)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1778)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1714)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1778)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1683)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1683)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1747)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1715)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1779)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1715)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1779)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1556)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1620)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1556)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1620)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1588)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1652)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1588)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1652)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1557)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1621)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1557)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1621)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1589)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1653)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1589)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1653)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1558)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1622)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1558)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1622)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1590)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1654)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1590)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1654)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1559)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1623)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1559)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1623)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1591)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1655)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1591)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1655)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1684)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1748)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1684)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1748)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1716)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1780)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1716)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1780)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1685)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1749)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1685)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1749)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1717)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1781)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1717)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1781)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1686)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1750)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1686)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1750)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1718)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1782)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1718)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1782)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1687)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1751)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1687)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1751)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1719)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1783)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1719)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1783)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1560)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1624)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1560)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1624)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1592)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1656)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1592)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1656)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1561)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1625)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1561)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1625)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1593)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1593)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1657)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1562)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1626)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1562)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1626)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1594)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1594)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1658)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1563)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1627)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1563)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1627)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1595)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1595)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1659)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1688)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1752)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1688)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1752)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1720)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1784)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1720)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1784)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1689)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1753)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1689)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1753)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1721)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1785)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1721)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1785)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1690)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1754)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1690)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1754)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1722)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1786)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1722)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1786)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1691)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1755)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1691)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1755)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1723)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1787)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1723)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1787)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1564)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1628)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1564)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1628)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1596)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1596)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1660)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1565)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1629)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1565)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1629)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1597)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1597)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1661)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1566)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1630)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1566)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1630)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1598)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1598)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1662)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1567)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1631)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1567)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1631)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1599)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1599)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1663)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1692)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1756)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1692)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1756)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1724)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1788)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1724)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1788)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1693)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1757)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1693)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1757)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1725)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1789)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1725)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1789)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1694)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1758)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1694)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1758)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1726)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1790)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1726)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1790)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1695)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1759)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1695)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1759)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1727)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1791)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1727)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1791)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc[((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2))] = conv2d_nhwc_local[0];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 256)] = conv2d_nhwc_local[8];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 1)] = conv2d_nhwc_local[1];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 257)] = conv2d_nhwc_local[9];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 128)] = conv2d_nhwc_local[2];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 384)] = conv2d_nhwc_local[10];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 129)] = conv2d_nhwc_local[3];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 385)] = conv2d_nhwc_local[11];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3584)] = conv2d_nhwc_local[4];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3840)] = conv2d_nhwc_local[12];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3585)] = conv2d_nhwc_local[5];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3841)] = conv2d_nhwc_local[13];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3712)] = conv2d_nhwc_local[6];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3968)] = conv2d_nhwc_local[14];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3713)] = conv2d_nhwc_local[7];
  conv2d_nhwc[(((((((((int)blockIdx.x) / 28) * 14336) + ((((int)threadIdx.x) >> 4) * 7168)) + (((((int)blockIdx.x) % 28) >> 2) * 512)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3969)] = conv2d_nhwc_local[15];
}


