
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[14];
  __shared__ float PadInput_shared[4704];
  __shared__ float weight_shared[3072];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  conv2d_nhwc_local[7] = 0.000000e+00f;
  conv2d_nhwc_local[8] = 0.000000e+00f;
  conv2d_nhwc_local[9] = 0.000000e+00f;
  conv2d_nhwc_local[10] = 0.000000e+00f;
  conv2d_nhwc_local[11] = 0.000000e+00f;
  conv2d_nhwc_local[12] = 0.000000e+00f;
  conv2d_nhwc_local[13] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) * 2)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)) + 3584))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)) + 7168))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)) + 10752))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)) + 14336))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1120)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)) + 17920))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1344)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)) + 21504))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((int)threadIdx.x)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 112)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 112) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 14336))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 336)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 336) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 28672))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 560)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 560) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 43008))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 784)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 784) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 57344))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 16) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1008)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 63504))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1568)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)) + 32))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)) + 3616))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2016)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)) + 7200))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2240)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)) + 10784))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2464)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)) + 14368))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2688)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)) + 17952))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2912)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)threadIdx.x) & 15) * 2)) + 21536))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 65536))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1136)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 112) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 65536))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1248)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 79872))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1360)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 336) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 65536))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1472)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 94208))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1584)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 560) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 65536))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1696)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 108544))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1808)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 784) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 65536))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1920)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 5) * 2048) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 122880))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 16) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2032)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) + 129040))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int rh_0_rw_0_rc_0_fused = 0; rh_0_rw_0_rc_0_fused < 14; ++rh_0_rw_0_rc_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1568) + (((int)threadIdx.x) * 2))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)threadIdx.x) >> 4) * 512) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 64))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1568) + (((int)threadIdx.x) * 2)) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)threadIdx.x) >> 4) * 512) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3648))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1568) + (((int)threadIdx.x) * 2)) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)threadIdx.x) >> 4) * 512) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 7232))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1568) + (((int)threadIdx.x) * 2)) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)threadIdx.x) >> 4) * 512) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 10816))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1568) + (((int)threadIdx.x) * 2)) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)threadIdx.x) >> 4) * 512) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 14400))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1568) + (((int)threadIdx.x) * 2)) + 1120)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)threadIdx.x) >> 4) * 512) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 17984))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1568) + (((int)threadIdx.x) * 2)) + 1344)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)threadIdx.x) >> 4) * 512) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 21568))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + ((int)threadIdx.x))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 65536) + ((((int)threadIdx.x) >> 5) * 2048)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 131072))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + ((int)threadIdx.x)) + 112)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 65536) + (((((int)threadIdx.x) + 112) >> 5) * 2048)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 131072))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + ((int)threadIdx.x)) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 65536) + ((((int)threadIdx.x) >> 5) * 2048)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 145408))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + ((int)threadIdx.x)) + 336)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 65536) + (((((int)threadIdx.x) + 336) >> 5) * 2048)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 131072))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + ((int)threadIdx.x)) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 65536) + ((((int)threadIdx.x) >> 5) * 2048)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 159744))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + ((int)threadIdx.x)) + 560)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 65536) + (((((int)threadIdx.x) + 560) >> 5) * 2048)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 131072))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + ((int)threadIdx.x)) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 65536) + ((((int)threadIdx.x) >> 5) * 2048)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 174080))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + ((int)threadIdx.x)) + 784)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 65536) + (((((int)threadIdx.x) + 784) >> 5) * 2048)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)) + 131072))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + ((int)threadIdx.x)) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 65536) + ((((int)threadIdx.x) >> 5) * 2048)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 188416))), "n"(4)
    );
  }
    if (((int)threadIdx.x) < 16) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1024) + ((int)threadIdx.x)) + 1008)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((rh_0_rw_0_rc_0_fused * 65536) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 194576))), "n"(4)
    );
  }
    }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224))] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224))] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 32)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 32)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 33)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 34)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 35)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 36)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 37)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 38)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 39)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 40)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 41)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 42)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 43)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 44)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 45)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 46)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 47)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 64)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 96)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 96)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 97)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 98)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 99)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 100)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 101)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 102)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 103)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 104)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 105)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 106)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 107)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 108)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 109)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 110)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 111)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 128)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 160)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 161)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 162)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 163)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 164)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 165)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 166)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 167)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 168)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 169)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 170)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 171)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 172)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 173)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 174)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 175)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 160)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 161)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 162)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 163)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 164)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 165)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 166)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 167)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 168)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 169)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 170)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 171)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 172)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 173)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 174)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 175)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 192)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 48)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 49)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 50)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 51)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 52)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 53)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 54)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 55)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 56)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 57)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 58)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 59)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 60)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 61)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 62)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 63)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 112)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 113)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 114)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 115)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 116)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 117)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 118)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 119)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 120)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 121)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 122)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 123)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 124)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 125)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 126)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 127)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 112)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 113)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 114)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 115)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 116)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 117)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 118)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 119)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 120)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 121)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 122)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 123)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 124)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 125)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 126)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 127)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 144)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 144)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 176)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 177)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 178)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 179)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 180)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 181)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 182)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 183)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 184)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 185)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 186)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 187)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 188)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 189)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 190)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 191)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 176)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 177)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 178)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 179)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 180)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 181)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 182)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 183)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 184)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 185)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 186)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 187)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 188)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 189)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 190)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 191)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 208)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 209)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 210)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 211)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 212)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 213)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 214)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 215)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 216)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 217)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 218)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 219)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 220)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 221)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 222)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 223)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 208)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 209)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 210)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 211)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 212)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 213)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 214)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 215)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 216)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 217)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 218)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 219)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 220)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 221)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 222)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1568) + ((((int)threadIdx.x) >> 4) * 224)) + 223)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1024) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3136)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3137)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3138)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3139)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3140)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3141)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3142)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3143)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3144)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3136)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3137)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3138)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3139)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3140)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3141)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3142)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3143)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3144)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3168)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3169)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3170)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3171)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3172)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3173)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3174)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3175)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3176)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3177)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3178)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3179)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3180)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3181)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3182)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3183)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3168)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3169)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3170)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3171)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3172)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3173)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3174)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3175)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3176)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3177)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3178)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3179)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3180)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3181)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3182)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3183)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3208)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3209)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3210)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3211)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3212)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3213)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3214)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3215)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3208)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3209)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3210)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3211)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3212)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3213)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3214)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3215)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3232)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3233)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3234)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3235)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3236)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3237)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3238)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3239)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3240)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3241)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3242)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3243)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3244)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3245)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3246)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3247)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3232)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3233)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3234)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3235)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3236)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3237)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3238)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3239)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3240)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3241)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3242)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3243)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3244)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3245)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3246)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3247)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3264)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3265)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3266)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3267)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3268)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3269)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3270)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3271)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3272)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3273)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3274)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3275)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3276)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3277)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3278)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3279)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3264)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3265)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3266)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3267)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3268)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3269)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3270)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3271)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3272)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3273)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3274)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3275)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3276)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3277)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3278)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3279)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3296)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3297)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3298)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3299)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3300)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3301)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3302)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3303)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3304)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3305)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3306)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3307)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3308)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3309)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3310)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3311)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3296)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3297)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3298)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3299)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3300)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3301)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3302)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3303)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3304)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3305)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3306)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3307)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3308)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3309)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3310)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3311)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3328)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3329)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3330)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3331)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3336)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3337)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3338)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3339)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3340)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3341)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3342)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3343)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3328)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3329)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3330)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3331)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3332)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3333)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3334)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3335)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3336)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3337)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3338)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3339)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3340)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3341)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3342)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3343)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3160)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3161)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3162)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3163)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3164)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3165)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3166)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3167)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3160)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3161)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3162)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3163)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3164)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3165)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3166)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3167)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3184)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3185)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3186)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3187)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3188)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3189)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3190)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3191)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3184)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3185)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3186)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3187)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3188)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3189)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3190)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3191)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3216)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3217)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3218)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3219)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3220)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3221)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3222)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3223)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3224)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3225)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3226)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3227)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3228)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3229)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3230)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3231)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3216)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3217)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3218)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3219)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3220)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3221)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3222)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3223)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3224)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3225)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3226)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3227)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3228)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3229)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3230)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3231)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3248)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3249)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3250)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3251)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3252)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3253)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3254)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3255)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3256)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3257)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3258)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3259)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3260)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3261)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3262)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3263)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3248)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3249)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3250)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3251)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3252)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3253)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3254)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3255)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3256)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3257)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3258)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3259)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3260)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3261)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3262)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3263)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3280)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3281)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3282)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3283)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3284)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3285)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3286)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3287)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3288)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3289)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3290)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3291)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3292)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3293)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3294)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3295)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3280)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3281)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3282)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3283)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3284)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3285)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3286)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3287)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3288)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3289)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3290)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3291)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3292)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3293)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3294)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3295)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3312)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3313)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3314)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3315)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3316)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3317)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3318)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3319)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3320)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3321)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3322)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3323)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3324)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3325)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3326)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3327)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3312)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3313)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3314)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3315)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3316)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3317)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3318)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3319)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3320)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3321)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3322)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3323)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3324)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3325)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3326)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3327)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3344)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3345)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3346)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3347)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3348)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3349)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3350)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3351)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3352)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3353)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3354)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3355)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3356)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3357)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3358)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3359)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3344)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3345)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3346)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3347)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3348)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3349)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3350)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3351)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3352)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3353)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3354)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3355)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3356)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3357)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3358)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3359)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((int)threadIdx.x) >> 4) * 224)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 1)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 2)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 4)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 5)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 6)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 7)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 8)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 9)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 10)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 11)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 12)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 13)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 14)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 15)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((int)threadIdx.x) >> 4) * 224)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 1)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 2)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 3)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 4)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 5)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 6)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 7)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 8)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 9)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 10)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 11)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 12)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 13)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 14)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 15)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 32)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 33)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 34)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 35)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 36)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 37)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 38)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 39)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 40)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 41)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 42)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 43)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 44)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 45)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 46)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 47)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 32)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 33)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 34)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 35)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 36)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 37)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 38)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 39)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 40)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 41)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 42)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 43)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 44)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 45)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 46)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 47)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 64)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 65)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 66)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 67)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 68)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 69)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 70)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 71)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 72)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 73)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 74)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 75)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 76)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 77)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 78)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 79)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 64)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 65)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 66)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 67)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 68)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 69)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 70)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 71)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 72)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 73)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 74)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 75)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 76)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 77)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 78)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 79)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 96)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 97)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 98)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 99)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 100)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 101)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 102)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 103)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 104)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 105)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 106)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 107)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 108)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 109)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 110)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 111)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 96)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 97)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 98)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 99)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 100)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 101)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 102)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 103)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 104)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 105)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 106)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 107)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 108)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 109)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 110)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 111)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 128)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 129)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 130)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 131)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 132)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 133)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 134)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 135)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 136)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 137)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 138)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 139)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 140)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 141)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 142)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 143)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 128)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 129)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 130)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 131)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 132)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 133)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 134)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 135)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 136)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 137)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 138)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 139)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 140)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 141)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 142)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 143)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 160)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 161)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 162)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 163)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 164)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 165)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 166)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 167)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 168)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 169)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 170)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 171)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 172)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 173)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 174)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 175)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 160)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 161)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 162)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 163)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 164)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 165)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 166)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 167)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 168)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 169)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 170)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 171)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 172)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 173)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 174)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 175)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 192)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 16)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 17)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 18)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 19)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 20)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 21)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 22)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 23)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 24)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 25)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 26)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 27)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 28)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 29)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 30)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 31)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 16)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 17)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 18)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 19)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 20)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 21)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 22)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 23)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 24)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 25)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 26)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 27)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 28)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 29)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 30)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 31)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 48)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 49)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 50)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 51)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 52)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 53)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 54)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 55)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 56)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 57)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 58)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 59)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 60)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 61)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 62)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 63)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 48)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 49)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 50)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 51)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 52)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 53)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 54)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 55)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 56)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 57)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 58)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 59)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 60)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 61)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 62)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 63)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 80)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 81)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 82)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 83)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 84)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 85)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 86)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 87)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 88)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 89)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 90)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 91)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 92)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 93)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 94)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 95)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 80)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 81)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 82)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 83)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 84)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 85)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 86)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 87)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 88)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 89)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 90)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 91)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 92)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 93)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 94)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 95)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 112)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 113)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 114)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 115)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 116)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 117)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 118)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 119)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 120)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 121)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 122)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 123)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 124)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 125)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 126)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 127)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 112)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 113)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 114)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 115)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 116)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 117)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 118)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 119)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 120)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 121)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 122)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 123)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 124)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 125)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 126)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 127)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 144)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 144)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 176)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 177)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 178)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 179)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 180)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 181)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 182)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 183)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 184)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 185)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 186)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 187)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 188)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 189)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 190)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 191)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 176)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 177)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 178)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 179)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 180)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 181)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 182)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 183)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 184)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 185)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 186)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 187)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 188)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 189)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 190)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 191)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 208)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 209)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 210)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 211)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 212)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 213)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 214)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 215)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 216)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 217)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 218)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 219)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 220)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 221)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 222)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 223)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 208)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 209)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 210)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 211)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 212)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 213)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 214)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 215)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 216)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 217)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 218)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 219)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 220)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 221)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 222)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 224) + 223)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc[((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2))] = conv2d_nhwc_local[0];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 1)] = conv2d_nhwc_local[1];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 2048)] = conv2d_nhwc_local[2];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 2049)] = conv2d_nhwc_local[3];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 4096)] = conv2d_nhwc_local[4];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 4097)] = conv2d_nhwc_local[5];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 6144)] = conv2d_nhwc_local[6];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 6145)] = conv2d_nhwc_local[7];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 8192)] = conv2d_nhwc_local[8];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 8193)] = conv2d_nhwc_local[9];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 10240)] = conv2d_nhwc_local[10];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 10241)] = conv2d_nhwc_local[11];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 12288)] = conv2d_nhwc_local[12];
  conv2d_nhwc[(((((((int)threadIdx.x) >> 4) * 14336) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 12289)] = conv2d_nhwc_local[13];
}


