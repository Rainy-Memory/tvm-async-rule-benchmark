
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) main_kernel0(float* __restrict__ inputs, float* __restrict__ weight, float* __restrict__ conv2d_nhwc) {
  float conv2d_nhwc_local[16];
  __shared__ float PadInput_shared[5280];
  __shared__ float weight_shared[6144];
  conv2d_nhwc_local[0] = 0.000000e+00f;
  conv2d_nhwc_local[8] = 0.000000e+00f;
  conv2d_nhwc_local[1] = 0.000000e+00f;
  conv2d_nhwc_local[9] = 0.000000e+00f;
  conv2d_nhwc_local[2] = 0.000000e+00f;
  conv2d_nhwc_local[10] = 0.000000e+00f;
  conv2d_nhwc_local[3] = 0.000000e+00f;
  conv2d_nhwc_local[11] = 0.000000e+00f;
  conv2d_nhwc_local[4] = 0.000000e+00f;
  conv2d_nhwc_local[12] = 0.000000e+00f;
  conv2d_nhwc_local[5] = 0.000000e+00f;
  conv2d_nhwc_local[13] = 0.000000e+00f;
  conv2d_nhwc_local[6] = 0.000000e+00f;
  conv2d_nhwc_local[14] = 0.000000e+00f;
  conv2d_nhwc_local[7] = 0.000000e+00f;
  conv2d_nhwc_local[15] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((int)threadIdx.x) * 2)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 1))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 1792))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 225)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 1793))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 3584))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 449)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 3585))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 5376))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 673)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 5377))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 7168))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 897)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 7169))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1120)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 8960))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1121)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 8961))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1344)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 10752))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1345)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 10753))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 96) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1568)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 12544))), "n"(4)
    );
  }
  }
  if (((int)threadIdx.x) < 96) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1569)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 12545))), "n"(4)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((int)threadIdx.x)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((int)threadIdx.x) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 112)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 112) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 224) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 336)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 336) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 16) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 896))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 560)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 560) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 672) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 784)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 784) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 16) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 1792))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1008)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1008) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1120)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1120) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1232)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1232) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 16) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1344)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 2688))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1456)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1456) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1568)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1568) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1680)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1680) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 16) & 63)))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 3584))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 1904)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) + 1904) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2016)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 2016) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 32))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1760)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 32))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1761)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 33))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1984)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 1824))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 1985)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 1825))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2208)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 3616))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2209)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 3617))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2432)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 5408))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2433)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 5409))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2656)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 7200))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2657)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 7201))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2880)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 8992))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 2881)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 8993))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 3104)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 10784))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 3105)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 10785))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 96) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 3328)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 12576))), "n"(4)
    );
  }
  }
  if (((int)threadIdx.x) < 96) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((int)threadIdx.x) * 2) + 3329)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + (((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + 12577))), "n"(4)
    );
  }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 4096))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2160)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 112) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 4096))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2272)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 224) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 4096))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2384)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 336) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 4096))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2496)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 4992))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2608)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 560) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 4096))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2720)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 672) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 4096))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2832)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 784) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 4096))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 2944)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 5888))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3056)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1008) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 4096))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3168)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1120) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 4096))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3280)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1232) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 4096))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3392)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 6784))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3504)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1456) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 4096))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3616)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1568) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 4096))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3728)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1680) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 4096))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3840)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((((int)threadIdx.x) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 7680))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 3952)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 1904) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 4096))), "n"(4)
    );
  }
  if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((int)threadIdx.x) + 4064)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + ((((((((int)threadIdx.x) + 2016) >> 6) * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 4128))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int rh_0_rw_0_rc_0_fused = 0; rh_0_rw_0_rc_0_fused < 6; ++rh_0_rw_0_rc_0_fused) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + ((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 64))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 1)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 65))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 1856))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 225)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 1857))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3648))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 449)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 3649))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 5440))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 673)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 5441))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 7232))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 897)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 7233))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 1120)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 9024))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 1121)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 9025))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 1344)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 10816))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 1345)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 10817))), "n"(4)
    );
  }
    if (((int)threadIdx.x) < 96) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 1568)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 12608))), "n"(4)
    );
  }
    }
    if (((int)threadIdx.x) < 96) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(PadInput_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 1760) + (((int)threadIdx.x) * 2)) + 1569)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(inputs + ((((((((int)blockIdx.x) >> 1) * 28672) + ((((int)threadIdx.x) >> 4) * 256)) + (rh_0_rw_0_rc_0_fused * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 12609))), "n"(4)
    );
  }
    }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + ((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 112)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 112) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 224)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 224) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 336)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 336) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 448)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 9088))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 560)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 560) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 672)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 672) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 784)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 784) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 896)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 9984))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 1008)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 1008) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 1120)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 1120) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 1232)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 1232) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 1344)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 10880))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 1456)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 1456) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 1568)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 1568) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 1680)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 1680) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 16) & 63)) + 8192))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 1792)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 11776))), "n"(4)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 1904)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 1904) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)) + 8192))), "n"(4)
    );
  }
    if (((int)threadIdx.x) < 32) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(weight_shared + (((((rh_0_rw_0_rc_0_fused + 2) % 3) * 2048) + ((int)threadIdx.x)) + 2016)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(weight + (((((rh_0_rw_0_rc_0_fused * 4096) + (((((int)threadIdx.x) + 2016) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 8224))), "n"(4)
    );
  }
    }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256))] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256))] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256))] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256))] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 1)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 2)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 3)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 64)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 64)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 65)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 66)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 67)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 128)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 128)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 129)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 130)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 131)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 192)] * weight_shared[(((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2))]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 192)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 64)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 96)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 65)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 193)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 97)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 128)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 160)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 129)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 194)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 161)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 192)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 224)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 193)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 195)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 225)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 4)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 5)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 6)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 7)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 68)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 69)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 70)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 71)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 132)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 133)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 134)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 135)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 256)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 288)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 257)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 196)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 289)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 320)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 352)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 321)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 197)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 353)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 384)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 416)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 385)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 198)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 417)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 448)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 480)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 449)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 199)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 481)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 8)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 9)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 10)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 11)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 72)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 73)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 74)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 75)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 136)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 137)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 138)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 139)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 512)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 544)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 513)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 200)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 545)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 576)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 608)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 577)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 201)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 609)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 640)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 672)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 641)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 202)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 673)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 704)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 736)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 705)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 203)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 737)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 12)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 13)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 14)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 15)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 76)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 77)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 78)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 79)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 140)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 141)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 142)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 143)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 768)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 800)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 769)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 204)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 801)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 832)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 864)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 833)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 205)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 865)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 896)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 928)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 897)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 206)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 929)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 960)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 992)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 961)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 207)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 993)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1024)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1056)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1025)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 16)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1057)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1088)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1120)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1089)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 17)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1121)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1152)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1184)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1153)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 18)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1185)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1216)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1248)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1217)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 19)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1249)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1024)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1056)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1025)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 80)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1057)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1088)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1120)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1089)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 81)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1121)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1152)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1184)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1153)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 82)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1185)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1216)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1248)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1217)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 83)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1249)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 144)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1024)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 144)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1056)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 144)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1025)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 144)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1057)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1088)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1120)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1089)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 145)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1121)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1152)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1184)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1153)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 146)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1185)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1216)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1248)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1217)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 147)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1249)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 208)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1024)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 208)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1056)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 208)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1025)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 208)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1057)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 209)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1088)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 209)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1120)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 209)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1089)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 209)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1121)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 210)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1152)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 210)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1184)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 210)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1153)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 210)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1185)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 211)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1216)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 211)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1248)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 211)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1217)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 211)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1249)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1280)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1312)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1281)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 20)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1313)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1344)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1376)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1345)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 21)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1377)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1408)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1440)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1409)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 22)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1441)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1472)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1504)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1473)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 23)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1505)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1280)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1312)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1281)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 84)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1313)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1344)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1376)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1345)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 85)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1377)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1408)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1440)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1409)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 86)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1441)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1472)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1504)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1473)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 87)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1505)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1280)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1312)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1281)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 148)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1313)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1344)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1376)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1345)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 149)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1377)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1408)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1440)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1409)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 150)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1441)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1472)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1504)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1473)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 151)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1505)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 212)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1280)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 212)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1312)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 212)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1281)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 212)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1313)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 213)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1344)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 213)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1376)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 213)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1345)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 213)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1377)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 214)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1408)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 214)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1440)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 214)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1409)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 214)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1441)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 215)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1472)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 215)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1504)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 215)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1473)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 215)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1505)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1536)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1568)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1537)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 24)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1569)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1600)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1632)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1601)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 25)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1633)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1664)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1696)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1665)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 26)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1697)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1728)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1760)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1729)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 27)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1761)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1536)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1568)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1537)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 88)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1569)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1600)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1632)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1601)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 89)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1633)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1664)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1696)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1665)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 90)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1697)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1728)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1760)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1729)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 91)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1761)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1536)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1568)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1537)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 152)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1569)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1600)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1632)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1601)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 153)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1633)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1664)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1696)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1665)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 154)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1697)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1728)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1760)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1729)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 155)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1761)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 216)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1536)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 216)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1568)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 216)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1537)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 216)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1569)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 217)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1600)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 217)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1632)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 217)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1601)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 217)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1633)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 218)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1664)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 218)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1696)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 218)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1665)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 218)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1697)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 219)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1728)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 219)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1760)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 219)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1729)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 219)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1761)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1792)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1824)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1793)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 28)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1825)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1856)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1888)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1857)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 29)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1889)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1920)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1952)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1921)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 30)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1953)]));
    conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1984)]));
    conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 2016)]));
    conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1985)]));
    conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 31)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 2017)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1792)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1824)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1793)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 92)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1825)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1856)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1888)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1857)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 93)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1889)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1920)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1952)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1921)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 94)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1953)]));
    conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1984)]));
    conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 2016)]));
    conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1985)]));
    conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 95)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 2017)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1792)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1824)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1793)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 156)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1825)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1856)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1888)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1857)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 157)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1889)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1920)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1952)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1921)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 158)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1953)]));
    conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1984)]));
    conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 2016)]));
    conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1985)]));
    conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 159)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 2017)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 220)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1792)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 220)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1824)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 220)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1793)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 220)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1825)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 221)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1856)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 221)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1888)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 221)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1857)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 221)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1889)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 222)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1920)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 222)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1952)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 222)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1921)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 222)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1953)]));
    conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 223)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1984)]));
    conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 223)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 2016)]));
    conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 223)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 1985)]));
    conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[((((rh_0_rw_0_rc_0_fused % 3) * 1760) + ((((int)threadIdx.x) >> 4) * 256)) + 223)] * weight_shared[((((rh_0_rw_0_rc_0_fused % 3) * 2048) + ((((int)threadIdx.x) & 15) * 2)) + 2017)]));
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[((((int)threadIdx.x) >> 4) * 256)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[((((int)threadIdx.x) >> 4) * 256)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[((((int)threadIdx.x) >> 4) * 256)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[((((int)threadIdx.x) >> 4) * 256)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 2)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 2)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 2)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 2)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 3)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 3)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 3)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 3)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 64)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 64)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 64)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 64)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 65)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 65)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 65)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 65)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 66)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 66)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 66)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 66)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 67)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 67)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 67)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 67)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 128)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 128)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 128)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 128)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 129)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 129)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 129)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 129)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 130)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 130)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 130)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 130)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 131)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 131)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 131)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 131)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 192)] * weight_shared[((((int)threadIdx.x) & 15) * 2)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 32)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 192)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 33)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 64)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 96)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 65)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 193)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 97)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 128)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 160)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 129)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 194)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 161)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 192)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 224)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 193)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 195)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 225)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 4)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 4)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 4)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 4)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 5)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 5)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 5)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 5)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 6)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 6)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 6)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 6)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 7)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 7)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 7)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 7)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 68)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 68)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 68)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 68)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 69)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 69)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 69)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 69)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 70)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 70)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 70)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 70)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 71)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 71)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 71)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 71)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 132)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 132)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 132)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 132)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 133)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 133)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 133)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 133)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 134)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 134)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 134)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 134)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 135)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 135)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 135)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 135)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 256)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 288)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 257)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 196)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 289)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 320)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 352)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 321)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 197)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 353)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 384)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 416)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 385)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 198)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 417)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 448)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 480)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 449)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 199)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 481)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 8)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 8)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 8)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 8)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 9)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 9)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 9)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 9)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 10)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 10)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 10)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 10)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 11)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 11)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 11)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 11)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 72)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 72)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 72)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 72)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 73)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 73)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 73)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 73)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 74)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 74)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 74)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 74)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 75)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 75)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 75)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 75)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 136)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 136)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 136)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 136)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 137)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 137)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 137)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 137)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 138)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 138)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 138)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 138)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 139)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 139)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 139)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 139)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 512)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 544)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 513)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 200)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 545)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 576)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 608)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 577)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 201)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 609)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 640)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 672)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 641)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 202)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 673)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 704)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 736)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 705)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 203)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 737)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 12)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 12)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 12)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 12)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 13)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 13)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 13)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 13)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 14)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 14)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 14)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 14)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 15)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 15)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 15)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 15)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 76)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 76)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 76)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 76)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 77)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 77)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 77)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 77)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 78)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 78)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 78)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 78)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 79)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 79)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 79)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 79)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 140)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 140)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 140)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 140)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 141)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 141)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 141)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 141)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 142)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 142)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 142)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 142)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 143)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 143)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 143)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 143)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 768)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 800)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 769)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 204)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 801)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 832)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 864)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 833)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 205)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 865)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 896)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 928)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 897)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 206)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 929)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 960)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 992)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 961)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 207)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 993)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 16)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 16)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 16)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 16)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 17)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 17)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 17)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 17)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 18)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 18)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 18)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 18)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 19)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 19)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 19)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 19)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 80)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 80)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 80)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 80)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 81)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 81)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 81)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 81)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 82)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 82)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 82)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 82)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 83)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 83)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 83)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 83)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 144)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 144)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 144)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 144)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 145)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 146)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 147)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 208)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1024)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 208)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1056)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 208)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1025)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 208)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1057)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 209)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1088)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 209)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1120)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 209)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1089)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 209)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1121)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 210)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1152)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 210)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1184)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 210)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1153)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 210)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1185)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 211)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1216)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 211)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1248)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 211)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1217)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 211)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1249)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 20)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 20)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 20)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 20)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 21)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 21)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 21)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 21)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 22)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 22)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 22)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 22)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 23)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 23)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 23)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 23)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 84)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 84)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 84)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 84)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 85)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 85)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 85)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 85)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 86)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 86)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 86)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 86)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 87)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 87)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 87)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 87)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 148)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 149)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 150)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 151)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 212)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1280)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 212)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1312)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 212)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1281)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 212)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1313)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 213)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1344)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 213)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1376)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 213)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1345)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 213)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1377)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 214)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1408)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 214)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1440)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 214)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1409)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 214)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1441)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 215)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1472)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 215)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1504)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 215)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1473)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 215)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1505)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 24)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 24)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 24)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 24)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 25)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 25)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 25)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 25)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 26)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 26)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 26)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 26)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 27)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 27)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 27)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 27)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 88)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 88)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 88)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 88)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 89)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 89)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 89)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 89)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 90)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 90)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 90)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 90)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 91)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 91)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 91)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 91)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 152)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 153)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 154)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 155)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 216)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1536)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 216)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1568)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 216)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1537)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 216)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1569)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 217)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1600)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 217)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1632)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 217)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1601)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 217)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1633)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 218)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1664)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 218)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1696)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 218)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1665)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 218)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1697)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 219)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1728)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 219)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1760)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 219)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1729)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 219)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1761)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 28)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 28)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 28)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 28)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 29)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 29)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 29)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 29)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 30)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 30)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 30)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 30)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 31)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 31)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 31)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 31)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 92)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 92)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 92)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 92)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 93)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 93)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 93)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 93)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 94)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 94)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 94)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 94)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 95)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 95)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 95)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 95)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 156)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 157)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 158)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 159)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 220)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1792)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 220)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1824)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 220)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1793)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 220)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1825)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 221)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1856)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 221)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1888)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 221)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1857)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 221)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1889)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 222)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1920)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 222)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1952)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 222)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1921)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 222)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1953)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 223)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1984)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 223)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2016)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 223)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 1985)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 223)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2017)]));
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1760)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1760)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1760)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1760)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1761)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1761)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1761)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1761)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1762)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1762)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1762)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1762)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1763)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1763)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1763)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1763)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1824)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1824)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1824)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1824)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1825)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1825)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1825)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1825)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1826)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1826)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1826)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1826)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1827)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1827)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1827)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1827)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1888)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1888)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1888)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1888)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1889)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1889)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1889)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1889)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1890)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1890)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1890)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1890)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1891)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1891)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1891)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1891)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1952)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2048)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1952)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2080)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1952)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2049)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1952)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2081)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1953)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2112)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1953)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2144)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1953)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2113)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1953)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2145)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1954)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2176)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1954)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2208)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1954)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2177)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1954)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2209)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1955)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2240)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1955)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2272)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1955)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2241)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1955)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2273)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1764)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1764)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1764)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1764)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1765)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1765)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1765)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1765)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1766)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1766)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1766)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1766)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1767)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1767)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1767)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1767)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1828)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1828)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1828)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1828)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1829)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1829)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1829)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1829)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1830)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1830)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1830)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1830)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1831)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1831)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1831)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1831)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1892)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1892)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1892)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1892)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1893)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1893)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1893)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1893)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1894)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1894)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1894)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1894)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1895)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1895)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1895)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1895)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1956)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2304)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1956)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2336)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1956)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2305)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1956)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2337)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1957)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2368)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1957)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2400)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1957)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2369)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1957)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2401)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1958)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2432)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1958)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2464)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1958)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2433)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1958)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2465)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1959)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2496)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1959)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2528)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1959)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2497)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1959)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2529)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1768)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1768)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1768)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1768)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1769)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1769)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1769)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1769)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1770)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1770)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1770)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1770)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1771)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1771)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1771)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1771)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1832)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1832)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1832)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1832)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1833)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1833)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1833)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1833)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1834)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1834)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1834)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1834)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1835)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1835)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1835)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1835)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1896)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1896)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1896)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1896)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1897)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1897)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1897)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1897)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1898)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1898)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1898)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1898)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1899)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1899)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1899)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1899)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1960)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2560)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1960)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2592)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1960)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2561)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1960)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2593)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1961)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2624)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1961)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2656)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1961)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2625)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1961)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2657)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1962)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2688)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1962)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2720)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1962)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2689)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1962)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2721)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1963)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2752)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1963)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2784)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1963)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2753)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1963)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2785)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1772)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1772)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1772)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1772)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1773)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1773)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1773)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1773)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1774)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1774)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1774)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1774)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1775)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1775)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1775)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1775)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1836)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1836)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1836)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1836)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1837)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1837)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1837)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1837)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1838)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1838)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1838)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1838)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1839)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1839)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1839)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1839)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1900)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1900)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1900)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1900)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1901)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1901)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1901)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1901)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1902)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1902)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1902)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1902)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1903)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1903)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1903)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1903)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1964)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2816)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1964)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2848)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1964)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2817)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1964)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2849)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1965)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2880)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1965)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2912)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1965)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2881)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1965)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2913)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1966)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2944)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1966)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2976)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1966)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2945)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1966)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 2977)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1967)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3008)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1967)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3040)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1967)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3009)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1967)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3041)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1776)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1776)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1776)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1776)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1777)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1777)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1777)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1777)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1778)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1778)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1778)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1778)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1779)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1779)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1779)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1779)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1840)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1840)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1840)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1840)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1841)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1841)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1841)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1841)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1842)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1842)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1842)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1842)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1843)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1843)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1843)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1843)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1904)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1904)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1904)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1904)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1905)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1905)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1905)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1905)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1906)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1906)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1906)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1906)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1907)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1907)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1907)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1907)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1968)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3072)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1968)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3104)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1968)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3073)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1968)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3105)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1969)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3136)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1969)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3168)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1969)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3137)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1969)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3169)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1970)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3200)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1970)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3232)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1970)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3201)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1970)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3233)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1971)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3264)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1971)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3296)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1971)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3265)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1971)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3297)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1780)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1780)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1780)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1780)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1781)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1781)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1781)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1781)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1782)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1782)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1782)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1782)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1783)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1783)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1783)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1783)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1844)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1844)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1844)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1844)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1845)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1845)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1845)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1845)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1846)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1846)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1846)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1846)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1847)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1847)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1847)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1847)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1908)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1908)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1908)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1908)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1909)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1909)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1909)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1909)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1910)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1910)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1910)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1910)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1911)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1911)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1911)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1911)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1972)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3328)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1972)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3360)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1972)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3329)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1972)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3361)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1973)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3392)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1973)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3424)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1973)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3393)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1973)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3425)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1974)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3456)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1974)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3488)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1974)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3457)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1974)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3489)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1975)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3520)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1975)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3552)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1975)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3521)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1975)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3553)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1784)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1784)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1784)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1784)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1785)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1785)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1785)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1785)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1786)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1786)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1786)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1786)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1787)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1787)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1787)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1787)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1848)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1848)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1848)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1848)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1849)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1849)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1849)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1849)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1850)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1850)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1850)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1850)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1851)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1851)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1851)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1851)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1912)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1912)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1912)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1912)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1913)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1913)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1913)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1913)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1914)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1914)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1914)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1914)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1915)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1915)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1915)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1915)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1976)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3584)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1976)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3616)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1976)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3585)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1976)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3617)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1977)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3648)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1977)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3680)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1977)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3649)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1977)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3681)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1978)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3712)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1978)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3744)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1978)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3713)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1978)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3745)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1979)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3776)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1979)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3808)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1979)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3777)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1979)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3809)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1788)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1788)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1788)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1788)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1789)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1789)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1789)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1789)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1790)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1790)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1790)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1790)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[0] = (conv2d_nhwc_local[0] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1791)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[8] = (conv2d_nhwc_local[8] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1791)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[1] = (conv2d_nhwc_local[1] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1791)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[9] = (conv2d_nhwc_local[9] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1791)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1852)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1852)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1852)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1852)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1853)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1853)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1853)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1853)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1854)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1854)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1854)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1854)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[2] = (conv2d_nhwc_local[2] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1855)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[10] = (conv2d_nhwc_local[10] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1855)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[3] = (conv2d_nhwc_local[3] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1855)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[11] = (conv2d_nhwc_local[11] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1855)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1916)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1916)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1916)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1916)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1917)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1917)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1917)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1917)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1918)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1918)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1918)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1918)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[4] = (conv2d_nhwc_local[4] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1919)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[12] = (conv2d_nhwc_local[12] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1919)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[5] = (conv2d_nhwc_local[5] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1919)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[13] = (conv2d_nhwc_local[13] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1919)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1980)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3840)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1980)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3872)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1980)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3841)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1980)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3873)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1981)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3904)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1981)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3936)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1981)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3905)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1981)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3937)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1982)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3968)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1982)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4000)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1982)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 3969)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1982)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4001)]));
  conv2d_nhwc_local[6] = (conv2d_nhwc_local[6] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1983)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4032)]));
  conv2d_nhwc_local[14] = (conv2d_nhwc_local[14] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1983)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4064)]));
  conv2d_nhwc_local[7] = (conv2d_nhwc_local[7] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1983)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4033)]));
  conv2d_nhwc_local[15] = (conv2d_nhwc_local[15] + (PadInput_shared[(((((int)threadIdx.x) >> 4) * 256) + 1983)] * weight_shared[(((((int)threadIdx.x) & 15) * 2) + 4065)]));
  conv2d_nhwc[(((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2))] = conv2d_nhwc_local[0];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)] = conv2d_nhwc_local[8];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)] = conv2d_nhwc_local[1];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)] = conv2d_nhwc_local[9];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 128)] = conv2d_nhwc_local[2];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 160)] = conv2d_nhwc_local[10];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 129)] = conv2d_nhwc_local[3];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 161)] = conv2d_nhwc_local[11];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 256)] = conv2d_nhwc_local[4];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 288)] = conv2d_nhwc_local[12];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 257)] = conv2d_nhwc_local[5];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 289)] = conv2d_nhwc_local[13];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 384)] = conv2d_nhwc_local[6];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 416)] = conv2d_nhwc_local[14];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 385)] = conv2d_nhwc_local[7];
  conv2d_nhwc[((((((((int)blockIdx.x) >> 1) * 3584) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 417)] = conv2d_nhwc_local[15];
}


